#include "hip/hip_runtime.h"
#include <stdio.h>

#include <assert.h>

#include "cudpp/cudpp.h"

#if	 __DEVICE_EMULATION__ 
#pragma comment (lib, "cudpp32d_emu.lib")
#else
#pragma comment (lib, "cudpp32.lib")
#endif

#include "cuda_macro.h"
#pragma comment (lib, "cutil32.lib ")      // link with my own library libfps

#pragma comment (lib, "cudart.lib")      // link with my own library libfps


static float pfModelViewMatrix_host[16];
static float pfProjectionMatrix_host[16];
static __constant__ float pfModelViewMatrix_constant[16];
static __constant__ float pfProjectionMatrix_constant[16];
static float4 *pf4LineCentroids_global;
static int *piSlabs_global;
static int *piLines_global;
static int *piSlabs_host;
static int *piLines_host;
static CUDPPHandle hScanPlan = 0;

__device__
float4 
F4MatrixDotVector
(
	float pfMatrix[],
	float4 f4Vector
)
{
	return make_float4(
		pfMatrix[0] * f4Vector.x + pfMatrix[4] * f4Vector.y + pfMatrix[8] * f4Vector.z + pfMatrix[12] * f4Vector.w,
		pfMatrix[1] * f4Vector.x + pfMatrix[5] * f4Vector.y + pfMatrix[9] * f4Vector.z + pfMatrix[13] * f4Vector.w,
		pfMatrix[2] * f4Vector.x + pfMatrix[6] * f4Vector.y + pfMatrix[10] * f4Vector.z + pfMatrix[14] * f4Vector.w,
		pfMatrix[3] * f4Vector.x + pfMatrix[7] * f4Vector.y + pfMatrix[11] * f4Vector.z + pfMatrix[15] * f4Vector.w
		);
}

__global__
void
_ComputeDepth_kernel
(
	float	fMinZ,
	float	fMaxZ,
	int		iNrOfSlabs,
	int		iNrOfLines,
	float4	pf4LineCentroids_global[],
	int		piSlabs_global[],
	int		piLines_global[]
)
{
	int iL = threadIdx.x + blockDim.x * blockIdx.x;	// index to the assigned line segment

	if( iL < iNrOfLines )
	{
		float4 f4Centroid_obj = pf4LineCentroids_global[iL];
		float4 f4Centroid_eye = F4MatrixDotVector(pfModelViewMatrix_constant,	f4Centroid_obj);
		float fDepth = f4Centroid_eye.z / f4Centroid_eye.w;

		int iSlab = int(float(iNrOfSlabs) *  (fDepth - fMinZ) / (fMaxZ - fMinZ));
		iSlab = min(max(iSlab, 0), iNrOfSlabs - 1);

		piSlabs_global[iL] = iSlab; 
		piLines_global[iL] = iL; 
	}
}

extern "C" {
void 
_ComputeDepthInit_cuda
(
	int iNrOfLines,
	float pfLineCentroids[]
)
{
	CUDA_SAFE_CALL_NO_SYNC(
		hipMalloc(	(void**)&piSlabs_global, sizeof(piSlabs_global[0]) * iNrOfLines) );
	CUDA_SAFE_CALL_NO_SYNC(
		hipMalloc(	(void**)&piLines_global, sizeof(piLines_global[0]) * iNrOfLines) );
	CUDA_SAFE_CALL_NO_SYNC(
		hipMalloc(	(void**)&pf4LineCentroids_global, sizeof(pf4LineCentroids_global[0]) * iNrOfLines) );

	CUDA_SAFE_CALL_NO_SYNC(
		hipHostMalloc(	(void**)&piSlabs_host, sizeof(piSlabs_host[0]) * iNrOfLines) );
	CUDA_SAFE_CALL_NO_SYNC(
		hipHostMalloc(	(void**)&piLines_host, sizeof(piLines_host[0]) * iNrOfLines) );

	float4 *pf4LineCentroids_host;
	CUDA_SAFE_CALL_NO_SYNC(
		hipHostMalloc(	(void**)&pf4LineCentroids_host, sizeof(pf4LineCentroids_host[0]) * iNrOfLines) );
	for(int l = 0; l < iNrOfLines; l++)
	{
		pf4LineCentroids_host[l] = 
			make_float4(
				pfLineCentroids[3 * l + 0],
				pfLineCentroids[3 * l + 1],
				pfLineCentroids[3 * l + 2],
				1.0);
	}

	CUDA_SAFE_CALL_NO_SYNC(
		hipMemcpy(	pf4LineCentroids_global, pf4LineCentroids_host, sizeof(pf4LineCentroids_global[0]) * iNrOfLines, hipMemcpyHostToDevice) );

	FREE_MEMORY_ON_HOST(pf4LineCentroids_host);

	CUDPPConfiguration	cConfig;
	cConfig.op =		CUDPP_ADD;
	cConfig.datatype =	CUDPP_UINT;
	cConfig.algorithm =	CUDPP_SORT_RADIX;
	cConfig.options =	CUDPP_OPTION_KEY_VALUE_PAIRS;

	assert( 
		CUDPP_SUCCESS  == cudppPlan(
			&hScanPlan,	
			cConfig, 
			iNrOfLines,
			1, 
			0) );

}

void
_ComputeDepth_cuda
(
	int iNrOfSlabs,

	double dMinZ,
	double dMaxZ,
	double pdModelViewMatrix[],
	double pdProjectionMatrix[],

	int iNrOfLines,
	int2 pi2Slabs[]
)
{
	for(int i = 0; i < 16; i++)
	{
		pfModelViewMatrix_host[i] =		float(pdModelViewMatrix[i]);
		pfProjectionMatrix_host[i] =	float(pdProjectionMatrix[i]);
	}
	CUDA_SAFE_CALL_NO_SYNC(
		hipMemcpyToSymbol(HIP_SYMBOL(pfModelViewMatrix_constant),	pfModelViewMatrix_host,	sizeof(pfModelViewMatrix_constant)) );
	CUDA_SAFE_CALL_NO_SYNC(
		hipMemcpyToSymbol(HIP_SYMBOL(pfProjectionMatrix_constant), pfProjectionMatrix_host, sizeof(pfProjectionMatrix_constant)) );

	int iNrOfThreads = 128;
	int iNrOfBlocks = int(ceilf(float(iNrOfLines)/float(iNrOfThreads)));

	_ComputeDepth_kernel<<<
		dim3(iNrOfBlocks, 1, 1),
		dim3(iNrOfThreads, 1, 1),
		0
	>>>
	(
		float(dMinZ),
		float(dMaxZ),
		iNrOfSlabs,
		iNrOfLines,
		pf4LineCentroids_global,
		piSlabs_global,
		piLines_global
	);
	CUT_CHECK_ERROR("_ComputeDepth_kernel() failed");

	cudppSort(
		hScanPlan,
		piSlabs_global,
		piLines_global,
		int(ceilf(log2f(float(iNrOfSlabs)))),
		iNrOfLines) ;
	CUT_CHECK_ERROR("cudppSort() failed");

	CUDA_SAFE_CALL_NO_SYNC(
		hipMemcpy(	piSlabs_host, piSlabs_global, sizeof(piSlabs_host[0]) * iNrOfLines, hipMemcpyDeviceToHost) );

	CUDA_SAFE_CALL_NO_SYNC(
		hipMemcpy(	piLines_host, piLines_global, sizeof(piLines_host[0]) * iNrOfLines, hipMemcpyDeviceToHost) );

	for(int l = 0; l < iNrOfLines; l++)
	{
		pi2Slabs[l].x = piSlabs_host[l];
		pi2Slabs[l].y = piLines_host[l];
	}
}

void 
_ComputeDeptpFree_cuda
(
)
{
	FREE_MEMORY( piSlabs_global );
	FREE_MEMORY( piLines_global );
	FREE_MEMORY_ON_HOST( piSlabs_host );
	FREE_MEMORY_ON_HOST( piLines_host );
	FREE_MEMORY( pf4LineCentroids_global );
	cudppDestroyPlan(hScanPlan);
}

} // extern "C"

/*

$Log: not supported by cvs2svn $
Revision 1.2  2009/12/31 02:00:30  leeten

[12/30/2009]
1. [ADD] Add the log section.


*/
