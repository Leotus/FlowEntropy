#include "hip/hip_runtime.h"
// ADD-BY-LEETEN 12/07/2009-BEGIN
__global__ 
static 
void 
_FlowDiffusion2D_kernel
(
	// INPUT
	float fAttenuation,

	int iVolumeWidth,
	int iVolumeHeight,

	hipPitchedPtr cDstPitchedPtr,

	// ADD-BY-LEETEN 2009/11/25-BEGIN
	hipPitchedPtr cErrorPitchedPtr
	// ADD-BY-LEETEN 2009/11/25-END
)
{
 	int iVoxelX = blockIdx.x * blockDim.x + threadIdx.x;
	int iVoxelY = blockIdx.y * blockDim.y + threadIdx.y;

	// compute the central differnece
	float4 f4Value	= tex2D(t2dSrc, iVoxelX,		iVoxelY);
	float4 f4PX		= tex2D(t2dSrc, min(iVoxelX + 1, iVolumeWidth - 1),	iVoxelY);
	float4 f4NX		= tex2D(t2dSrc, max(iVoxelX - 1, 0),				iVoxelY);
	float4 f4PY		= tex2D(t2dSrc, iVoxelX,		min(iVoxelY + 1, iVolumeHeight - 1));
	float4 f4NY		= tex2D(t2dSrc, iVoxelX,		max(iVoxelY - 1, 0)				);

	float4 f4WeightOffset = tex3D(t3dWeightOffset, iVoxelX,	iVoxelY, 0);

	float4 f4Result;

	f4Result = make_float4(
		f4WeightOffset.w * f4Value.x + (f4PX.x + f4NX.x + f4PY.x + f4NY.x - 4.0f * f4Value.x) * fAttenuation + f4WeightOffset.x,
		f4WeightOffset.w * f4Value.y + (f4PX.y + f4NX.y + f4PY.y + f4NY.y - 4.0f * f4Value.y) * fAttenuation + f4WeightOffset.y,
		f4WeightOffset.w * f4Value.z + (f4PX.z + f4NX.z + f4PY.z + f4NY.z - 4.0f * f4Value.z) * fAttenuation + f4WeightOffset.z,
		0);

	if( iVoxelX < iVolumeWidth && iVoxelY < iVolumeHeight )
	{
		*ADDRESS_2D(
			float4, cDstPitchedPtr.ptr, 
			sizeof(float4), cDstPitchedPtr.pitch, 
			iVoxelX, iVoxelY) = f4Result;

		// ADD-BY-LEETEN 12/16/2009-BEGIN
		#if	CHECK_ERROR_CONVERGENCE_BY_CUDPP
		// ADD-BY-LEETEN 12/16/2009-END
		float4 f4Diff;
		f4Diff.x = f4Value.x - f4Result.x;
		f4Diff.y = f4Value.y - f4Result.y;
		f4Diff.z = f4Value.z - f4Result.z;
		float fDiff = f4Diff.x * f4Diff.x + f4Diff.y * f4Diff.y + f4Diff.z * f4Diff.z;
		*ADDRESS_2D(
			float, cErrorPitchedPtr.ptr, 
			sizeof(float), cErrorPitchedPtr.pitch, 
			iVoxelX, iVoxelY) = fDiff;
		// ADD-BY-LEETEN 12/16/2009-BEGIN
		#endif	// #if	CHECK_ERROR_CONVERGENCE_BY_CUDPP
		// ADD-BY-LEETEN 12/16/2009-END
	}
}

/*

$Log: not supported by cvs2svn $

*/
