#include "hip/hip_runtime.h"
#include "FlowDiffusion_cuda.h"

enum {
	UPDATE_DIR_X, 
	UPDATE_DIR_Y, 
	UPDATE_DIR_Z, 
};

enum {
	HISTOGRAM_OP_SUB_SLICE = -1, 
	HISTOGRAM_OP_ADD_SLICE = +1, 
};

						// a texture tha represents the src. bin volume
						// this textrue defines the input for the kernel
static texture<int, 2, hipReadModeElementType> t2dSrcBinVolume;
						// a texture tha represents the dst. bin volume
static texture<int, 2, hipReadModeElementType> t2dDstBinVolume;
static texture<int, 1, hipReadModeElementType> t1dHistogram;

					// a volume of integer, where each element is the joint histogram of both the src. and dst. bin volume
					// here the joint of two bins mean aan integer that represent both numbers
static hipPitchedPtr cSrcDstBinVolumePtr_global;	

// ADD-BY-LEETEN 12/18/2009-BEGIN
static texture<int, 2, hipReadModeElementType> t2dActiveVoxelHistorgrams;

// MOD-BY-LEETEN 12/18/2009-FROM
	// static texture<unsigned int, 1, hipReadModeElementType> t1dActiveVoxelSortedNeighbors;
// TO:
static texture<unsigned int, 2, hipReadModeElementType> t2dActiveVoxelSortedNeighbors;
// MOD-BY-LEETEN 12/18/2009-END
// ADD-BY-LEETEN 12/18/2009-END


// the kernel to update the histogram 
__device__ 
int IMirrorCoord(int iCoord, int iSize)
{
	int iMirroredCoord = iCoord;
	if( iCoord < 0 )
		iMirroredCoord = -iCoord;
	if( iCoord > iSize - 1 )
		iMirroredCoord = iSize - (iCoord - (iSize - 1));
	return iMirroredCoord;
}

// before calling this kernel, the input volume should be bound to the texture t2dBinVolume

__global__ 
void
_UpdateSliceToHistogram_kernel
(
	int3 i3Center, 
	int	iUpdateDir, 
	int iHisotgramOp, 

	int3 i3UDir, 
	int3 i3VDir,
	int iUKernelSize, 
	int iVKernelSize,

	int3 i3VolumeSize,

	int iNrOfBins,
	int *piHistorgram_global
)
{
 	int iX = blockIdx.x * blockDim.x + threadIdx.x;
 	int iY = blockIdx.y * blockDim.y + threadIdx.y;
	int3 i3Point;
	i3Point.x = i3Center.x + (iX - iUKernelSize) * i3UDir.x + (iY - iVKernelSize) * i3VDir.x;
	i3Point.y = i3Center.y + (iX - iUKernelSize) * i3UDir.y + (iY - iVKernelSize) * i3VDir.y;
	i3Point.z = i3Center.z + (iX - iUKernelSize) * i3UDir.z + (iY - iVKernelSize) * i3VDir.z;

	// read the bin
	int3 i3TexCoord;
	i3TexCoord.x = IMirrorCoord(i3Point.x, i3VolumeSize.x);
	i3TexCoord.y = IMirrorCoord(i3Point.y, i3VolumeSize.y);
	i3TexCoord.z = IMirrorCoord(i3Point.z, i3VolumeSize.z);

	if( iX < 2 * iUKernelSize + 1 && iY < 2 * iVKernelSize + 1)
	{
		int iSrcBin = tex2D(t2dSrcBinVolume, i3TexCoord.x, i3TexCoord.y + i3TexCoord.z * i3VolumeSize.y);

		// update the joint histogram
		atomicAdd(&piHistorgram_global[iSrcBin], iHisotgramOp );
	}
}

// comptue p log p for each bin
// the final entropy will be summed later via CUDPP
__global__ 
void 
_ComputeProbDotLogProb_kernel
(
	int iNrOfElements,

	int iNrOfBins,
	float *pfLogHistogram_global
)
{
	int iBin = blockIdx.x * blockDim.x + threadIdx.x;
	int iCount = tex1D(t1dHistogram, iBin);
	float fProbDotLogProb = 0.0f;
	if( iCount )
	{
		float fCount = float(iCount);
		fProbDotLogProb = log2(fCount) * fCount;
	}
	float fNrOfElements = float(iNrOfElements);
	fProbDotLogProb = log2(fNrOfElements)/float(iNrOfBins) - fProbDotLogProb / fNrOfElements;

	if( iBin < iNrOfBins )
	{
		pfLogHistogram_global[iBin] = fProbDotLogProb;
	}
}

__global__
void 
_JointSrcDst_kernel
(
	int iNrOfSrcBins, 
	int iNrOfDstbins,

	int3 i3VolumeSize,
	hipPitchedPtr cJointBinVolume_pitched
)
{
	int iVoxelX = blockIdx.x * blockDim.x + threadIdx.x;
	int iVoxelYZ = blockIdx.y * blockDim.y + threadIdx.y;
	int iVoxelY = (iVoxelYZ % i3VolumeSize.y);
	int iVoxelZ = (iVoxelYZ / i3VolumeSize.y);

	int iSrcBin = tex2D(t2dSrcBinVolume, iVoxelX,	iVoxelY + iVoxelZ * i3VolumeSize.y);
	int iDstBin = tex2D(t2dDstBinVolume, iVoxelX,	iVoxelY + iVoxelZ * i3VolumeSize.y);
	int iJointBin = iSrcBin + iDstBin * iNrOfSrcBins;

	if( iVoxelX < i3VolumeSize.x && iVoxelY < i3VolumeSize.y && iVoxelZ < i3VolumeSize.z )  
		*ADDRESS_2D(
			int, cJointBinVolume_pitched.ptr, 
			sizeof(int), cJointBinVolume_pitched.pitch, 
			iVoxelX, iVoxelY + iVoxelZ * i3VolumeSize.y) = iJointBin;
}

__global__ 
static 
void 
// MOD-BY-LEETEN 12/18/2009-FROM:
	// _Vector3DToVolume_kernel
// TO:
_Vector3DToBinVolume_kernel
// MOD-BY-LEETEN 12/18/2009-END
(
	int iVolumeWidth,
	int iVolumeHeight,
	int iVolumeDepth,

	int iNrOfYBlocks,
	int iBlockZSize,

	// texture<float4, 2, hipReadModeElementType> t2dSrc,
	hipPitchedPtr cBinVolumePtr_global
)
{
 	int iVoxelX = blockIdx.x * blockDim.x + threadIdx.x;
	int iVoxelY = (blockIdx.y % iNrOfYBlocks) * blockDim.y + threadIdx.y;
	int iBeginZ = (blockIdx.y / iNrOfYBlocks) * iBlockZSize;
	int iEndZ = min(iBeginZ + iBlockZSize, iVolumeDepth);

	for(int z = iBeginZ; z < iEndZ; z++)
	{
		float4 f4Vector = tex2D(t2dVectorVolume, iVoxelX,		iVoxelY + z				* iVolumeHeight);;
		
		float fLength	= sqrt(f4Vector.x * f4Vector.x + f4Vector.y * f4Vector.y + f4Vector.z * f4Vector.z);
		float fTheta	= 0.0f;
		float fPhi		= 0.0f;
		int iBin = 0;

		if( 0 < fLength )
		{
			f4Vector.x /= fLength;
			f4Vector.y /= fLength;
			f4Vector.z /= fLength;
			fTheta = ( 0.0f == f4Vector.x && 0.0f == f4Vector.y )?0.0f:(float(M_PI)+(atan2(f4Vector.y, f4Vector.x)));
			fTheta /= 2.0f * float(M_PI);

			float fLength2D = sqrt(f4Vector.x * f4Vector.x + f4Vector.y * f4Vector.y);
			fPhi = ((0.0f == fLength2D)&&(0.0f == f4Vector.z))?0.0f:fabs(float(M_PI)/2.0f-(atan2(f4Vector.z, fLength2D)));
			fPhi /= float(M_PI);
			iBin = tex2D(t2dAngleMap, fPhi, fTheta);
		}

		if( iVoxelX < iVolumeWidth && iVoxelY < iVolumeHeight )
		{
			*ADDRESS_2D(
				int,			cBinVolumePtr_global.ptr, 
				sizeof(int),	cBinVolumePtr_global.pitch, 
				iVoxelX, iVoxelY + z * iVolumeHeight) = iBin;
		}
	}
}

// ADD-BY-LEETEN 12/18/2009-BEGIN
__device__
void
_UpdateHistogramEntry_device
(
	int iActiveVoxelId,

	int iVoxelX, 
	int iVoxelY, 
	int iVoxelZ,
	int iXOffset,
	int iYOffset,
	int iZOffset,

	int iValue,

	int iNrOfBins,

	int3 i3VolumeSize,
	hipPitchedPtr cActiveVoxelsHistorgram_pitched
)
{
	int iX, iY, iZ;
	iX = iVoxelX + iXOffset;
	iY = iVoxelY + iYOffset;
	iZ = iVoxelZ + iZOffset;

	int3 i3TexCoord;
	i3TexCoord.x = IMirrorCoord(iX, i3VolumeSize.x);
	i3TexCoord.y = IMirrorCoord(iY, i3VolumeSize.y);
	i3TexCoord.z = IMirrorCoord(iZ, i3VolumeSize.z);

	int iSrcBin = tex2D(t2dSrcBinVolume, i3TexCoord.x, i3TexCoord.y + i3TexCoord.z * i3VolumeSize.y);

						// update the joint histogram
	if( iSrcBin  < iNrOfBins )
		atomicAdd(
			ADDRESS_2D(
				int,			cActiveVoxelsHistorgram_pitched.ptr, 
				sizeof(int),	cActiveVoxelsHistorgram_pitched.pitch, 
				iActiveVoxelId, iSrcBin),
			iValue );
}

__global__ 
static 
void 
_UpdateHistogram_kernel
(
	int3 i3BlockCorner,

	int3 i3KernelSize,

	int3 i3VolumeSize,

	int iNrOfBins,
	hipPitchedPtr cActiveVoxelsHistorgram_pitched
)
{
	int iActiveVoxelX = blockIdx.x * blockDim.x + threadIdx.x;
	#if	0		// MOD-BY-LEETEN 12/19/2009-FROM:
		int iActiveVoxelZ = blockIdx.y * blockDim.y + threadIdx.y;
		int iActiveVoxelId = iActiveVoxelX + iActiveVoxelZ * gridDim.x * blockDim.x;
 		int iVoxelX = i3BlockCorner.x + iActiveVoxelX;
		int iVoxelY = i3BlockCorner.y;
		int iVoxelZ = i3BlockCorner.z + iActiveVoxelZ;
	#else		// MOD-BY-LEETEN 12/19/2009-TO:

	int iActiveVoxelY = blockIdx.y * blockDim.y + threadIdx.y;
	int iActiveVoxelId = iActiveVoxelX + iActiveVoxelY * gridDim.x * blockDim.x;
 	int iVoxelX = i3BlockCorner.x + iActiveVoxelX;
	int iVoxelY = i3BlockCorner.y + iActiveVoxelY;
	int iVoxelZ = i3BlockCorner.z;

	#endif		// MOD-BY-LEETEN 12/19/2009-END


	if( iVoxelX < i3VolumeSize.x && iVoxelY < i3VolumeSize.y && iVoxelZ < i3VolumeSize.z  )

		#if	0		// MOD-BY-LEETEN 12/19/2009-FROM:

			for(int iYDir = -1; iYDir <= +1; iYDir+=2)
			{
				int iYOffset = i3KernelSize.y * iYDir;
				switch (iYDir)
				{
				case -1:	iYOffset--;		break;
				}
				for(int		iZOffset = -i3KernelSize.z; iZOffset <= i3KernelSize.z; iZOffset++)
					for(int iXOffset = -i3KernelSize.x; iXOffset <= i3KernelSize.x; iXOffset++)
					{
						_UpdateHistogramEntry_device
						(
							iActiveVoxelId,	
							iVoxelX, iVoxelY, iVoxelZ,
							iXOffset,iYOffset,iZOffset,
							iYDir,
							iNrOfBins,
							i3VolumeSize, 
							cActiveVoxelsHistorgram_pitched
						);
					}
			}

		#else

		for(int iZDir = -1; iZDir <= +1; iZDir+=2)
		{
			int iZOffset = i3KernelSize.z * iZDir;
			switch (iZDir)
			{
			case -1:	iZOffset--;		break;
			}
			for(int		iYOffset = -i3KernelSize.y; iYOffset <= i3KernelSize.y; iYOffset++)
				for(int iXOffset = -i3KernelSize.x; iXOffset <= i3KernelSize.x; iXOffset++)
				{
					_UpdateHistogramEntry_device
					(
						iActiveVoxelId,	
						iVoxelX, iVoxelY, iVoxelZ,
						iXOffset,iYOffset,iZOffset,
						iZDir,
						iNrOfBins,
						i3VolumeSize, 
						cActiveVoxelsHistorgram_pitched
					);
				}
		}
		#endif
}

__global__ 
static 
void 
_CreateHistogram_kernel
(
	int3 i3BlockCorner,

	int3 i3KernelSize,

	int3 i3VolumeSize,

	int iNrOfBins,
	hipPitchedPtr cActiveVoxelsHistorgram_pitched
)
{
	int iActiveVoxelX = blockIdx.x * blockDim.x + threadIdx.x;
	#if	0		// MOD-BY-LEETEN 12/19/2009-FROM:
		int iActiveVoxelZ = blockIdx.y * blockDim.y + threadIdx.y;
		int iActiveVoxelId = iActiveVoxelX + iActiveVoxelZ * gridDim.x * blockDim.x;
 		int iVoxelX = i3BlockCorner.x + iActiveVoxelX;
		int iVoxelY = i3BlockCorner.y;
		int iVoxelZ = i3BlockCorner.z + iActiveVoxelZ;
	#else		// MOD-BY-LEETEN 12/19/2009-TO:

	int iActiveVoxelY = blockIdx.y * blockDim.y + threadIdx.y;
	int iActiveVoxelId = iActiveVoxelX + iActiveVoxelY * gridDim.x * blockDim.x;
 	int iVoxelX = i3BlockCorner.x + iActiveVoxelX;
	int iVoxelY = i3BlockCorner.y + iActiveVoxelY;
	int iVoxelZ = i3BlockCorner.z;

	#endif		// MOD-BY-LEETEN 12/19/2009-END

	if( iVoxelX < i3VolumeSize.x && iVoxelY < i3VolumeSize.y && iVoxelZ < i3VolumeSize.z  )
		for(int				iZOffset = -i3KernelSize.z; iZOffset <= i3KernelSize.z; iZOffset++)
			for(int			iYOffset = -i3KernelSize.y; iYOffset <= i3KernelSize.y; iYOffset++)
				for(int		iXOffset = -i3KernelSize.x; iXOffset <= i3KernelSize.x; iXOffset++)
				{
					_UpdateHistogramEntry_device
					(
						iActiveVoxelId,	
						iVoxelX, iVoxelY, iVoxelZ,
						iXOffset,iYOffset,iZOffset,
						+1,
						iNrOfBins,
						i3VolumeSize, 
						cActiveVoxelsHistorgram_pitched
					);
				}
}

__global__ 
void 
_ComputeEntropy_kernel
(
	int3 i3BlockCorner,
	int iNrOfBins,
	int3 i3KernelSize,
	int3 i3VolumeSize,
	hipPitchedPtr cEntropyVolume_pitched
)
{
	int iActiveVoxelX = blockIdx.x * blockDim.x + threadIdx.x;
	#if	0	// MOD-BY-LEETEN 12/19/2009-FROM:
		int iActiveVoxelZ = blockIdx.y * blockDim.y + threadIdx.y;
		int iActiveVoxelId = iActiveVoxelX + iActiveVoxelZ * gridDim.x * blockDim.x;
 		int iVoxelX = i3BlockCorner.x + iActiveVoxelX;
		int iVoxelY = i3BlockCorner.y;
		int iVoxelZ = i3BlockCorner.z + iActiveVoxelZ;
	#else	// MOD-BY-LEETEN 12/19/2009-TO:

	int iActiveVoxelY = blockIdx.y * blockDim.y + threadIdx.y;
	int iActiveVoxelId = iActiveVoxelX + iActiveVoxelY * gridDim.x * blockDim.x;
 	int iVoxelX = i3BlockCorner.x + iActiveVoxelX;
	int iVoxelY = i3BlockCorner.y + iActiveVoxelY;
	int iVoxelZ = i3BlockCorner.z;

	#endif	// MOD-BY-LEETEN 12/19/2009-END

	int iNrOfNeighbors = (2 * i3KernelSize.x + 1) * (2 * i3KernelSize.y + 1) * (2 * i3KernelSize.z + 1);
	float fNrOfNeighbors = float(iNrOfNeighbors);

	float fEntropy = 0.0f;
	for(int b = 0; b < iNrOfBins; b++)
	{
		int iCount = tex2D(t2dActiveVoxelHistorgrams, iActiveVoxelId, b);
		if( iCount )
		{
			/*
			float fProb = float(iCount) / fNrOfNeighbors;
			fEntropy += fProb * log2(fProb);
			*/
			float fCount = float(iCount);
			fEntropy += fCount * log2(fCount);
		}
	}
	// fEntropy *= -1.0;
	fEntropy = -fEntropy / fNrOfNeighbors + log2(fNrOfNeighbors);
	fEntropy = max(0.0, fEntropy);

	*ADDRESS_2D(
		float, cEntropyVolume_pitched.ptr, 
		sizeof(float), cEntropyVolume_pitched.pitch, 
		iVoxelX, iVoxelY + iVoxelZ * i3VolumeSize.y) = fEntropy;
}

__global__ 
void
_ComputeEntropyOnSortedNeighbors_kernel
(
	int3 i3BlockCorner,
	int iNrOfBins,
	int3 i3KernelSize,
	unsigned int *puActiveVoxelSortedNeighbors_global,
	int3 i3VolumeSize,
	hipPitchedPtr cEntropyVolume_pitched
)
{
	int iActiveVoxelX = blockIdx.x * blockDim.x + threadIdx.x;
	#if	0	// MOD-BY-LEETEN 12/19/2009-FROM:
		int iActiveVoxelZ = blockIdx.y * blockDim.y + threadIdx.y;
		int iActiveVoxelId = iActiveVoxelX + iActiveVoxelZ * gridDim.x * blockDim.x;
 		int iVoxelX = i3BlockCorner.x + iActiveVoxelX;
		int iVoxelY = i3BlockCorner.y;
		int iVoxelZ = i3BlockCorner.z + iActiveVoxelZ;
	#else	// MOD-BY-LEETEN 12/19/2009-TO:

	int iActiveVoxelY = blockIdx.y * blockDim.y + threadIdx.y;
	int iActiveVoxelId = iActiveVoxelX + iActiveVoxelY * gridDim.x * blockDim.x;
 	int iVoxelX = i3BlockCorner.x + iActiveVoxelX;
	int iVoxelY = i3BlockCorner.y + iActiveVoxelY;
	int iVoxelZ = i3BlockCorner.z;

	#endif	// MOD-BY-LEETEN 12/19/2009-END

	int iNrOfNeighbors = (2 * i3KernelSize.x + 1) * (2 * i3KernelSize.y + 1) * (2 * i3KernelSize.z + 1);
	float fNrOfNeighbors = float(iNrOfNeighbors);

	float fEntropy = 0.0f;
	unsigned int uPrevBin = 0;
	float fCount = 0.0f; 

	// ADD-BY-LEETEN 12/18/2009-BEGIN
	unsigned int uNrOfActiveVoxels = gridDim.x * blockDim.x * gridDim.y * blockDim.y;
	unsigned int uTexIndex = iNrOfNeighbors * iActiveVoxelId;
	// ADD-BY-LEETEN 12/18/2009-END

	for(int n = 0; n < iNrOfNeighbors; n++)
	{
		#if	0	// MOD-BY-LEETEN 12/18/2009-FROM:
			unsigned int uBin = tex1Dfetch(t1dActiveVoxelSortedNeighbors, iNrOfNeighbors * iActiveVoxelId + n);
			// unsigned int uBin = puActiveVoxelSortedNeighbors_global[iNrOfNeighbors * iActiveVoxelId + n];
			// uBin = uBin % unsigned int (iNrOfBins);
		#else	// MOD-BY-LEETEN 12/18/2009-TO:
		float fTexCoordX = float(uTexIndex % uNrOfActiveVoxels);
		float fTexCoordY = float(uTexIndex / uNrOfActiveVoxels);
		unsigned int uBin = tex2D(t2dActiveVoxelSortedNeighbors, fTexCoordX, fTexCoordY);
		uTexIndex++;
		#endif	// MOD-BY-LEETEN 12/18/2009-END

		if( n == 0 || uBin == uPrevBin )
		{
			fCount += 1.0f;
		}
		else
		{
			if( fCount > 0.0f )
				fEntropy += fCount * log2(fCount);
			
			fCount = 1.0f;
		}
		uPrevBin = uBin;
	}
	if( fCount > 0.0f )
		fEntropy += fCount * log2(fCount);

	fEntropy = -fEntropy / fNrOfNeighbors + log2(fNrOfNeighbors);
	fEntropy = max(fEntropy, 0.0f);

	*ADDRESS_2D(
		float, cEntropyVolume_pitched.ptr, 
		sizeof(float), cEntropyVolume_pitched.pitch, 
		iVoxelX, iVoxelY + iVoxelZ * i3VolumeSize.y) = fEntropy;
}

__global__ 
void 
_CollectNeighbors_kernel
(
	int3 i3BlockCorner,
	int3 i3KernelSize,
	int3 i3VolumeSize,
	int iNrOfBins,
	hipPitchedPtr cActiveVoxelNeighbors_pitched
)
{
	int iActiveVoxelX = blockIdx.x * blockDim.x + threadIdx.x;
	#if	0	// MOD-BY-LEETEN 12/19/2009-FROM:
		int iActiveVoxelZ = blockIdx.y * blockDim.y + threadIdx.y;
		int iActiveVoxelId = iActiveVoxelX + iActiveVoxelZ * gridDim.x * blockDim.x;
 		int iVoxelX = i3BlockCorner.x + iActiveVoxelX;
		int iVoxelY = i3BlockCorner.y;
		int iVoxelZ = i3BlockCorner.z + iActiveVoxelZ;
		int iNrOfXZNeighbors = (2 * i3KernelSize.x + 1) * (2 * i3KernelSize.z + 1);
	#else	// MOD-BY-LEETEN 12/19/2009-TO:
	int iActiveVoxelY = blockIdx.y * blockDim.y + threadIdx.y;
	int iActiveVoxelId = iActiveVoxelX + iActiveVoxelY * gridDim.x * blockDim.x;
 	int iVoxelX = i3BlockCorner.x + iActiveVoxelX;
	int iVoxelY = i3BlockCorner.y + iActiveVoxelY;
	int iVoxelZ = i3BlockCorner.z;
	int iNrOfXYNeighbors = (2 * i3KernelSize.x + 1) * (2 * i3KernelSize.y + 1);
	#endif	// MOD-BY-LEETEN 12/19/2009-END

	// MOD-BY-LEETEN 12/19/2009-FROM:
		// if ( 0 == iVoxelY )
	// TO:
	if ( 0 == iVoxelZ )
	// MOD-BY-LEETEN 12/19/2009-END
	{
		#if	0	// MOD-BY-LEETEN 12/19/2009-FROM:
			for(int			iOffset = 0,	iYOffset = -i3KernelSize.y; iYOffset <= i3KernelSize.y; iYOffset++)
				for(int						iZOffset = -i3KernelSize.z; iZOffset <= i3KernelSize.z; iZOffset++)
					for(int					iXOffset = -i3KernelSize.x; iXOffset <= i3KernelSize.x; iXOffset++, iOffset++)
		#else	// MOD-BY-LEETEN 12/19/2009-TO:
		for(int	iOffset = 0,iZOffset = -i3KernelSize.z; iZOffset <= i3KernelSize.z; iZOffset++)
			for(int			iYOffset = -i3KernelSize.y; iYOffset <= i3KernelSize.y; iYOffset++)
				for(int		iXOffset = -i3KernelSize.x; iXOffset <= i3KernelSize.x; iXOffset++, iOffset++)
		#endif	// MOD-BY-LEETEN 12/19/2009-END
				{
					int iX, iY, iZ;
					iX = iVoxelX + iXOffset;
					iY = iVoxelY + iYOffset;
					iZ = iVoxelZ + iZOffset;

					int3 i3TexCoord;
					i3TexCoord.x = IMirrorCoord(iX, i3VolumeSize.x);
					i3TexCoord.y = IMirrorCoord(iY, i3VolumeSize.y);
					i3TexCoord.z = IMirrorCoord(iZ, i3VolumeSize.z);

					int iSrcBin = tex2D(t2dSrcBinVolume, i3TexCoord.x, i3TexCoord.y + i3TexCoord.z * i3VolumeSize.y);

					*ADDRESS_2D(
						unsigned int,			cActiveVoxelNeighbors_pitched.ptr, 
						sizeof(unsigned int),	cActiveVoxelNeighbors_pitched.pitch, 
						iActiveVoxelId, iOffset) = unsigned int(iActiveVoxelId * iNrOfBins + iSrcBin);
				}
	}
	else
	{
		#if	0	// MOD-BY-LEETEN 12/19/2009-FROM:
			int iYOffset = i3KernelSize.y;
			for(int		iXZOffset = 0,	iZOffset = -i3KernelSize.z; iZOffset <= i3KernelSize.z; iZOffset++)
				for(int					iXOffset = -i3KernelSize.x; iXOffset <= i3KernelSize.x; iXOffset++, iXZOffset++)
		#else	// MOD-BY-LEETEN 12/19/2009-TO:
		int iZOffset = i3KernelSize.z;
		for(int		iXYOffset = 0,	iYOffset = -i3KernelSize.y; iYOffset <= i3KernelSize.y; iYOffset++)
			for(int					iXOffset = -i3KernelSize.x; iXOffset <= i3KernelSize.x; iXOffset++, iXYOffset++)
		#endif	// MOD-BY-LEETEN 12/19/2009-END
			{
				int iX, iY, iZ;
				iX = iVoxelX + iXOffset;
				iY = iVoxelY + iYOffset;
				iZ = iVoxelZ + iZOffset;

				int3 i3TexCoord;
				i3TexCoord.x = IMirrorCoord(iX, i3VolumeSize.x);
				i3TexCoord.y = IMirrorCoord(iY, i3VolumeSize.y);
				i3TexCoord.z = IMirrorCoord(iZ, i3VolumeSize.z);

				int iSrcBin = tex2D(t2dSrcBinVolume, i3TexCoord.x, i3TexCoord.y + i3TexCoord.z * i3VolumeSize.y);

				#if	0	// MOD-BY-LEETEN 12/19/2009-FROM:
					int iModY = iVoxelY % (2 * i3KernelSize.y + 1);
					int iOffset = (0 == iModY)?(2 * i3KernelSize.y):(iModY - 1);
					iOffset = iOffset * iNrOfXZNeighbors + iXZOffset;
				#else	// MOD-BY-LEETEN 12/19/2009-TO:
				int iModZ = iVoxelZ % (2 * i3KernelSize.z + 1);
				int iOffset = (0 == iModZ)?(2 * i3KernelSize.z):(iModZ - 1);
				iOffset = iOffset * iNrOfXYNeighbors + iXYOffset;
				#endif	// MOD-BY-LEETEN 12/19/2009-END
				*ADDRESS_2D(
					unsigned int,			cActiveVoxelNeighbors_pitched.ptr, 
					sizeof(unsigned int),	cActiveVoxelNeighbors_pitched.pitch, 
					iActiveVoxelId, iOffset) = unsigned (iActiveVoxelId * iNrOfBins + iSrcBin);
			}
	}
}
// ADD-BY-LEETEN 12/18/2009-END


/*

$Log: not supported by cvs2svn $
Revision 1.1  2009/12/17 17:44:56  leeten

[12/17/2009]
1. [1ST] First time checkin.


*/
