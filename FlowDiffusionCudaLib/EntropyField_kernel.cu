#include "hip/hip_runtime.h"
#include "FlowDiffusion_cuda.h"

enum {
	UPDATE_DIR_X, 
	UPDATE_DIR_Y, 
	UPDATE_DIR_Z, 
};

enum {
	HISTOGRAM_OP_SUB_SLICE = -1, 
	HISTOGRAM_OP_ADD_SLICE = +1, 
};

						// a texture tha represents the src. bin volume
						// this textrue defines the input for the kernel
static texture<int, 2, hipReadModeElementType> t2dSrcBinVolume;
						// a texture tha represents the dst. bin volume
static texture<int, 2, hipReadModeElementType> t2dDstBinVolume;
static texture<int, 1, hipReadModeElementType> t1dHistogram;

					// a volume of integer, where each element is the joint histogram of both the src. and dst. bin volume
					// here the joint of two bins mean aan integer that represent both numbers
static hipPitchedPtr cSrcDstBinVolumePtr_global;	

// ADD-BY-LEETEN 12/18/2009-BEGIN
static texture<int, 2, hipReadModeElementType> t2dActiveVoxelHistorgrams;

// MOD-BY-LEETEN 12/18/2009-FROM
	// static texture<unsigned int, 1, hipReadModeElementType> t1dActiveVoxelSortedNeighbors;
// TO:
static texture<unsigned int, 2, hipReadModeElementType> t2dActiveVoxelSortedNeighbors;
// MOD-BY-LEETEN 12/18/2009-END
// ADD-BY-LEETEN 12/18/2009-END


// the kernel to update the histogram 
__device__ 
int IMirrorCoord(int iCoord, int iSize)
{
	int iMirroredCoord = iCoord;
	if( iCoord < 0 )
		iMirroredCoord = -iCoord;
	if( iCoord > iSize - 1 )
		iMirroredCoord = iSize - (iCoord - (iSize - 1));
	return iMirroredCoord;
}

// before calling this kernel, the input volume should be bound to the texture t2dBinVolume

// DEL-BY-LEETEN 12/23/2009-BEGIN
/*
__global__ 
void
_UpdateSliceToHistogram_kernel
(
	int3 i3Center, 
	int	iUpdateDir, 
	int iHisotgramOp, 

	int3 i3UDir, 
	int3 i3VDir,
	int iUKernelSize, 
	int iVKernelSize,

	int3 i3VolumeSize,

	int iNrOfBins,
	int *piHistorgram_global
)
{
 	int iX = blockIdx.x * blockDim.x + threadIdx.x;
 	int iY = blockIdx.y * blockDim.y + threadIdx.y;
	int3 i3Point;
	i3Point.x = i3Center.x + (iX - iUKernelSize) * i3UDir.x + (iY - iVKernelSize) * i3VDir.x;
	i3Point.y = i3Center.y + (iX - iUKernelSize) * i3UDir.y + (iY - iVKernelSize) * i3VDir.y;
	i3Point.z = i3Center.z + (iX - iUKernelSize) * i3UDir.z + (iY - iVKernelSize) * i3VDir.z;

	// read the bin
	int3 i3TexCoord;
	i3TexCoord.x = IMirrorCoord(i3Point.x, i3VolumeSize.x);
	i3TexCoord.y = IMirrorCoord(i3Point.y, i3VolumeSize.y);
	i3TexCoord.z = IMirrorCoord(i3Point.z, i3VolumeSize.z);

	if( iX < 2 * iUKernelSize + 1 && iY < 2 * iVKernelSize + 1)
	{
		int iSrcBin = tex2D(t2dSrcBinVolume, i3TexCoord.x, i3TexCoord.y + i3TexCoord.z * i3VolumeSize.y);

		// update the joint histogram
		atomicAdd(&piHistorgram_global[iSrcBin], iHisotgramOp );
	}
}

// comptue p log p for each bin
// the final entropy will be summed later via CUDPP
__global__ 
void 
_ComputeProbDotLogProb_kernel
(
	int iNrOfElements,

	int iNrOfBins,
	float *pfLogHistogram_global
)
{
	int iBin = blockIdx.x * blockDim.x + threadIdx.x;
	int iCount = tex1D(t1dHistogram, iBin);
	float fProbDotLogProb = 0.0f;
	if( iCount )
	{
		float fCount = float(iCount);
		fProbDotLogProb = log2(fCount) * fCount;
	}
	float fNrOfElements = float(iNrOfElements);
	fProbDotLogProb = log2(fNrOfElements)/float(iNrOfBins) - fProbDotLogProb / fNrOfElements;

	if( iBin < iNrOfBins )
	{
		pfLogHistogram_global[iBin] = fProbDotLogProb;
	}
}
*/
// DEL-BY-LEETEN 12/23/2009-END


__global__
void 
_JointSrcDst_kernel
(
	int iNrOfSrcBins, 
	int iNrOfDstbins,

	int3 i3VolumeSize,
	hipPitchedPtr cJointBinVolume_pitched
)
{
	int iVoxelX = blockIdx.x * blockDim.x + threadIdx.x;
	int iVoxelYZ = blockIdx.y * blockDim.y + threadIdx.y;
	int iVoxelY = (iVoxelYZ % i3VolumeSize.y);
	int iVoxelZ = (iVoxelYZ / i3VolumeSize.y);

	int iSrcBin = tex2D(t2dSrcBinVolume, iVoxelX,	iVoxelY + iVoxelZ * i3VolumeSize.y);
	int iDstBin = tex2D(t2dDstBinVolume, iVoxelX,	iVoxelY + iVoxelZ * i3VolumeSize.y);
	int iJointBin = iSrcBin + iDstBin * iNrOfSrcBins;

	if( iVoxelX < i3VolumeSize.x && iVoxelY < i3VolumeSize.y && iVoxelZ < i3VolumeSize.z )  
		*ADDRESS_2D(
			int, cJointBinVolume_pitched.ptr, 
			sizeof(int), cJointBinVolume_pitched.pitch, 
			iVoxelX, iVoxelY + iVoxelZ * i3VolumeSize.y) = iJointBin;
}

__global__ 
static 
void 
// MOD-BY-LEETEN 12/18/2009-FROM:
	// _Vector3DToVolume_kernel
// TO:
_Vector3DToBinVolume_kernel
// MOD-BY-LEETEN 12/18/2009-END
(
	int iVolumeWidth,
	int iVolumeHeight,
	int iVolumeDepth,

	int iNrOfYBlocks,
	int iBlockZSize,

	// texture<float4, 2, hipReadModeElementType> t2dSrc,
	hipPitchedPtr cBinVolumePtr_global
)
{
 	int iVoxelX = blockIdx.x * blockDim.x + threadIdx.x;
	int iVoxelY = (blockIdx.y % iNrOfYBlocks) * blockDim.y + threadIdx.y;
	int iBeginZ = (blockIdx.y / iNrOfYBlocks) * iBlockZSize;
	int iEndZ = min(iBeginZ + iBlockZSize, iVolumeDepth);

	for(int z = iBeginZ; z < iEndZ; z++)
	{
		float4 f4Vector = tex2D(t2dVectorVolume, iVoxelX,		iVoxelY + z				* iVolumeHeight);;
		
		float fLength	= sqrt(f4Vector.x * f4Vector.x + f4Vector.y * f4Vector.y + f4Vector.z * f4Vector.z);
		float fTheta	= 0.0f;
		float fPhi		= 0.0f;
		int iBin = 0;

		if( 0 < fLength )
		{
			f4Vector.x /= fLength;
			f4Vector.y /= fLength;
			f4Vector.z /= fLength;
			fTheta = ( 0.0f == f4Vector.x && 0.0f == f4Vector.y )?0.0f:(float(M_PI)+(atan2(f4Vector.y, f4Vector.x)));
			fTheta /= 2.0f * float(M_PI);

			float fLength2D = sqrt(f4Vector.x * f4Vector.x + f4Vector.y * f4Vector.y);
			fPhi = ((0.0f == fLength2D)&&(0.0f == f4Vector.z))?0.0f:fabs(float(M_PI)/2.0f-(atan2(f4Vector.z, fLength2D)));
			fPhi /= float(M_PI);
			iBin = tex2D(t2dAngleMap, fPhi, fTheta);
		}

		if( iVoxelX < iVolumeWidth && iVoxelY < iVolumeHeight )
		{
			*ADDRESS_2D(
				int,			cBinVolumePtr_global.ptr, 
				sizeof(int),	cBinVolumePtr_global.pitch, 
				iVoxelX, iVoxelY + z * iVolumeHeight) = iBin;
		}
	}
}


/*

$Log: not supported by cvs2svn $
Revision 1.2  2009/12/20 03:21:45  leeten

[12/19/2009]
1. [MOD] Change the 1D texture (t1dActiveVoxelHistorgrams) to a 2D texture (t2dActiveVoxelHistorgrams).
2. [MOD] Change the function name _Vector3DToVolume_kernel to _Vector3DToBinVolume_kernel.
3. [ADD] Define the kernel and device functions to compute the histogram on GPUs.
4. [ADD] Define the kernel and device functions to compute the entropy of each voxel by scanning the sorted bins of its neighboring regions.

Revision 1.1  2009/12/17 17:44:56  leeten

[12/17/2009]
1. [1ST] First time checkin.


*/
