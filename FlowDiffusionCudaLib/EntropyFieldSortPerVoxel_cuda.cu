
#include <hip/hip_runtime.h>
#if	COMPUTE_ENTROPY_VOLUME_SORT_PER_VOXEL_CUDA

__global__ 
void 
_CollectNeighborsHorizontally_kernel
(
	int3 i3BlockCorner,
	int3 i3KernelSize,
	int3 i3VolumeSize,
	int3 i3Steps,
	cudaPitchedPtr cActiveVoxelNeighbors_pitched
)
{
	int iActiveVoxelX = blockIdx.x;
	int iActiveVoxelY = blockIdx.y / i3Steps.y;
	int iActiveVoxelZ = blockIdx.y % i3Steps.y;
	int iActiveVoxelId = blockIdx.x + blockIdx.y * gridDim.x;
 	int iVoxelX = i3BlockCorner.x + iActiveVoxelX;
	int iVoxelY = i3BlockCorner.y + iActiveVoxelY;
	int iVoxelZ = i3BlockCorner.z + iActiveVoxelZ;
	int iNrOfNeighbors = (i3KernelSize.x * 2 + 1) * (i3KernelSize.y * 2 + 1) * (i3KernelSize.z * 2 + 1);

	for(int n = threadIdx.x; n < iNrOfNeighbors; n += blockDim.x)
	{
		int iXOffset	= n % (i3KernelSize.x * 2 + 1);
		int iXYOffset	= n / (i3KernelSize.x * 2 + 1); 
		int iYOffset	= iXYOffset % (i3KernelSize.y * 2 + 1);
		int iZOffset	= iXYOffset / (i3KernelSize.y * 2 + 1);
		iXOffset -= i3KernelSize.x;
		iYOffset -= i3KernelSize.y;
		iZOffset -= i3KernelSize.z;

		int iX = iVoxelX + iXOffset;
		int iY = iVoxelY + iYOffset;
		int iZ = iVoxelZ + iZOffset;

		int3 i3TexCoord;
		i3TexCoord.x = IMirrorCoord(iX, i3VolumeSize.x);
		i3TexCoord.y = IMirrorCoord(iY, i3VolumeSize.y);
		i3TexCoord.z = IMirrorCoord(iZ, i3VolumeSize.z);

		int iSrcBin = tex2D(t2dSrcBinVolume, i3TexCoord.x, i3TexCoord.y + i3TexCoord.z * i3VolumeSize.y);

			*ADDRESS_2D(
				unsigned int,			cActiveVoxelNeighbors_pitched.ptr, 
				sizeof(unsigned int),	cActiveVoxelNeighbors_pitched.pitch, 
				n, 
				iActiveVoxelId) 
					= unsigned int(iSrcBin);
		}
}

void 
_ComputeEntropyVolumePerVoxel_cuda
(
						// res. of the neighboring region
	int3 i3KernelSize,

						// the histogram
	int iNrOfBins,
	int *piHistogram_global,
	float *pfLogHistogram_global,

						// res. of the volume																
	int3 i3VolumeSize,	
						// bin volume																	
	cudaPitchedPtr		cBinVolume_pitched,								
	cudaPitchedPtr		cEntropyVolume_pitched
)
{
CLOCK_INIT(_ComputeEntropyVolume_PRINT_TIMING, __FUNCTION__ ": ");

CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_TIMING);
	const int iNrOfNeighbors = (2 * i3KernelSize.x + 1) * (2 * i3KernelSize.y + 1) * (2 * i3KernelSize.z + 1);

	int3 i3Step;

	int iRequiredMemoryInBytesPerVoxel = iNrOfNeighbors * sizeof(int);
	int iMaxVoxelsInMemory = MAX_MEMORY_SIZE / iRequiredMemoryInBytesPerVoxel;
	i3Step.x = min(i3VolumeSize.x, iMaxVoxelsInMemory);
	i3Step.y = min(i3VolumeSize.y, max(iMaxVoxelsInMemory / i3Step.x, 1));
	i3Step.z = min(i3VolumeSize.z, max(iMaxVoxelsInMemory / (i3Step.x * i3Step.y), 1));

	dim3 v3Blk = dim3(MAX_THREADS, 1);
	dim3 v3Grid = dim3(i3Step.x, i3Step.y * i3Step.z);

	fprintf(stderr, "#BLOCKS = %d x %d\n", v3Grid.x, v3Grid.y);

	cudaPitchedPtr cActiveVoxelNeighbors_pitched;
	cActiveVoxelNeighbors_pitched.xsize = iNrOfNeighbors;
	cActiveVoxelNeighbors_pitched.ysize = i3Step.x * i3Step.y * i3Step.z;
	CUDA_SAFE_CALL_NO_SYNC(
		cudaMallocPitch(
			(void**)&cActiveVoxelNeighbors_pitched.ptr, 
			&cActiveVoxelNeighbors_pitched.pitch,
			cActiveVoxelNeighbors_pitched.xsize * sizeof(int), 
			cActiveVoxelNeighbors_pitched.ysize)	);

	// bind the input vin volume to the texture that represents the src. bin volume 
	t2dSrcBinVolume.addressMode[0] = cudaAddressModeClamp;
	t2dSrcBinVolume.addressMode[1] = cudaAddressModeClamp;
	t2dSrcBinVolume.filterMode =	cudaFilterModePoint;
	t2dSrcBinVolume.normalized =	false;
	CUDA_SAFE_CALL_NO_SYNC(
		cudaBindTexture2D(
			0, 
			t2dSrcBinVolume, 
			cBinVolume_pitched.ptr, 
			cudaCreateChannelDesc<int>(),
			cVolumeExtent_array.width, 
			cVolumeExtent_array.height * cVolumeExtent_array.depth, 
			cBinVolume_pitched.pitch) );

	// bind the histogram as a texture
	t2dActiveVoxelSortedNeighbors.addressMode[0] = cudaAddressModeClamp;
	t2dActiveVoxelSortedNeighbors.addressMode[1] = cudaAddressModeClamp;
	t2dActiveVoxelSortedNeighbors.filterMode =	cudaFilterModePoint;
	t2dActiveVoxelSortedNeighbors.normalized =	false;

	CUDA_SAFE_CALL_NO_SYNC(
		cudaBindTexture2D(
			0, 
			t2dActiveVoxelSortedNeighbors, 
			cActiveVoxelNeighbors_pitched.ptr, 
			cudaCreateChannelDesc<unsigned int>(),
			cActiveVoxelNeighbors_pitched.xsize,
			cActiveVoxelNeighbors_pitched.ysize, 
			cActiveVoxelNeighbors_pitched.pitch) );

	CUDPPHandle hScanPlan = 0;

	CUDPPConfiguration	cConfig;
	cConfig.op =		CUDPP_ADD;
	cConfig.datatype =	CUDPP_UINT;
	cConfig.algorithm =	CUDPP_SORT_RADIX;
	cConfig.options =	CUDPP_OPTION_KEYS_ONLY;

	assert( 
		CUDPP_SUCCESS  == cudppPlan(
			&hScanPlan,	
			cConfig, 
			cActiveVoxelNeighbors_pitched.xsize * cActiveVoxelNeighbors_pitched.ysize,
			1, 
			0) );

CLOCK_END(_ComputeEntropyVolume_PRINT_TIMING, false);

CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_TIMING);

	for(int			z = 0; z < i3VolumeSize.z; z += i3Step.z)
		for(int		y = 0; y < i3VolumeSize.y; y += i3Step.y)
			for(int	x = 0; x < i3VolumeSize.x; x += i3Step.x)
			{
				CLOCK_INIT(_ComputeEntropyVolume_PRINT_LOOP_TIMING, __FUNCTION__ " (main loop): ");

				CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_LOOP_TIMING);

				int3 i3BlockCorner = make_int3(x, y, z);
				_CollectNeighborsHorizontally_kernel<<<v3Grid, v3Blk, 0>>>
				(
					i3BlockCorner,
					i3KernelSize,
					i3VolumeSize,
					i3Step,
					cActiveVoxelNeighbors_pitched
				);
				CUT_CHECK_ERROR("_CollectNeighborsHorizontally_kernel() failed");

CLOCK_END(_ComputeEntropyVolume_PRINT_LOOP_TIMING, false);

CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_LOOP_TIMING);

/*
				for(int n = 0; n < v3Grid.x * v3Grid.y / 4096; n++)
					cudppSort(
						hScanPlan,
						(unsigned int*)cActiveVoxelNeighbors_pitched.ptr + 4096 * cActiveVoxelNeighbors_pitched.xsize * n,
						NULL,
						12,	// RADIX_SORT_BITS,
						4096 * cActiveVoxelNeighbors_pitched.xsize) ;
*/
					cudppSort(
						hScanPlan,
						cActiveVoxelNeighbors_pitched.ptr,
						NULL,
						18,	// RADIX_SORT_BITS,
						cActiveVoxelNeighbors_pitched.xsize * cActiveVoxelNeighbors_pitched.ysize) ;
				CUT_CHECK_ERROR("cudppSort() failed");
CLOCK_END(_ComputeEntropyVolume_PRINT_LOOP_TIMING, false);

CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_LOOP_TIMING);

CLOCK_END(_ComputeEntropyVolume_PRINT_LOOP_TIMING, false);

#if	0
CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_LOOP_TIMING);
				_ComputeEntropyOnSortedNeighbors_kernel<<<v3Grid, v3Blk, 0>>>
				(
					i3BlockCorner,
					iNrOfBins,
					i3KernelSize,
					(unsigned int *)cActiveVoxelNeighbors_pitched.ptr,
					i3VolumeSize,
					cEntropyVolume_pitched
				);
				CUT_CHECK_ERROR("_ComputeEntropyOnSortedNeighbors_kernel() failed");
CLOCK_END(_ComputeEntropyVolume_PRINT_LOOP_TIMING, false);
#endif

CLOCK_PRINT(_ComputeEntropyVolume_PRINT_LOOP_TIMING);
			}


CLOCK_END(_ComputeEntropyVolume_PRINT_TIMING, false);

CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_TIMING);

	cudppDestroyPlan(hScanPlan);
	FREE_MEMORY(cActiveVoxelNeighbors_pitched.ptr);

CLOCK_END(_ComputeEntropyVolume_PRINT_TIMING, false);

CLOCK_PRINT(_ComputeEntropyVolume_PRINT_TIMING);
}

#endif	// #if	COMPUTE_ENTROPY_VOLUME_SORT_PER_VOXEL_CUDA


/*

$Log: not supported by cvs2svn $

*/
