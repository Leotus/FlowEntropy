
#include <hip/hip_runtime.h>
#if	COMPUTE_ENTROPY_VOLUME_ON_SPARSE_HISTOGRAM_CUDA

static texture<int, 2, cudaReadModeElementType> t2dActiveVoxelSparseHistorgrams;

__global__ 
static 
void 
_CreateSparseHistogram_kernel
(
	int3 i3BlockCorner,

	int3 i3KernelSize,

	int3 i3VolumeSize,

	int iNrOfBins,
	cudaPitchedPtr cActiveVoxelsHistorgram_pitched
)
{
	int iActiveVoxelX = blockIdx.x * blockDim.x + threadIdx.x;
	int iActiveVoxelY = blockIdx.y * blockDim.y + threadIdx.y;
	int iActiveVoxelId = iActiveVoxelX + iActiveVoxelY * gridDim.x * blockDim.x;
 	int iVoxelX = i3BlockCorner.x + iActiveVoxelX;
	int iVoxelY = i3BlockCorner.y + iActiveVoxelY;
	int iVoxelZ = i3BlockCorner.z;
//	int iNrOfSparseMatrixEntries = (2 * i3KernelSize.x + 1) * (2 * i3KernelSize.y + 1) * (2 * i3KernelSize.z + 1);

	if( iVoxelX < i3VolumeSize.x && iVoxelY < i3VolumeSize.y && iVoxelZ < i3VolumeSize.z  )
	{
		int iSparseMatrixOffset_words = cActiveVoxelsHistorgram_pitched.pitch / sizeof(int);
		int *piBase_global = ADDRESS_2D(
			int,			cActiveVoxelsHistorgram_pitched.ptr, 
			sizeof(int),	cActiveVoxelsHistorgram_pitched.pitch, 
			iActiveVoxelId, 0);
		int iNrOfEntries = 0;

		for(int				iZOffset = -i3KernelSize.z; iZOffset <= i3KernelSize.z; iZOffset++)
			for(int			iYOffset = -i3KernelSize.y; iYOffset <= i3KernelSize.y; iYOffset++)
				for(int		iXOffset = -i3KernelSize.x; iXOffset <= i3KernelSize.x; iXOffset++)
				{
					int iX, iY, iZ;
					iX = iVoxelX + iXOffset;
					iY = iVoxelY + iYOffset;
					iZ = iVoxelZ + iZOffset;

					int3 i3TexCoord;
					i3TexCoord.x = IMirrorCoord(iX, i3VolumeSize.x);
					i3TexCoord.y = IMirrorCoord(iY, i3VolumeSize.y);
					i3TexCoord.z = IMirrorCoord(iZ, i3VolumeSize.z);

					int iSrcBin = tex2D(t2dSrcBinVolume, i3TexCoord.x, i3TexCoord.y + i3TexCoord.z * i3VolumeSize.y);

					int b;
					for(b = 0; b < iNrOfEntries; b++)
					{
						int *piEntry_global = 
							&piBase_global[(1 + 2 * b) * iSparseMatrixOffset_words];

						if( piEntry_global[0] == iSrcBin )
						{
							atomicAdd(
								&piEntry_global[iSparseMatrixOffset_words],
								1);
							break;
						}
					}
					if( b  == iNrOfEntries )
					{
						piBase_global[
							(1 + 2 * b) * iSparseMatrixOffset_words] 
							= iSrcBin;

						piBase_global[
							(1 + 2 * b + 1) * iSparseMatrixOffset_words] 
							= 1;

						iNrOfEntries++;
					}
				}
		piBase_global[0] = iNrOfEntries;
	}
}

__global__ 
void 
_ComputeEntropyOnSparseMatrix_kernel
(
	int3 i3BlockCorner,
	int iNrOfBins,
	int3 i3KernelSize,
	int3 i3VolumeSize,
	cudaPitchedPtr cEntropyVolume_pitched
)
{
	int iActiveVoxelX = blockIdx.x * blockDim.x + threadIdx.x;
	int iActiveVoxelY = blockIdx.y * blockDim.y + threadIdx.y;
	int iActiveVoxelId = iActiveVoxelX + iActiveVoxelY * gridDim.x * blockDim.x;
 	int iVoxelX = i3BlockCorner.x + iActiveVoxelX;
	int iVoxelY = i3BlockCorner.y + iActiveVoxelY;
	int iVoxelZ = i3BlockCorner.z;

	int iNrOfNeighbors = (2 * i3KernelSize.x + 1) * (2 * i3KernelSize.y + 1) * (2 * i3KernelSize.z + 1);
	float fNrOfNeighbors = float(iNrOfNeighbors);

	float fEntropy = 0.0f;
	int iNrOfEntries = tex2D(t2dActiveVoxelSparseHistorgrams, iActiveVoxelId, 0);
	for(int e = 0; e < iNrOfEntries; e++)
	{
		float fCount = float(tex2D(t2dActiveVoxelSparseHistorgrams, iActiveVoxelId, 1 + 2 * e + 1));
		fEntropy += fCount * log2(fCount);
	}
	fEntropy = -fEntropy / fNrOfNeighbors + log2(fNrOfNeighbors);
	fEntropy = max(0.0, fEntropy);

	*ADDRESS_2D(
		float, cEntropyVolume_pitched.ptr, 
		sizeof(float), cEntropyVolume_pitched.pitch, 
		iVoxelX, iVoxelY + iVoxelZ * i3VolumeSize.y) = fEntropy;
}

//////////////////////////////////////////////////////////////////////////////
void 
_ComputeEntropyVolumeOnSparseHistogram_cuda
(
						// res. of the neighboring region
	int3 i3KernelSize,

						// the histogram
	int iNrOfBins,
	int *piHistogram_global,
	float *pfLogHistogram_global,

						// res. of the volume																
	int3 i3VolumeSize,	
						// bin volume																	
	cudaPitchedPtr		cBinVolume_pitched,								
	cudaPitchedPtr		cEntropyVolume_pitched
)
{
CLOCK_INIT(_ComputeEntropyVolume_PRINT_TIMING, __FUNCTION__ ": ");

CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_TIMING);
	dim3 v3Blk = dim3(BLOCK_DIM_X, BLOCK_DIM_Y);
	const unsigned int iMaxMemorySpace = MAX_MEMORY_SIZE;
	int iNrOfNeighbors = (2 * i3KernelSize.x + 1) * (2 * i3KernelSize.y + 1) * (2 * i3KernelSize.z + 1);
	int iSparseMatrixMemorySize_words = 1 + 2 * iNrOfNeighbors; // 1 as the counter; each entry incldues 2 values
	int iSparseMatrixMemorySize_bytes = iSparseMatrixMemorySize_words * sizeof(int);
	int iMemorySpacePerBlock_bytes = iSparseMatrixMemorySize_bytes * v3Blk.x * v3Blk.y;
	int iMaxNrOfBlocks = int(floorf(float(iMaxMemorySpace) / float(iMemorySpacePerBlock_bytes)));	// each elements take 2 values (entry, pair)
/*
	int iNrOfXBlocks = int(ceilf(float(i3VolumeSize.x) / float(v3Blk.x)));
	int iNrOfYBlocks = int(ceilf(float(i3VolumeSize.y) / float(v3Blk.y)));
	iMaxNrOfBlocks = min(iMaxNrOfBlocks, iNrOfXBlocks * iNrOfYBlocks);
	iNrOfXBlocks = min(iNrOfXBlocks, iMaxNrOfBlocks);
	iNrOfYBlocks = int(ceilf(float(iMaxNrOfBlocks) / float(iNrOfXBlocks)));
*/
	int iNrOfXBlocks = 
		min(
			iMaxNrOfBlocks, 
			int(ceilf(float(i3VolumeSize.x) / float(v3Blk.x))));
	int iNrOfYBlocks = 
		min(
			max(1, iMaxNrOfBlocks / iNrOfXBlocks), 
			int(ceilf(float(i3VolumeSize.y) / float(v3Blk.y))));
	dim3 v3Grid = dim3(iNrOfXBlocks, iNrOfYBlocks);

	fprintf(stderr, "MEM = %d MB; #BLOCKS = %d x %d\n", iMaxMemorySpace/(1<<20), v3Grid.x, v3Grid.y);

	cudaPitchedPtr cActiveVoxelHistorgrams_pitched;
	cActiveVoxelHistorgrams_pitched.xsize = v3Grid.x * v3Grid.y * v3Blk.x * v3Blk.y;
	cActiveVoxelHistorgrams_pitched.ysize = iSparseMatrixMemorySize_words;
	CUDA_SAFE_CALL_NO_SYNC(
		cudaMallocPitch(
			(void**)&cActiveVoxelHistorgrams_pitched.ptr, 
			&cActiveVoxelHistorgrams_pitched.pitch,
			cActiveVoxelHistorgrams_pitched.xsize * sizeof(int), 
			cActiveVoxelHistorgrams_pitched.ysize)	);

	// bind the input vin volume to the texture that represents the src. bin volume 
	t2dSrcBinVolume.addressMode[0] = cudaAddressModeClamp;
	t2dSrcBinVolume.addressMode[1] = cudaAddressModeClamp;
	t2dSrcBinVolume.filterMode =	cudaFilterModePoint;
	t2dSrcBinVolume.normalized =	false;
	CUDA_SAFE_CALL_NO_SYNC(
		cudaBindTexture2D(
			0, 
			t2dSrcBinVolume, 
			cBinVolume_pitched.ptr, 
			cudaCreateChannelDesc<int>(),
			cVolumeExtent_array.width, 
			cVolumeExtent_array.height * cVolumeExtent_array.depth, 
			cBinVolume_pitched.pitch) );

	// bind the histogram as a texture
	t2dActiveVoxelSparseHistorgrams.addressMode[0] = cudaAddressModeClamp;
	t2dActiveVoxelSparseHistorgrams.addressMode[1] = cudaAddressModeClamp;
	t2dActiveVoxelSparseHistorgrams.filterMode =	cudaFilterModePoint;
	t2dActiveVoxelSparseHistorgrams.normalized =	false;
	CUDA_SAFE_CALL_NO_SYNC(
		cudaBindTexture2D(
			0, 
			t2dActiveVoxelSparseHistorgrams, 
			cActiveVoxelHistorgrams_pitched.ptr, 
			cudaCreateChannelDesc<int>(),
			cActiveVoxelHistorgrams_pitched.xsize, 
			cActiveVoxelHistorgrams_pitched.ysize,
			cActiveVoxelHistorgrams_pitched.pitch) );

CLOCK_END(_ComputeEntropyVolume_PRINT_TIMING, false);

CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_TIMING);

	for(int			y = 0; y < i3VolumeSize.y; y += v3Grid.y * v3Blk.y)
		for(int		x = 0; x < i3VolumeSize.x; x += v3Grid.x * v3Blk.x)
			for(int z = 0; z < i3VolumeSize.z; z++)
			{
				CLOCK_INIT(_ComputeEntropyVolume_PRINT_LOOP_TIMING, __FUNCTION__ " (main loop): ");

				CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_LOOP_TIMING);

				int3 i3BlockCorner = make_int3(x, y, z);

				// reset the counters
				CUDA_SAFE_CALL_NO_SYNC(
					cudaMemset2D(
						cActiveVoxelHistorgrams_pitched.ptr, 
						cActiveVoxelHistorgrams_pitched.pitch,
						0, 
						cActiveVoxelHistorgrams_pitched.pitch, 
						1)	);

				CLOCK_END(_ComputeEntropyVolume_PRINT_LOOP_TIMING, false);

				CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_LOOP_TIMING);

				_CreateSparseHistogram_kernel<<<v3Grid, v3Blk, 0>>>
				(
					i3BlockCorner,
					i3KernelSize,
					i3VolumeSize,
					iNrOfBins,
					cActiveVoxelHistorgrams_pitched
				);
				CUT_CHECK_ERROR("_CreateSparseHistogram_kernel() failed");

				CLOCK_END(_ComputeEntropyVolume_PRINT_LOOP_TIMING, false);

				CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_LOOP_TIMING);

				_ComputeEntropyOnSparseMatrix_kernel<<<v3Grid, v3Blk, 0>>>
				(
					i3BlockCorner,
					iNrOfBins,
					i3KernelSize,
					i3VolumeSize,
					cEntropyVolume_pitched
				);
				CUT_CHECK_ERROR("_ComputeEntropyOnSparseMatrix_kernel() failed");

				CLOCK_END(_ComputeEntropyVolume_PRINT_LOOP_TIMING, false);

				CLOCK_PRINT(_ComputeEntropyVolume_PRINT_LOOP_TIMING);
			}

CLOCK_END(_ComputeEntropyVolume_PRINT_TIMING, false);

CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_TIMING);

	FREE_MEMORY(cActiveVoxelHistorgrams_pitched.ptr);

CLOCK_END(_ComputeEntropyVolume_PRINT_TIMING, false);

CLOCK_PRINT(_ComputeEntropyVolume_PRINT_TIMING);
}

#endif	// #if		COMPUTE_ENTROPY_VOLUME_ON_SPARSE_HISTOGRAM_CUDA

/*

$Log: not supported by cvs2svn $

*/
