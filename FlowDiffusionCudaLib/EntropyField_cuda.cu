
#include <hip/hip_runtime.h>
#if	COMPUTE_ENTROPY_VOLUME_CUDA

#define NR_OF_THREADS_PER_MARGIN	1

static texture<float, 2, cudaReadModeElementType> t2dActiveVoxelMarginalEntropy;

__device__
void
_UpdateHistogramEntry_device
(
	int iActiveVoxelId,

	int iVoxelX, 
	int iVoxelY, 
	int iVoxelZ,
	int iXOffset,
	int iYOffset,
	int iZOffset,

	int iValue,

	int iNrOfBins,

	int3 i3VolumeSize,
	cudaPitchedPtr cActiveVoxelsHistorgram_pitched
)
{
	int iX, iY, iZ;
	iX = iVoxelX + iXOffset;
	iY = iVoxelY + iYOffset;
	iZ = iVoxelZ + iZOffset;

	int3 i3TexCoord;
	i3TexCoord.x = IMirrorCoord(iX, i3VolumeSize.x);
	i3TexCoord.y = IMirrorCoord(iY, i3VolumeSize.y);
	i3TexCoord.z = IMirrorCoord(iZ, i3VolumeSize.z);

	int iSrcBin = tex2D(t2dSrcBinVolume, i3TexCoord.x, i3TexCoord.y + i3TexCoord.z * i3VolumeSize.y);

						// update the joint histogram
	if( iSrcBin  < iNrOfBins )
		atomicAdd(
			ADDRESS_2D(
				int,			cActiveVoxelsHistorgram_pitched.ptr, 
				sizeof(int),	cActiveVoxelsHistorgram_pitched.pitch, 
				iActiveVoxelId, iSrcBin),
			iValue );
}


__global__ 
static 
void 
_UpdateHistogram_kernel
(
	int3 i3BlockCorner,

	int3 i3KernelSize,

	int3 i3VolumeSize,

	int iNrOfBins,
	cudaPitchedPtr cActiveVoxelsHistorgram_pitched
)
{
	int iActiveVoxelX = blockIdx.x * blockDim.x + threadIdx.x;
	int iActiveVoxelY = blockIdx.y * blockDim.y + threadIdx.y;
	int iActiveVoxelId = iActiveVoxelX + iActiveVoxelY * gridDim.x * blockDim.x;
 	int iVoxelX = i3BlockCorner.x + iActiveVoxelX;
	int iVoxelY = i3BlockCorner.y + iActiveVoxelY;
	int iVoxelZ = i3BlockCorner.z;

	if( iVoxelX < i3VolumeSize.x && iVoxelY < i3VolumeSize.y && iVoxelZ < i3VolumeSize.z  )

		for(int iZDir = -1; iZDir <= +1; iZDir+=2)
		{
			int iZOffset = i3KernelSize.z * iZDir;
			switch (iZDir)
			{
			case -1:	iZOffset--;		break;
			}
			for(int		iYOffset = -i3KernelSize.y; iYOffset <= i3KernelSize.y; iYOffset++)
				for(int iXOffset = -i3KernelSize.x; iXOffset <= i3KernelSize.x; iXOffset++)
				{
					_UpdateHistogramEntry_device
					(
						iActiveVoxelId,	
						iVoxelX, iVoxelY, iVoxelZ,
						iXOffset,iYOffset,iZOffset,
						iZDir,
						iNrOfBins,
						i3VolumeSize, 
						cActiveVoxelsHistorgram_pitched
					);
				}
		}
}

__global__ 
static 
void 
_CreateHistogram_kernel
(
	int3 i3BlockCorner,

	int3 i3KernelSize,

	int3 i3VolumeSize,

	int iNrOfBins,
	cudaPitchedPtr cActiveVoxelsHistorgram_pitched
)
{
	int iActiveVoxelX = blockIdx.x * blockDim.x + threadIdx.x;
	int iActiveVoxelY = blockIdx.y * blockDim.y + threadIdx.y;
	int iActiveVoxelId = iActiveVoxelX + iActiveVoxelY * gridDim.x * blockDim.x;
 	int iVoxelX = i3BlockCorner.x + iActiveVoxelX;
	int iVoxelY = i3BlockCorner.y + iActiveVoxelY;
	int iVoxelZ = i3BlockCorner.z;

	if( iVoxelX < i3VolumeSize.x && iVoxelY < i3VolumeSize.y && iVoxelZ < i3VolumeSize.z  )
		for(int				iZOffset = -i3KernelSize.z; iZOffset <= i3KernelSize.z; iZOffset++)
			for(int			iYOffset = -i3KernelSize.y; iYOffset <= i3KernelSize.y; iYOffset++)
				for(int		iXOffset = -i3KernelSize.x; iXOffset <= i3KernelSize.x; iXOffset++)
				{
					_UpdateHistogramEntry_device
					(
						iActiveVoxelId,	
						iVoxelX, iVoxelY, iVoxelZ,
						iXOffset,iYOffset,iZOffset,
						+1,
						iNrOfBins,
						i3VolumeSize, 
						cActiveVoxelsHistorgram_pitched
					);
				}
}

__global__ 
void 
_ComputeEntropy_kernel
(
	int3 i3BlockCorner,
	int iNrOfBins,
	int3 i3KernelSize,
	int3 i3VolumeSize,
	cudaPitchedPtr cEntropyVolume_pitched
)
{
	int iActiveVoxelX = blockIdx.x * blockDim.x + threadIdx.x;
	int iActiveVoxelY = blockIdx.y * blockDim.y + threadIdx.y;
	int iActiveVoxelId = iActiveVoxelX + iActiveVoxelY * gridDim.x * blockDim.x;
 	int iVoxelX = i3BlockCorner.x + iActiveVoxelX;
	int iVoxelY = i3BlockCorner.y + iActiveVoxelY;
	int iVoxelZ = i3BlockCorner.z;

	int iNrOfNeighbors = (2 * i3KernelSize.x + 1) * (2 * i3KernelSize.y + 1) * (2 * i3KernelSize.z + 1);
	float fNrOfNeighbors = float(iNrOfNeighbors);

	float fEntropy = 0.0f;
	for(int b = 0; b < iNrOfBins; b++)
	{
		int iCount = tex2D(t2dActiveVoxelHistorgrams, iActiveVoxelId, b);
		if( iCount )
		{
			/*
			float fProb = float(iCount) / fNrOfNeighbors;
			fEntropy += fProb * log2(fProb);
			*/
			float fCount = float(iCount);
			fEntropy += fCount * log2(fCount);
		}
	}
	// fEntropy *= -1.0;
	fEntropy = -fEntropy / fNrOfNeighbors + log2(fNrOfNeighbors);
	fEntropy = max(0.0, fEntropy);

	*ADDRESS_2D(
		float, cEntropyVolume_pitched.ptr, 
		sizeof(float), cEntropyVolume_pitched.pitch, 
		iVoxelX, iVoxelY + iVoxelZ * i3VolumeSize.y) = fEntropy;
}

__global__ 
void 
_ComputeEntropyHierarchically_kernel
(
	int3 i3BlockCorner,
	int iNrOfBins,
	int3 i3KernelSize,
	int3 i3VolumeSize,
	int iGridDimX, 
	int iGridDimY, 
	int iLevel,
	cudaPitchedPtr cActiveVoxelsHistorgram_pitched,
	cudaPitchedPtr cEntropyVolume_pitched
)
{
	int iBlockIdxX = blockIdx.x % iGridDimX;
	int iBlockIdxY = blockIdx.x / iGridDimX;
	int iActiveVoxelX = iBlockIdxX * blockDim.x + threadIdx.x;
	int iActiveVoxelY = iBlockIdxY * blockDim.y + threadIdx.y;
	int iActiveVoxelId = iActiveVoxelX + iActiveVoxelY * iGridDimX * blockDim.x;
	int iToBeUpdatedBin = blockIdx.y * iLevel;

	int iCount =		( iToBeUpdatedBin < iNrOfBins )?tex2D(t2dActiveVoxelHistorgrams, iActiveVoxelId, iToBeUpdatedBin):0;
	int iNextCount =	( iToBeUpdatedBin + iLevel < iNrOfBins )?tex2D(t2dActiveVoxelHistorgrams, iActiveVoxelId, iToBeUpdatedBin + iLevel):0;
	int iSum = iCount + iNextCount;

	*ADDRESS_2D(
		int,			cActiveVoxelsHistorgram_pitched.ptr, 
		sizeof(int),	cActiveVoxelsHistorgram_pitched.pitch, 
		iActiveVoxelId, iToBeUpdatedBin) = iSum;
}

__global__ 
void 
_ComputeMarginalEntropy_kernel
(
	int3 i3BlockCorner,
	int iNrOfBins,
	int3 i3KernelSize,
	int3 i3VolumeSize,
	int iGridDimX, 
	int iGridDimY, 
	int iNrOfMarginalBins,
	int iNrOfThreadsPerMagin,
	cudaPitchedPtr cActiveVoxelMarginalEntropy_pitched
)
{
	int iBlockIdxX = blockIdx.x % iGridDimX;
	int iBlockIdxY = blockIdx.x / iGridDimX;
	int iActiveVoxelX = iBlockIdxX * blockDim.x + threadIdx.x;
	int iActiveVoxelY = iBlockIdxY * blockDim.y + threadIdx.y;
	int iActiveVoxelId = iActiveVoxelX + iActiveVoxelY * iGridDimX * blockDim.x;
	int iMargin =			blockIdx.y / iNrOfThreadsPerMagin;
	int iThreadIdInMargin = blockIdx.y % iNrOfThreadsPerMagin;

	int iBase = iMargin * iNrOfMarginalBins + iThreadIdInMargin * iNrOfMarginalBins / iNrOfThreadsPerMagin;
	float fEntropy = 0.0f;
	for(int b = 0; b < iNrOfMarginalBins / iNrOfThreadsPerMagin; b++)
	{
		int iCount = tex2D(t2dActiveVoxelHistorgrams, iActiveVoxelId, iBase + b);
		if( iCount )
		{
			float fCount = float(iCount);
			fEntropy += fCount * log2(fCount);
		}
	}
	*ADDRESS_2D(
		float,			cActiveVoxelMarginalEntropy_pitched.ptr, 
		sizeof(float),	cActiveVoxelMarginalEntropy_pitched.pitch, 
		iActiveVoxelId, blockIdx.y) = fEntropy;
}

__global__ 
void 
_SumMarginalEntropy_kernel
(
	int3 i3BlockCorner,
	int iNrOfBins,
	int3 i3KernelSize,
	int3 i3VolumeSize,
	int iNrOfMargins,
	int iNrOfThreadsPerMagin,
	cudaPitchedPtr cEntropyVolume_pitched
)
{
	int iActiveVoxelX = blockIdx.x * blockDim.x + threadIdx.x;
	int iActiveVoxelY = blockIdx.y * blockDim.y + threadIdx.y;
	int iActiveVoxelId = iActiveVoxelX + iActiveVoxelY * gridDim.x * blockDim.x;
 	int iVoxelX = i3BlockCorner.x + iActiveVoxelX;
	int iVoxelY = i3BlockCorner.y + iActiveVoxelY;
	int iVoxelZ = i3BlockCorner.z;

	int iNrOfNeighbors = (2 * i3KernelSize.x + 1) * (2 * i3KernelSize.y + 1) * (2 * i3KernelSize.z + 1);
	float fNrOfNeighbors = float(iNrOfNeighbors);

	float fEntropy = 0.0f;
	for(int b = 0; b < iNrOfMargins * iNrOfThreadsPerMagin; b++)
	{
		float fMarginalEntropy = tex2D(t2dActiveVoxelMarginalEntropy, iActiveVoxelId, b);
		fEntropy += fMarginalEntropy;
	}

	fEntropy = -fEntropy / fNrOfNeighbors + log2(fNrOfNeighbors);
	fEntropy = max(0.0, fEntropy);

	*ADDRESS_2D(
		float, cEntropyVolume_pitched.ptr, 
		sizeof(float), cEntropyVolume_pitched.pitch, 
		iVoxelX, iVoxelY + iVoxelZ * i3VolumeSize.y) = fEntropy;
}

//////////////////////////////////////////////////////////////////////////////
void 
_ComputeEntropyVolume_cuda
(
						// res. of the neighboring region
	int3 i3KernelSize,

						// the histogram
	int iNrOfBins,
	int *piHistogram_global,
	float *pfLogHistogram_global,

						// res. of the volume																
	int3 i3VolumeSize,	
						// bin volume																	
	cudaPitchedPtr		cBinVolume_pitched,								
	cudaPitchedPtr		cEntropyVolume_pitched
)
{
CLOCK_INIT(_ComputeEntropyVolume_PRINT_TIMING, __FUNCTION__ ": ");

CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_TIMING);
	dim3 v3Blk = dim3(BLOCK_DIM_X, BLOCK_DIM_Y);
	const unsigned int iMaxMemorySpace = MAX_MEMORY_SIZE;
	int iMaxNrOfBlocks = int(floorf(float(iMaxMemorySpace) / float(sizeof(int) * iNrOfBins * v3Blk.x * v3Blk.y)));
	int iNrOfXBlocks = int(ceilf(float(i3VolumeSize.x) / float(v3Blk.x)));
	int iNrOfYBlocks = int(ceilf(float(i3VolumeSize.y) / float(v3Blk.y)));
	iMaxNrOfBlocks = min(iMaxNrOfBlocks, iNrOfXBlocks * iNrOfYBlocks);
	iNrOfXBlocks = min(iNrOfXBlocks, iMaxNrOfBlocks);
	iNrOfYBlocks = int(ceilf(float(iMaxNrOfBlocks) / float(iNrOfXBlocks)));
	dim3 v3Grid = dim3(iNrOfXBlocks, iNrOfYBlocks);
	fprintf(stderr, "MEM = %d MB; #BLOCKS = %d x %d\n", iMaxMemorySpace/(1<<20), v3Grid.x, v3Grid.y);

	cudaPitchedPtr cActiveVoxelHistorgrams_pitched;
	cActiveVoxelHistorgrams_pitched.xsize = v3Grid.x * v3Grid.y * v3Blk.x * v3Blk.y;
	cActiveVoxelHistorgrams_pitched.ysize = iNrOfBins;
	CUDA_SAFE_CALL_NO_SYNC(
		cudaMallocPitch(
			(void**)&cActiveVoxelHistorgrams_pitched.ptr, 
			&cActiveVoxelHistorgrams_pitched.pitch,
			cActiveVoxelHistorgrams_pitched.xsize * sizeof(int), 
			cActiveVoxelHistorgrams_pitched.ysize)	);

	CUDA_SAFE_CALL_NO_SYNC(
		cudaMemset2D(
			cActiveVoxelHistorgrams_pitched.ptr, 
			cActiveVoxelHistorgrams_pitched.pitch,
			0, 
			cActiveVoxelHistorgrams_pitched.xsize * sizeof(int), 
			cActiveVoxelHistorgrams_pitched.ysize)	);

	// bind the input vin volume to the texture that represents the src. bin volume 
	t2dSrcBinVolume.addressMode[0] = cudaAddressModeClamp;
	t2dSrcBinVolume.addressMode[1] = cudaAddressModeClamp;
	t2dSrcBinVolume.filterMode =	cudaFilterModePoint;
	t2dSrcBinVolume.normalized =	false;
	CUDA_SAFE_CALL_NO_SYNC(
		cudaBindTexture2D(
			0, 
			t2dSrcBinVolume, 
			cBinVolume_pitched.ptr, 
			cudaCreateChannelDesc<int>(),
			cVolumeExtent_array.width, 
			cVolumeExtent_array.height * cVolumeExtent_array.depth, 
			cBinVolume_pitched.pitch) );

	// bind the histogram as a texture
	t2dActiveVoxelHistorgrams.addressMode[0] = cudaAddressModeClamp;
	t2dActiveVoxelHistorgrams.addressMode[1] = cudaAddressModeClamp;
	t2dActiveVoxelHistorgrams.filterMode =	cudaFilterModePoint;
	t2dActiveVoxelHistorgrams.normalized =	false;
	CUDA_SAFE_CALL_NO_SYNC(
		cudaBindTexture2D(
			0, 
			t2dActiveVoxelHistorgrams, 
			cActiveVoxelHistorgrams_pitched.ptr, 
			cudaCreateChannelDesc<int>(),
			cActiveVoxelHistorgrams_pitched.xsize, 
			cActiveVoxelHistorgrams_pitched.ysize,
			cActiveVoxelHistorgrams_pitched.pitch) );

	int iNrOfMarginalBins = int(sqrtf(float(iNrOfBins)));
	int iNrOfMargins = int(ceilf(float(iNrOfBins)/float(iNrOfMarginalBins)));

	cudaPitchedPtr cActiveVoxelMarginalEntropy_pitched;
	cActiveVoxelMarginalEntropy_pitched.xsize = v3Grid.x * v3Grid.y * v3Blk.x * v3Blk.y;
	cActiveVoxelMarginalEntropy_pitched.ysize = iNrOfMargins * NR_OF_THREADS_PER_MARGIN;
	CUDA_SAFE_CALL_NO_SYNC(
		cudaMallocPitch(
			(void**)&cActiveVoxelMarginalEntropy_pitched.ptr, 
			&cActiveVoxelMarginalEntropy_pitched.pitch,
			cActiveVoxelMarginalEntropy_pitched.xsize * sizeof(float), 
			cActiveVoxelMarginalEntropy_pitched.ysize)	);

	CUDA_SAFE_CALL_NO_SYNC(
		cudaMemset2D(
			cActiveVoxelMarginalEntropy_pitched.ptr, 
			cActiveVoxelMarginalEntropy_pitched.pitch,
			0, 
			cActiveVoxelMarginalEntropy_pitched.pitch, 
			cActiveVoxelMarginalEntropy_pitched.ysize)	);

	t2dActiveVoxelMarginalEntropy.addressMode[0] = cudaAddressModeClamp;
	t2dActiveVoxelMarginalEntropy.addressMode[1] = cudaAddressModeClamp;
	t2dActiveVoxelMarginalEntropy.filterMode =	cudaFilterModePoint;
	t2dActiveVoxelMarginalEntropy.normalized =	false;
	CUDA_SAFE_CALL_NO_SYNC(
		cudaBindTexture2D(
			0, 
			t2dActiveVoxelMarginalEntropy, 
			cActiveVoxelMarginalEntropy_pitched.ptr, 
			cudaCreateChannelDesc<float>(),
			cActiveVoxelMarginalEntropy_pitched.xsize, 
			cActiveVoxelMarginalEntropy_pitched.ysize,
			cActiveVoxelMarginalEntropy_pitched.pitch) );

CLOCK_END(_ComputeEntropyVolume_PRINT_TIMING, false);

CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_TIMING);

	for(int			y = 0; y < i3VolumeSize.y; y += v3Grid.y * v3Blk.y)
		for(int		x = 0; x < i3VolumeSize.x; x += v3Grid.x * v3Blk.x)
			for(int z = 0; z < i3VolumeSize.z; z++)
			{
				int3 i3BlockCorner = make_int3(x, y, z);

				CLOCK_INIT(_ComputeEntropyVolume_PRINT_LOOP_TIMING, __FUNCTION__ " (main loop): ");

				CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_LOOP_TIMING);

				if( 0 == z )
				{
					_CreateHistogram_kernel<<<v3Grid, v3Blk, 0>>>
					(
						i3BlockCorner,
						i3KernelSize,
						i3VolumeSize,
						iNrOfBins,
						cActiveVoxelHistorgrams_pitched
					);
					CUT_CHECK_ERROR("_CreateHistogram_kernel() failed");
				}
				else
				{
					_UpdateHistogram_kernel<<<v3Grid, v3Blk, 0>>>
					(
						i3BlockCorner,
						i3KernelSize,
						i3VolumeSize,
						iNrOfBins,
						cActiveVoxelHistorgrams_pitched
					);
					CUT_CHECK_ERROR("_UpdateHistogram_kernel() failed");
				}

				CLOCK_END(_ComputeEntropyVolume_PRINT_LOOP_TIMING, false);

				CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_LOOP_TIMING);

				#if		SCANNING_METHOD	== SCANNING_METHOD_SCAN_WHOLE_HISTOGRAM					
				_ComputeEntropy_kernel<<<v3Grid, v3Blk, 0>>>
				(
					i3BlockCorner,
					iNrOfBins,
					i3KernelSize,
					i3VolumeSize,
					cEntropyVolume_pitched
				);
				CUT_CHECK_ERROR("_ComputeEntropy_kernel() failed");
				#endif	// #if		SCANNING_METHOD	== SCANNING_METHOD_SCAN_WHOLE_HISTOGRAM

				#if		SCANNING_METHOD	== SCANNING_METHOD_HIERARCHICAL_SCAN
				int iNrOfLevels = int(floorf(log2f(float(iNrOfBins))));
				int iNrOfThreadsPerVoxel = iNrOfBins/2;
				for(int l = 1; l < iNrOfLevels; l++)
				{
					_ComputeEntropyHierarchically_kernel<<<
						dim3(v3Grid.x * v3Grid.y, iNrOfThreadsPerVoxel),
						v3Blk, 0
						>>>
					(
						i3BlockCorner,
						iNrOfBins,
						i3KernelSize,
						i3VolumeSize,
						v3Grid.x,
						v3Grid.y,
						l,
						cActiveVoxelHistorgrams_pitched,
						cEntropyVolume_pitched
					);

					iNrOfThreadsPerVoxel /= 2;
				}
				CUT_CHECK_ERROR("_ComputeEntropyHierarchically_kernel() failed");
				#endif	// #if	SCANNING_METHOD	== SCANNING_METHOD_HIERARCHICAL_SCAN

				#if	SCANNING_METHOD	== SCANNING_METHOD_SCAN_ROWS_IN_PARALLEL
				_ComputeMarginalEntropy_kernel<<<
					dim3(v3Grid.x * v3Grid.y, NR_OF_THREADS_PER_MARGIN * iNrOfMargins), v3Blk, 0
					>>>
				(
					i3BlockCorner,
					iNrOfBins,
					i3KernelSize,
					i3VolumeSize,
					v3Grid.x,
					v3Grid.y,
					iNrOfMarginalBins,
					NR_OF_THREADS_PER_MARGIN,
					cActiveVoxelMarginalEntropy_pitched
				);
				CUT_CHECK_ERROR("_ComputeMarginalEntropy_kernel() failed");

				CLOCK_END(_ComputeEntropyVolume_PRINT_LOOP_TIMING, false);

				CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_LOOP_TIMING);

				_SumMarginalEntropy_kernel<<<v3Grid, v3Blk, 0>>>
				(
					i3BlockCorner,
					iNrOfBins,
					i3KernelSize,
					i3VolumeSize,
					iNrOfMargins,
					NR_OF_THREADS_PER_MARGIN,
					cEntropyVolume_pitched
				);
				CUT_CHECK_ERROR("_SumMarginalEntropy_kernel() failed");
				#endif	// #if	SCANNING_METHOD	== SCANNING_METHOD_SCAN_ROWS_IN_PARALLEL

				CLOCK_END(_ComputeEntropyVolume_PRINT_LOOP_TIMING, false);

				CLOCK_PRINT(_ComputeEntropyVolume_PRINT_LOOP_TIMING);
			}

CLOCK_END(_ComputeEntropyVolume_PRINT_TIMING, false);

CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_TIMING);

	FREE_MEMORY(cActiveVoxelHistorgrams_pitched.ptr);

	FREE_MEMORY(cActiveVoxelMarginalEntropy_pitched.ptr);

CLOCK_END(_ComputeEntropyVolume_PRINT_TIMING, false);

CLOCK_PRINT(_ComputeEntropyVolume_PRINT_TIMING);
}

#endif	// #if	COMPUTE_ENTROPY_VOLUME_CUDA

/*

$Log: not supported by cvs2svn $

*/
