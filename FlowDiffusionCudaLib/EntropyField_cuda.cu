
#include <hip/hip_runtime.h>
#if	COMPUTE_ENTROPY_VOLUME_CUDA

// ADD-BY-LEETEN 12/23/2009-BEGIN
#define NR_OF_THREADS_PER_MARGIN	1

static texture<float, 2, cudaReadModeElementType> t2dActiveVoxelMarginalEntropy;
// ADD-BY-LEETEN 12/23/2009-END

__device__
void
_UpdateHistogramEntry_device
(
	int iActiveVoxelId,

	int iVoxelX, 
	int iVoxelY, 
	int iVoxelZ,
	int iXOffset,
	int iYOffset,
	int iZOffset,

	int iValue,

	int iNrOfBins,

	int3 i3VolumeSize,
	cudaPitchedPtr cActiveVoxelsHistorgram_pitched
)
{
	int iX, iY, iZ;
	iX = iVoxelX + iXOffset;
	iY = iVoxelY + iYOffset;
	iZ = iVoxelZ + iZOffset;

	int3 i3TexCoord;
	i3TexCoord.x = IMirrorCoord(iX, i3VolumeSize.x);
	i3TexCoord.y = IMirrorCoord(iY, i3VolumeSize.y);
	i3TexCoord.z = IMirrorCoord(iZ, i3VolumeSize.z);

	int iSrcBin = tex2D(t2dSrcBinVolume, i3TexCoord.x, i3TexCoord.y + i3TexCoord.z * i3VolumeSize.y);

						// update the joint histogram
	if( iSrcBin  < iNrOfBins )
		atomicAdd(
			ADDRESS_2D(
				int,			cActiveVoxelsHistorgram_pitched.ptr, 
				sizeof(int),	cActiveVoxelsHistorgram_pitched.pitch, 
				iActiveVoxelId, iSrcBin),
			iValue );
}


__global__ 
static 
void 
_UpdateHistogram_kernel
(
	int3 i3BlockCorner,

	int3 i3KernelSize,

	int3 i3VolumeSize,

	int iNrOfBins,
	cudaPitchedPtr cActiveVoxelsHistorgram_pitched
)
{
	int iActiveVoxelX = blockIdx.x * blockDim.x + threadIdx.x;
	#if	0		// MOD-BY-LEETEN 12/19/2009-FROM:
		int iActiveVoxelZ = blockIdx.y * blockDim.y + threadIdx.y;
		int iActiveVoxelId = iActiveVoxelX + iActiveVoxelZ * gridDim.x * blockDim.x;
 		int iVoxelX = i3BlockCorner.x + iActiveVoxelX;
		int iVoxelY = i3BlockCorner.y;
		int iVoxelZ = i3BlockCorner.z + iActiveVoxelZ;
	#else		// MOD-BY-LEETEN 12/19/2009-TO:

	int iActiveVoxelY = blockIdx.y * blockDim.y + threadIdx.y;
	int iActiveVoxelId = iActiveVoxelX + iActiveVoxelY * gridDim.x * blockDim.x;
 	int iVoxelX = i3BlockCorner.x + iActiveVoxelX;
	int iVoxelY = i3BlockCorner.y + iActiveVoxelY;
	int iVoxelZ = i3BlockCorner.z;

	#endif		// MOD-BY-LEETEN 12/19/2009-END


	if( iVoxelX < i3VolumeSize.x && iVoxelY < i3VolumeSize.y && iVoxelZ < i3VolumeSize.z  )

		#if	0		// MOD-BY-LEETEN 12/19/2009-FROM:

			for(int iYDir = -1; iYDir <= +1; iYDir+=2)
			{
				int iYOffset = i3KernelSize.y * iYDir;
				switch (iYDir)
				{
				case -1:	iYOffset--;		break;
				}
				for(int		iZOffset = -i3KernelSize.z; iZOffset <= i3KernelSize.z; iZOffset++)
					for(int iXOffset = -i3KernelSize.x; iXOffset <= i3KernelSize.x; iXOffset++)
					{
						_UpdateHistogramEntry_device
						(
							iActiveVoxelId,	
							iVoxelX, iVoxelY, iVoxelZ,
							iXOffset,iYOffset,iZOffset,
							iYDir,
							iNrOfBins,
							i3VolumeSize, 
							cActiveVoxelsHistorgram_pitched
						);
					}
			}

		#else

		for(int iZDir = -1; iZDir <= +1; iZDir+=2)
		{
			int iZOffset = i3KernelSize.z * iZDir;
			switch (iZDir)
			{
			case -1:	iZOffset--;		break;
			}
			for(int		iYOffset = -i3KernelSize.y; iYOffset <= i3KernelSize.y; iYOffset++)
				for(int iXOffset = -i3KernelSize.x; iXOffset <= i3KernelSize.x; iXOffset++)
				{
					_UpdateHistogramEntry_device
					(
						iActiveVoxelId,	
						iVoxelX, iVoxelY, iVoxelZ,
						iXOffset,iYOffset,iZOffset,
						iZDir,
						iNrOfBins,
						i3VolumeSize, 
						cActiveVoxelsHistorgram_pitched
					);
				}
		}
		#endif
}

__global__ 
static 
void 
_CreateHistogram_kernel
(
	int3 i3BlockCorner,

	int3 i3KernelSize,

	int3 i3VolumeSize,

	int iNrOfBins,
	cudaPitchedPtr cActiveVoxelsHistorgram_pitched
)
{
	int iActiveVoxelX = blockIdx.x * blockDim.x + threadIdx.x;
	#if	0		// MOD-BY-LEETEN 12/19/2009-FROM:
		int iActiveVoxelZ = blockIdx.y * blockDim.y + threadIdx.y;
		int iActiveVoxelId = iActiveVoxelX + iActiveVoxelZ * gridDim.x * blockDim.x;
 		int iVoxelX = i3BlockCorner.x + iActiveVoxelX;
		int iVoxelY = i3BlockCorner.y;
		int iVoxelZ = i3BlockCorner.z + iActiveVoxelZ;
	#else		// MOD-BY-LEETEN 12/19/2009-TO:

	int iActiveVoxelY = blockIdx.y * blockDim.y + threadIdx.y;
	int iActiveVoxelId = iActiveVoxelX + iActiveVoxelY * gridDim.x * blockDim.x;
 	int iVoxelX = i3BlockCorner.x + iActiveVoxelX;
	int iVoxelY = i3BlockCorner.y + iActiveVoxelY;
	int iVoxelZ = i3BlockCorner.z;

	#endif		// MOD-BY-LEETEN 12/19/2009-END

	if( iVoxelX < i3VolumeSize.x && iVoxelY < i3VolumeSize.y && iVoxelZ < i3VolumeSize.z  )
		for(int				iZOffset = -i3KernelSize.z; iZOffset <= i3KernelSize.z; iZOffset++)
			for(int			iYOffset = -i3KernelSize.y; iYOffset <= i3KernelSize.y; iYOffset++)
				for(int		iXOffset = -i3KernelSize.x; iXOffset <= i3KernelSize.x; iXOffset++)
				{
					_UpdateHistogramEntry_device
					(
						iActiveVoxelId,	
						iVoxelX, iVoxelY, iVoxelZ,
						iXOffset,iYOffset,iZOffset,
						+1,
						iNrOfBins,
						i3VolumeSize, 
						cActiveVoxelsHistorgram_pitched
					);
				}
}

__global__ 
void 
_ComputeEntropy_kernel
(
	int3 i3BlockCorner,
	int iNrOfBins,
	int3 i3KernelSize,
	int3 i3VolumeSize,
	cudaPitchedPtr cEntropyVolume_pitched
)
{
	int iActiveVoxelX = blockIdx.x * blockDim.x + threadIdx.x;
	#if	0	// MOD-BY-LEETEN 12/19/2009-FROM:
		int iActiveVoxelZ = blockIdx.y * blockDim.y + threadIdx.y;
		int iActiveVoxelId = iActiveVoxelX + iActiveVoxelZ * gridDim.x * blockDim.x;
 		int iVoxelX = i3BlockCorner.x + iActiveVoxelX;
		int iVoxelY = i3BlockCorner.y;
		int iVoxelZ = i3BlockCorner.z + iActiveVoxelZ;
	#else	// MOD-BY-LEETEN 12/19/2009-TO:

	int iActiveVoxelY = blockIdx.y * blockDim.y + threadIdx.y;
	int iActiveVoxelId = iActiveVoxelX + iActiveVoxelY * gridDim.x * blockDim.x;
 	int iVoxelX = i3BlockCorner.x + iActiveVoxelX;
	int iVoxelY = i3BlockCorner.y + iActiveVoxelY;
	int iVoxelZ = i3BlockCorner.z;

	#endif	// MOD-BY-LEETEN 12/19/2009-END

	int iNrOfNeighbors = (2 * i3KernelSize.x + 1) * (2 * i3KernelSize.y + 1) * (2 * i3KernelSize.z + 1);
	float fNrOfNeighbors = float(iNrOfNeighbors);

	float fEntropy = 0.0f;
	for(int b = 0; b < iNrOfBins; b++)
	{
		int iCount = tex2D(t2dActiveVoxelHistorgrams, iActiveVoxelId, b);
		if( iCount )
		{
			/*
			float fProb = float(iCount) / fNrOfNeighbors;
			fEntropy += fProb * log2(fProb);
			*/
			float fCount = float(iCount);
			fEntropy += fCount * log2(fCount);
		}
	}
	// fEntropy *= -1.0;
	fEntropy = -fEntropy / fNrOfNeighbors + log2(fNrOfNeighbors);
	fEntropy = max(0.0, fEntropy);

	*ADDRESS_2D(
		float, cEntropyVolume_pitched.ptr, 
		sizeof(float), cEntropyVolume_pitched.pitch, 
		iVoxelX, iVoxelY + iVoxelZ * i3VolumeSize.y) = fEntropy;
}

// ADD-BY-LEETEN 12/23/2009-BEGIN
__global__ 
void 
_ComputeEntropyHierarchically_kernel
(
	int3 i3BlockCorner,
	int iNrOfBins,
	int3 i3KernelSize,
	int3 i3VolumeSize,
	int iGridDimX, 
	int iGridDimY, 
	int iLevel,
	cudaPitchedPtr cActiveVoxelsHistorgram_pitched,
	cudaPitchedPtr cEntropyVolume_pitched
)
{
	int iBlockIdxX = blockIdx.x % iGridDimX;
	int iBlockIdxY = blockIdx.x / iGridDimX;
	int iActiveVoxelX = iBlockIdxX * blockDim.x + threadIdx.x;
	int iActiveVoxelY = iBlockIdxY * blockDim.y + threadIdx.y;
	int iActiveVoxelId = iActiveVoxelX + iActiveVoxelY * iGridDimX * blockDim.x;
	int iToBeUpdatedBin = blockIdx.y * iLevel;

	int iCount =		( iToBeUpdatedBin < iNrOfBins )?tex2D(t2dActiveVoxelHistorgrams, iActiveVoxelId, iToBeUpdatedBin):0;
	int iNextCount =	( iToBeUpdatedBin + iLevel < iNrOfBins )?tex2D(t2dActiveVoxelHistorgrams, iActiveVoxelId, iToBeUpdatedBin + iLevel):0;
	int iSum = iCount + iNextCount;

	*ADDRESS_2D(
		int,			cActiveVoxelsHistorgram_pitched.ptr, 
		sizeof(int),	cActiveVoxelsHistorgram_pitched.pitch, 
		iActiveVoxelId, iToBeUpdatedBin) = iSum;
}

__global__ 
void 
_ComputeMarginalEntropy_kernel
(
	int3 i3BlockCorner,
	int iNrOfBins,
	int3 i3KernelSize,
	int3 i3VolumeSize,
	int iGridDimX, 
	int iGridDimY, 
	int iNrOfMarginalBins,
	int iNrOfThreadsPerMagin,
	cudaPitchedPtr cActiveVoxelMarginalEntropy_pitched
)
{
	int iBlockIdxX = blockIdx.x % iGridDimX;
	int iBlockIdxY = blockIdx.x / iGridDimX;
	int iActiveVoxelX = iBlockIdxX * blockDim.x + threadIdx.x;
	int iActiveVoxelY = iBlockIdxY * blockDim.y + threadIdx.y;
	int iActiveVoxelId = iActiveVoxelX + iActiveVoxelY * iGridDimX * blockDim.x;
	int iMargin =			blockIdx.y / iNrOfThreadsPerMagin;
	int iThreadIdInMargin = blockIdx.y % iNrOfThreadsPerMagin;

	int iBase = iMargin * iNrOfMarginalBins + iThreadIdInMargin * iNrOfMarginalBins / iNrOfThreadsPerMagin;
	float fEntropy = 0.0f;
	for(int b = 0; b < iNrOfMarginalBins / iNrOfThreadsPerMagin; b++)
	{
		int iCount = tex2D(t2dActiveVoxelHistorgrams, iActiveVoxelId, iBase + b);
		if( iCount )
		{
			float fCount = float(iCount);
			fEntropy += fCount * log2(fCount);
		}
	}
	*ADDRESS_2D(
		float,			cActiveVoxelMarginalEntropy_pitched.ptr, 
		sizeof(float),	cActiveVoxelMarginalEntropy_pitched.pitch, 
		iActiveVoxelId, blockIdx.y) = fEntropy;
}

__global__ 
void 
_SumMarginalEntropy_kernel
(
	int3 i3BlockCorner,
	int iNrOfBins,
	int3 i3KernelSize,
	int3 i3VolumeSize,
	int iNrOfMargins,
	int iNrOfThreadsPerMagin,
	cudaPitchedPtr cEntropyVolume_pitched
)
{
	int iActiveVoxelX = blockIdx.x * blockDim.x + threadIdx.x;
	int iActiveVoxelY = blockIdx.y * blockDim.y + threadIdx.y;
	int iActiveVoxelId = iActiveVoxelX + iActiveVoxelY * gridDim.x * blockDim.x;
 	int iVoxelX = i3BlockCorner.x + iActiveVoxelX;
	int iVoxelY = i3BlockCorner.y + iActiveVoxelY;
	int iVoxelZ = i3BlockCorner.z;

	int iNrOfNeighbors = (2 * i3KernelSize.x + 1) * (2 * i3KernelSize.y + 1) * (2 * i3KernelSize.z + 1);
	float fNrOfNeighbors = float(iNrOfNeighbors);

	float fEntropy = 0.0f;
	for(int b = 0; b < iNrOfMargins * iNrOfThreadsPerMagin; b++)
	{
		float fMarginalEntropy = tex2D(t2dActiveVoxelMarginalEntropy, iActiveVoxelId, b);
		fEntropy += fMarginalEntropy;
	}

	fEntropy = -fEntropy / fNrOfNeighbors + log2(fNrOfNeighbors);
	fEntropy = max(0.0, fEntropy);

	*ADDRESS_2D(
		float, cEntropyVolume_pitched.ptr, 
		sizeof(float), cEntropyVolume_pitched.pitch, 
		iVoxelX, iVoxelY + iVoxelZ * i3VolumeSize.y) = fEntropy;
}
// ADD-BY-LEETEN 12/23/2009-END

//////////////////////////////////////////////////////////////////////////////
void 
_ComputeEntropyVolume_cuda
(
						// res. of the neighboring region
	int3 i3KernelSize,

						// the histogram
	int iNrOfBins,
	int *piHistogram_global,
	float *pfLogHistogram_global,

						// res. of the volume																
	int3 i3VolumeSize,	
						// bin volume																	
	cudaPitchedPtr		cBinVolume_pitched,								
	cudaPitchedPtr		cEntropyVolume_pitched
)
{
CLOCK_INIT(_ComputeEntropyVolume_PRINT_TIMING, __FUNCTION__ ": ");

CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_TIMING);
	dim3 v3Blk = dim3(BLOCK_DIM_X, BLOCK_DIM_Y);
	const unsigned int iMaxMemorySpace = MAX_MEMORY_SIZE;
	int iMaxNrOfBlocks = int(floorf(float(iMaxMemorySpace) / float(sizeof(int) * iNrOfBins * v3Blk.x * v3Blk.y)));
	int iNrOfXBlocks = int(ceilf(float(i3VolumeSize.x) / float(v3Blk.x)));
	#if	0	// MOD-BY-LEETEN 12/19/2009-FROM:
		int iNrOfZBlocks = int(ceilf(float(i3VolumeSize.z) / float(v3Blk.y)));
		iMaxNrOfBlocks = min(iMaxNrOfBlocks, iNrOfXBlocks * iNrOfZBlocks);
		iNrOfXBlocks = min(iNrOfXBlocks, iMaxNrOfBlocks);
		iNrOfZBlocks = int(ceilf(float(iMaxNrOfBlocks) / float(iNrOfXBlocks)));
		dim3 v3Grid = dim3(iNrOfXBlocks, iNrOfZBlocks);
	#else	// MOD-BY-LEETEN 12/19/2009-TO:
	int iNrOfYBlocks = int(ceilf(float(i3VolumeSize.y) / float(v3Blk.y)));
	iMaxNrOfBlocks = min(iMaxNrOfBlocks, iNrOfXBlocks * iNrOfYBlocks);
	iNrOfXBlocks = min(iNrOfXBlocks, iMaxNrOfBlocks);
	iNrOfYBlocks = int(ceilf(float(iMaxNrOfBlocks) / float(iNrOfXBlocks)));
	dim3 v3Grid = dim3(iNrOfXBlocks, iNrOfYBlocks);
	#endif	// MOD-BY-LEETEN 12/19/2009-END
	fprintf(stderr, "MEM = %d MB; #BLOCKS = %d x %d\n", iMaxMemorySpace/(1<<20), v3Grid.x, v3Grid.y);

	cudaPitchedPtr cActiveVoxelHistorgrams_pitched;
	cActiveVoxelHistorgrams_pitched.xsize = v3Grid.x * v3Grid.y * v3Blk.x * v3Blk.y;
	cActiveVoxelHistorgrams_pitched.ysize = iNrOfBins;
	CUDA_SAFE_CALL_NO_SYNC(
		cudaMallocPitch(
			(void**)&cActiveVoxelHistorgrams_pitched.ptr, 
			&cActiveVoxelHistorgrams_pitched.pitch,
			cActiveVoxelHistorgrams_pitched.xsize * sizeof(int), 
			cActiveVoxelHistorgrams_pitched.ysize)	);

	CUDA_SAFE_CALL_NO_SYNC(
		cudaMemset2D(
			cActiveVoxelHistorgrams_pitched.ptr, 
			cActiveVoxelHistorgrams_pitched.pitch,
			0, 
			cActiveVoxelHistorgrams_pitched.xsize * sizeof(int), 
			cActiveVoxelHistorgrams_pitched.ysize)	);

	// bind the input vin volume to the texture that represents the src. bin volume 
	t2dSrcBinVolume.addressMode[0] = cudaAddressModeClamp;
	t2dSrcBinVolume.addressMode[1] = cudaAddressModeClamp;
	t2dSrcBinVolume.filterMode =	cudaFilterModePoint;
	t2dSrcBinVolume.normalized =	false;
	CUDA_SAFE_CALL_NO_SYNC(
		cudaBindTexture2D(
			0, 
			t2dSrcBinVolume, 
			cBinVolume_pitched.ptr, 
			cudaCreateChannelDesc<int>(),
			cVolumeExtent_array.width, 
			cVolumeExtent_array.height * cVolumeExtent_array.depth, 
			cBinVolume_pitched.pitch) );

	// bind the histogram as a texture
	t2dActiveVoxelHistorgrams.addressMode[0] = cudaAddressModeClamp;
	t2dActiveVoxelHistorgrams.addressMode[1] = cudaAddressModeClamp;
	t2dActiveVoxelHistorgrams.filterMode =	cudaFilterModePoint;
	t2dActiveVoxelHistorgrams.normalized =	false;
	CUDA_SAFE_CALL_NO_SYNC(
		cudaBindTexture2D(
			0, 
			t2dActiveVoxelHistorgrams, 
			cActiveVoxelHistorgrams_pitched.ptr, 
			cudaCreateChannelDesc<int>(),
			cActiveVoxelHistorgrams_pitched.xsize, 
			cActiveVoxelHistorgrams_pitched.ysize,
			cActiveVoxelHistorgrams_pitched.pitch) );

	// ADD-BY-LEETEN 12/23/2009-BEGIN
	int iNrOfMarginalBins = int(sqrtf(float(iNrOfBins)));
	int iNrOfMargins = int(ceilf(float(iNrOfBins)/float(iNrOfMarginalBins)));

	cudaPitchedPtr cActiveVoxelMarginalEntropy_pitched;
	cActiveVoxelMarginalEntropy_pitched.xsize = v3Grid.x * v3Grid.y * v3Blk.x * v3Blk.y;
	cActiveVoxelMarginalEntropy_pitched.ysize = iNrOfMargins * NR_OF_THREADS_PER_MARGIN;
	CUDA_SAFE_CALL_NO_SYNC(
		cudaMallocPitch(
			(void**)&cActiveVoxelMarginalEntropy_pitched.ptr, 
			&cActiveVoxelMarginalEntropy_pitched.pitch,
			cActiveVoxelMarginalEntropy_pitched.xsize * sizeof(float), 
			cActiveVoxelMarginalEntropy_pitched.ysize)	);

	CUDA_SAFE_CALL_NO_SYNC(
		cudaMemset2D(
			cActiveVoxelMarginalEntropy_pitched.ptr, 
			cActiveVoxelMarginalEntropy_pitched.pitch,
			0, 
			cActiveVoxelMarginalEntropy_pitched.pitch, 
			cActiveVoxelMarginalEntropy_pitched.ysize)	);

	t2dActiveVoxelMarginalEntropy.addressMode[0] = cudaAddressModeClamp;
	t2dActiveVoxelMarginalEntropy.addressMode[1] = cudaAddressModeClamp;
	t2dActiveVoxelMarginalEntropy.filterMode =	cudaFilterModePoint;
	t2dActiveVoxelMarginalEntropy.normalized =	false;
	CUDA_SAFE_CALL_NO_SYNC(
		cudaBindTexture2D(
			0, 
			t2dActiveVoxelMarginalEntropy, 
			cActiveVoxelMarginalEntropy_pitched.ptr, 
			cudaCreateChannelDesc<float>(),
			cActiveVoxelMarginalEntropy_pitched.xsize, 
			cActiveVoxelMarginalEntropy_pitched.ysize,
			cActiveVoxelMarginalEntropy_pitched.pitch) );
	// ADD-BY-LEETEN 12/23/2009-END

CLOCK_END(_ComputeEntropyVolume_PRINT_TIMING, false);

CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_TIMING);

	#if	0	// MOD-BY-LEETEN 12/19/2009-FROM:
		for(int			z = 0; z < i3VolumeSize.z; z += v3Grid.y * v3Blk.y)
			for(int		x = 0; x < i3VolumeSize.x; x += v3Grid.x * v3Blk.x)
				for(int y = 0; y < i3VolumeSize.y; y++)
	#else	// MOD-BY-LEETEN 12/19/2009-TO:
	for(int			y = 0; y < i3VolumeSize.y; y += v3Grid.y * v3Blk.y)
		for(int		x = 0; x < i3VolumeSize.x; x += v3Grid.x * v3Blk.x)
			for(int z = 0; z < i3VolumeSize.z; z++)
	#endif	// MOD-BY-LEETEN 12/19/2009-END
			{
				int3 i3BlockCorner = make_int3(x, y, z);

				// ADD-BY-LEETEN 12/23/2009-BEGIN
				CLOCK_INIT(_ComputeEntropyVolume_PRINT_LOOP_TIMING, __FUNCTION__ " (main loop): ");

				CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_LOOP_TIMING);
				// ADD-BY-LEETEN 12/23/2009-END

				// MOD-BY-LEETEN 12/19/2009-FROM:
					// if( 0 == y )
				// TO:
				if( 0 == z )
				// MOD-BY-LEETEN 12/19/2009-END
				{
					_CreateHistogram_kernel<<<v3Grid, v3Blk, 0>>>
					(
						i3BlockCorner,
						i3KernelSize,
						i3VolumeSize,
						iNrOfBins,
						cActiveVoxelHistorgrams_pitched
					);
					CUT_CHECK_ERROR("_CreateHistogram_kernel() failed");
				}
				else
				{
					_UpdateHistogram_kernel<<<v3Grid, v3Blk, 0>>>
					(
						i3BlockCorner,
						i3KernelSize,
						i3VolumeSize,
						iNrOfBins,
						cActiveVoxelHistorgrams_pitched
					);
					CUT_CHECK_ERROR("_UpdateHistogram_kernel() failed");
				}

				// ADD-BY-LEETEN 12/23/2009-BEGIN
				CLOCK_END(_ComputeEntropyVolume_PRINT_LOOP_TIMING, false);

				CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_LOOP_TIMING);
				// ADD-BY-LEETEN 12/23/2009-END

				#if		SCANNING_METHOD	== SCANNING_METHOD_SCAN_WHOLE_HISTOGRAM					// ADD-BY-LEETEN 12/23/2009
				_ComputeEntropy_kernel<<<v3Grid, v3Blk, 0>>>
				(
					i3BlockCorner,
					iNrOfBins,
					i3KernelSize,
					i3VolumeSize,
					cEntropyVolume_pitched
				);
				CUT_CHECK_ERROR("_ComputeEntropy_kernel() failed");
				#endif	// #if		SCANNING_METHOD	== SCANNING_METHOD_SCAN_WHOLE_HISTOGRAM		// ADD-BY-LEETEN 12/23/2009

				// ADD-BY-LEETEN 12/23/2009-BEGIN
				#if		SCANNING_METHOD	== SCANNING_METHOD_HIERARCHICAL_SCAN
				int iNrOfLevels = int(floorf(log2f(float(iNrOfBins))));
				int iNrOfThreadsPerVoxel = iNrOfBins/2;
				for(int l = 1; l < iNrOfLevels; l++)
				{
					_ComputeEntropyHierarchically_kernel<<<
						dim3(v3Grid.x * v3Grid.y, iNrOfThreadsPerVoxel),
						v3Blk, 0
						>>>
					(
						i3BlockCorner,
						iNrOfBins,
						i3KernelSize,
						i3VolumeSize,
						v3Grid.x,
						v3Grid.y,
						l,
						cActiveVoxelHistorgrams_pitched,
						cEntropyVolume_pitched
					);

					iNrOfThreadsPerVoxel /= 2;
				}
				CUT_CHECK_ERROR("_ComputeEntropyHierarchically_kernel() failed");
				#endif	// #if	SCANNING_METHOD	== SCANNING_METHOD_HIERARCHICAL_SCAN

				#if	SCANNING_METHOD	== SCANNING_METHOD_SCAN_ROWS_IN_PARALLEL
				_ComputeMarginalEntropy_kernel<<<
					dim3(v3Grid.x * v3Grid.y, NR_OF_THREADS_PER_MARGIN * iNrOfMargins), v3Blk, 0
					>>>
				(
					i3BlockCorner,
					iNrOfBins,
					i3KernelSize,
					i3VolumeSize,
					v3Grid.x,
					v3Grid.y,
					iNrOfMarginalBins,
					NR_OF_THREADS_PER_MARGIN,
					cActiveVoxelMarginalEntropy_pitched
				);
				CUT_CHECK_ERROR("_ComputeMarginalEntropy_kernel() failed");

				CLOCK_END(_ComputeEntropyVolume_PRINT_LOOP_TIMING, false);

				CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_LOOP_TIMING);

				_SumMarginalEntropy_kernel<<<v3Grid, v3Blk, 0>>>
				(
					i3BlockCorner,
					iNrOfBins,
					i3KernelSize,
					i3VolumeSize,
					iNrOfMargins,
					NR_OF_THREADS_PER_MARGIN,
					cEntropyVolume_pitched
				);
				CUT_CHECK_ERROR("_SumMarginalEntropy_kernel() failed");
				#endif	// #if	SCANNING_METHOD	== SCANNING_METHOD_SCAN_ROWS_IN_PARALLEL
				// ADD-BY-LEETEN 12/23/2009-END

				// ADD-BY-LEETEN 12/23/2009-BEGIN
				CLOCK_END(_ComputeEntropyVolume_PRINT_LOOP_TIMING, false);

				CLOCK_PRINT(_ComputeEntropyVolume_PRINT_LOOP_TIMING);
				// ADD-BY-LEETEN 12/23/2009-END
			}

CLOCK_END(_ComputeEntropyVolume_PRINT_TIMING, false);

CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_TIMING);

	FREE_MEMORY(cActiveVoxelHistorgrams_pitched.ptr);

	// ADD-BY-LEETEN 12/23/2009-BEGIN
	FREE_MEMORY(cActiveVoxelMarginalEntropy_pitched.ptr);
	// ADD-BY-LEETEN 12/23/2009-END

CLOCK_END(_ComputeEntropyVolume_PRINT_TIMING, false);

CLOCK_PRINT(_ComputeEntropyVolume_PRINT_TIMING);
}
// ADD-BY-LEETEN 12/18/2009-END

#endif	// #if	COMPUTE_ENTROPY_VOLUME_CUDA

/*

$Log: not supported by cvs2svn $

*/
