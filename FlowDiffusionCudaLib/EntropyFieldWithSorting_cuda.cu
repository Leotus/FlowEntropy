
#include <hip/hip_runtime.h>
#if	COMPUTE_ENTROPY_VOLUME_WITH_SORTING_CUDA

__global__ 
void
_ComputeEntropyOnSortedNeighbors_kernel
(
	int3 i3BlockCorner,
	int iNrOfBins,
	int3 i3KernelSize,
	unsigned int *puActiveVoxelSortedNeighbors_global,
	int3 i3VolumeSize,
	cudaPitchedPtr cEntropyVolume_pitched
)
{
	int iActiveVoxelX = blockIdx.x * blockDim.x + threadIdx.x;
	int iActiveVoxelY = blockIdx.y * blockDim.y + threadIdx.y;
	int iActiveVoxelId = iActiveVoxelX + iActiveVoxelY * gridDim.x * blockDim.x;
 	int iVoxelX = i3BlockCorner.x + iActiveVoxelX;
	int iVoxelY = i3BlockCorner.y + iActiveVoxelY;
	int iVoxelZ = i3BlockCorner.z;

	int iNrOfNeighbors = (2 * i3KernelSize.x + 1) * (2 * i3KernelSize.y + 1) * (2 * i3KernelSize.z + 1);
	float fNrOfNeighbors = float(iNrOfNeighbors);

	float fEntropy = 0.0f;
	unsigned int uPrevBin = 0;
	float fCount = 0.0f; 

	unsigned int uNrOfActiveVoxels = gridDim.x * blockDim.x * gridDim.y * blockDim.y;
	unsigned int uTexIndex = iNrOfNeighbors * iActiveVoxelId;

	for(int n = 0; n < iNrOfNeighbors; n++)
	{
		float fTexCoordX = float(uTexIndex % uNrOfActiveVoxels);
		float fTexCoordY = float(uTexIndex / uNrOfActiveVoxels);
		unsigned int uBin = tex2D(t2dActiveVoxelSortedNeighbors, fTexCoordX, fTexCoordY);
		uTexIndex++;

		if( n == 0 || uBin == uPrevBin )
		{
			fCount += 1.0f;
		}
		else
		{
			if( fCount > 0.0f )
				fEntropy += fCount * log2(fCount);
			
			fCount = 1.0f;
		}
		uPrevBin = uBin;
	}
	if( fCount > 0.0f )
		fEntropy += fCount * log2(fCount);

	fEntropy = -fEntropy / fNrOfNeighbors + log2(fNrOfNeighbors);
	fEntropy = max(fEntropy, 0.0f);

	if( iVoxelX < i3VolumeSize.x && iVoxelY < i3VolumeSize.y && iVoxelZ < i3VolumeSize.z )
		*ADDRESS_2D(
			float, cEntropyVolume_pitched.ptr, 
			sizeof(float), cEntropyVolume_pitched.pitch, 
			iVoxelX, iVoxelY + iVoxelZ * i3VolumeSize.y) = fEntropy;
}

__global__ 
void 
_CollectNeighbors_kernel
(
	int3 i3BlockCorner,
	int3 i3KernelSize,
	int3 i3VolumeSize,
	int iNrOfBins,
	cudaPitchedPtr cActiveVoxelNeighbors_pitched
)
{
	int iActiveVoxelX = blockIdx.x * blockDim.x + threadIdx.x;
	int iActiveVoxelY = blockIdx.y * blockDim.y + threadIdx.y;
	int iActiveVoxelId = iActiveVoxelX + iActiveVoxelY * gridDim.x * blockDim.x;
 	int iVoxelX = i3BlockCorner.x + iActiveVoxelX;
	int iVoxelY = i3BlockCorner.y + iActiveVoxelY;
	int iVoxelZ = i3BlockCorner.z;

	{
		for(int	iOffset = 0,iZOffset = -i3KernelSize.z; iZOffset <= i3KernelSize.z; iZOffset++)
			for(int			iYOffset = -i3KernelSize.y; iYOffset <= i3KernelSize.y; iYOffset++)
				for(int		iXOffset = -i3KernelSize.x; iXOffset <= i3KernelSize.x; iXOffset++, iOffset++)
				{
					int iX, iY, iZ;
					iX = iVoxelX + iXOffset;
					iY = iVoxelY + iYOffset;
					iZ = iVoxelZ + iZOffset;

					int3 i3TexCoord;
					i3TexCoord.x = IMirrorCoord(iX, i3VolumeSize.x);
					i3TexCoord.y = IMirrorCoord(iY, i3VolumeSize.y);
					i3TexCoord.z = IMirrorCoord(iZ, i3VolumeSize.z);

					int iSrcBin = tex2D(t2dSrcBinVolume, i3TexCoord.x, i3TexCoord.y + i3TexCoord.z * i3VolumeSize.y);

					*ADDRESS_2D(
						unsigned int,			cActiveVoxelNeighbors_pitched.ptr, 
						sizeof(unsigned int),	cActiveVoxelNeighbors_pitched.pitch, 
						iActiveVoxelId, iOffset) = unsigned int(iActiveVoxelId * iNrOfBins + iSrcBin);
				}
	}
}

void 
_ComputeEntropyVolumeWithSorting_cuda
(
						// res. of the neighboring region
	int3 i3KernelSize,

						// the histogram
	int iNrOfBins,
	int *piHistogram_global,
	float *pfLogHistogram_global,

						// res. of the volume																
	int3 i3VolumeSize,	
						// bin volume																	
	cudaPitchedPtr		cBinVolume_pitched,								
	cudaPitchedPtr		cEntropyVolume_pitched
)
{
CLOCK_INIT(_ComputeEntropyVolume_PRINT_TIMING, __FUNCTION__ ": ");

CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_TIMING);
	const int iNrOfRows = (2 * i3KernelSize.x + 1) * (2 * i3KernelSize.y + 1) * (2 * i3KernelSize.z + 1);

	dim3 v3Blk = dim3(BLOCK_DIM_X, BLOCK_DIM_Y);
	/*
	const unsigned int iMaxMemorySpace = MAX_MEMORY_SIZE;
	int iMaxNrOfBlocks = int(floorf(
		float(iMaxMemorySpace/2) / 
		float(sizeof(int) * iNrOfRows * v3Blk.x * v3Blk.y)));
	*/
	int iNrOfXBlocks = int(ceilf(float(i3VolumeSize.x) / float(v3Blk.x)));
	int iNrOfYBlocks = int(ceilf(float(i3VolumeSize.y) / float(v3Blk.y)));
	int iMaxNrOfThreads = int(floor(double(1<<(RADIX_SORT_BITS-1)) / double(iNrOfBins)));
	int iMaxNrOfBlocks	= int(floorf(float(iMaxNrOfThreads) / float(v3Blk.x * v3Blk.y)));
	iMaxNrOfBlocks = min(iMaxNrOfBlocks, iNrOfXBlocks * iNrOfYBlocks);
	iNrOfXBlocks = min(iNrOfXBlocks, iMaxNrOfBlocks);
	iNrOfYBlocks = int(ceilf(float(iMaxNrOfBlocks) / float(iNrOfXBlocks)));
	dim3 v3Grid = dim3(iNrOfXBlocks, iNrOfYBlocks);
	fprintf(stderr, "#BLOCKS = %d x %d\n", v3Grid.x, v3Grid.y);

	cudaPitchedPtr cActiveVoxelNeighbors_pitched;
	cActiveVoxelNeighbors_pitched.xsize = v3Grid.x * v3Grid.y * v3Blk.x * v3Blk.y;
	cActiveVoxelNeighbors_pitched.ysize = iNrOfRows;
	CUDA_SAFE_CALL_NO_SYNC(
		cudaMallocPitch(
			(void**)&cActiveVoxelNeighbors_pitched.ptr, 
			&cActiveVoxelNeighbors_pitched.pitch,
			cActiveVoxelNeighbors_pitched.xsize * sizeof(int), 
			cActiveVoxelNeighbors_pitched.ysize)	);

	CUDA_SAFE_CALL_NO_SYNC(
		cudaMemset2D(
			cActiveVoxelNeighbors_pitched.ptr, 
			cActiveVoxelNeighbors_pitched.pitch,
			0, 
			cActiveVoxelNeighbors_pitched.xsize * sizeof(int), 
			cActiveVoxelNeighbors_pitched.ysize)	);

	// bind the input vin volume to the texture that represents the src. bin volume 
	t2dSrcBinVolume.addressMode[0] = cudaAddressModeClamp;
	t2dSrcBinVolume.addressMode[1] = cudaAddressModeClamp;
	t2dSrcBinVolume.filterMode =	cudaFilterModePoint;
	t2dSrcBinVolume.normalized =	false;
	CUDA_SAFE_CALL_NO_SYNC(
		cudaBindTexture2D(
			0, 
			t2dSrcBinVolume, 
			cBinVolume_pitched.ptr, 
			cudaCreateChannelDesc<int>(),
			cVolumeExtent_array.width, 
			cVolumeExtent_array.height * cVolumeExtent_array.depth, 
			cBinVolume_pitched.pitch) );

	// bind the histogram as a texture
	t2dActiveVoxelSortedNeighbors.addressMode[0] = cudaAddressModeClamp;
	t2dActiveVoxelSortedNeighbors.addressMode[1] = cudaAddressModeClamp;
	t2dActiveVoxelSortedNeighbors.filterMode =	cudaFilterModePoint;
	t2dActiveVoxelSortedNeighbors.normalized =	false;

	CUDA_SAFE_CALL_NO_SYNC(
		cudaBindTexture2D(
			0, 
			t2dActiveVoxelSortedNeighbors, 
			cActiveVoxelNeighbors_pitched.ptr, 
			cudaCreateChannelDesc<unsigned int>(),
			cActiveVoxelNeighbors_pitched.xsize,
			cActiveVoxelNeighbors_pitched.ysize, 
			cActiveVoxelNeighbors_pitched.pitch) );

	CUDPPHandle hScanPlan = 0;

	CUDPPConfiguration	cConfig;
	cConfig.op =		CUDPP_ADD;
	cConfig.datatype =	CUDPP_UINT;
	cConfig.algorithm =	CUDPP_SORT_RADIX;
	cConfig.options =	CUDPP_OPTION_KEYS_ONLY;

	assert( 
		CUDPP_SUCCESS  == cudppPlan(
			&hScanPlan,	
			cConfig, 
			cActiveVoxelNeighbors_pitched.pitch * cActiveVoxelNeighbors_pitched.ysize / sizeof(unsigned int),
			1, 
			0) );

CLOCK_END(_ComputeEntropyVolume_PRINT_TIMING, false);

CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_TIMING);

	for(int			y = 0; y < i3VolumeSize.y; y += v3Grid.y * v3Blk.y)
		for(int		x = 0; x < i3VolumeSize.x; x += v3Grid.x * v3Blk.x)
			for(int	z = 0; z < i3VolumeSize.z; z ++ )
			{
				CLOCK_INIT(_ComputeEntropyVolume_PRINT_LOOP_TIMING, __FUNCTION__ " (main loop): ");

				CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_LOOP_TIMING);

				int3 i3BlockCorner = make_int3(x, y, z);
				_CollectNeighbors_kernel<<<v3Grid, v3Blk, 0>>>
				(
					i3BlockCorner,
					i3KernelSize,
					i3VolumeSize,
					iNrOfBins,
					cActiveVoxelNeighbors_pitched
				);
				CUT_CHECK_ERROR("_CollectNeighbors_kernel() failed");

CLOCK_END(_ComputeEntropyVolume_PRINT_LOOP_TIMING, false);

CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_LOOP_TIMING);

				// call cudpp to sort the buffer _CollectNeighborsToBeSorted_kernel
				cudppSort(
					hScanPlan,
					cActiveVoxelNeighbors_pitched.ptr,
					NULL,
					RADIX_SORT_BITS,
					cActiveVoxelNeighbors_pitched.pitch * cActiveVoxelNeighbors_pitched.ysize / sizeof(unsigned int)) ;
				CUT_CHECK_ERROR("cudppSort() failed");
CLOCK_END(_ComputeEntropyVolume_PRINT_LOOP_TIMING, false);

CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_LOOP_TIMING);
				// 
				_ComputeEntropyOnSortedNeighbors_kernel<<<v3Grid, v3Blk, 0>>>
				(
					i3BlockCorner,
					iNrOfBins,
					i3KernelSize,
					(unsigned int *)cActiveVoxelNeighbors_pitched.ptr,
					i3VolumeSize,
					cEntropyVolume_pitched
				);
				CUT_CHECK_ERROR("_ComputeEntropyOnSortedNeighbors_kernel() failed");
CLOCK_END(_ComputeEntropyVolume_PRINT_LOOP_TIMING, false);

CLOCK_PRINT(_ComputeEntropyVolume_PRINT_LOOP_TIMING);
			}


CLOCK_END(_ComputeEntropyVolume_PRINT_TIMING, false);

CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_TIMING);

	cudppDestroyPlan(hScanPlan);
	FREE_MEMORY(cActiveVoxelNeighbors_pitched.ptr);

CLOCK_END(_ComputeEntropyVolume_PRINT_TIMING, false);

CLOCK_PRINT(_ComputeEntropyVolume_PRINT_TIMING);
}

#endif	// #if COMPUTE_ENTROPY_VOLUME_WITH_SORTING_CUDA

/*

$Log: not supported by cvs2svn $
Revision 1.1  2009/12/27 19:03:06  leeten

[12/27/2009]
1. [1ST] First time checkin. Ths file define the host and kernel functions to compute the netropy field by sorting all the neighbors of all threads via CUDPP.


*/
