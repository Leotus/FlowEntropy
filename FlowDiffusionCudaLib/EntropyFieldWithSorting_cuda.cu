
#include <hip/hip_runtime.h>
#if	COMPUTE_ENTROPY_VOLUME_WITH_SORTING_CUDA

__global__ 
void
_ComputeEntropyOnSortedNeighbors_kernel
(
	int3 i3BlockCorner,
	int iNrOfBins,
	int3 i3KernelSize,
	unsigned int *puActiveVoxelSortedNeighbors_global,
	int3 i3VolumeSize,
	cudaPitchedPtr cEntropyVolume_pitched
)
{
	int iActiveVoxelX = blockIdx.x * blockDim.x + threadIdx.x;
	#if	0	// MOD-BY-LEETEN 12/19/2009-FROM:
		int iActiveVoxelZ = blockIdx.y * blockDim.y + threadIdx.y;
		int iActiveVoxelId = iActiveVoxelX + iActiveVoxelZ * gridDim.x * blockDim.x;
 		int iVoxelX = i3BlockCorner.x + iActiveVoxelX;
		int iVoxelY = i3BlockCorner.y;
		int iVoxelZ = i3BlockCorner.z + iActiveVoxelZ;
	#else	// MOD-BY-LEETEN 12/19/2009-TO:

	int iActiveVoxelY = blockIdx.y * blockDim.y + threadIdx.y;
	int iActiveVoxelId = iActiveVoxelX + iActiveVoxelY * gridDim.x * blockDim.x;
 	int iVoxelX = i3BlockCorner.x + iActiveVoxelX;
	int iVoxelY = i3BlockCorner.y + iActiveVoxelY;
	int iVoxelZ = i3BlockCorner.z;

	#endif	// MOD-BY-LEETEN 12/19/2009-END

	int iNrOfNeighbors = (2 * i3KernelSize.x + 1) * (2 * i3KernelSize.y + 1) * (2 * i3KernelSize.z + 1);
	float fNrOfNeighbors = float(iNrOfNeighbors);

	float fEntropy = 0.0f;
	unsigned int uPrevBin = 0;
	float fCount = 0.0f; 

	// ADD-BY-LEETEN 12/18/2009-BEGIN
	unsigned int uNrOfActiveVoxels = gridDim.x * blockDim.x * gridDim.y * blockDim.y;
	unsigned int uTexIndex = iNrOfNeighbors * iActiveVoxelId;
	// ADD-BY-LEETEN 12/18/2009-END

	for(int n = 0; n < iNrOfNeighbors; n++)
	{
		#if	0	// MOD-BY-LEETEN 12/18/2009-FROM:
			unsigned int uBin = tex1Dfetch(t1dActiveVoxelSortedNeighbors, iNrOfNeighbors * iActiveVoxelId + n);
			// unsigned int uBin = puActiveVoxelSortedNeighbors_global[iNrOfNeighbors * iActiveVoxelId + n];
			// uBin = uBin % unsigned int (iNrOfBins);
		#else	// MOD-BY-LEETEN 12/18/2009-TO:
		float fTexCoordX = float(uTexIndex % uNrOfActiveVoxels);
		float fTexCoordY = float(uTexIndex / uNrOfActiveVoxels);
		unsigned int uBin = tex2D(t2dActiveVoxelSortedNeighbors, fTexCoordX, fTexCoordY);
		uTexIndex++;
		#endif	// MOD-BY-LEETEN 12/18/2009-END

		if( n == 0 || uBin == uPrevBin )
		{
			fCount += 1.0f;
		}
		else
		{
			if( fCount > 0.0f )
				fEntropy += fCount * log2(fCount);
			
			fCount = 1.0f;
		}
		uPrevBin = uBin;
	}
	if( fCount > 0.0f )
		fEntropy += fCount * log2(fCount);

	fEntropy = -fEntropy / fNrOfNeighbors + log2(fNrOfNeighbors);
	fEntropy = max(fEntropy, 0.0f);

	// ADD-BY-LEETEN 12/20/2009-BEGIN
	if( iVoxelX < i3VolumeSize.x && iVoxelY < i3VolumeSize.y && iVoxelZ < i3VolumeSize.z )
	// ADD-BY-LEETEN 12/20/2009-END
		*ADDRESS_2D(
			float, cEntropyVolume_pitched.ptr, 
			sizeof(float), cEntropyVolume_pitched.pitch, 
			iVoxelX, iVoxelY + iVoxelZ * i3VolumeSize.y) = fEntropy;
}

__global__ 
void 
_CollectNeighbors_kernel
(
	int3 i3BlockCorner,
	int3 i3KernelSize,
	int3 i3VolumeSize,
	int iNrOfBins,
	cudaPitchedPtr cActiveVoxelNeighbors_pitched
)
{
	int iActiveVoxelX = blockIdx.x * blockDim.x + threadIdx.x;
	#if	0	// MOD-BY-LEETEN 12/19/2009-FROM:
		int iActiveVoxelZ = blockIdx.y * blockDim.y + threadIdx.y;
		int iActiveVoxelId = iActiveVoxelX + iActiveVoxelZ * gridDim.x * blockDim.x;
 		int iVoxelX = i3BlockCorner.x + iActiveVoxelX;
		int iVoxelY = i3BlockCorner.y;
		int iVoxelZ = i3BlockCorner.z + iActiveVoxelZ;
		int iNrOfXZNeighbors = (2 * i3KernelSize.x + 1) * (2 * i3KernelSize.z + 1);
	#else	// MOD-BY-LEETEN 12/19/2009-TO:
	int iActiveVoxelY = blockIdx.y * blockDim.y + threadIdx.y;
	int iActiveVoxelId = iActiveVoxelX + iActiveVoxelY * gridDim.x * blockDim.x;
 	int iVoxelX = i3BlockCorner.x + iActiveVoxelX;
	int iVoxelY = i3BlockCorner.y + iActiveVoxelY;
	int iVoxelZ = i3BlockCorner.z;
	// DEL-BY-LEETEN 12/20/2009-BEGIN
		// int iNrOfXYNeighbors = (2 * i3KernelSize.x + 1) * (2 * i3KernelSize.y + 1);
	// DEL-BY-LEETEN 12/20/2009-END
	#endif	// MOD-BY-LEETEN 12/19/2009-END

	// MOD-BY-LEETEN 12/19/2009-FROM:
		// if ( 0 == iVoxelY )
	// TO:
	// DEL-BY-LEETEN 12/20/2009-BEGIN
		// if ( 0 == iVoxelZ )
	// DEL-BY-LEETEN 12/20/2009-END
	// MOD-BY-LEETEN 12/19/2009-END
	{
		#if	0	// MOD-BY-LEETEN 12/19/2009-FROM:
			for(int			iOffset = 0,	iYOffset = -i3KernelSize.y; iYOffset <= i3KernelSize.y; iYOffset++)
				for(int						iZOffset = -i3KernelSize.z; iZOffset <= i3KernelSize.z; iZOffset++)
					for(int					iXOffset = -i3KernelSize.x; iXOffset <= i3KernelSize.x; iXOffset++, iOffset++)
		#else	// MOD-BY-LEETEN 12/19/2009-TO:
		for(int	iOffset = 0,iZOffset = -i3KernelSize.z; iZOffset <= i3KernelSize.z; iZOffset++)
			for(int			iYOffset = -i3KernelSize.y; iYOffset <= i3KernelSize.y; iYOffset++)
				for(int		iXOffset = -i3KernelSize.x; iXOffset <= i3KernelSize.x; iXOffset++, iOffset++)
		#endif	// MOD-BY-LEETEN 12/19/2009-END
				{
					int iX, iY, iZ;
					iX = iVoxelX + iXOffset;
					iY = iVoxelY + iYOffset;
					iZ = iVoxelZ + iZOffset;

					int3 i3TexCoord;
					i3TexCoord.x = IMirrorCoord(iX, i3VolumeSize.x);
					i3TexCoord.y = IMirrorCoord(iY, i3VolumeSize.y);
					i3TexCoord.z = IMirrorCoord(iZ, i3VolumeSize.z);

					int iSrcBin = tex2D(t2dSrcBinVolume, i3TexCoord.x, i3TexCoord.y + i3TexCoord.z * i3VolumeSize.y);

					*ADDRESS_2D(
						unsigned int,			cActiveVoxelNeighbors_pitched.ptr, 
						sizeof(unsigned int),	cActiveVoxelNeighbors_pitched.pitch, 
						iActiveVoxelId, iOffset) = unsigned int(iActiveVoxelId * iNrOfBins + iSrcBin);
				}
	}
	#if	0	// DEL-BY-LEETEN 12/20/2009-BEGIN
	else
	{
		// ADD-BY-LEETEN 12/20/2009-BEGIN
		int iNrOfXYNeighbors = (2 * i3KernelSize.x + 1) * (2 * i3KernelSize.y + 1);
		// ADD-BY-LEETEN 12/20/2009-END
		#if	0	// MOD-BY-LEETEN 12/19/2009-FROM:
			int iYOffset = i3KernelSize.y;
			for(int		iXZOffset = 0,	iZOffset = -i3KernelSize.z; iZOffset <= i3KernelSize.z; iZOffset++)
				for(int					iXOffset = -i3KernelSize.x; iXOffset <= i3KernelSize.x; iXOffset++, iXZOffset++)
		#else	// MOD-BY-LEETEN 12/19/2009-TO:
		int iZOffset = i3KernelSize.z;
		for(int		iXYOffset = 0,	iYOffset = -i3KernelSize.y; iYOffset <= i3KernelSize.y; iYOffset++)
			for(int					iXOffset = -i3KernelSize.x; iXOffset <= i3KernelSize.x; iXOffset++, iXYOffset++)
		#endif	// MOD-BY-LEETEN 12/19/2009-END
			{
				int iX, iY, iZ;
				iX = iVoxelX + iXOffset;
				iY = iVoxelY + iYOffset;
				iZ = iVoxelZ + iZOffset;

				int3 i3TexCoord;
				i3TexCoord.x = IMirrorCoord(iX, i3VolumeSize.x);
				i3TexCoord.y = IMirrorCoord(iY, i3VolumeSize.y);
				i3TexCoord.z = IMirrorCoord(iZ, i3VolumeSize.z);

				int iSrcBin = tex2D(t2dSrcBinVolume, i3TexCoord.x, i3TexCoord.y + i3TexCoord.z * i3VolumeSize.y);

				#if	0	// MOD-BY-LEETEN 12/19/2009-FROM:
					int iModY = iVoxelY % (2 * i3KernelSize.y + 1);
					int iOffset = (0 == iModY)?(2 * i3KernelSize.y):(iModY - 1);
					iOffset = iOffset * iNrOfXZNeighbors + iXZOffset;
				#else	// MOD-BY-LEETEN 12/19/2009-TO:
				int iModZ = iVoxelZ % (2 * i3KernelSize.z + 1);
				int iOffset = (0 == iModZ)?(2 * i3KernelSize.z):(iModZ - 1);
				iOffset = iOffset * iNrOfXYNeighbors + iXYOffset;
				#endif	// MOD-BY-LEETEN 12/19/2009-END
				*ADDRESS_2D(
					unsigned int,			cActiveVoxelNeighbors_pitched.ptr, 
					sizeof(unsigned int),	cActiveVoxelNeighbors_pitched.pitch, 
					iActiveVoxelId, iOffset) = unsigned (iActiveVoxelId * iNrOfBins + iSrcBin);
			}
	}
	#endif	// DEL-BY-LEETEN 12/20/2009-END
}
// ADD-BY-LEETEN 12/18/2009-END

void 
_ComputeEntropyVolumeWithSorting_cuda
(
						// res. of the neighboring region
	int3 i3KernelSize,

						// the histogram
	int iNrOfBins,
	int *piHistogram_global,
	float *pfLogHistogram_global,

						// res. of the volume																
	int3 i3VolumeSize,	
						// bin volume																	
	cudaPitchedPtr		cBinVolume_pitched,								
	cudaPitchedPtr		cEntropyVolume_pitched
)
{
CLOCK_INIT(_ComputeEntropyVolume_PRINT_TIMING, __FUNCTION__ ": ");

CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_TIMING);
	const int iNrOfRows = (2 * i3KernelSize.x + 1) * (2 * i3KernelSize.y + 1) * (2 * i3KernelSize.z + 1);

	dim3 v3Blk = dim3(BLOCK_DIM_X, BLOCK_DIM_Y);
	/*
	const unsigned int iMaxMemorySpace = MAX_MEMORY_SIZE;
	int iMaxNrOfBlocks = int(floorf(
		float(iMaxMemorySpace/2) / 
		float(sizeof(int) * iNrOfRows * v3Blk.x * v3Blk.y)));
	*/
	int iNrOfXBlocks = int(ceilf(float(i3VolumeSize.x) / float(v3Blk.x)));
	#if	0	// MOD-BY-LEETEN 12/19/2009-FROM:
	#else	// MOD-BY-LEETEN 12/19/2009-TO:
	int iNrOfYBlocks = int(ceilf(float(i3VolumeSize.y) / float(v3Blk.y)));
	int iMaxNrOfThreads = int(floor(double(1<<(RADIX_SORT_BITS-1)) / double(iNrOfBins)));
	int iMaxNrOfBlocks	= int(floorf(float(iMaxNrOfThreads) / float(v3Blk.x * v3Blk.y)));
	iMaxNrOfBlocks = min(iMaxNrOfBlocks, iNrOfXBlocks * iNrOfYBlocks);
	iNrOfXBlocks = min(iNrOfXBlocks, iMaxNrOfBlocks);
	iNrOfYBlocks = int(ceilf(float(iMaxNrOfBlocks) / float(iNrOfXBlocks)));
	dim3 v3Grid = dim3(iNrOfXBlocks, iNrOfYBlocks);
	#endif	// MOD-BY-LEETEN 12/19/2009-END
	fprintf(stderr, "#BLOCKS = %d x %d\n", v3Grid.x, v3Grid.y);

	cudaPitchedPtr cActiveVoxelNeighbors_pitched;
	cActiveVoxelNeighbors_pitched.xsize = v3Grid.x * v3Grid.y * v3Blk.x * v3Blk.y;
	cActiveVoxelNeighbors_pitched.ysize = iNrOfRows;
	CUDA_SAFE_CALL_NO_SYNC(
		cudaMallocPitch(
			(void**)&cActiveVoxelNeighbors_pitched.ptr, 
			&cActiveVoxelNeighbors_pitched.pitch,
			cActiveVoxelNeighbors_pitched.xsize * sizeof(int), 
			cActiveVoxelNeighbors_pitched.ysize)	);

	CUDA_SAFE_CALL_NO_SYNC(
		cudaMemset2D(
			cActiveVoxelNeighbors_pitched.ptr, 
			cActiveVoxelNeighbors_pitched.pitch,
			0, 
			cActiveVoxelNeighbors_pitched.xsize * sizeof(int), 
			cActiveVoxelNeighbors_pitched.ysize)	);

	#if	0	// DEL-BY-LEETEN 12/20/2009-BEGIN
		cudaPitchedPtr cActiveVoxelSortedNeighbors_pitched = cActiveVoxelNeighbors_pitched;
		CUDA_SAFE_CALL_NO_SYNC(
			cudaMallocPitch(
				(void**)&cActiveVoxelSortedNeighbors_pitched.ptr, 
				&cActiveVoxelSortedNeighbors_pitched.pitch,
				cActiveVoxelSortedNeighbors_pitched.xsize * sizeof(int), 
				cActiveVoxelSortedNeighbors_pitched.ysize)	);
	#endif	// DEL-BY-LEETEN 12/20/2009-END

	// bind the input vin volume to the texture that represents the src. bin volume 
	t2dSrcBinVolume.addressMode[0] = cudaAddressModeClamp;
	t2dSrcBinVolume.addressMode[1] = cudaAddressModeClamp;
	t2dSrcBinVolume.filterMode =	cudaFilterModePoint;
	t2dSrcBinVolume.normalized =	false;
	CUDA_SAFE_CALL_NO_SYNC(
		cudaBindTexture2D(
			0, 
			t2dSrcBinVolume, 
			cBinVolume_pitched.ptr, 
			cudaCreateChannelDesc<int>(),
			cVolumeExtent_array.width, 
			cVolumeExtent_array.height * cVolumeExtent_array.depth, 
			cBinVolume_pitched.pitch) );

	// bind the histogram as a texture
	#if	0	// MOD-BY-LEETEN 12/18/2009-FROM:
	#else	// MOD-BY-LEETEN 12/18/2009-TO:
	t2dActiveVoxelSortedNeighbors.addressMode[0] = cudaAddressModeClamp;
	t2dActiveVoxelSortedNeighbors.addressMode[1] = cudaAddressModeClamp;
	t2dActiveVoxelSortedNeighbors.filterMode =	cudaFilterModePoint;
	t2dActiveVoxelSortedNeighbors.normalized =	false;

	#if	0	// DEL-BY-LEETEN 12/20/2009-BEGIN
		CUDA_SAFE_CALL_NO_SYNC(
			cudaBindTexture2D(
				0, 
				t2dActiveVoxelSortedNeighbors, 
				cActiveVoxelSortedNeighbors_pitched.ptr, 
				cudaCreateChannelDesc<unsigned int>(),
				cActiveVoxelSortedNeighbors_pitched.xsize,
				cActiveVoxelSortedNeighbors_pitched.ysize, 
				cActiveVoxelSortedNeighbors_pitched.pitch) );
	#else	// DEL-BY-LEETEN 12/20/2009-END
	CUDA_SAFE_CALL_NO_SYNC(
		cudaBindTexture2D(
			0, 
			t2dActiveVoxelSortedNeighbors, 
			cActiveVoxelNeighbors_pitched.ptr, 
			cudaCreateChannelDesc<unsigned int>(),
			cActiveVoxelNeighbors_pitched.xsize,
			cActiveVoxelNeighbors_pitched.ysize, 
			cActiveVoxelNeighbors_pitched.pitch) );
#endif
	#endif	// MOD-BY-LEETEN 12/18/2009-END

	CUDPPHandle hScanPlan = 0;

	CUDPPConfiguration	cConfig;
	cConfig.op =		CUDPP_ADD;
	cConfig.datatype =	CUDPP_UINT;
	cConfig.algorithm =	CUDPP_SORT_RADIX;
	cConfig.options =	CUDPP_OPTION_KEYS_ONLY;

	#if	0	// MOD-BY-LEETEN 12/20/2009-FROM:
		assert( 
			CUDPP_SUCCESS  == cudppPlan(
				&hScanPlan,	
				cConfig, 
				cActiveVoxelSortedNeighbors_pitched.pitch * cActiveVoxelSortedNeighbors_pitched.ysize / sizeof(unsigned int),
				1, 
				0) );
	#else	// MOD-BY-LEETEN 12/20/2009-TO:
	assert( 
		CUDPP_SUCCESS  == cudppPlan(
			&hScanPlan,	
			cConfig, 
			cActiveVoxelNeighbors_pitched.pitch * cActiveVoxelNeighbors_pitched.ysize / sizeof(unsigned int),
			1, 
			0) );
	#endif	// MOD-BY-LEETEN 12/20/2009-END

CLOCK_END(_ComputeEntropyVolume_PRINT_TIMING, false);

CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_TIMING);

	#if	0	// MOD-BY-LEETEN 12/19/2009-FROM:
	#else	// MOD-BY-LEETEN 12/19/2009-TO:
	for(int			y = 0; y < i3VolumeSize.y; y += v3Grid.y * v3Blk.y)
		for(int		x = 0; x < i3VolumeSize.x; x += v3Grid.x * v3Blk.x)
			for(int	z = 0; z < i3VolumeSize.z; z ++ )
	#endif	// MOD-BY-LEETEN 12/19/2009-END
			{
				CLOCK_INIT(_ComputeEntropyVolume_PRINT_LOOP_TIMING, __FUNCTION__ " (main loop): ");

				CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_LOOP_TIMING);

				int3 i3BlockCorner = make_int3(x, y, z);
				_CollectNeighbors_kernel<<<v3Grid, v3Blk, 0>>>
				(
					i3BlockCorner,
					i3KernelSize,
					i3VolumeSize,
					iNrOfBins,
					cActiveVoxelNeighbors_pitched
				);
				CUT_CHECK_ERROR("_CollectNeighbors_kernel() failed");

				#if	0	// TEST-DEBUG
											unsigned int *puActiveVoxelNeighbors_host;
											CUDA_SAFE_CALL_NO_SYNC(
												cudaMallocHost(
													(void**)&puActiveVoxelNeighbors_host,
													cActiveVoxelNeighbors_pitched.pitch * cActiveVoxelSortedNeighbors_pitched.ysize) );

											CUDA_SAFE_CALL_NO_SYNC(
												cudaMemcpy(
													puActiveVoxelNeighbors_host, 
													cActiveVoxelNeighbors_pitched.ptr,
													cActiveVoxelSortedNeighbors_pitched.pitch * cActiveVoxelSortedNeighbors_pitched.ysize,
													cudaMemcpyDeviceToHost) );
											for(unsigned int p = 0,	n = 0; n < cActiveVoxelSortedNeighbors_pitched.ysize; n++)
												for(unsigned int	i = 0; i < cActiveVoxelNeighbors_pitched.pitch / sizeof(unsigned int); i++, p++)
												{
													unsigned int uTemp = puActiveVoxelNeighbors_host[p];
													unsigned int uId = uTemp / unsigned int(iNrOfBins);
													unsigned int uBin = uTemp % unsigned int(iNrOfBins);

													fprintf(stderr, "%d, %d, %d\n", n, uId, uBin);
												}
											FREE_MEMORY_ON_HOST(puActiveVoxelNeighbors_host);
				#endif

CLOCK_END(_ComputeEntropyVolume_PRINT_LOOP_TIMING, false);

#if	0	// DEL-BY-LEETEN 12/20/2009-BEGIN
	CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_LOOP_TIMING);

					// copy the memory from the buffer _CollectNeighbors_kernel to another buffer _CollectNeighborsToBeSorted_kernel
					#if	0	// MOD-BY-LEETEN 12/18/2009-FROM:
						CUDA_SAFE_CALL_NO_SYNC(
							cudaMemcpy(
								cActiveVoxelSortedNeighbors_pitched.ptr, 
								cActiveVoxelNeighbors_pitched.ptr, 
								cActiveVoxelSortedNeighbors_pitched.pitch * cActiveVoxelSortedNeighbors_pitched.ysize,
								cudaMemcpyDeviceToDevice));
					#else	// MOD-BY-LEETEN 12/18/2009-TO:
					CUDA_SAFE_CALL_NO_SYNC(
						cudaMemcpy2D(
							cActiveVoxelSortedNeighbors_pitched.ptr, 
							cActiveVoxelSortedNeighbors_pitched.pitch,
							cActiveVoxelNeighbors_pitched.ptr, 
							cActiveVoxelNeighbors_pitched.pitch,
							cActiveVoxelSortedNeighbors_pitched.pitch,
							cActiveVoxelSortedNeighbors_pitched.ysize,
							cudaMemcpyDeviceToDevice) );
					#endif	// MOD-BY-LEETEN 12/18/2009-END

	CLOCK_END(_ComputeEntropyVolume_PRINT_LOOP_TIMING, false);
#endif	// DEL-BY-LEETEN 12/20/2009-END

CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_LOOP_TIMING);

				// call cudpp to sort the buffer _CollectNeighborsToBeSorted_kernel
				#if	0	// MOD-BY-LEETEN 12/20/2009-FROM:
					cudppSort(
						hScanPlan,
						cActiveVoxelSortedNeighbors_pitched.ptr,
						NULL,
						RADIX_SORT_BITS,
						cActiveVoxelSortedNeighbors_pitched.pitch * cActiveVoxelSortedNeighbors_pitched.ysize / sizeof(unsigned int)) ;
				#else	// MOD-BY-LEETEN 12/20/2009-TO:
				cudppSort(
					hScanPlan,
					cActiveVoxelNeighbors_pitched.ptr,
					NULL,
					RADIX_SORT_BITS,
					cActiveVoxelNeighbors_pitched.pitch * cActiveVoxelNeighbors_pitched.ysize / sizeof(unsigned int)) ;
				#endif	// MOD-BY-LEETEN 12/20/2009-END
				CUT_CHECK_ERROR("cudppSort() failed");
CLOCK_END(_ComputeEntropyVolume_PRINT_LOOP_TIMING, false);

CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_LOOP_TIMING);
				// 
				#if	0	// MOD-BY-LEETEN 12/20/2009-FROM:
					_ComputeEntropyOnSortedNeighbors_kernel<<<v3Grid, v3Blk, 0>>>
					(
						i3BlockCorner,
						iNrOfBins,
						i3KernelSize,
						(unsigned int *)cActiveVoxelSortedNeighbors_pitched.ptr,
						i3VolumeSize,
						cEntropyVolume_pitched
					);
				#else	// MOD-BY-LEETEN 12/20/2009-TO:
				_ComputeEntropyOnSortedNeighbors_kernel<<<v3Grid, v3Blk, 0>>>
				(
					i3BlockCorner,
					iNrOfBins,
					i3KernelSize,
					(unsigned int *)cActiveVoxelNeighbors_pitched.ptr,
					i3VolumeSize,
					cEntropyVolume_pitched
				);
				#endif	// MOD-BY-LEETEN 12/20/2009-END
				CUT_CHECK_ERROR("_ComputeEntropyOnSortedNeighbors_kernel() failed");
CLOCK_END(_ComputeEntropyVolume_PRINT_LOOP_TIMING, false);

CLOCK_PRINT(_ComputeEntropyVolume_PRINT_LOOP_TIMING);
			}


CLOCK_END(_ComputeEntropyVolume_PRINT_TIMING, false);

CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_TIMING);

	cudppDestroyPlan(hScanPlan);
	FREE_MEMORY(cActiveVoxelNeighbors_pitched.ptr);

CLOCK_END(_ComputeEntropyVolume_PRINT_TIMING, false);

CLOCK_PRINT(_ComputeEntropyVolume_PRINT_TIMING);
}

#endif	// #if COMPUTE_ENTROPY_VOLUME_WITH_SORTING_CUDA

/*

$Log: not supported by cvs2svn $

*/
