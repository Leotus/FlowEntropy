#include "hip/hip_runtime.h"
////////////////////////////////////////////
	#include <stdio.h>
	#include <>

	#if	0	// DEL-BY-LEETEN 2009/12/17-BEGIN
	#endif	// DEL-BY-LEETEN 2009/12/17-END


	// ADD-BY-LEETEN 12/07/2009-BEGIN
	#include "FlowDiffusion_cuda.h"
	#include "FlowDiffusion.h"
	// ADD-BY-LEETEN 12/07/2009-END	

////////////////////////////////////////////
	static hipExtent cVolumeExtent;
	static float4 *pf4Volume_host;
	static hipPitchedPtr pcVolumePtrs_global[2];
	static texture<float4, 2, hipReadModeElementType> t2dSrc;
	// ADD-BY-LEETEN 10/02/2009-BEGIN
						// array and texture reference for the weight and offset volume
	static hipExtent cVolumeExtent_array;
	static float4 *pf4WeightOffsetVolume_host;
	static hipArray *cWeightOffsetVolume_array = 0;
	static texture<float4, 3, hipReadModeElementType> t3dWeightOffset;
    static hipChannelFormatDesc cWeightOffsetChannelDesc = hipCreateChannelDesc<float4>();
	// ADD-BY-LEETEN 10/02/2009-END

	// ADD-BY-LEETEN 2009/11/25-BEGIN
	#if	CHECK_ERROR_CONVERGENCE_BY_CUDPP
    static CUDPPConfiguration cConfig;
    static CUDPPHandle cScanplan = 0;
	#endif	// #if	CHECK_ERROR_CONVERGENCE_BY_CUDPP

	// ADD-BY-LEETEN 12/18/2009-BEGIN
	#if	CHECK_ERROR_CONVERGENCE_BY_CUDPP				
	// ADD-BY-LEETEN 12/18/2009-END
	static hipExtent cErrorVolumeExtent;
	// ADD-BY-LEETEN 12/18/2009-BEGIN
	#endif	// #if	CHECK_ERROR_CONVERGENCE_BY_CUDPP	
	// ADD-BY-LEETEN 12/18/2009-END

	static hipPitchedPtr cErrorVolume_device;
	static hipPitchedPtr cErrorSum_device;
	// ADD-BY-LEETEN 2009/11/25-END

	// ADD-BY-LEETEN 12/14/2009-BEGIN
	int iNrOfVoxels;

	static texture<int, 2, hipReadModeElementType> t2dAngleMap;
	static hipArray *pcAngleMap_array;	// cuda array to hold the Gaussian kernels

	static hipExtent cBinVolumeExtent;
	#if	0	// MOD-BY-LEETEN 12/17/2009-FROM:
		static hipPitchedPtr cSrcBinVolumePtr_global;
		static hipPitchedPtr cDstBinVolumePtr_global;
	#else	// MOD-BY-LEETEN 12/17/2009-TO:
	static hipPitchedPtr cSrcBinVolume_pitched;
	static hipPitchedPtr cDstBinVolume_pitched;
	#endif	// MOD-BY-LEETEN 12/17/2009-END

	#if	0	// MOD-BY-LEETEN 2009/12/17-FROM:
		static int *piSrcAngleBinVolume_host;
		static int *piDstAngleBinVolume_host;
	#else	// MOD-BY-LEETEN 2009/12/17-TO:
						// a temp. volume in the host side to stored the 
	static int *piBinVolume_host;
	#endif	// MOD-BY-LEETEN 2009/12/17-END

						// a texture tha represents a vector field to be converted into bins 
	static texture<float4, 2, hipReadModeElementType> t2dVectorVolume;
	// ADD-BY-LEETEN 12/14/2009-END

////////////////////////////////////////////
// ADD-BY-LEETEN 12/07/2009-BEGIN
#if	0	// MOD-BY-LEETEN 2009/12/17-FROM:
#else	// MOD-BY-LEETEN 2009/12/17-TO:
	#include "FlowDiffusion2D_kernel.cu"
#endif	// MOD-BY-LEETEN 2009/12/17-END

// ADD-BY-LEETEN 12/07/2009-END

#if	0		// DEL-BY-LEETEN 2009/12/17-BEGIN
#endif	// DEL-BY-LEETEN 2009/12/17-END

// ADD-BY-LEETEN 12/17/2009-BEGIN
////////////////////////////////////////////

#include "EntropyField_kernel.cu"

	// ADD-BY-LEETEN 12/23/2009-BEGIN
	#if	COMPUTE_ENTROPY_VOLUME_HOST
		#include	"EntropyField_host.cu"
	#endif

	#if	COMPUTE_ENTROPY_VOLUME_CUDA
		#include	"EntropyField_cuda.cu"
	#endif

	#if	COMPUTE_ENTROPY_VOLUME_WITH_SORTING_CUDA	
		#include	"EntropyFieldWithSorting_cuda.cu"
	#endif

	#if	COMPUTE_ENTROPY_VOLUME_PER_SCANLINE_CUDA
		#include	"EntropyFieldPerScanline_cuda.cu"
	#endif

	#if	COMPUTE_ENTROPY_VOLUME_SORT_PER_VOXEL_CUDA	
		#include	"EntropyFieldSortPerVoxel_cuda.cu"
	#endif

	#if	COMPUTE_ENTROPY_VOLUME_ON_SPARSE_HISTOGRAM_CUDA	
		#include	"EntropyFieldOnSparseHistogram_cuda.cu"
	#endif

	#if	COMPUTE_ENTROPY_VOLUME_WITH_MARGINAL_HISTOGRAM	
		#include	"EntropyFieldWithMarginalHistogram_cuda.cu"
	#endif

	// ADD-BY-LEETEN 12/29/2009-BEGIN
	#if	COMPUTE_ENTROPY_VOLUME_WITH_SORTING_IN_VECTORS_CUDA	
		#include	"EntropyFieldWithSortingInVectors_cuda.cu"
	#endif
	// ADD-BY-LEETEN 12/29/2009-END

	// ADD-BY-LEETEN 12/23/2009-END

////////////////////////////////////////////

#if	0	// MOD-BY-LEETEN 01/27/2010-FROM:
	void
	_GetSrcEntropyVolume
	(
		int iNrOfBins,
		int iKernelWidth, int iKernelHeight, int iKernelDepth
	)
	{
		int *piHistogram_global;
		CUDA_SAFE_CALL_NO_SYNC(
			hipMalloc(	(void**)&piHistogram_global, sizeof(piHistogram_global[0]) * iNrOfBins) );
		CUDA_SAFE_CALL_NO_SYNC(
			hipMemset(	(void*)piHistogram_global, 0, sizeof(piHistogram_global[0]) * iNrOfBins) );

		float *pfLogHistogram_global;
		CUDA_SAFE_CALL_NO_SYNC(
			hipMalloc(	(void**)&pfLogHistogram_global, sizeof(pfLogHistogram_global[0]) * iNrOfBins) );
		CUDA_SAFE_CALL_NO_SYNC(
			hipMemset(	(void*)pfLogHistogram_global, 0, sizeof(pfLogHistogram_global[0]) * iNrOfBins) );

		hipExtent cEntropyVolumeExtent	= make_hipExtent(
			cVolumeExtent_array.width * sizeof(float),
			cVolumeExtent_array.height,
			cVolumeExtent_array.depth);
		hipPitchedPtr cEntropyVolume_pitched;
		CUDA_SAFE_CALL( 
			hipMalloc3D(&cEntropyVolume_pitched, cEntropyVolumeExtent) );

		// DEL-BY-LEETEN 12/18/2009-BEGIN
			// CUDPPHandle hScanPlanEntropy = CInitEntropyField(iNrOfBins);
		// DEL-BY-LEETEN 12/18/2009-END

		#if	0	// MOD-BY-LEETEN 12/18/2009-FROM:
			_ComputeEntropyVolume
			(
								// res. of the neighboring region
				make_int3(iKernelWidth, iKernelHeight, iKernelDepth),

								// the histogram
				iNrOfBins,
				piHistogram_global,
				pfLogHistogram_global,

				hScanPlanEntropy,
								// res. of the volume																
				make_int3(cVolumeExtent_array.width, cVolumeExtent_array.height, cVolumeExtent_array.depth),

								// bin volume																	
				cSrcBinVolume_pitched,
				cEntropyVolume_pitched
			);
		#else	// MOD-BY-LEETEN 12/18/2009-TO:

		#if	COMPUTE_ENTROPY_VOLUME == COMPUTE_ENTROPY_VOLUME_CUDA
			_ComputeEntropyVolume_cuda
		#endif

		#if	COMPUTE_ENTROPY_VOLUME == COMPUTE_ENTROPY_VOLUME_HOST
			_ComputeEntropyVolume_host
		#endif

		#if	COMPUTE_ENTROPY_VOLUME == COMPUTE_ENTROPY_VOLUME_WITH_SORTING_CUDA
			_ComputeEntropyVolumeWithSorting_cuda
		#endif	

		// ADD-BY-LEETEN 12/20/2009-BEGIN
		#if	COMPUTE_ENTROPY_VOLUME == COMPUTE_ENTROPY_VOLUME_PER_SCANLINE_CUDA	
			_ComputeEntropyVolumePerScanline_cuda
		#endif	
		// ADD-BY-LEETEN 12/20/2009-END

		// ADD-BY-LEETEN 12/23/2009-BEGIN
		#if	COMPUTE_ENTROPY_VOLUME == COMPUTE_ENTROPY_VOLUME_SORT_PER_VOXEL_CUDA	
			_ComputeEntropyVolumePerVoxel_cuda
		#endif	

		#if	COMPUTE_ENTROPY_VOLUME == COMPUTE_ENTROPY_VOLUME_ON_SPARSE_HISTOGRAM_CUDA
			_ComputeEntropyVolumeOnSparseHistogram_cuda
		#endif	
		// ADD-BY-LEETEN 12/23/2009-END

		// ADD-BY-LEETEN 12/29/2009-BEGIN
		#if	COMPUTE_ENTROPY_VOLUME == COMPUTE_ENTROPY_VOLUME_WITH_SORTING_IN_VECTORS_CUDA
			_ComputeEntropyVolumeWithSortingInVectors_cuda
		#endif	
		// ADD-BY-LEETEN 12/29/2009-END

		(
							// res. of the neighboring region
			make_int3(iKernelWidth, iKernelHeight, iKernelDepth),

							// the histogram
			iNrOfBins,
			piHistogram_global,
			pfLogHistogram_global,

							// res. of the volume																
			make_int3(int(cVolumeExtent_array.width), int(cVolumeExtent_array.height), int(cVolumeExtent_array.depth)),

							// bin volume																	
			cSrcBinVolume_pitched,
			cEntropyVolume_pitched
		);
		#endif	// MOD-BY-LEETEN 12/18/2009-END

		// ADD-BY-LEETEN 12/19/2009-BEGIN
		float *pfEntropyVolume_host;
		CUDA_SAFE_CALL(
			hipHostMalloc(
				(void**)&pfEntropyVolume_host,
				sizeof(pfEntropyVolume_host[0]) * iNrOfVoxels) );
		CUDA_SAFE_CALL(
			hipMemcpy2D(
				pfEntropyVolume_host, 
				cVolumeExtent_array.width * sizeof(pfEntropyVolume_host[0]),
				cEntropyVolume_pitched.ptr,
				cEntropyVolume_pitched.pitch,
				cVolumeExtent_array.width * sizeof(pfEntropyVolume_host[0]),
				cVolumeExtent_array.height * cVolumeExtent_array.depth,
				hipMemcpyDeviceToHost) );

		#if	1	// TEST-DEBUG
		FILE *fpFile;

		#if	0		// DEL-BY-LEETEN 01/03/2010-BEGIN
			// MOD-BY-LEETEN 12/20/2009-FROM:
				// fpFile = fopen( __FUNCTION__ "_" SRC_ENTROPY_VOLUME_POSTFIX ".txt", "wt");
			// TO:
			fpFile = fopen( __FUNCTION__ "_" ENTROPY_VOLUME_POSTFIX ".txt", "wt");
			// MOD-BY-LEETEN 12/20/2009-END
			assert(fpFile);

			for(int	v = 0,		z = 0; z < int(cVolumeExtent_array.depth);	z++)
				for(int			y = 0; y < int(cVolumeExtent_array.height);	y++)
					for(int		x = 0; x < int(cVolumeExtent_array.width);	x++, v++)
						fprintf(fpFile, "E(%d, %d, %d) = %.4f\n", x, y, z, pfEntropyVolume_host[v]);

			fclose(fpFile);
		#endif		// DEL-BY-LEETEN 01/03/2010-END

		// ADD-BY-LEETEN 12/28/2009-BEGIN
		// MOD-BY-LEETEN 01/03/2010-FROM:
			// fpFile = fopen( __FUNCTION__ "_" ENTROPY_VOLUME_POSTFIX ".bin", "wb");
		// TO:
		extern char* g_filename;
		static char szInputFilename[1024+1];
		static char szOutputFilename[1024+1];
		char *szInputFilenameDot;
		strcpy(szInputFilename, g_filename);
		if( NULL != (szInputFilenameDot = strrchr(szInputFilename, '.')) )
			*szInputFilenameDot = '\0';
		sprintf(szOutputFilename, "%s_entropy_" ENTROPY_VOLUME_POSTFIX ".bin", szInputFilename);
		fpFile = fopen(szOutputFilename, "wb");
		// MOD-BY-LEETEN 01/03/2010-END
		assert(fpFile);
		fwrite(&cVolumeExtent_array.width,	sizeof(cVolumeExtent_array.width), 1, fpFile);
		fwrite(&cVolumeExtent_array.height, sizeof(cVolumeExtent_array.height), 1, fpFile);
		fwrite(&cVolumeExtent_array.depth,	sizeof(cVolumeExtent_array.depth), 1, fpFile);
		fwrite(pfEntropyVolume_host, 
			sizeof(pfEntropyVolume_host[0]), 
			cVolumeExtent_array.depth * cVolumeExtent_array.height * cVolumeExtent_array.width, 
			fpFile);
		fclose(fpFile);
		// ADD-BY-LEETEN 12/28/2009-END

		#endif
		FREE_MEMORY_ON_HOST(pfEntropyVolume_host);
		// ADD-BY-LEETEN 12/19/2009-END

		FREE_MEMORY(piHistogram_global);
		FREE_MEMORY(pfLogHistogram_global);
		FREE_MEMORY(cEntropyVolume_pitched.ptr);
		// DEL-BY-LEETEN 12/18/2009-BEGIN
			// cudppDestroyPlan(hScanPlanEntropy);
		// DEL-BY-LEETEN 12/18/2009-END
	}

#else	// MOD-BY-LEETEN 01/27/2010-TO:

void
_ComputeSrcEntropyVolume
(
	int iNrOfBins,
	int iKernelWidth, int iKernelHeight, int iKernelDepth,
	float *pfEntropyVolume_host
)
{
	int *piHistogram_global;
	CUDA_SAFE_CALL_NO_SYNC(
		hipMalloc(	(void**)&piHistogram_global, sizeof(piHistogram_global[0]) * iNrOfBins) );
	CUDA_SAFE_CALL_NO_SYNC(
		hipMemset(	(void*)piHistogram_global, 0, sizeof(piHistogram_global[0]) * iNrOfBins) );

	float *pfLogHistogram_global;
	CUDA_SAFE_CALL_NO_SYNC(
		hipMalloc(	(void**)&pfLogHistogram_global, sizeof(pfLogHistogram_global[0]) * iNrOfBins) );
	CUDA_SAFE_CALL_NO_SYNC(
		hipMemset(	(void*)pfLogHistogram_global, 0, sizeof(pfLogHistogram_global[0]) * iNrOfBins) );

	hipExtent cEntropyVolumeExtent	= make_hipExtent(
		cVolumeExtent_array.width * sizeof(float),
		cVolumeExtent_array.height,
		cVolumeExtent_array.depth);
	hipPitchedPtr cEntropyVolume_pitched;
	CUDA_SAFE_CALL( 
		hipMalloc3D(&cEntropyVolume_pitched, cEntropyVolumeExtent) );

	#if	COMPUTE_ENTROPY_VOLUME == COMPUTE_ENTROPY_VOLUME_CUDA
		_ComputeEntropyVolume_cuda
	#endif

	#if	COMPUTE_ENTROPY_VOLUME == COMPUTE_ENTROPY_VOLUME_HOST
		_ComputeEntropyVolume_host
	#endif

	#if	COMPUTE_ENTROPY_VOLUME == COMPUTE_ENTROPY_VOLUME_WITH_SORTING_CUDA
		_ComputeEntropyVolumeWithSorting_cuda
	#endif	

	// ADD-BY-LEETEN 12/20/2009-BEGIN
	#if	COMPUTE_ENTROPY_VOLUME == COMPUTE_ENTROPY_VOLUME_PER_SCANLINE_CUDA	
		_ComputeEntropyVolumePerScanline_cuda
	#endif	
	// ADD-BY-LEETEN 12/20/2009-END

	// ADD-BY-LEETEN 12/23/2009-BEGIN
	#if	COMPUTE_ENTROPY_VOLUME == COMPUTE_ENTROPY_VOLUME_SORT_PER_VOXEL_CUDA	
		_ComputeEntropyVolumePerVoxel_cuda
	#endif	

	#if	COMPUTE_ENTROPY_VOLUME == COMPUTE_ENTROPY_VOLUME_ON_SPARSE_HISTOGRAM_CUDA
		_ComputeEntropyVolumeOnSparseHistogram_cuda
	#endif	
	// ADD-BY-LEETEN 12/23/2009-END

	// ADD-BY-LEETEN 12/29/2009-BEGIN
	#if	COMPUTE_ENTROPY_VOLUME == COMPUTE_ENTROPY_VOLUME_WITH_SORTING_IN_VECTORS_CUDA
		_ComputeEntropyVolumeWithSortingInVectors_cuda
	#endif	
	// ADD-BY-LEETEN 12/29/2009-END

	(
						// res. of the neighboring region
		make_int3(iKernelWidth, iKernelHeight, iKernelDepth),

						// the histogram
		iNrOfBins,
		piHistogram_global,
		pfLogHistogram_global,

						// res. of the volume																
		make_int3(int(cVolumeExtent_array.width), int(cVolumeExtent_array.height), int(cVolumeExtent_array.depth)),

						// bin volume																	
		cSrcBinVolume_pitched,
		cEntropyVolume_pitched
	);

	if( NULL != pfEntropyVolume_host )
		CUDA_SAFE_CALL(
			hipMemcpy2D(
				pfEntropyVolume_host, 
				cVolumeExtent_array.width * sizeof(pfEntropyVolume_host[0]),
				cEntropyVolume_pitched.ptr,
				cEntropyVolume_pitched.pitch,
				cVolumeExtent_array.width * sizeof(pfEntropyVolume_host[0]),
				cVolumeExtent_array.height * cVolumeExtent_array.depth,
				hipMemcpyDeviceToHost) );

	FREE_MEMORY(piHistogram_global);
	FREE_MEMORY(pfLogHistogram_global);
	FREE_MEMORY(cEntropyVolume_pitched.ptr);
}	
#endif	// MOD-BY-LEETEN 01/27/2010-END

#if	0	// MOD-BY-LEETEN 02/02/2010-FROM:
	void
	_GetJointEntropyVolume
	(
		int iNrOfSrcBins,
		int iNrOfDstBins,
		int iKernelWidth, int iKernelHeight, int iKernelDepth
	)
	{
		int iNrOfBins = iNrOfSrcBins * iNrOfDstBins;

		int *piHistogram_global;
		CUDA_SAFE_CALL_NO_SYNC(
			hipMalloc(	(void**)&piHistogram_global, sizeof(piHistogram_global[0]) * iNrOfBins) );
		CUDA_SAFE_CALL_NO_SYNC(
			hipMemset(	(void*)piHistogram_global, 0, sizeof(piHistogram_global[0]) * iNrOfBins) );

		float *pfLogHistogram_global;
		CUDA_SAFE_CALL_NO_SYNC(
			hipMalloc(	(void**)&pfLogHistogram_global, sizeof(pfLogHistogram_global[0]) * iNrOfBins) );
		CUDA_SAFE_CALL_NO_SYNC(
			hipMemset(	(void*)pfLogHistogram_global, 0, sizeof(pfLogHistogram_global[0]) * iNrOfBins) );

		hipExtent cEntropyVolumeExtent	= make_hipExtent(
			cVolumeExtent_array.width * sizeof(float),
			cVolumeExtent_array.height,
			cVolumeExtent_array.depth);
		hipPitchedPtr cEntropyVolume_pitched;
		CUDA_SAFE_CALL( 
			hipMalloc3D(&cEntropyVolume_pitched, cEntropyVolumeExtent) );

		// DEL-BY-LEETEN 12/18/2009-BEGIN
			// CUDPPHandle hScanPlanEntropy = CInitEntropyField(iNrOfBins);
		// DEL-BY-LEETEN 12/18/2009-END

		// bind the bin volume as a 2D texture
		t2dSrcBinVolume.addressMode[0] = hipAddressModeClamp;
		t2dSrcBinVolume.addressMode[1] = hipAddressModeClamp;
		t2dSrcBinVolume.filterMode =	hipFilterModePoint;
		t2dSrcBinVolume.normalized =	false;
		CUDA_SAFE_CALL_NO_SYNC(
			hipBindTexture2D(
				0, 
				t2dSrcBinVolume, 
				cSrcBinVolume_pitched.ptr, 
				hipCreateChannelDesc<int>(),
				cVolumeExtent_array.width, 
				cVolumeExtent_array.height * cVolumeExtent_array.depth, 
				cSrcBinVolume_pitched.pitch) );

		t2dDstBinVolume.addressMode[0] = hipAddressModeClamp;
		t2dDstBinVolume.addressMode[1] = hipAddressModeClamp;
		t2dDstBinVolume.filterMode =	hipFilterModePoint;
		t2dDstBinVolume.normalized =	false;
		CUDA_SAFE_CALL_NO_SYNC(
			hipBindTexture2D(
				0, 
				t2dDstBinVolume, 
				cDstBinVolume_pitched.ptr, 
				hipCreateChannelDesc<int>(),
				cVolumeExtent_array.width, 
				cVolumeExtent_array.height * cVolumeExtent_array.depth, 
				cDstBinVolume_pitched.pitch) );

		// bind the histogram as a 1D texture
		// ...
		hipPitchedPtr cJointBinVolume_pitched;
		CUDA_SAFE_CALL( 
			hipMalloc3D(
				&cJointBinVolume_pitched, 
				make_hipExtent(
					cVolumeExtent_array.width * sizeof(int),
					cVolumeExtent_array.height,
					cVolumeExtent_array.depth) ) );

		dim3 v3Blk, v3Grid;
		v3Blk = dim3(BLOCK_DIM_X, BLOCK_DIM_Y);
		v3Grid = dim3(
			size_t(ceilf(float(cVolumeExtent_array.width) / float(v3Blk.x))),
			size_t(ceilf(float(cVolumeExtent_array.height * cVolumeExtent_array.depth) / float(v3Blk.y)))
		);

		_JointSrcDst_kernel<<<v3Grid, v3Blk, 0>>>
		(
			iNrOfSrcBins, 
			iNrOfDstBins,

			make_int3(
				int(cVolumeExtent_array.width), 
				int(cVolumeExtent_array.height), 
				int(cVolumeExtent_array.depth)),

			cJointBinVolume_pitched
		);
		CUT_CHECK_ERROR("_JointSrcDst_kernel() failed");

	// MOD-BY-LEETEN 12/18/2009-FROM:
		// _ComputeEntropyVolume
	// TO:

		#if	COMPUTE_ENTROPY_VOLUME == COMPUTE_ENTROPY_VOLUME_CUDA
			_ComputeEntropyVolume_cuda
		#endif

		#if	COMPUTE_ENTROPY_VOLUME == COMPUTE_ENTROPY_VOLUME_HOST
			_ComputeEntropyVolume_host
		#endif

		#if	COMPUTE_ENTROPY_VOLUME == COMPUTE_ENTROPY_VOLUME_WITH_SORTING_CUDA
			_ComputeEntropyVolumeWithSorting_cuda
		#endif

		// ADD-BY-LEETEN 12/20/2009-BEGIN
		#if	COMPUTE_ENTROPY_VOLUME == COMPUTE_ENTROPY_VOLUME_PER_SCANLINE_CUDA	
			_ComputeEntropyVolumePerScanline_cuda
		#endif	
		// ADD-BY-LEETEN 12/20/2009-END

		// ADD-BY-LEETEN 12/23/2009-END
		#if	COMPUTE_ENTROPY_VOLUME == COMPUTE_ENTROPY_VOLUME_SORT_PER_VOXEL_CUDA	
			_ComputeEntropyVolumePerVoxel_cuda
		#endif	

		#if	COMPUTE_ENTROPY_VOLUME == COMPUTE_ENTROPY_VOLUME_ON_SPARSE_HISTOGRAM_CUDA
			_ComputeEntropyVolumeOnSparseHistogram_cuda
		#endif	 

		#if	COMPUTE_ENTROPY_VOLUME == COMPUTE_ENTROPY_VOLUME_WITH_MARGINAL_HISTOGRAM
			_ComputeEntropyVolumeWithMarginalHistogram_cuda
		#endif	
		// ADD-BY-LEETEN 12/23/2009-END

		#if	COMPUTE_ENTROPY_VOLUME == COMPUTE_ENTROPY_VOLUME_WITH_SORTING_IN_VECTORS_CUDA
			_ComputeEntropyVolumeWithSortingInVectors_cuda
		#endif	
	// MOD-BY-LEETEN 12/18/2009-END
		(
							// res. of the neighboring region
			make_int3(iKernelWidth, iKernelHeight, iKernelDepth),

							// the histogram
			iNrOfBins,
			piHistogram_global,
			pfLogHistogram_global,

			// DEL-BY-LEETEN 12/18/2009-BEGIN
				// hScanPlanEntropy,
			// DEL-BY-LEETEN 12/18/2009-END

							// res. of the volume																
			make_int3(int(cVolumeExtent_array.width), int(cVolumeExtent_array.height), int(cVolumeExtent_array.depth)),

							// bin volume																	
			cJointBinVolume_pitched,
			cEntropyVolume_pitched
		);

		// ADD-BY-LEETEN 12/19/2009-BEGIN
		float *pfEntropyVolume_host;
		CUDA_SAFE_CALL(
			hipHostMalloc(
				(void**)&pfEntropyVolume_host,
				sizeof(pfEntropyVolume_host[0]) * iNrOfVoxels) );
		CUDA_SAFE_CALL_NO_SYNC(
			hipMemcpy2D(
				pfEntropyVolume_host, 
				cVolumeExtent_array.width * sizeof(pfEntropyVolume_host[0]),
				cEntropyVolume_pitched.ptr,
				cEntropyVolume_pitched.pitch,
				cVolumeExtent_array.width * sizeof(pfEntropyVolume_host[0]),
				cVolumeExtent_array.height * cVolumeExtent_array.depth,
				hipMemcpyDeviceToHost) );
		 /*
		for(int	v = 0,		z = 0; z < i3VolumeSize.z; z++)
			for(int			y = 0; y < i3VolumeSize.y; y++)
				for(int		x = 0; x < i3VolumeSize.x; x++, v++)
					fprintf(stderr, "E(%d, %d, %d) = %f\n", x, y, z, pfEntropyVolume_host[v]);
		*/
		#if	0	// TEST-DEBUG
		FILE *fpFile;
		fpFile = fopen( __FUNCTION__ "_" ENTROPY_VOLUME_POSTFIX ".txt", "wt");
		assert(fpFile);

		for(int	v = 0,		z = 0; z < int(cVolumeExtent_array.depth);	z++)
			for(int			y = 0; y < int(cVolumeExtent_array.height);	y++)
				for(int		x = 0; x < int(cVolumeExtent_array.width);	x++, v++)
					fprintf(fpFile, "E(%d, %d, %d) = %.4f\n", x, y, z, pfEntropyVolume_host[v]);

		fclose(fpFile);

		// ADD-BY-LEETEN 12/28/2009-BEGIN
		fpFile = fopen( __FUNCTION__ "_" ENTROPY_VOLUME_POSTFIX ".bin", "wb");
		assert(fpFile);
		fwrite(&cVolumeExtent_array.width,	sizeof(cVolumeExtent_array.width), 1, fpFile);
		fwrite(&cVolumeExtent_array.height, sizeof(cVolumeExtent_array.height), 1, fpFile);
		fwrite(&cVolumeExtent_array.depth,	sizeof(cVolumeExtent_array.depth), 1, fpFile);
		fwrite(
			pfEntropyVolume_host, 
			sizeof(pfEntropyVolume_host[0]), 
			cVolumeExtent_array.depth * cVolumeExtent_array.height * cVolumeExtent_array.width,
			fpFile);
		fclose(fpFile);
		// ADD-BY-LEETEN 12/28/2009-END

		#endif

		FREE_MEMORY_ON_HOST(pfEntropyVolume_host);
		// ADD-BY-LEETEN 12/19/2009-END

		FREE_MEMORY(cJointBinVolume_pitched.ptr);
		FREE_MEMORY(piHistogram_global);
		FREE_MEMORY(pfLogHistogram_global);
		FREE_MEMORY(cEntropyVolume_pitched.ptr);
		// DEL-BY-LEETEN 12/18/2009-BEGIN
			// cudppDestroyPlan(hScanPlanEntropy);
		// DEL-BY-LEETEN 12/18/2009-END
	}
#else	// MOD-BY-LEETEN 02/02/2010-TO:
void
_ComputeJointEntropyVolume
(
	int iNrOfSrcBins,
	int iNrOfDstBins,
	int iKernelWidth, int iKernelHeight, int iKernelDepth,
	float *pfEntropyVolume_host
)
{
	int iNrOfBins = iNrOfSrcBins * iNrOfDstBins;

	int *piHistogram_global;
	CUDA_SAFE_CALL_NO_SYNC(
		hipMalloc(	(void**)&piHistogram_global, sizeof(piHistogram_global[0]) * iNrOfBins) );
	CUDA_SAFE_CALL_NO_SYNC(
		hipMemset(	(void*)piHistogram_global, 0, sizeof(piHistogram_global[0]) * iNrOfBins) );

	float *pfLogHistogram_global;
	CUDA_SAFE_CALL_NO_SYNC(
		hipMalloc(	(void**)&pfLogHistogram_global, sizeof(pfLogHistogram_global[0]) * iNrOfBins) );
	CUDA_SAFE_CALL_NO_SYNC(
		hipMemset(	(void*)pfLogHistogram_global, 0, sizeof(pfLogHistogram_global[0]) * iNrOfBins) );

	hipExtent cEntropyVolumeExtent	= make_hipExtent(
		cVolumeExtent_array.width * sizeof(float),
		cVolumeExtent_array.height,
		cVolumeExtent_array.depth);
	hipPitchedPtr cEntropyVolume_pitched;
	CUDA_SAFE_CALL( 
		hipMalloc3D(&cEntropyVolume_pitched, cEntropyVolumeExtent) );

	// DEL-BY-LEETEN 12/18/2009-BEGIN
		// CUDPPHandle hScanPlanEntropy = CInitEntropyField(iNrOfBins);
	// DEL-BY-LEETEN 12/18/2009-END

	// bind the bin volume as a 2D texture
	t2dSrcBinVolume.addressMode[0] = hipAddressModeClamp;
	t2dSrcBinVolume.addressMode[1] = hipAddressModeClamp;
	t2dSrcBinVolume.filterMode =	hipFilterModePoint;
	t2dSrcBinVolume.normalized =	false;
	CUDA_SAFE_CALL_NO_SYNC(
		hipBindTexture2D(
			0, 
			t2dSrcBinVolume, 
			cSrcBinVolume_pitched.ptr, 
			hipCreateChannelDesc<int>(),
			cVolumeExtent_array.width, 
			cVolumeExtent_array.height * cVolumeExtent_array.depth, 
			cSrcBinVolume_pitched.pitch) );

	t2dDstBinVolume.addressMode[0] = hipAddressModeClamp;
	t2dDstBinVolume.addressMode[1] = hipAddressModeClamp;
	t2dDstBinVolume.filterMode =	hipFilterModePoint;
	t2dDstBinVolume.normalized =	false;
	CUDA_SAFE_CALL_NO_SYNC(
		hipBindTexture2D(
			0, 
			t2dDstBinVolume, 
			cDstBinVolume_pitched.ptr, 
			hipCreateChannelDesc<int>(),
			cVolumeExtent_array.width, 
			cVolumeExtent_array.height * cVolumeExtent_array.depth, 
			cDstBinVolume_pitched.pitch) );

	// bind the histogram as a 1D texture
	// ...
	hipPitchedPtr cJointBinVolume_pitched;
	CUDA_SAFE_CALL( 
		hipMalloc3D(
			&cJointBinVolume_pitched, 
			make_hipExtent(
				cVolumeExtent_array.width * sizeof(int),
				cVolumeExtent_array.height,
				cVolumeExtent_array.depth) ) );

	dim3 v3Blk, v3Grid;
	v3Blk = dim3(BLOCK_DIM_X, BLOCK_DIM_Y);
	v3Grid = dim3(
		size_t(ceilf(float(cVolumeExtent_array.width) / float(v3Blk.x))),
		size_t(ceilf(float(cVolumeExtent_array.height * cVolumeExtent_array.depth) / float(v3Blk.y)))
	);

	_JointSrcDst_kernel<<<v3Grid, v3Blk, 0>>>
	(
		iNrOfSrcBins, 
		iNrOfDstBins,

		make_int3(
			int(cVolumeExtent_array.width), 
			int(cVolumeExtent_array.height), 
			int(cVolumeExtent_array.depth)),

		cJointBinVolume_pitched
	);
	CUT_CHECK_ERROR("_JointSrcDst_kernel() failed");

	#if	COMPUTE_ENTROPY_VOLUME == COMPUTE_ENTROPY_VOLUME_CUDA
		_ComputeEntropyVolume_cuda
	#endif

	#if	COMPUTE_ENTROPY_VOLUME == COMPUTE_ENTROPY_VOLUME_HOST
		_ComputeEntropyVolume_host
	#endif

	#if	COMPUTE_ENTROPY_VOLUME == COMPUTE_ENTROPY_VOLUME_WITH_SORTING_CUDA
		_ComputeEntropyVolumeWithSorting_cuda
	#endif

	// ADD-BY-LEETEN 12/20/2009-BEGIN
	#if	COMPUTE_ENTROPY_VOLUME == COMPUTE_ENTROPY_VOLUME_PER_SCANLINE_CUDA	
		_ComputeEntropyVolumePerScanline_cuda
	#endif	
	// ADD-BY-LEETEN 12/20/2009-END

	// ADD-BY-LEETEN 12/23/2009-END
	#if	COMPUTE_ENTROPY_VOLUME == COMPUTE_ENTROPY_VOLUME_SORT_PER_VOXEL_CUDA	
		_ComputeEntropyVolumePerVoxel_cuda
	#endif	

	#if	COMPUTE_ENTROPY_VOLUME == COMPUTE_ENTROPY_VOLUME_ON_SPARSE_HISTOGRAM_CUDA
		_ComputeEntropyVolumeOnSparseHistogram_cuda
	#endif	 

	#if	COMPUTE_ENTROPY_VOLUME == COMPUTE_ENTROPY_VOLUME_WITH_MARGINAL_HISTOGRAM
		_ComputeEntropyVolumeWithMarginalHistogram_cuda
	#endif	
	// ADD-BY-LEETEN 12/23/2009-END

	#if	COMPUTE_ENTROPY_VOLUME == COMPUTE_ENTROPY_VOLUME_WITH_SORTING_IN_VECTORS_CUDA
		_ComputeEntropyVolumeWithSortingInVectors_cuda
	#endif	
	(
						// res. of the neighboring region
		make_int3(iKernelWidth, iKernelHeight, iKernelDepth),

						// the histogram
		iNrOfBins,
		piHistogram_global,
		pfLogHistogram_global,

						// res. of the volume																
		make_int3(int(cVolumeExtent_array.width), int(cVolumeExtent_array.height), int(cVolumeExtent_array.depth)),

						// bin volume																	
		cJointBinVolume_pitched,
		cEntropyVolume_pitched
	);

	if( NULL != pfEntropyVolume_host )
		CUDA_SAFE_CALL_NO_SYNC(
			hipMemcpy2D(
				pfEntropyVolume_host, 
				cVolumeExtent_array.width * sizeof(pfEntropyVolume_host[0]),
				cEntropyVolume_pitched.ptr,
				cEntropyVolume_pitched.pitch,
				cVolumeExtent_array.width * sizeof(pfEntropyVolume_host[0]),
				cVolumeExtent_array.height * cVolumeExtent_array.depth,
				hipMemcpyDeviceToHost) );

	FREE_MEMORY(cJointBinVolume_pitched.ptr);
	FREE_MEMORY(piHistogram_global);
	FREE_MEMORY(pfLogHistogram_global);
	FREE_MEMORY(cEntropyVolume_pitched.ptr);
}
#endif	// MOD-BY-LEETEN 02/02/2010-END

// ADD-BY-LEETEN 12/17/2009-END

////////////////////////////////////////////
#if	0	// MOD-BY-LEETEN 12/16/2009-FROM:
#else	// MOD-BY-LEETEN 12/16/2009-TO:
	#include "FlowDiffusion3D_kernel.cu"
#endif	// MOD-BY-LEETEN 12/16/2009-END

extern "C" {

// MOD-BY-LEETEN 12/07/2009-FROM:
	// void _FlowFusionFree()
// TO:
void 
_FlowDiffusionFree()
// MOD-BY-LEETEN 12/07/2009-END
{
	FREE_MEMORY_ON_HOST(pf4Volume_host);
	for(int i = 0; i < 2; i++)
		FREE_MEMORY(pcVolumePtrs_global[i].ptr);
	FREE_ARRAY(cWeightOffsetVolume_array);
	FREE_MEMORY_ON_HOST(pf4WeightOffsetVolume_host);
	// ADD-BY-LEETEN 2009/11/25-BEGIN
	FREE_MEMORY(cErrorVolume_device.ptr);
	FREE_MEMORY(cErrorSum_device.ptr);

	#if	CHECK_ERROR_CONVERGENCE_BY_CUDPP
	cudppDestroyPlan(cScanplan);
	#endif	// #if	CHECK_ERROR_CONVERGENCE_BY_CUDPP
	// ADD-BY-LEETEN 2009/11/25-END

	// ADD-BY-LEETEN 12/14/2009-BEGIN
	FREE_ARRAY(pcAngleMap_array);	

	FREE_MEMORY(cSrcBinVolume_pitched.ptr);
	FREE_MEMORY(cDstBinVolume_pitched.ptr);
	// ADD-BY-LEETEN 12/17/2009-BEGIN
	FREE_MEMORY(cSrcDstBinVolumePtr_global.ptr);
	// ADD-BY-LEETEN 12/17/2009-END

	#if	0	// MOD-BY-LEETEN 12/16/2009-FROM:
		FREE_MEMORY_ON_HOST(piSrcAngleBinVolume_host);
		FREE_MEMORY_ON_HOST(piDstAngleBinVolume_host);
	#else	// MOD-BY-LEETEN 12/16/2009-TO:

	// DEL-BY-LEETEN 02/02/2010		FREE_MEMORY_ON_HOST(piBinVolume_host);
	#endif	// MOD-BY-LEETEN 12/16/2009-END
	// ADD-BY-LEETEN 12/14/2009-END
}

void
// MOD-BY-LEETEN 12/07/2009-FROM:
	// _FlowFusionInit(
// TO:
_FlowDiffusionInit(
// MOD-BY-LEETEN 12/07/2009-END
	int iVolumeWidth,
	int iVolumeHeight,
	int iVolumeDepth
)
{
	// MOD-BY-LEETEN 12/07/2009-FROM:
		// CLOCK_INIT(PRINT_FLOW_DIFFUSION_TIMING, "_FlowFusionInit(): ");
	// TO:
	CLOCK_INIT(PRINT_FLOW_DIFFUSION_TIMING, __FUNCTION__ ": ");
	// MOD-BY-LEETEN 12/07/2009-END

	// MOD-BY-LEETEN 12/14/2009-FROM:
		// int iNrOfVoxels = iVolumeWidth * iVolumeHeight * iVolumeDepth;
	// TO:
	iNrOfVoxels = iVolumeWidth * iVolumeHeight * iVolumeDepth;
	// MOD-BY-LEETEN 12/14/2009-END

	cVolumeExtent	= make_hipExtent(
		iVolumeWidth * sizeof(float4),
		iVolumeHeight,
		iVolumeDepth);

	// allocate a local copy in the CPU side
	CLOCK_BEGIN(PRINT_FLOW_DIFFUSION_TIMING);
	CUDA_SAFE_CALL(
		hipHostMalloc(
			(void**)&pf4Volume_host,
			sizeof(pf4Volume_host[0]) * iNrOfVoxels) );
	CLOCK_END(PRINT_FLOW_DIFFUSION_TIMING, true);

	// Allocate 2 volumes on the  GPU's global memory.
	// in each iteration, one of them will be used as the source
	// 3D texture, and the other will be served as the dst.
	CLOCK_BEGIN(PRINT_FLOW_DIFFUSION_TIMING);

	for(int i = 0; i < 2; i++)
	{
		CUDA_SAFE_CALL( 
			hipMalloc3D(&pcVolumePtrs_global[i], cVolumeExtent) );
	}

	// ADD-BY-LEETEN 2009/11/25-BEGIN
	// ADD-BY-LEETEN 2009/12/17-BEGIN
	#if	CHECK_ERROR_CONVERGENCE_BY_CUDPP	
	// ADD-BY-LEETEN 2009/12/17-END
	cErrorVolumeExtent	= make_hipExtent(
		// MOD-BY-LEETEN 12/16/2009-FROM:
			// iVolumeWidth,
		// TO:
		iVolumeWidth * sizeof(float),
		// MOD-BY-LEETEN 12/16/2009-END
		iVolumeHeight,
		iVolumeDepth);
	CUDA_SAFE_CALL( 
		hipMalloc3D(&cErrorVolume_device,	cErrorVolumeExtent) );
	CUDA_SAFE_CALL( 
		hipMalloc3D(&cErrorSum_device,		cErrorVolumeExtent) );

	// DEL-BY-LEETEN 12/16/2009-BEGIN
	// #if	CHECK_ERROR_CONVERGENCE_BY_CUDPP
	// DEL-BY-LEETEN 12/16/2009-END
	cConfig.op = CUDPP_ADD;
	// cConfig.op = CUDPP_MAX;
	cConfig.datatype = CUDPP_FLOAT;
	cConfig.algorithm = CUDPP_SCAN;
	#if	0	// MOD-BY-LEETEN 12/16/2009-FROM:
		cConfig.options = CUDPP_OPTION_FORWARD | CUDPP_OPTION_INCLUSIVE;
		cudppPlan(&cScanplan, cConfig, iVolumeWidth * iVolumeHeight * iVolumeDepth, iVolumeHeight * iVolumeDepth, cErrorVolume_device.pitch);  
	#else	// MOD-BY-LEETEN 12/16/2009-TO:
	cConfig.options = CUDPP_OPTION_BACKWARD | CUDPP_OPTION_INCLUSIVE;
	assert(CUDPP_SUCCESS  == cudppPlan(&cScanplan, cConfig, iVolumeWidth * iVolumeHeight * iVolumeDepth, iVolumeHeight * iVolumeDepth, cErrorVolume_device.pitch) );  
	#endif	// MOD-BY-LEETEN 12/16/2009-END
	// DEL-BY-LEETEN 2009/12/17-BEGIN
	// #endif	// #if	CHECK_ERROR_CONVERGENCE_BY_CUDPP
	// DEL-BY-LEETEN 2009/12/17-END
	// ADD-BY-LEETEN 2009/12/17-BEGIN
	#endif	// #if	CHECK_ERROR_CONVERGENCE_BY_CUDPP
	// ADD-BY-LEETEN 2009/12/17-END
	// ADD-BY-LEETEN 2009/11/25-END

	// ADD-BY-LEETEN 10/02/2009-BEGIN
	cVolumeExtent_array = make_hipExtent(
		iVolumeWidth,
		iVolumeHeight,
		iVolumeDepth);

	CUDA_SAFE_CALL( 
		hipMalloc3DArray(&cWeightOffsetVolume_array, &cWeightOffsetChannelDesc, cVolumeExtent_array) );

	CUDA_SAFE_CALL(
		hipHostMalloc(
			(void**)&pf4WeightOffsetVolume_host,
			sizeof(pf4WeightOffsetVolume_host[0]) * iNrOfVoxels) );

	// ADD-BY-LEETEN 10/02/2009-END
	CLOCK_END(PRINT_FLOW_DIFFUSION_TIMING, true);

	CLOCK_PRINT(PRINT_FLOW_DIFFUSION_TIMING);
}

// ADD-BY-LEETEN 12/14/2009-BEGIN
void 
_FlowDiffusionSetAngleMap(int *piAngleMap, int iNrOfPhis, int iNrOfThetas)
{
	// allocate an array on the GPU side

	// upload the angle map to the araray

	// setup the texture for the angle map
	/*
    t2dAngleMap.addressMode[0] =	hipAddressModeClamp;
    t2dAngleMap.addressMode[1] =	hipAddressModeClamp;
    t2dAngleMap.filterMode =		hipFilterModePoint;
    t2dAngleMap.normalized =		true;
	*/
	SETUP_ARRAY(pcAngleMap_array, iNrOfPhis, iNrOfThetas, 32, 0, 0, 0, hipChannelFormatKindSigned);	
	BIND_ARRAY_AS_TEXTURE(
		t2dAngleMap, pcAngleMap_array, 
		hipAddressModeClamp, hipAddressModeClamp, hipFilterModePoint, true);	

    CUDA_SAFE_CALL( 
		hipMemcpy2DToArray(
			pcAngleMap_array, 
			0, 
			0, 
			(void*)piAngleMap, 
			iNrOfPhis * sizeof(piAngleMap), 
			iNrOfPhis * sizeof(piAngleMap), 
			iNrOfThetas, 
			hipMemcpyHostToDevice) );

	// check whether the volume extent has been setup
	assert( NULL != pcVolumePtrs_global[0].ptr );

	// allocate the volume of bins in the global memorty on GPUs
	cBinVolumeExtent	= make_hipExtent(
		cVolumeExtent_array.width * sizeof(int),
		cVolumeExtent_array.height,
		cVolumeExtent_array.depth);
	CUDA_SAFE_CALL( 
		hipMalloc3D(&cSrcBinVolume_pitched, cBinVolumeExtent) );
	CUDA_SAFE_CALL( 
		hipMalloc3D(&cDstBinVolume_pitched, cBinVolumeExtent) );

	// ADD-BY-LEETEN 12/17/2009-BEGIN
	CUDA_SAFE_CALL( 
		hipMalloc3D(&cSrcDstBinVolumePtr_global, cBinVolumeExtent) );

	t2dSrcBinVolume.addressMode[0] = hipAddressModeClamp;
	t2dSrcBinVolume.addressMode[1] = hipAddressModeClamp;
	t2dSrcBinVolume.filterMode =	hipFilterModePoint;
	t2dSrcBinVolume.normalized =	false;

	t2dDstBinVolume.addressMode[0] = hipAddressModeClamp;
	t2dDstBinVolume.addressMode[1] = hipAddressModeClamp;
	t2dDstBinVolume.filterMode =	hipFilterModePoint;
	t2dDstBinVolume.normalized =	false;
	// ADD-BY-LEETEN 12/17/2009-END


	// allocate the volume of bins in the host side
	#if	0	// MOD-BY-LEETEN 2009/12/17-FROM:
	#else	// MOD-BY-LEETEN 2009/12/17-TO:
	#if	0	// DEL-BY-LEETEN 02/02/2010-BEGIN
		CUDA_SAFE_CALL(
			hipHostMalloc(
				(void**)&piBinVolume_host,
				sizeof(piBinVolume_host[0]) * iNrOfVoxels) );
	#endif	// DEL-BY-LEETEN 02/02/2010-END
	#endif	// MOD-BY-LEETEN 2009/12/17-END

	// setup the src. texture for bin lookup
	t2dVectorVolume.addressMode[0] = hipAddressModeClamp;
	t2dVectorVolume.addressMode[1] = hipAddressModeClamp;
	t2dVectorVolume.filterMode =	hipFilterModePoint;
	t2dVectorVolume.normalized =	false;
}

void
_Vector3DToVolume(
	int iVolumeWidth,
	int iVolumeHeight,
	int iVolumeDepth,
	int iBlockZSize,
	hipPitchedPtr cVolumePtr_global,	// input
	hipPitchedPtr cBinVolumePtr_global	// output
)
{
	CUDA_SAFE_CALL_NO_SYNC(
		hipBindTexture2D(
			0, 
			t2dVectorVolume, 
			cVolumePtr_global.ptr, 
			hipCreateChannelDesc<float4>(),
			iVolumeWidth, 
			iVolumeHeight * iVolumeDepth, 
			cVolumePtr_global.pitch) );

	dim3 v3Blk = dim3(BLOCK_DIM_X, BLOCK_DIM_Y);
	dim3 v3Grid = dim3(
		(unsigned int)ceilf((float)iVolumeWidth	 / (float)v3Blk.x),
		unsigned int(ceilf( float(iVolumeHeight) / float(v3Blk.y) ) ) *
		unsigned int(ceilf( float(iVolumeDepth) /  float(iBlockZSize) ) ) );
	// MOD-BY-LEETEN 12/18/2009-FROM:
		// _Vector3DToVolume_kernel<<<v3Grid, v3Blk, 0>>>
	// TO:
	_Vector3DToBinVolume_kernel<<<v3Grid, v3Blk, 0>>>
	// MOD-BY-LEETEN 12/18/2009-END
	(
		iVolumeWidth,
		iVolumeHeight,
		iVolumeDepth,
		int(ceilf(float(iVolumeHeight) / float(BLOCK_DIM_Y))),
		iBlockZSize,
		cBinVolumePtr_global
	);
	CUT_CHECK_ERROR("_Vector3DToVolume_kernel() failed");
}

void
_ComputeSrcBinVolume
(
	int iVolumeWidth,
	int iVolumeHeight,
	int iVolumeDepth,
	int iBlockZSize,
	float *pfSrcVolume
)
{
	CLOCK_INIT(SHOW_COMPUTE_SRC_BIN_VOLUME_TIMING, __FUNCTION__ ": ");

	CLOCK_BEGIN(SHOW_COMPUTE_SRC_BIN_VOLUME_TIMING);
	for(int v = 0,		d = 0; d < iVolumeDepth;	d++)
		for(int			h = 0; h < iVolumeHeight;	h++)
			for(int		w = 0; w < iVolumeWidth;	w++, v++)
			{
				pf4Volume_host[v].x = pfSrcVolume[v * 3 + 0];
				pf4Volume_host[v].y = pfSrcVolume[v * 3 + 1];
				pf4Volume_host[v].z = pfSrcVolume[v * 3 + 2];
				pf4Volume_host[v].w = 0.0f;
			}
	CLOCK_END(SHOW_COMPUTE_SRC_BIN_VOLUME_TIMING, false);

	CLOCK_BEGIN(SHOW_COMPUTE_SRC_BIN_VOLUME_TIMING);
    hipMemcpy3DParms cCopyParamsHostToDevice = {0};

    cCopyParamsHostToDevice.srcPtr   = make_hipPitchedPtr(
		(void*)pf4Volume_host, 
		cVolumeExtent.width, 
		iVolumeWidth * sizeof(float4), 
		iVolumeHeight);
    cCopyParamsHostToDevice.dstPtr		= pcVolumePtrs_global[0];
    cCopyParamsHostToDevice.extent		= cVolumeExtent;
    cCopyParamsHostToDevice.kind		= hipMemcpyHostToDevice;
    CUDA_SAFE_CALL( hipMemcpy3D(&cCopyParamsHostToDevice) );  

	CLOCK_END(SHOW_COMPUTE_SRC_BIN_VOLUME_TIMING, false);

	CLOCK_BEGIN(SHOW_COMPUTE_SRC_BIN_VOLUME_TIMING);
	_Vector3DToVolume(
		iVolumeWidth,
		iVolumeHeight,
		iVolumeDepth,
		iBlockZSize,
		pcVolumePtrs_global[0],
		cSrcBinVolume_pitched);
	CLOCK_END(SHOW_COMPUTE_SRC_BIN_VOLUME_TIMING, false);

	CLOCK_PRINT(SHOW_COMPUTE_SRC_BIN_VOLUME_TIMING);
}

void 
_GetSrcBinVolume(int *piBinVolume)
{
	// ADD-BY-LEETEN 02/02/2010-BEGIN
	CUDA_SAFE_CALL(
		hipHostMalloc(
			(void**)&piBinVolume_host,
			sizeof(piBinVolume_host[0]) * iNrOfVoxels) );
	// ADD-BY-LEETEN 02/02/2010-END

	hipMemcpy3DParms cCopyParamsDeviceToHost = {0};
	cCopyParamsDeviceToHost.srcPtr		= cSrcBinVolume_pitched;
	cCopyParamsDeviceToHost.dstPtr		= make_hipPitchedPtr(
		// MOD-BY-LEETEN 2009/12/17-FROM:
			// (void*)piSrcAngleBinVolume_host, 
		// TO:
		(void*)piBinVolume_host, 
		// MOD-BY-LEETEN 2009/12/17-END
		cVolumeExtent_array.width * sizeof(int),	// cSrcBinVolume_pitched.width, 
		cVolumeExtent_array.width * sizeof(int), 
		cVolumeExtent_array.height);
	cCopyParamsDeviceToHost.extent		= cBinVolumeExtent;
	cCopyParamsDeviceToHost.kind		= hipMemcpyDeviceToHost;

	CUDA_SAFE_CALL( hipMemcpy3D(&cCopyParamsDeviceToHost) );  

	// MOD-BY-LEETEN 2009/12/17-FROM:
		// memcpy(piBinVolume, piSrcAngleBinVolume_host, sizeof(piBinVolume) * iNrOfVoxels);
	// TO:
	memcpy(piBinVolume, piBinVolume_host, sizeof(piBinVolume) * iNrOfVoxels);
	// MOD-BY-LEETEN 2009/12/17-END

	// ADD-BY-LEETEN 02/02/2010-BEGIN
	FREE_MEMORY_ON_HOST(piBinVolume_host);
	// ADD-BY-LEETEN 02/02/2010-END
}

void 
_GetDstBinVolume(int *piBinVolume)
{
	// ADD-BY-LEETEN 02/02/2010-BEGIN
	CUDA_SAFE_CALL(
		hipHostMalloc(
			(void**)&piBinVolume_host,
			sizeof(piBinVolume_host[0]) * iNrOfVoxels) );
	// ADD-BY-LEETEN 02/02/2010-END


	hipMemcpy3DParms cCopyParamsDeviceToHost = {0};
	cCopyParamsDeviceToHost.srcPtr		= cDstBinVolume_pitched;
	cCopyParamsDeviceToHost.dstPtr		= make_hipPitchedPtr(
		// MOD-BY-LEETEN 2009/12/17-FROM:
			// (void*)piDstAngleBinVolume_host,		
		// TO:
		(void*)piBinVolume_host, 
		// MOD-BY-LEETEN 2009/12/17-END

		cVolumeExtent_array.width * sizeof(int),	// cDstBinVolume_pitched.width, 
		cVolumeExtent_array.width * sizeof(int), 
		cVolumeExtent_array.height);
	cCopyParamsDeviceToHost.extent		= cBinVolumeExtent;
	cCopyParamsDeviceToHost.kind		= hipMemcpyDeviceToHost;

	CUDA_SAFE_CALL( hipMemcpy3D(&cCopyParamsDeviceToHost) );  

	// MOD-BY-LEETEN 2009/12/17-FROM:
		// memcpy(piBinVolume, piDstAngleBinVolume_host, sizeof(piBinVolume) * iNrOfVoxels);
	// TO:
	memcpy(piBinVolume, piBinVolume_host, sizeof(piBinVolume) * iNrOfVoxels);
	// MOD-BY-LEETEN 2009/12/17-END

	// ADD-BY-LEETEN 02/02/2010-BEGIN
	FREE_MEMORY_ON_HOST(piBinVolume_host);
	// ADD-BY-LEETEN 02/02/2010-END
}

// ADD-BY-LEETEN 12/14/2009-END

void
// MOD-BY-LEETEN 12/07/2009-FROM:
	// _FlowFusion(
// TO:
_FlowDiffusion(
// MOD-BY-LEETEN 12/07/2009-END
	float fAttenuation,
	int iNrOfIterations,
	int iVolumeWidth,
	int iVolumeHeight,
	int iVolumeDepth,
	// ADD-BY-LEETEN 10/02/2009-BEGIN
	int iBlockZSize,
	float *pfWeightVolume,
	// MOD-BY-LEETEN 2009/11/10-FROM:
		// float *pfOffsetVolume,
	// TO:
	float *pfXOffsetVolume,
	float *pfYOffsetVolume,
	float *pfZOffsetVolume,
	// MOD-BY-LEETEN 2009/11/10-END
	// ADD-BY-LEETEN 10/02/2009-END
	float *pfSrcVolume,
	float *pfDstVolume,
	// ADD-BY-LEETEN 2009/11/10-BEGIN
	int *piFlagVolume
	// ADD-BY-LEETEN 2009/11/10-END
)
{
	// ADD-BY-LEETEN 12/07/2009-BEGIN
	#if	CHECK_ERROR_CONVERGENCE_BY_CUDPP
	// MOD-BY-LEETEN 02/06/2010-FROM:
		// iNrOfIterations = 4 * max(max(iVolumeWidth, iVolumeHeight), iVolumeDepth);
	// TO:
	// MOD-BY-LEETEN 03/18/2010-FROM:
		// iNrOfIterations = 1024 * max(max(iVolumeWidth, iVolumeHeight), iVolumeDepth);
	// TO:
	iNrOfIterations = 9 * max(max(iVolumeWidth, iVolumeHeight), iVolumeDepth);
	// MOD-BY-LEETEN 03/18/2010-END
	// MOD-BY-LEETEN 02/06/2010-END
	// LOG(printf("Warning! iNrOfIterations is changed to %d", iNrOfIterations));
	#endif
	// ADD-BY-LEETEN 12/07/2009-END
	

	// MOD-BY-LEETEN 12/07/2009-FROM:
		// CLOCK_INIT(PRINT_FLOW_DIFFUSION_TIMING, "_FlowFusion(): ");
	// TO:
	// MOD-BY-LEETEN 12/14/2009-FROM:
		// CLOCK_INIT(PRINT_FLOW_DIFFUSION_TIMING, __FUNCTION__);
	// TO:
	CLOCK_INIT(PRINT_FLOW_DIFFUSION_TIMING, __FUNCTION__ ": ");
	// MOD-BY-LEETEN 12/07/2009-END

	// MOD-BY-LEETEN 10/02/2009-FROM:
		// dim3 v3Blk = dim3(16, 8);
	// TO:
	dim3 v3Blk = dim3(BLOCK_DIM_X, BLOCK_DIM_Y);
	// MOD-BY-LEETEN 10/02/2009-END
	dim3 v3Grid = dim3(
		(unsigned int)ceilf((float)iVolumeWidth	 / (float)v3Blk.x),
		// ADD-BY-LEETEN 11/04/2009-BEGIN
		#if					DIFFUSION_BY_FOR_LOOP_ON_HOST	
		// ADD-BY-LEETEN 11/04/2009-END

		(unsigned int)ceilf((float)iVolumeHeight / (float)v3Blk.y));

		// ADD-BY-LEETEN 11/04/2009-BEGIN
		#else	// #if		DIFFUSION_BY_FOR_LOOP_ON_HOST
		unsigned int(ceilf( float(iVolumeHeight) / float(v3Blk.y) ) ) *
		unsigned int(ceilf( float(iVolumeDepth) /  float(iBlockZSize) ) ) );
		#endif	// #if		DIFFUSION_BY_FOR_LOOP_ON_HOST
		// ADD-BY-LEETEN 11/04/2009-END

	// ADD-BY-LEETEN 10/02/2009-END

	// convert the #channels in the src volume from 3 to 4
	CLOCK_BEGIN(PRINT_FLOW_DIFFUSION_TIMING);
	for(int v = 0,		d = 0; d < iVolumeDepth;	d++)
		for(int			h = 0; h < iVolumeHeight;	h++)
			for(int		w = 0; w < iVolumeWidth;	w++, v++)
			{
				pf4Volume_host[v].x = pfSrcVolume[v * 3 + 0];
				pf4Volume_host[v].y = pfSrcVolume[v * 3 + 1];
				pf4Volume_host[v].z = pfSrcVolume[v * 3 + 2];

				// ADD-BY-LEETEN 2009/11/10-BEGIN
				if( piFlagVolume )
					pf4Volume_host[v].w = float(piFlagVolume[v]);
				else
				// ADD-BY-LEETEN 2009/11/10-END
				pf4Volume_host[v].w = 0.0f;
			}

	// ADD-BY-LEETEN 10/02/2009-BEGIN
	for(int v = 0,		d = 0; d < iVolumeDepth;	d++)
		for(int			h = 0; h < iVolumeHeight;	h++)
			for(int		w = 0; w < iVolumeWidth;	w++, v++)
			{
				#if	0	// MOD-BY-LEETEN 2009/11/10-FROM:
				#else	// MOD-BY-LEETEN 2009/11/10-TO:
				// ADD-BY-LEETEN 12/07/2009-BEGIN
				if( pfXOffsetVolume )
				// ADD-BY-LEETEN 12/07/2009-END
				pf4WeightOffsetVolume_host[v].x = pfXOffsetVolume[v];
				// ADD-BY-LEETEN 12/07/2009-BEGIN
				if( pfYOffsetVolume )
				// ADD-BY-LEETEN 12/07/2009-END
				pf4WeightOffsetVolume_host[v].y = pfYOffsetVolume[v];
				// ADD-BY-LEETEN 12/07/2009-BEGIN
				if( pfZOffsetVolume )
				// ADD-BY-LEETEN 12/07/2009-END
				pf4WeightOffsetVolume_host[v].z = pfZOffsetVolume[v];
				pf4WeightOffsetVolume_host[v].w = 1.0f - pfWeightVolume[v];
				#endif	// MOD-BY-LEETEN 2009/11/10-END
			}
	// ADD-BY-LEETEN 10/02/2009-END

	CLOCK_END(PRINT_FLOW_DIFFUSION_TIMING, true);

	// Upload the input volume to the first volume on the GPU
	CLOCK_BEGIN(PRINT_FLOW_DIFFUSION_TIMING);

    hipMemcpy3DParms cCopyParamsHostToDevice = {0};

    cCopyParamsHostToDevice.srcPtr   = make_hipPitchedPtr(
		(void*)pf4Volume_host, 
		cVolumeExtent.width, 
		iVolumeWidth * sizeof(float4), 
		iVolumeHeight);
    cCopyParamsHostToDevice.dstPtr		= pcVolumePtrs_global[0];
    cCopyParamsHostToDevice.extent		= cVolumeExtent;
    cCopyParamsHostToDevice.kind		= hipMemcpyHostToDevice;
    CUDA_SAFE_CALL( hipMemcpy3D(&cCopyParamsHostToDevice) );  

	// ADD-BY-LEETEN 10/02/2009-BEGIN
    hipMemcpy3DParms cCopyParamsHostToArray = {0};
    cCopyParamsHostToArray.srcPtr   = make_hipPitchedPtr(
		(void*)pf4WeightOffsetVolume_host, 
		iVolumeWidth * sizeof(float4), 
		iVolumeWidth, 
		iVolumeHeight);
    cCopyParamsHostToArray.dstArray = cWeightOffsetVolume_array;
    cCopyParamsHostToArray.extent   = cVolumeExtent_array;
    cCopyParamsHostToArray.kind     = hipMemcpyHostToDevice;
    CUDA_SAFE_CALL( 
		hipMemcpy3D(&cCopyParamsHostToArray) );  

    t3dWeightOffset.addressMode[0] = hipAddressModeClamp;
    t3dWeightOffset.addressMode[1] = hipAddressModeClamp;
    t3dWeightOffset.addressMode[2] = hipAddressModeClamp;
    t3dWeightOffset.filterMode		= hipFilterModePoint;
    t3dWeightOffset.normalized		= false; 

    // bind array to 3D texture
    CUDA_SAFE_CALL(
		hipBindTextureToArray(t3dWeightOffset, cWeightOffsetVolume_array, cWeightOffsetChannelDesc));
	// ADD-BY-LEETEN 10/02/2009-END

	CLOCK_END(PRINT_FLOW_DIFFUSION_TIMING, true);

	// calcuate the fusion operator on GPUs
	CLOCK_BEGIN(PRINT_FLOW_DIFFUSION_TIMING);

    t2dSrc.addressMode[0] = hipAddressModeClamp;
    t2dSrc.addressMode[1] = hipAddressModeClamp;
    t2dSrc.filterMode =		hipFilterModePoint;
    t2dSrc.normalized =		false;

	int iSrc = 0;

	for(int i = 0; 
			i < iNrOfIterations; 
			i++,				iSrc = 1 - iSrc)
	{
		// bind the src. volume as the 3D texture

		hipChannelFormatDesc cChannelDesc =
			hipCreateChannelDesc<float4>();

		CUDA_SAFE_CALL_NO_SYNC(
			hipBindTexture2D(
				0, 
				t2dSrc, 
				pcVolumePtrs_global[iSrc].ptr, 
				cChannelDesc,
				iVolumeWidth, 
				iVolumeHeight * iVolumeDepth, 
				pcVolumePtrs_global[iSrc].pitch) );


		// ADD-BY-LEETEN 12/07/2009-BEGIN
		if( iVolumeDepth <= 2 )
			_FlowDiffusion2D_kernel<<<v3Grid, v3Blk, 0>>>
			(
				fAttenuation,
				iVolumeWidth,
				iVolumeHeight,
				pcVolumePtrs_global[1 - iSrc],
				cErrorVolume_device
			);	
		else
		{
		// ADD-BY-LEETEN 12/07/2009-END

		// launch the kernel to compute the diffusion operator for 1 iteration
		// MOD-BY-LEETEN 11/04/2009-FROM:
			// #if	0	// MOD-BY-LEETEN 10/02/2009-FROM:
		// TO:
		#if	DIFFUSION_BY_FOR_LOOP_ON_HOST
		// MOD-BY-LEETEN 11/04/2009-END

			for(int z = 0; z < iVolumeDepth; z++)
				// MOD-BY-LEETEN 12/07/2009-FROM:
					// _FlowFusion_kernel<<<v3Grid, v3Blk, 0>>>
				// TO:
				_FlowDiffusion3D_kernel<<<v3Grid, v3Blk, 0>>>
				// MOD-BY-LEETEN 12/07/2009-END
				(
					fAttenuation,
					iVolumeWidth,
					iVolumeHeight,
					iVolumeDepth,
					z,

					// output
					pcVolumePtrs_global[1 - iSrc],
					// ADD-BY-LEETEN 2009/11/25-BEGIN
					cErrorVolume_device
					// ADD-BY-LEETEN 2009/11/25-END
				);	
		// MOD-BY-LEETEN 11/04/2009-FROM:
			// #else	// MOD-BY-LEETEN 10/02/2009-TO:
		// TO:
		#else	// #if	DIFFUSION_BY_FOR_LOOP_ON_HOST
		// MOD-BY-LEETEN 11/04/2009-END

		// _FlowFusion_kernel<<<v3Grid, v3Blk, v3Blk.x * v3Blk.y * sizeof(float4)>>>
		// MOD-BY-LEETEN 12/07/2009-FROM:
			// _FlowFusion_kernel<<<v3Grid, v3Blk, 0>>>
		// TO:
		_FlowDiffusion3D_kernel<<<v3Grid, v3Blk, 0>>>
		// MOD-BY-LEETEN 12/07/2009-END
		(
			fAttenuation,
			iVolumeWidth,
			iVolumeHeight,
			iVolumeDepth,
			0,
			// ADD-BY-LEETEN 11/04/2009-BEGIN
			int(ceilf(float(iVolumeHeight) / float(BLOCK_DIM_Y))),
			iBlockZSize,
			// ADD-BY-LEETEN 11/04/2009-END
			// output
			pcVolumePtrs_global[1 - iSrc],
			// ADD-BY-LEETEN 2009/11/25-BEGIN
			cErrorVolume_device
			// ADD-BY-LEETEN 2009/11/25-END
		);
		#endif	// MOD-BY-LEETEN 10/02/2009-END
		// ADD-BY-LEETEN 12/07/2009-BEGIN
		}
		// ADD-BY-LEETEN 12/07/2009-END
		CUT_CHECK_ERROR("_FlowFusion_kernel() failed");

		// ADD-BY-LEETEN 2009/11/25-BEGIN
		#if	CHECK_ERROR_CONVERGENCE_BY_CUDPP
		cudppScan(cScanplan, cErrorSum_device.ptr, cErrorVolume_device.ptr, iVolumeWidth * iVolumeHeight * iVolumeDepth);

		float fError;
		#if	0	// MOD-BY-LEETEN 2009/12/17-FROM:
			CUDA_SAFE_CALL( 
				hipMemcpy(
					&fError, 
					ADDRESS_2D(
						float, cErrorSum_device.ptr, 
						sizeof(float), cErrorSum_device.pitch, 
						iVolumeWidth - 1, iVolumeHeight - 1 + (iVolumeDepth - 1 ) * iVolumeHeight),
					sizeof(float), 
					hipMemcpyDeviceToHost));
		#else	// MOD-BY-LEETEN 2009/12/17-TO:
	    CUDA_SAFE_CALL_NO_SYNC( 
			hipMemcpy(
				&fError, 
				cErrorSum_device.ptr, 
				sizeof(float), 
				hipMemcpyDeviceToHost));
		#endif	// MOD-BY-LEETEN 2009/12/17-END

		/*
		fError /= float(iVolumeWidth * iVolumeHeight * iVolumeDepth * 3);
		fError = sqrtf(fError);
		*/
		#if	0	// MOD-BY-LEETEN 2009/12/17-FROM:
			fprintf(stderr, "Error: %f\n", fError);
			if( fError < 1e-10 )
				break;
		#else	// MOD-BY-LEETEN 2009/12/17-TO:
		fError /= float(iVolumeWidth * iVolumeHeight * iVolumeDepth * 3);
		fError = sqrtf(fError);
		// fprintf(stderr, "Error: %f\n", fError);

		static float fPrevError = -1.0f;
		#if	0	// MOD-BY-LEETEN 02/06/2010-FROM:
			float fErrorDif = fabsf(fPrevError - fError);
			if( i > max(max(iVolumeWidth, iVolumeHeight), iVolumeDepth) && (fErrorDif/fPrevError < 0.05f || fErrorDif < 0.000001f) )
		#else	// MOD-BY-LEETEN 02/06/2010-TO:
		float fErrorRate = fError/fPrevError;
		if( i > max(max(iVolumeWidth, iVolumeHeight), iVolumeDepth) && 
			(DIFFUSION_CONVERGE_THRESHOLD <= fErrorRate && fErrorRate <= 1.00f ) )
		#endif	// MOD-BY-LEETEN 02/06/2010-END
		{
			printf("\t#iters = %d;", i);
			break;
		}
		// ADD-BY-LEETEN 03/10/2010-BEGIN
		// MOD-BY-LEETEN 03/18/2010-FROM:
			// if( fErrorRate > 1.0f )
		// TO: 
		if( i > 1 && fErrorRate > 1.0f )
		// MOD-BY-LEETEN 03/18/2010-END
		{
			printf("Error increase. Stop.");
			break;
		}
		// ADD-BY-LEETEN 03/10/2010-END
		fPrevError = fError;
		#endif	// MOD-BY-LEETEN 2009/12/17-END

		#endif	// #if	CHECK_ERROR_CONVERGENCE_BY_CUDPP
		// ADD-BY-LEETEN 2009/11/25-END

	}

	// ADD-BY-LEETEN 12/14/2009-BEGIN
	if( iVolumeDepth > 2 )
	{
		_Vector3DToVolume(
			iVolumeWidth,
			iVolumeHeight,
			iVolumeDepth,
			iBlockZSize,
			pcVolumePtrs_global[iSrc],
			cDstBinVolume_pitched);
			// piDstAngleBinVolume_host);
	}
	// ADD-BY-LEETEN 12/14/2009-END

	CLOCK_END(PRINT_FLOW_DIFFUSION_TIMING, true);

	int iDst = iSrc;

	// read the result from latest iteration back to the CPU side
	CLOCK_BEGIN(PRINT_FLOW_DIFFUSION_TIMING);

	hipMemcpy3DParms cCopyParamsDeviceToHost = {0};
    cCopyParamsDeviceToHost.srcPtr		= pcVolumePtrs_global[iDst];
    cCopyParamsDeviceToHost.dstPtr		= make_hipPitchedPtr(
		(void*)pf4Volume_host, 
		cVolumeExtent.width, 
		iVolumeWidth * sizeof(float4), 
		iVolumeHeight);
    cCopyParamsDeviceToHost.extent		= cVolumeExtent;
    cCopyParamsDeviceToHost.kind		= hipMemcpyDeviceToHost;

	CUDA_SAFE_CALL( hipMemcpy3D(&cCopyParamsDeviceToHost) );  

	CLOCK_END(PRINT_FLOW_DIFFUSION_TIMING, true);

	// convert the #channels in the dst volume from 4 to 3
	CLOCK_BEGIN(PRINT_FLOW_DIFFUSION_TIMING);
	for(int v = 0,		d = 0; d < iVolumeDepth;	d++)
		for(int			h = 0; h < iVolumeHeight;	h++)
			for(int		w = 0; w < iVolumeWidth;	w++, v++)
			{
				pfDstVolume[v * 3 + 0] = pf4Volume_host[v].x;
				pfDstVolume[v * 3 + 1] = pf4Volume_host[v].y;
				pfDstVolume[v * 3 + 2] = pf4Volume_host[v].z;
			}
	CLOCK_END(PRINT_FLOW_DIFFUSION_TIMING, true);

	CLOCK_PRINT(PRINT_FLOW_DIFFUSION_TIMING);
}

} // extern "C"

/*

$Log: not supported by cvs2svn $
Revision 1.12  2010/03/10 20:19:44  leeten

[03/10/2010]
1. [ADD] Stop the iteration if the error increases.

Revision 1.11  2010/02/09 00:40:48  leeten

[02/08/2010]
1. [MOD] Change the #iterations to 1024 * max(max(iVolumeWidth, iVolumeHeight), iVolumeDepth) in order to gurantee the converge of the diffusion.
2. [MOD] Change the checking of converge.

Revision 1.10  2010/02/05 01:39:53  leeten

[02/02/2010]
1. [MOD] Change the name of the macro from PRINT_FLOW_FUSION_TIMING to PRINT_FLOW_DIFFUSION_TIMING .

Revision 1.9  2010/02/02 23:41:39  leeten

[02/02/2010]
1. [MOD] Change the name of the function _GetJointEntropyVolume to _ComputeJointEntropyVolume. Besides, one more paramter is added to specify the dst. for the result on the host side.

Revision 1.8  2010/01/27 22:07:27  leeten

[01/27/2010]
1. [MOD] Add the declaration of the new _ComputeSrcEntropyVolume() and remove the function _GetSrcEntropyVolume.

Revision 1.7  2010/01/04 18:13:36  leeten

[01/04/2010]
1. [ADD] Dump the entropy field in to a file in binay format.

Revision 1.6  2009/12/31 02:37:02  leeten

[12/30/2009]
1. [ADD] Dump the entropy field into a binary file.

Revision 1.5  2009/12/27 19:19:21  leeten

[12/27/2009]
1. [DEL] Move the files for entropy computation into different files.

Revision 1.4  2009/12/20 03:31:22  leeten

[12/19/2009]
1. [DEL] remove deleted code segments.
2. [MOD] the variable cErrorvolume_exten is not declared if CHECK_ERROR_CONVERGENCE_BY_CUDPP is not given.
3. [MOD] Change the scanning order when subdividing the volume into blocks: before the blocks are subdsived along the XZ plane, while now the XY plane is subdivided instead.
4. [DEBUG] Change the #neighboring from i3KernelSize.x * i3KernelSize.y * i3KernelSize.z to (2 * i3KernelSize.x + 1) *  (2 * i3KernelSize.y + 1) * (2 * i3KernelSize.z + 1) )
5. [ADD] Define a function _ComputeEntropyVolumeWithSorting_cuda to compute the entropy field without computing the histogram.
6. [ADD] Define a function _ComputeEntropyVolume_cuda to compute the entropy field based on the histogram.
7. [ADD] Define a function _ComputeEntropyVolume_host  to compute the entropy field by CPUs.
8. [MOD] Modfy the functions _GetSrcEntropyVolume and _GetJointEntropyVolume s.t. different function will be called depending on the value of the preprocessor COMPUTE_ENTROPY_VOLUME
COMPUTE_ENTROPY_VOLUME_CUDA: _ComputeEntropyVolumeWithSorting_cuda
COMPUTE_ENTROPY_VOLUME_WITH_SORTING_CUDA: _ComputeEntropyVolumeWithSorting_cuda
COMPUTE_ENTROPY_VOLUME_HOST: _ComputeEntropyVolume_host

Revision 1.3  2009/12/17 20:20:32  leeten

[12/17/2009]
1. [MOD] Change the preprocessor USE_CUDPP to CHECK_ERROR_CONVERGENCE_BY_CUDPP.
2. [MOD] Move several preprocessors to a new header FlowDiffusion_cuda.h
3. [MOD] Change the variable t2dBinVolumeSrc to t2dVectorVolume.
4. [MOD] Move CUDA kernels for 2D diffusion to FlowDiffusion2D_kernel.cu.
5. [MOD] Move CUDA kernels for 3D diffusion to FlowDiffusion3D_kernel.cu.
6. [MOD] Move CUDA kernels for 3D entropy field computation to EntropyField_kernel.cu.
7. [ADD] Define host functions to compute the entropy field by CPUs.
8. [DEL] Remove useless codesegments.

Revision 1.2  2009/12/15 20:05:57  leeten

[12/15/2009]
1. [ADD] Define new function _FlowDiffusionSetAngleMap() to setup the lookup table that converts a vector to a discrete bin.
2. [ADD] Define new function _GetSrcBinVolume() to download the bin of the orginal input vector field.
3. [ADD] Define new function _GetDstBinVolume() to download the bin for the diffused vector field.
4. [ADD] Define new function _ComputeSrcBinVolume() to download the bin of the orginal input vector field.
5. [ADD] Define a host function _Vector3DToVolume to convert each vector in a vector field into a bin.
6. [ADD] Define a kernel function _Vector3DToVolume_kernel() to convert each vector in a vector field into a bin on GPUs.

2. [ADD] Define new functions

Revision 1.1.1.1  2009/12/07 20:04:02  leeten

[12/07/2009]
1. [1ST] First time checkin.

Revision 1.3  2009/12/05 21:19:51  leeten

[12/05/2009]
1. [ADD] Add a 3D texture glWeightOffset to store the weights and offsets.
2. [ADD] Specify the flag donochange to the w channel in the 3D textures.
3. [ADD] Output the error to a 3D volume.

Revision 1.2  2009/11/04 19:11:13  leeten

[2009/11/04]
1. [ADD] Add a new preprocessor USE_SHARED_MEMORY to decide whether the shared memory is utilized.
2. [ADD] Add a new preprocessor DIFFUSION_BY_FOR_LOOP_ON_HOSTto decide whether the diffusion is executed via a for loop on the host side to scan throught all XY layers.
If it is not zero, the volume will be divided along the Z direction as well, and each block will be assigned as a single block on CUDA.
3. [ADD] Add 2 new preprocessors BLOCK_DIM_X and BLOCK_DIM_Y to control the block dimension.
4. [ADD] Add 1 array and 1 3D textrue for the weight/offset volume.

Revision 1.1.1.1  2009/11/02 15:30:56  leeten

[11/02/2009]
1 [1ST] Firs time checkin.


*/
