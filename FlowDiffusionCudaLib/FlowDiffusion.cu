#include "hip/hip_runtime.h"
////////////////////////////////////////////
	#include <stdio.h>
	#include <>
	// ADD-BY-LEETEN 2009/11/25-BEGIN
	#define USE_CUDPP	0

	#if	USE_CUDPP	
		#include "cudpp/cudpp.h"
		#pragma comment (lib, "cudpp32.lib")
	#endif
	// ADD-BY-LEETEN 2009/11/25-END

	#include "cuda_macro.h"

	// ADD-BY-LEETEN 12/14/2009-BEGIN
	#include "liblog.h"
	using namespace std;

	#define	M_PI	3.1415926535897932384626433832795f
	// ADD-BY-LEETEN 12/14/2009-END

	#include "libbuf.h"

////////////////////////////////////////////
	#define PRINT_FLOW_FUSION_TIMING	1
	#define USE_SHARED_MEMORY			0

	// ADD-BY-LEETEN 12/14/2009-BEGIN
	#define SHOW_COMPUTE_SRC_BIN_VOLUME_TIMING	1	
	// ADD-BY-LEETEN 12/14/2009-END

	// ADD-BY-LEETEN 11/04/2009-BEGIN
						// if this preprocessor is non zero, the volume is scanned via a for loop on the host
	#define DIFFUSION_BY_FOR_LOOP_ON_HOST		0
	// ADD-BY-LEETEN 11/04/2009-END

	// ADD-BY-LEETEN 10/02/2009-BEGIN
	#define BLOCK_DIM_X	16
	#define BLOCK_DIM_Y	12
	// ADD-BY-LEETEN 10/02/2009-END

////////////////////////////////////////////
	static hipExtent cVolumeExtent;
	static float4 *pf4Volume_host;
	static hipPitchedPtr pcVolumePtrs_global[2];
	static texture<float4, 2, hipReadModeElementType> t2dSrc;
	// ADD-BY-LEETEN 10/02/2009-BEGIN
						// array and texture reference for the weight and offset volume
	static hipExtent cVolumeExtent_array;
	static float4 *pf4WeightOffsetVolume_host;
	static hipArray *cWeightOffsetVolume_array = 0;
	static texture<float4, 3, hipReadModeElementType> t3dWeightOffset;
    static hipChannelFormatDesc cWeightOffsetChannelDesc = hipCreateChannelDesc<float4>();
	// ADD-BY-LEETEN 10/02/2009-END

	// ADD-BY-LEETEN 2009/11/25-BEGIN
	#if	USE_CUDPP
    static CUDPPConfiguration cConfig;
    static CUDPPHandle cScanplan = 0;
	#endif	// #if	USE_CUDPP

	static hipExtent cErrorVolumeExtent;
	static hipPitchedPtr cErrorVolume_device;
	static hipPitchedPtr cErrorSum_device;
	// ADD-BY-LEETEN 2009/11/25-END

	// ADD-BY-LEETEN 12/14/2009-BEGIN
	int iNrOfVoxels;

	static texture<int, 2, hipReadModeElementType> t2dAngleMap;
	static hipArray *pcAngleMap_array;	// cuda array to hold the Gaussian kernels

	static hipExtent cBinVolumeExtent;
	static hipPitchedPtr cSrcBinVolumePtr_global;
	static hipPitchedPtr cDstBinVolumePtr_global;
	static int *piSrcAngleBinVolume_host;
	static int *piDstAngleBinVolume_host;

	static texture<float4, 2, hipReadModeElementType> t2dBinVolumeSrc;
	// ADD-BY-LEETEN 12/14/2009-END

// ADD-BY-LEETEN 12/07/2009-BEGIN
__global__ 
static 
void 
_FlowDiffusion2D_kernel
(
	// INPUT
	float fAttenuation,

	int iVolumeWidth,
	int iVolumeHeight,

	hipPitchedPtr cDstPitchedPtr,

	// ADD-BY-LEETEN 2009/11/25-BEGIN
	hipPitchedPtr cErrorPitchedPtr
	// ADD-BY-LEETEN 2009/11/25-END
)
{
 	int iVoxelX = blockIdx.x * blockDim.x + threadIdx.x;
	int iVoxelY = blockIdx.y * blockDim.y + threadIdx.y;

	// compute the central differnece
	float4 f4Value	= tex2D(t2dSrc, iVoxelX,		iVoxelY);
	float4 f4PX		= tex2D(t2dSrc, min(iVoxelX + 1, iVolumeWidth - 1),	iVoxelY);
	float4 f4NX		= tex2D(t2dSrc, max(iVoxelX - 1, 0),				iVoxelY);
	float4 f4PY		= tex2D(t2dSrc, iVoxelX,		min(iVoxelY + 1, iVolumeHeight - 1));
	float4 f4NY		= tex2D(t2dSrc, iVoxelX,		max(iVoxelY - 1, 0)				);

	float4 f4WeightOffset = tex3D(t3dWeightOffset, iVoxelX,	iVoxelY, 0);

	float4 f4Result;

	f4Result = make_float4(
		f4WeightOffset.w * f4Value.x + (f4PX.x + f4NX.x + f4PY.x + f4NY.x - 4.0f * f4Value.x) * fAttenuation + f4WeightOffset.x,
		f4WeightOffset.w * f4Value.y + (f4PX.y + f4NX.y + f4PY.y + f4NY.y - 4.0f * f4Value.y) * fAttenuation + f4WeightOffset.y,
		f4WeightOffset.w * f4Value.z + (f4PX.z + f4NX.z + f4PY.z + f4NY.z - 4.0f * f4Value.z) * fAttenuation + f4WeightOffset.z,
		0);

	if( iVoxelX < iVolumeWidth && iVoxelY < iVolumeHeight )
	{
		*ADDRESS_2D(
			float4, cDstPitchedPtr.ptr, 
			sizeof(float4), cDstPitchedPtr.pitch, 
			iVoxelX, iVoxelY) = f4Result;

		float4 f4Diff;
		f4Diff.x = f4Value.x - f4Result.x;
		f4Diff.y = f4Value.y - f4Result.y;
		f4Diff.z = f4Value.z - f4Result.z;
		float fDiff = f4Diff.x * f4Diff.x + f4Diff.y * f4Diff.y + f4Diff.z * f4Diff.z;
		*ADDRESS_2D(
			float, cErrorPitchedPtr.ptr, 
			sizeof(float), cErrorPitchedPtr.pitch, 
			iVoxelX, iVoxelY) = fDiff;
	}
}

// ADD-BY-LEETEN 12/07/2009-END

// ADD-BY-LEETEN 12/14/2009-BEGIN
__global__ 
static 
void 
// MOD-BY-LEETEN 12/07/2009-FROM:
	// _FlowFusion_kernel
// TO:
_Vector3DToVolume_kernel
// MOD-BY-LEETEN 12/07/2009-END
(
	int iVolumeWidth,
	int iVolumeHeight,
	int iVolumeDepth,

	int iNrOfYBlocks,
	int iBlockZSize,

	// texture<float4, 2, hipReadModeElementType> t2dSrc,
	hipPitchedPtr cBinVolumePtr_global
)
{
 	int iVoxelX = blockIdx.x * blockDim.x + threadIdx.x;
	int iVoxelY = (blockIdx.y % iNrOfYBlocks) * blockDim.y + threadIdx.y;
	int iBeginZ = (blockIdx.y / iNrOfYBlocks) * iBlockZSize;
	int iEndZ = min(iBeginZ + iBlockZSize, iVolumeDepth);

	for(int z = iBeginZ; z < iEndZ; z++)
	{
		float4 f4Vector = tex2D(t2dBinVolumeSrc, iVoxelX,		iVoxelY + z				* iVolumeHeight);;
		
		float fLength	= sqrt(f4Vector.x * f4Vector.x + f4Vector.y * f4Vector.y + f4Vector.z * f4Vector.z);
		float fTheta	= 0.0f;
		float fPhi		= 0.0f;
		int iBin = 0;

		if( 0 < fLength )
		{
			f4Vector.x /= fLength;
			f4Vector.y /= fLength;
			f4Vector.z /= fLength;
			fTheta = ( 0.0f == f4Vector.x && 0.0f == f4Vector.y )?0.0f:(M_PI+(atan2(f4Vector.y, f4Vector.x)));
			fTheta /= 2.0f * M_PI;

			float fLength2D = sqrt(f4Vector.x * f4Vector.x + f4Vector.y * f4Vector.y);
			fPhi = ((0.0f == fLength2D)&&(0.0f == f4Vector.z))?0.0f:fabs(M_PI/2.0f-(atan2(f4Vector.z, fLength2D)));
			fPhi /= M_PI;
			iBin = tex2D(t2dAngleMap, fPhi, fTheta);
		}

		if( iVoxelX < iVolumeWidth && iVoxelY < iVolumeHeight )
		{
			*ADDRESS_2D(
				int,			cBinVolumePtr_global.ptr, 
				sizeof(int),	cBinVolumePtr_global.pitch, 
				iVoxelX, iVoxelY + z * iVolumeHeight) = iBin;
		}
	}
}
// ADD-BY-LEETEN 12/14/2009-END

////////////////////////////////////////////
__global__ 
static 
void 
// MOD-BY-LEETEN 12/07/2009-FROM:
	// _FlowFusion_kernel
// TO:
_FlowDiffusion3D_kernel
// MOD-BY-LEETEN 12/07/2009-END
(
	// INPUT
	float fAttenuation,

	int iVolumeWidth,
	int iVolumeHeight,
	int iVolumeDepth,
	int iZ,

	// ADD-BY-LEETEN 11/04/2009-BEGIN
	#if				!DIFFUSION_BY_FOR_LOOP_ON_HOST
	// ADD-BY-LEETEN 11/04/2009-END

	int iNrOfYBlocks,
	int iBlockZSize,

	// ADD-BY-LEETEN 11/04/2009-BEGIN
	#endif // #if	!DIFFUSION_BY_FOR_LOOP_ON_HOST
	// ADD-BY-LEETEN 11/04/2009-END

	// output
	hipPitchedPtr cDstPitchedPtr,

	// ADD-BY-LEETEN 2009/11/25-BEGIN
	hipPitchedPtr cErrorPitchedPtr
	// ADD-BY-LEETEN 2009/11/25-END
)
{
 	int iVoxelX = blockIdx.x * blockDim.x + threadIdx.x;
	// ADD-BY-LEETEN 11/04/2009-BEGIN
	#if				DIFFUSION_BY_FOR_LOOP_ON_HOST
	// ADD-BY-LEETEN 11/04/2009-BEGIN

	int iVoxelY = blockIdx.y * blockDim.y + threadIdx.y;

	// ADD-BY-LEETEN 11/04/2009-BEGIN
	#else	// #if	DIFFUSION_BY_FOR_LOOP_ON_HOST
	int iVoxelY = (blockIdx.y % iNrOfYBlocks) * blockDim.y + threadIdx.y;
	int iBeginZ = (blockIdx.y / iNrOfYBlocks) * iBlockZSize;
	int iEndZ = min(iBeginZ + iBlockZSize, iVolumeDepth);
	#endif	// #if	DIFFUSION_BY_FOR_LOOP_ON_HOST
	// ADD-BY-LEETEN 11/04/2009-END

	#if	0	// DEL-BY-LEETEN 12/07/2009-BEGIN
		// ADD-BY-LEETEN 11/04/2009-BEGIN
		float fAttenuationDividedBy6 = fAttenuation / 6.0f;
		// ADD-BY-LEETEN 11/04/2009-END
	#endif	// DEL-BY-LEETEN 12/07/2009-END

	// MOD-BY-LEETEN 11/04/2009-FROM:
		// #if	0	// MOD-BY-LEETEN 10/02/2009-FROM:
	// TO:
	#if	DIFFUSION_BY_FOR_LOOP_ON_HOST
	// MOD-BY-LEETEN 11/04/2009-END

		// compute the central differnece
		float4 f4Value	= tex2D(t2dSrc, iVoxelX,		iVoxelY + iZ * iVolumeHeight);
		float4 f4PX		= tex2D(t2dSrc, iVoxelX + 1,	iVoxelY + iZ * iVolumeHeight);
		float4 f4NX		= tex2D(t2dSrc, iVoxelX - 1,	iVoxelY + iZ * iVolumeHeight);
		float4 f4PY		= tex2D(t2dSrc, iVoxelX,		min(iVoxelY + 1, iVolumeHeight - 1) + iZ * iVolumeHeight);
		float4 f4NY		= tex2D(t2dSrc, iVoxelX,		max(iVoxelY - 1, 0)					+ iZ * iVolumeHeight);
		float4 f4PZ		= tex2D(t2dSrc, iVoxelX,		iVoxelY + min(iZ + 1, iVolumeDepth - 1) * iVolumeHeight);
		float4 f4NZ		= tex2D(t2dSrc, iVoxelX,		iVoxelY + max(iZ - 1, 0)				* iVolumeHeight);

	#if	0	// MOD-BY-LEETEN 11/04/2009-FROM:
	#else	// MOD-BY-LEETEN 11/04/2009-TO:
		float4 f4WeightOffset = tex3D(t3dWeightOffset, iVoxelX,	iVoxelY, iZ);

		#if	0	// MOD-BY-LEETEN 2009/11/25-FROM:
		#else	// MOD-BY-LEETEN 2009/11/25-TO:
		float4 f4Result;

		#if	0	// DEL-BY-LEETEN 12/07/2009-BEGIN
			if( 0.0f != f4Value.w || 
				iZ == 0		 || iZ == iVolumeDepth - 1 || 
				iVoxelX == 0 || iVoxelX == iVolumeWidth - 1 ||
				iVoxelY == 0 || iVoxelY == iVolumeHeight - 1 )
				f4Result = f4Value;
			else
		#endif	// DEL-BY-LEETEN 12/07/2009-END

		#if	0	// MOD-BY-LEETEN 12/07/2009-FROM:
			f4Result = make_float4(
				f4WeightOffset.w * f4Value.x + (f4PX.x + f4NX.x + f4PY.x + f4NY.x + f4PZ.x + f4NZ.x - 6.0f * f4Value.x) * fAttenuationDividedBy6 + f4WeightOffset.x,
				f4WeightOffset.w * f4Value.y + (f4PX.y + f4NX.y + f4PY.y + f4NY.y + f4PZ.y + f4NZ.y - 6.0f * f4Value.y) * fAttenuationDividedBy6 + f4WeightOffset.y,
				f4WeightOffset.w * f4Value.z + (f4PX.z + f4NX.z + f4PY.z + f4NY.z + f4PZ.z + f4NZ.z - 6.0f * f4Value.z) * fAttenuationDividedBy6 + f4WeightOffset.z,
				0);
		#else	// MOD-BY-LEETEN 12/07/2009-TO:
		f4Result = make_float4(
			f4WeightOffset.w * f4Value.x + (f4PX.x + f4NX.x + f4PY.x + f4NY.x + f4PZ.x + f4NZ.x - 6.0f * f4Value.x) * fAttenuation + f4WeightOffset.x,
			f4WeightOffset.w * f4Value.y + (f4PX.y + f4NX.y + f4PY.y + f4NY.y + f4PZ.y + f4NZ.y - 6.0f * f4Value.y) * fAttenuation + f4WeightOffset.y,
			f4WeightOffset.w * f4Value.z + (f4PX.z + f4NX.z + f4PY.z + f4NY.z + f4PZ.z + f4NZ.z - 6.0f * f4Value.z) * fAttenuation + f4WeightOffset.z,
			0);
		#endif	// MOD-BY-LEETEN 12/07/2009-END

		if( iVoxelX < iVolumeWidth && iVoxelY < iVolumeHeight )
		{
			*ADDRESS_2D(
				float4, cDstPitchedPtr.ptr, 
				sizeof(float4), cDstPitchedPtr.pitch, 
				iVoxelX, iVoxelY + iZ * iVolumeHeight) = f4Result;

			float4 f4Diff;
			f4Diff.x = f4Value.x - f4Result.x;
			f4Diff.y = f4Value.y - f4Result.y;
			f4Diff.z = f4Value.z - f4Result.z;
			float fDiff = f4Diff.x * f4Diff.x + f4Diff.y * f4Diff.y + f4Diff.z * f4Diff.z;
			*ADDRESS_2D(
				float, cErrorPitchedPtr.ptr, 
				sizeof(float), cErrorPitchedPtr.pitch, 
				iVoxelX, iVoxelY + iZ * iVolumeHeight) = fDiff;
		}
	#endif	// MOD-BY-LEETEN 2009/11/25-END

	#endif	// MOD-BY-LEETEN 11/04/2009-END

	// MOD-BY-LEETEN 11/04/2009-FROM:
		// #else	// MOD-BY-LEETEN 10/02/2009-TO:
	// TO:
	#else	// #if	DIFFUSION_BY_FOR_LOOP_ON_HOST
	// MOD-BY-LEETEN 11/04/2009-END

	float4 f4Value;
	float4 f4PX;
	float4 f4NX;
	float4 f4PY;
	float4 f4NY;
	float4 f4PZ;
	float4 f4NZ;

	#if	0	// MOD-BY-LEETEN 11/04/2009-FROM:
	#else	// MOD-BY-LEETEN 11/04/2009-TO:

	float4 *pf4DstPtr = ADDRESS_2D(
							float4, cDstPitchedPtr.ptr, 
							sizeof(float4), cDstPitchedPtr.pitch, 
							iVoxelX, iVoxelY + iBeginZ * iVolumeHeight);

	// ADD-BY-LEETEN 2009/11/25-BEGIN
	float *pfErrorPtr = ADDRESS_2D(
							float, cErrorPitchedPtr.ptr, 
							sizeof(float), cErrorPitchedPtr.pitch, 
							iVoxelX, iVoxelY + iBeginZ * iVolumeHeight);
	// ADD-BY-LEETEN 2009/11/25-END

	#endif	// MOD-BY-LEETEN 11/04/2009-END

		#if	0	// MOD-BY-LEETEN 11/04/2009-FROM:
		#else	// MOD-BY-LEETEN 11/04/2009-TO:
		f4Value		= tex2D(t2dSrc, iVoxelX,		iVoxelY + iBeginZ				* iVolumeHeight);
		f4NZ		= tex2D(t2dSrc, iVoxelX,		iVoxelY + max(iBeginZ - 1, 0)	* iVolumeHeight);
		#endif	// MOD-BY-LEETEN 11/04/2009-END

		for(int z = iBeginZ; z < iEndZ; z++, f4NZ = f4Value, f4Value = f4PZ)
		{
			// ADD-BY-LEETEN 10/02/2009-BEGIN
			float4 f4WeightOffset = tex3D(t3dWeightOffset, iVoxelX,	iVoxelY, z);
			// ADD-BY-LEETEN 10/02/2009-END
			// f4Value		= tex2D(t2dSrc, iVoxelX,		iVoxelY + z * iVolumeHeight);
			#if	!USE_SHARED_MEMORY
			f4PX		= tex2D(t2dSrc, iVoxelX + 1,	iVoxelY + z * iVolumeHeight);
			f4NX		= tex2D(t2dSrc, iVoxelX - 1,	iVoxelY + z * iVolumeHeight);
			f4PY		= tex2D(t2dSrc, iVoxelX,		min(iVoxelY + 1, iVolumeHeight - 1) + z * iVolumeHeight);
			f4NY		= tex2D(t2dSrc, iVoxelX,		max(iVoxelY - 1, 0)					+ z * iVolumeHeight);

			#else	// #if	!USE_SHARED_MEMORY
			#endif	// #if	!USE_SHARED_MEMORY
			f4PZ		= tex2D(t2dSrc, iVoxelX,		iVoxelY + min(z + 1, iVolumeDepth - 1) * iVolumeHeight);
			// f4NZ		= tex2D(t2dSrc, iVoxelX,		iVoxelY + max(z - 1, 0)				* iVolumeHeight);

			// store the result back to the dst.
			#if	0	// MOD-BY-LEETEN 2009/11/10-FROM:
			#else	// MOD-BY-LEETEN 2009/11/10-TO:
			float4 f4Result;

			#if	0	// DEL-BY-LEETEN 12/07/2009-BEGIN
				if( 0.0f != f4Value.w || 
					z == 0 || z == iVolumeDepth - 1 || 
					iVoxelX == 0 || iVoxelX == iVolumeWidth - 1 ||
					iVoxelY == 0 || iVoxelY == iVolumeHeight - 1 )
					f4Result = f4Value;
				else
			#endif	// DEL-BY-LEETEN 12/07/2009-END	

				#if	0	// MOD-BY-LEETEN 12/07/2009-FROM:
					f4Result = make_float4(
						f4WeightOffset.w * f4Value.x + (f4PX.x + f4NX.x + f4PY.x + f4NY.x + f4PZ.x + f4NZ.x - 6.0f * f4Value.x) * fAttenuationDividedBy6 + f4WeightOffset.x,
						f4WeightOffset.w * f4Value.y + (f4PX.y + f4NX.y + f4PY.y + f4NY.y + f4PZ.y + f4NZ.y - 6.0f * f4Value.y) * fAttenuationDividedBy6 + f4WeightOffset.y,
						f4WeightOffset.w * f4Value.z + (f4PX.z + f4NX.z + f4PY.z + f4NY.z + f4PZ.z + f4NZ.z - 6.0f * f4Value.z) * fAttenuationDividedBy6 + f4WeightOffset.z,
						0);
				#else	// MOD-BY-LEETEN 12/07/2009-TO:
				f4Result = make_float4(	
					f4WeightOffset.w * f4Value.x + (f4PX.x + f4NX.x + f4PY.x + f4NY.x + f4PZ.x + f4NZ.x - 6.0f * f4Value.x) * fAttenuation + f4WeightOffset.x,
					f4WeightOffset.w * f4Value.y + (f4PX.y + f4NX.y + f4PY.y + f4NY.y + f4PZ.y + f4NZ.y - 6.0f * f4Value.y) * fAttenuation + f4WeightOffset.y,
					f4WeightOffset.w * f4Value.z + (f4PX.z + f4NX.z + f4PY.z + f4NY.z + f4PZ.z + f4NZ.z - 6.0f * f4Value.z) * fAttenuation + f4WeightOffset.z,
					0.0);
				#endif	// MOD-BY-LEETEN 12/07/2009-END
			#endif	// MOD-BY-LEETEN 2009/11/10-TO:

			// ADD-BY-LEETEN 11/04/2009-BEGIN
			if( iVoxelX < iVolumeWidth && iVoxelY < iVolumeHeight )
			// ADD-BY-LEETEN 11/04/2009-END
			{
				pf4DstPtr[0] = f4Result;

				// ADD-BY-LEETEN 2009/11/25-BEGIN
				float4 f4Diff;
				f4Diff.x = f4Value.x - f4Result.x;
				f4Diff.y = f4Value.y - f4Result.y;
				f4Diff.z = f4Value.z - f4Result.z;
				float fDiff;
				// float fDiff = f4Diff.x * f4Diff.x + f4Diff.y * f4Diff.y + f4Diff.z * f4Diff.z;
				fDiff = max(abs(f4Diff.x), max(abs(f4Diff.y), abs(f4Diff.z)));
				pfErrorPtr[0] = fDiff;
				// ADD-BY-LEETEN 2009/11/25-END
			}

			pf4DstPtr += iVolumeHeight * cDstPitchedPtr.pitch / sizeof(float4);

			// ADD-BY-LEETEN 2009/11/25-BEGIN
			pfErrorPtr += iVolumeHeight * cErrorPitchedPtr.pitch / sizeof(float);
			// ADD-BY-LEETEN 2009/11/25-END
		}
	// DEL-BY-LEETEN 11/04/2009-BEGIN
		// }
	// DEL-BY-LEETEN 11/04/2009-END

	#endif	// MOD-BY-LEETEN 10/02/2009-END
}


extern "C" {

// MOD-BY-LEETEN 12/07/2009-FROM:
	// void _FlowFusionFree()
// TO:
void 
_FlowDiffusionFree()
// MOD-BY-LEETEN 12/07/2009-END
{
	FREE_MEMORY_ON_HOST(pf4Volume_host);
	for(int i = 0; i < 2; i++)
		FREE_MEMORY(pcVolumePtrs_global[i].ptr);
	FREE_ARRAY(cWeightOffsetVolume_array);
	FREE_MEMORY_ON_HOST(pf4WeightOffsetVolume_host);
	// ADD-BY-LEETEN 2009/11/25-BEGIN
	FREE_MEMORY(cErrorVolume_device.ptr);
	FREE_MEMORY(cErrorSum_device.ptr);

	#if	USE_CUDPP
	cudppDestroyPlan(cScanplan);
	#endif	// #if	USE_CUDPP
	// ADD-BY-LEETEN 2009/11/25-END

	// ADD-BY-LEETEN 12/14/2009-BEGIN
	FREE_ARRAY(pcAngleMap_array);	

	FREE_MEMORY(cSrcBinVolumePtr_global.ptr);
	FREE_MEMORY(cDstBinVolumePtr_global.ptr);
	FREE_MEMORY_ON_HOST(piSrcAngleBinVolume_host);
	FREE_MEMORY_ON_HOST(piDstAngleBinVolume_host);
	// ADD-BY-LEETEN 12/14/2009-END
}

void
// MOD-BY-LEETEN 12/07/2009-FROM:
	// _FlowFusionInit(
// TO:
_FlowDiffusionInit(
// MOD-BY-LEETEN 12/07/2009-END
	int iVolumeWidth,
	int iVolumeHeight,
	int iVolumeDepth
)
{
	// MOD-BY-LEETEN 12/07/2009-FROM:
		// CLOCK_INIT(PRINT_FLOW_FUSION_TIMING, "_FlowFusionInit(): ");
	// TO:
	CLOCK_INIT(PRINT_FLOW_FUSION_TIMING, __FUNCTION__ ": ");
	// MOD-BY-LEETEN 12/07/2009-END

	// MOD-BY-LEETEN 12/14/2009-FROM:
		// int iNrOfVoxels = iVolumeWidth * iVolumeHeight * iVolumeDepth;
	// TO:
	iNrOfVoxels = iVolumeWidth * iVolumeHeight * iVolumeDepth;
	// MOD-BY-LEETEN 12/14/2009-END

	cVolumeExtent	= make_hipExtent(
		iVolumeWidth * sizeof(float4),
		iVolumeHeight,
		iVolumeDepth);
	// allocate a local copy in the CPU side
	CLOCK_BEGIN(PRINT_FLOW_FUSION_TIMING);
	CUDA_SAFE_CALL(
		hipHostMalloc(
			(void**)&pf4Volume_host,
			sizeof(pf4Volume_host[0]) * iNrOfVoxels) );
	CLOCK_END(PRINT_FLOW_FUSION_TIMING, true);

	// Allocate 2 volumes on the  GPU's global memory.
	// in each iteration, one of them will be used as the source
	// 3D texture, and the other will be served as the dst.
	CLOCK_BEGIN(PRINT_FLOW_FUSION_TIMING);

	for(int i = 0; i < 2; i++)
	{
		CUDA_SAFE_CALL( 
			hipMalloc3D(&pcVolumePtrs_global[i], cVolumeExtent) );
	}

	// ADD-BY-LEETEN 2009/11/25-BEGIN
	cErrorVolumeExtent	= make_hipExtent(
		iVolumeWidth,
		iVolumeHeight,
		iVolumeDepth);
	CUDA_SAFE_CALL( 
		hipMalloc3D(&cErrorVolume_device,	cErrorVolumeExtent) );
	CUDA_SAFE_CALL( 
		hipMalloc3D(&cErrorSum_device,		cErrorVolumeExtent) );

	#if	USE_CUDPP
	// cConfig.op = CUDPP_ADD;
	cConfig.op = CUDPP_MAX;
	cConfig.datatype = CUDPP_FLOAT;
	cConfig.algorithm = CUDPP_SCAN;
	cConfig.options = CUDPP_OPTION_FORWARD | CUDPP_OPTION_INCLUSIVE;

	cudppPlan(&cScanplan, cConfig, iVolumeWidth * iVolumeHeight * iVolumeDepth, iVolumeHeight * iVolumeDepth, cErrorVolume_device.pitch);  
	#endif	// #if	USE_CUDPP
	// ADD-BY-LEETEN 2009/11/25-END

	// ADD-BY-LEETEN 10/02/2009-BEGIN
	cVolumeExtent_array = make_hipExtent(
		iVolumeWidth,
		iVolumeHeight,
		iVolumeDepth);

	CUDA_SAFE_CALL( 
		hipMalloc3DArray(&cWeightOffsetVolume_array, &cWeightOffsetChannelDesc, cVolumeExtent_array) );

	CUDA_SAFE_CALL(
		hipHostMalloc(
			(void**)&pf4WeightOffsetVolume_host,
			sizeof(pf4WeightOffsetVolume_host[0]) * iNrOfVoxels) );
	// ADD-BY-LEETEN 10/02/2009-END

	CLOCK_END(PRINT_FLOW_FUSION_TIMING, true);

	CLOCK_PRINT(PRINT_FLOW_FUSION_TIMING);
}

// ADD-BY-LEETEN 12/14/2009-BEGIN
void 
_FlowDiffusionSetAngleMap(int *piAngleMap, int iNrOfPhis, int iNrOfThetas)
{
	// allocate an array on the GPU side

	// upload the angle map to the araray

	// setup the texture for the angle map
	/*
    t2dAngleMap.addressMode[0] =	hipAddressModeClamp;
    t2dAngleMap.addressMode[1] =	hipAddressModeClamp;
    t2dAngleMap.filterMode =		hipFilterModePoint;
    t2dAngleMap.normalized =		true;
	*/
	SETUP_ARRAY(pcAngleMap_array, iNrOfPhis, iNrOfThetas, 32, 0, 0, 0, hipChannelFormatKindSigned);	
	BIND_ARRAY_AS_TEXTURE(
		t2dAngleMap, pcAngleMap_array, 
		hipAddressModeClamp, hipAddressModeClamp, hipFilterModePoint, true);	

    CUDA_SAFE_CALL( 
		hipMemcpy2DToArray(
			pcAngleMap_array, 
			0, 
			0, 
			(void*)piAngleMap, 
			iNrOfPhis * sizeof(piAngleMap), 
			iNrOfPhis * sizeof(piAngleMap), 
			iNrOfThetas, 
			hipMemcpyHostToDevice) );

	// check whether the volume extent has been setup
	assert( NULL != pcVolumePtrs_global[0].ptr );

	// allocate the volume of bins in the global memorty on GPUs
	cBinVolumeExtent	= make_hipExtent(
		cVolumeExtent_array.width * sizeof(int),
		cVolumeExtent_array.height,
		cVolumeExtent_array.depth);
	CUDA_SAFE_CALL( 
		hipMalloc3D(&cSrcBinVolumePtr_global, cBinVolumeExtent) );
	CUDA_SAFE_CALL( 
		hipMalloc3D(&cDstBinVolumePtr_global, cBinVolumeExtent) );

	// allocate the volume of bins in the host side
	CUDA_SAFE_CALL(
		hipHostMalloc(
			(void**)&piSrcAngleBinVolume_host,
			sizeof(piSrcAngleBinVolume_host[0]) * iNrOfVoxels) );

	CUDA_SAFE_CALL(
		hipHostMalloc(
			(void**)&piDstAngleBinVolume_host,
			sizeof(piDstAngleBinVolume_host[0]) * iNrOfVoxels) );

	// setup the src. texture for bin lookup
	t2dBinVolumeSrc.addressMode[0] = hipAddressModeClamp;
	t2dBinVolumeSrc.addressMode[1] = hipAddressModeClamp;
	t2dBinVolumeSrc.filterMode =	hipFilterModePoint;
	t2dBinVolumeSrc.normalized =	false;
}

void
_Vector3DToVolume(
	int iVolumeWidth,
	int iVolumeHeight,
	int iVolumeDepth,
	int iBlockZSize,
	hipPitchedPtr cVolumePtr_global,	// input
	hipPitchedPtr cBinVolumePtr_global	// output
)
{
	CUDA_SAFE_CALL_NO_SYNC(
		hipBindTexture2D(
			0, 
			t2dBinVolumeSrc, 
			cVolumePtr_global.ptr, 
			hipCreateChannelDesc<float4>(),
			iVolumeWidth, 
			iVolumeHeight * iVolumeDepth, 
			cVolumePtr_global.pitch) );

	dim3 v3Blk = dim3(BLOCK_DIM_X, BLOCK_DIM_Y);
	dim3 v3Grid = dim3(
		(unsigned int)ceilf((float)iVolumeWidth	 / (float)v3Blk.x),
		unsigned int(ceilf( float(iVolumeHeight) / float(v3Blk.y) ) ) *
		unsigned int(ceilf( float(iVolumeDepth) /  float(iBlockZSize) ) ) );

	_Vector3DToVolume_kernel<<<v3Grid, v3Blk, 0>>>
	(
		iVolumeWidth,
		iVolumeHeight,
		iVolumeDepth,
		int(ceilf(float(iVolumeHeight) / float(BLOCK_DIM_Y))),
		iBlockZSize,
		cBinVolumePtr_global
	);
	CUT_CHECK_ERROR("_Vector3DToVolume_kernel() failed");
}

void
_ComputeSrcBinVolume
(
	int iVolumeWidth,
	int iVolumeHeight,
	int iVolumeDepth,
	int iBlockZSize,
	float *pfSrcVolume
)
{
	CLOCK_INIT(SHOW_COMPUTE_SRC_BIN_VOLUME_TIMING, __FUNCTION__ ": ");

	CLOCK_BEGIN(SHOW_COMPUTE_SRC_BIN_VOLUME_TIMING);
	for(int v = 0,		d = 0; d < iVolumeDepth;	d++)
		for(int			h = 0; h < iVolumeHeight;	h++)
			for(int		w = 0; w < iVolumeWidth;	w++, v++)
			{
				pf4Volume_host[v].x = pfSrcVolume[v * 3 + 0];
				pf4Volume_host[v].y = pfSrcVolume[v * 3 + 1];
				pf4Volume_host[v].z = pfSrcVolume[v * 3 + 2];
				pf4Volume_host[v].w = 0.0f;
			}
	CLOCK_END(SHOW_COMPUTE_SRC_BIN_VOLUME_TIMING, false);

	CLOCK_BEGIN(SHOW_COMPUTE_SRC_BIN_VOLUME_TIMING);
    hipMemcpy3DParms cCopyParamsHostToDevice = {0};

    cCopyParamsHostToDevice.srcPtr   = make_hipPitchedPtr(
		(void*)pf4Volume_host, 
		cVolumeExtent.width, 
		iVolumeWidth * sizeof(float4), 
		iVolumeHeight);
    cCopyParamsHostToDevice.dstPtr		= pcVolumePtrs_global[0];
    cCopyParamsHostToDevice.extent		= cVolumeExtent;
    cCopyParamsHostToDevice.kind		= hipMemcpyHostToDevice;
    CUDA_SAFE_CALL( hipMemcpy3D(&cCopyParamsHostToDevice) );  

	CLOCK_END(SHOW_COMPUTE_SRC_BIN_VOLUME_TIMING, false);

	CLOCK_BEGIN(SHOW_COMPUTE_SRC_BIN_VOLUME_TIMING);
	_Vector3DToVolume(
		iVolumeWidth,
		iVolumeHeight,
		iVolumeDepth,
		iBlockZSize,
		pcVolumePtrs_global[0],
		cSrcBinVolumePtr_global);
	CLOCK_END(SHOW_COMPUTE_SRC_BIN_VOLUME_TIMING, false);

	CLOCK_PRINT(SHOW_COMPUTE_SRC_BIN_VOLUME_TIMING);
}

void 
_GetSrcBinVolume(int *piBinVolume)
{
	hipMemcpy3DParms cCopyParamsDeviceToHost = {0};
	cCopyParamsDeviceToHost.srcPtr		= cSrcBinVolumePtr_global;
	cCopyParamsDeviceToHost.dstPtr		= make_hipPitchedPtr(
		(void*)piSrcAngleBinVolume_host, 
		cVolumeExtent_array.width * sizeof(int),	// cSrcBinVolumePtr_global.width, 
		cVolumeExtent_array.width * sizeof(int), 
		cVolumeExtent_array.height);
	cCopyParamsDeviceToHost.extent		= cBinVolumeExtent;
	cCopyParamsDeviceToHost.kind		= hipMemcpyDeviceToHost;

	CUDA_SAFE_CALL( hipMemcpy3D(&cCopyParamsDeviceToHost) );  

	memcpy(piBinVolume, piSrcAngleBinVolume_host, sizeof(piBinVolume) * iNrOfVoxels);
}

void 
_GetDstBinVolume(int *piBinVolume)
{
	hipMemcpy3DParms cCopyParamsDeviceToHost = {0};
	cCopyParamsDeviceToHost.srcPtr		= cDstBinVolumePtr_global;
	cCopyParamsDeviceToHost.dstPtr		= make_hipPitchedPtr(
		(void*)piDstAngleBinVolume_host, 
		cVolumeExtent_array.width * sizeof(int),	// cDstBinVolumePtr_global.width, 
		cVolumeExtent_array.width * sizeof(int), 
		cVolumeExtent_array.height);
	cCopyParamsDeviceToHost.extent		= cBinVolumeExtent;
	cCopyParamsDeviceToHost.kind		= hipMemcpyDeviceToHost;

	CUDA_SAFE_CALL( hipMemcpy3D(&cCopyParamsDeviceToHost) );  

	memcpy(piBinVolume, piDstAngleBinVolume_host, sizeof(piBinVolume) * iNrOfVoxels);
}

// ADD-BY-LEETEN 12/14/2009-END

void
// MOD-BY-LEETEN 12/07/2009-FROM:
	// _FlowFusion(
// TO:
_FlowDiffusion(
// MOD-BY-LEETEN 12/07/2009-END
	float fAttenuation,
	int iNrOfIterations,
	int iVolumeWidth,
	int iVolumeHeight,
	int iVolumeDepth,
	// ADD-BY-LEETEN 10/02/2009-BEGIN
	int iBlockZSize,
	float *pfWeightVolume,
	// MOD-BY-LEETEN 2009/11/10-FROM:
		// float *pfOffsetVolume,
	// TO:
	float *pfXOffsetVolume,
	float *pfYOffsetVolume,
	float *pfZOffsetVolume,
	// MOD-BY-LEETEN 2009/11/10-END
	// ADD-BY-LEETEN 10/02/2009-END
	float *pfSrcVolume,
	float *pfDstVolume,
	// ADD-BY-LEETEN 2009/11/10-BEGIN
	int *piFlagVolume
	// ADD-BY-LEETEN 2009/11/10-END
)
{
	// MOD-BY-LEETEN 12/07/2009-FROM:
		// CLOCK_INIT(PRINT_FLOW_FUSION_TIMING, "_FlowFusion(): ");
	// TO:
	// MOD-BY-LEETEN 12/14/2009-FROM:
		// CLOCK_INIT(PRINT_FLOW_FUSION_TIMING, __FUNCTION__);
	// TO:
	CLOCK_INIT(PRINT_FLOW_FUSION_TIMING, __FUNCTION__ ": ");
	// MOD-BY-LEETEN 12/07/2009-END

	#if	0	// DEL-BY-LEETEN 12/07/2009-BEGIN
		// ADD-BY-LEETEN 2009/11/10-BEGIN
		fAttenuation *= 6.0f;
		// ADD-BY-LEETEN 2009/11/10-END
	#endif	// DEL-BY-LEETEN 12/07/2009-END

	// MOD-BY-LEETEN 10/02/2009-FROM:
		// dim3 v3Blk = dim3(16, 8);
	// TO:
	dim3 v3Blk = dim3(BLOCK_DIM_X, BLOCK_DIM_Y);
	// MOD-BY-LEETEN 10/02/2009-END
	dim3 v3Grid = dim3(
		(unsigned int)ceilf((float)iVolumeWidth	 / (float)v3Blk.x),
		// ADD-BY-LEETEN 11/04/2009-BEGIN
		#if					DIFFUSION_BY_FOR_LOOP_ON_HOST	
		// ADD-BY-LEETEN 11/04/2009-END

		(unsigned int)ceilf((float)iVolumeHeight / (float)v3Blk.y));

		// ADD-BY-LEETEN 11/04/2009-BEGIN
		#else	// #if		DIFFUSION_BY_FOR_LOOP_ON_HOST
		unsigned int(ceilf( float(iVolumeHeight) / float(v3Blk.y) ) ) *
		unsigned int(ceilf( float(iVolumeDepth) /  float(iBlockZSize) ) ) );
		#endif	// #if		DIFFUSION_BY_FOR_LOOP_ON_HOST
		// ADD-BY-LEETEN 11/04/2009-END

	// convert the #channels in the src volume from 3 to 4
	CLOCK_BEGIN(PRINT_FLOW_FUSION_TIMING);
	for(int v = 0,		d = 0; d < iVolumeDepth;	d++)
		for(int			h = 0; h < iVolumeHeight;	h++)
			for(int		w = 0; w < iVolumeWidth;	w++, v++)
			{
				pf4Volume_host[v].x = pfSrcVolume[v * 3 + 0];
				pf4Volume_host[v].y = pfSrcVolume[v * 3 + 1];
				pf4Volume_host[v].z = pfSrcVolume[v * 3 + 2];

				// ADD-BY-LEETEN 2009/11/10-BEGIN
				if( piFlagVolume )
					pf4Volume_host[v].w = float(piFlagVolume[v]);
				else
				// ADD-BY-LEETEN 2009/11/10-END
				pf4Volume_host[v].w = 0.0f;
			}

	// ADD-BY-LEETEN 10/02/2009-BEGIN
	for(int v = 0,		d = 0; d < iVolumeDepth;	d++)
		for(int			h = 0; h < iVolumeHeight;	h++)
			for(int		w = 0; w < iVolumeWidth;	w++, v++)
			{
				#if	0	// MOD-BY-LEETEN 2009/11/10-FROM:
				#else	// MOD-BY-LEETEN 2009/11/10-TO:
				// ADD-BY-LEETEN 12/07/2009-BEGIN
				if( pfXOffsetVolume )
				// ADD-BY-LEETEN 12/07/2009-END
				pf4WeightOffsetVolume_host[v].x = pfXOffsetVolume[v];
				// ADD-BY-LEETEN 12/07/2009-BEGIN
				if( pfYOffsetVolume )
				// ADD-BY-LEETEN 12/07/2009-END
				pf4WeightOffsetVolume_host[v].y = pfYOffsetVolume[v];
				// ADD-BY-LEETEN 12/07/2009-BEGIN
				if( pfZOffsetVolume )
				// ADD-BY-LEETEN 12/07/2009-END
				pf4WeightOffsetVolume_host[v].z = pfZOffsetVolume[v];
				pf4WeightOffsetVolume_host[v].w = 1.0f - pfWeightVolume[v];
				#endif	// MOD-BY-LEETEN 2009/11/10-END
			}
	// ADD-BY-LEETEN 10/02/2009-END

	CLOCK_END(PRINT_FLOW_FUSION_TIMING, true);

	// Upload the input volume to the first volume on the GPU
	CLOCK_BEGIN(PRINT_FLOW_FUSION_TIMING);

    hipMemcpy3DParms cCopyParamsHostToDevice = {0};

    cCopyParamsHostToDevice.srcPtr   = make_hipPitchedPtr(
		(void*)pf4Volume_host, 
		cVolumeExtent.width, 
		iVolumeWidth * sizeof(float4), 
		iVolumeHeight);
    cCopyParamsHostToDevice.dstPtr		= pcVolumePtrs_global[0];
    cCopyParamsHostToDevice.extent		= cVolumeExtent;
    cCopyParamsHostToDevice.kind		= hipMemcpyHostToDevice;
    CUDA_SAFE_CALL( hipMemcpy3D(&cCopyParamsHostToDevice) );  

	// ADD-BY-LEETEN 10/02/2009-BEGIN
    hipMemcpy3DParms cCopyParamsHostToArray = {0};
    cCopyParamsHostToArray.srcPtr   = make_hipPitchedPtr(
		(void*)pf4WeightOffsetVolume_host, 
		iVolumeWidth * sizeof(float4), 
		iVolumeWidth, 
		iVolumeHeight);
    cCopyParamsHostToArray.dstArray = cWeightOffsetVolume_array;
    cCopyParamsHostToArray.extent   = cVolumeExtent_array;
    cCopyParamsHostToArray.kind     = hipMemcpyHostToDevice;
    CUDA_SAFE_CALL( 
		hipMemcpy3D(&cCopyParamsHostToArray) );  

    t3dWeightOffset.addressMode[0] = hipAddressModeClamp;
    t3dWeightOffset.addressMode[1] = hipAddressModeClamp;
    t3dWeightOffset.addressMode[2] = hipAddressModeClamp;
    t3dWeightOffset.filterMode		= hipFilterModePoint;
    t3dWeightOffset.normalized		= false; 

    // bind array to 3D texture
    CUDA_SAFE_CALL(
		hipBindTextureToArray(t3dWeightOffset, cWeightOffsetVolume_array, cWeightOffsetChannelDesc));
	// ADD-BY-LEETEN 10/02/2009-END

	CLOCK_END(PRINT_FLOW_FUSION_TIMING, true);

	// calcuate the fusion operator on GPUs
	CLOCK_BEGIN(PRINT_FLOW_FUSION_TIMING);

    t2dSrc.addressMode[0] = hipAddressModeClamp;
    t2dSrc.addressMode[1] = hipAddressModeClamp;
    t2dSrc.filterMode =		hipFilterModePoint;
    t2dSrc.normalized =		false;

	int iSrc = 0;

	for(int i = 0; 
			i < iNrOfIterations; 
			i++,				iSrc = 1 - iSrc)
	{
		// bind the src. volume as the 3D texture

		hipChannelFormatDesc cChannelDesc =
			hipCreateChannelDesc<float4>();

		CUDA_SAFE_CALL_NO_SYNC(
			hipBindTexture2D(
				0, 
				t2dSrc, 
				pcVolumePtrs_global[iSrc].ptr, 
				cChannelDesc,
				iVolumeWidth, 
				iVolumeHeight * iVolumeDepth, 
				pcVolumePtrs_global[iSrc].pitch) );


		// ADD-BY-LEETEN 12/07/2009-BEGIN
		if( iVolumeDepth <= 2 )
			_FlowDiffusion2D_kernel<<<v3Grid, v3Blk, 0>>>
			(
				fAttenuation,
				iVolumeWidth,
				iVolumeHeight,
				pcVolumePtrs_global[1 - iSrc],
				cErrorVolume_device
			);	
		else
		{
		// ADD-BY-LEETEN 12/07/2009-END

		// launch the kernel to compute the diffusion operator for 1 iteration
		// MOD-BY-LEETEN 11/04/2009-FROM:
			// #if	0	// MOD-BY-LEETEN 10/02/2009-FROM:
		// TO:
		#if	DIFFUSION_BY_FOR_LOOP_ON_HOST
		// MOD-BY-LEETEN 11/04/2009-END

			for(int z = 0; z < iVolumeDepth; z++)
				// MOD-BY-LEETEN 12/07/2009-FROM:
					// _FlowFusion_kernel<<<v3Grid, v3Blk, 0>>>
				// TO:
				_FlowDiffusion3D_kernel<<<v3Grid, v3Blk, 0>>>
				// MOD-BY-LEETEN 12/07/2009-END
				(
					fAttenuation,
					iVolumeWidth,
					iVolumeHeight,
					iVolumeDepth,
					z,

					// output
					pcVolumePtrs_global[1 - iSrc],
					// ADD-BY-LEETEN 2009/11/25-BEGIN
					cErrorVolume_device
					// ADD-BY-LEETEN 2009/11/25-END
				);	
		// MOD-BY-LEETEN 11/04/2009-FROM:
			// #else	// MOD-BY-LEETEN 10/02/2009-TO:
		// TO:
		#else	// #if	DIFFUSION_BY_FOR_LOOP_ON_HOST
		// MOD-BY-LEETEN 11/04/2009-END

		// _FlowFusion_kernel<<<v3Grid, v3Blk, v3Blk.x * v3Blk.y * sizeof(float4)>>>
		// MOD-BY-LEETEN 12/07/2009-FROM:
			// _FlowFusion_kernel<<<v3Grid, v3Blk, 0>>>
		// TO:
		_FlowDiffusion3D_kernel<<<v3Grid, v3Blk, 0>>>
		// MOD-BY-LEETEN 12/07/2009-END
		(
			fAttenuation,
			iVolumeWidth,
			iVolumeHeight,
			iVolumeDepth,
			0,
			// ADD-BY-LEETEN 11/04/2009-BEGIN
			int(ceilf(float(iVolumeHeight) / float(BLOCK_DIM_Y))),
			iBlockZSize,
			// ADD-BY-LEETEN 11/04/2009-END
			// output
			pcVolumePtrs_global[1 - iSrc],
			// ADD-BY-LEETEN 2009/11/25-BEGIN
			cErrorVolume_device
			// ADD-BY-LEETEN 2009/11/25-END
		);
		#endif	// MOD-BY-LEETEN 10/02/2009-END
		// ADD-BY-LEETEN 12/07/2009-BEGIN
		}
		// ADD-BY-LEETEN 12/07/2009-END
		CUT_CHECK_ERROR("_FlowFusion_kernel() failed");

		// ADD-BY-LEETEN 2009/11/25-BEGIN
		#if	USE_CUDPP
		cudppScan(cScanplan, cErrorSum_device.ptr, cErrorVolume_device.ptr, iVolumeWidth * iVolumeHeight * iVolumeDepth);

		float fError;
	    CUDA_SAFE_CALL( 
			hipMemcpy(
				&fError, 
				ADDRESS_2D(
					float, cErrorSum_device.ptr, 
					sizeof(float), cErrorSum_device.pitch, 
					iVolumeWidth - 1, iVolumeHeight - 1 + (iVolumeDepth - 1 ) * iVolumeHeight),
				sizeof(float), 
				hipMemcpyDeviceToHost));

		/*
		fError /= float(iVolumeWidth * iVolumeHeight * iVolumeDepth * 3);
		fError = sqrtf(fError);
		*/
		fprintf(stderr, "Error: %f\n", fError);

		if( fError < 1e-10 )
			break;
		#endif	// #if	USE_CUDPP
		// ADD-BY-LEETEN 2009/11/25-END

	}

	// ADD-BY-LEETEN 12/14/2009-BEGIN
	if( iVolumeDepth > 2 )
	{
		_Vector3DToVolume(
			iVolumeWidth,
			iVolumeHeight,
			iVolumeDepth,
			iBlockZSize,
			pcVolumePtrs_global[iSrc],
			cDstBinVolumePtr_global);
			// piDstAngleBinVolume_host);
	}
	// ADD-BY-LEETEN 12/14/2009-END

	CLOCK_END(PRINT_FLOW_FUSION_TIMING, true);

	int iDst = iSrc;

	// read the result from latest iteration back to the CPU side
	CLOCK_BEGIN(PRINT_FLOW_FUSION_TIMING);

	hipMemcpy3DParms cCopyParamsDeviceToHost = {0};
    cCopyParamsDeviceToHost.srcPtr		= pcVolumePtrs_global[iDst];
    cCopyParamsDeviceToHost.dstPtr		= make_hipPitchedPtr(
		(void*)pf4Volume_host, 
		cVolumeExtent.width, 
		iVolumeWidth * sizeof(float4), 
		iVolumeHeight);
    cCopyParamsDeviceToHost.extent		= cVolumeExtent;
    cCopyParamsDeviceToHost.kind		= hipMemcpyDeviceToHost;

	CUDA_SAFE_CALL( hipMemcpy3D(&cCopyParamsDeviceToHost) );  

	CLOCK_END(PRINT_FLOW_FUSION_TIMING, true);

	// convert the #channels in the dst volume from 4 to 3
	CLOCK_BEGIN(PRINT_FLOW_FUSION_TIMING);
	for(int v = 0,		d = 0; d < iVolumeDepth;	d++)
		for(int			h = 0; h < iVolumeHeight;	h++)
			for(int		w = 0; w < iVolumeWidth;	w++, v++)
			{
				pfDstVolume[v * 3 + 0] = pf4Volume_host[v].x;
				pfDstVolume[v * 3 + 1] = pf4Volume_host[v].y;
				pfDstVolume[v * 3 + 2] = pf4Volume_host[v].z;
			}
	CLOCK_END(PRINT_FLOW_FUSION_TIMING, true);

	CLOCK_PRINT(PRINT_FLOW_FUSION_TIMING);
}

#if	0	// DEL-BY-LEETEN 12/07/2009-BEGIN
	void
	_FlowFusionCPU(
		float fAttenuation,
		int iNrOfIterations,
		int iVolumeWidth,
		int iVolumeHeight,
		int iVolumeDepth,
		// ADD-BY-LEETEN 10/02/2009-BEGIN
		float *pfWeightVolume,
		float *pfOffsetVolume,
		// ADD-BY-LEETEN 10/02/2009-END
		float *pfSrcVolume,
		float *pfDstVolume
	)
	{
		CLOCK_INIT(PRINT_FLOW_FUSION_TIMING, "_FlowFusionCPU(): ");

		CLOCK_BEGIN(PRINT_FLOW_FUSION_TIMING);
		int iNrOfVoxels = iVolumeWidth * iVolumeHeight * iVolumeDepth;

		TBuffer<float> ppfTempVolume[2];
		for(int i = 0; i < 2; i++)
			ppfTempVolume[i].alloc(3 * iNrOfVoxels);
		memcpy(
			&ppfTempVolume[0][0], 
			&pfSrcVolume[0], 
			sizeof(ppfTempVolume[0][0]) * ppfTempVolume[0].USize());
		CLOCK_END(PRINT_FLOW_FUSION_TIMING, true);

		// convert the #channels in the src volume from 3 to 4
		CLOCK_BEGIN(PRINT_FLOW_FUSION_TIMING);
		int iSrc = 0;
		for(int i = 0; 
				i < iNrOfIterations; 
				i++,				iSrc = 1 - iSrc)
		{
			int iDst = 1 - iSrc;
			for(int v = 0,		d = 0; d < iVolumeDepth;	d++)
				for(int			h = 0; h < iVolumeHeight;	h++)
					for(int		w = 0; w < iVolumeWidth;	w++, v++)
					{
						for(int c = 0; c < 3; c++)
						{
							float fV =	ppfTempVolume[iSrc][c + 3 * v];
							#if	0	// MOD-BY-LEETEN 10/02/2009-FROM:
								float fPX = ppfTempVolume[iSrc][c + 3 * (min(w+1, iVolumeWidth-1) + h * iVolumeWidth + d * iVolumeWidth * iVolumeHeight)];
								float fNX = ppfTempVolume[iSrc][c + 3 * (max(w-1, 0) + h * iVolumeWidth + d * iVolumeWidth * iVolumeHeight)];
								float fPY = ppfTempVolume[iSrc][c + 3 * (w + min(h+1, iVolumeHeight-1) * iVolumeWidth + d * iVolumeWidth * iVolumeHeight)];
								float fNY = ppfTempVolume[iSrc][c + 3 * (w + max(h-1, 0) * iVolumeWidth + d * iVolumeWidth * iVolumeHeight)];
								float fPZ = ppfTempVolume[iSrc][c + 3 * (w + h * iVolumeWidth + min(d+1, iVolumeDepth-1) * iVolumeWidth * iVolumeHeight)];
								float fNZ = ppfTempVolume[iSrc][c + 3 * (w + h * iVolumeWidth + max(d-1, 0) * iVolumeWidth * iVolumeHeight)];
							#else	// MOD-BY-LEETEN 10/02/2009-TO:
							float fPX = ppfTempVolume[iSrc][c + 3 * (v + ((w+1<iVolumeWidth)?1:0))];
							float fNX = ppfTempVolume[iSrc][c + 3 * (v - ((w-1>=0)?1:0))];
							float fPY = ppfTempVolume[iSrc][c + 3 * (v + ((h+1<iVolumeHeight)?iVolumeWidth:0))];
							float fNY = ppfTempVolume[iSrc][c + 3 * (v - ((h-1>=0)?iVolumeWidth:0))];
							float fPZ = ppfTempVolume[iSrc][c + 3 * (v + ((d+1<iVolumeDepth) ? (iVolumeWidth * iVolumeHeight) : 0))];
							float fNZ = ppfTempVolume[iSrc][c + 3 * (v - ((d-1>=0) ? (iVolumeWidth * iVolumeHeight) : 0))];
							#endif	// MOD-BY-LEETEN 10/02/2009-END
							ppfTempVolume[iDst][c + 3 * v] = 
								// MOD-BY-LEETEN 10/02/2009-FROM:
									// fV + (fPX + fNX + fPY + fNY + fPZ + fNZ - 6.0f * fV) * fAttenuation / 6.0f;
								// TO:
								pfWeightVolume[v] * fV + (fPX + fNX + fPY + fNY + fPZ + fNZ - 6.0f * fV) * fAttenuation / 6.0f + pfOffsetVolume[c + 3 * v];
								// MOD-BY-LEETEN 10/02/2009-END
						}
					}
		}
		CLOCK_END(PRINT_FLOW_FUSION_TIMING, true);

		CLOCK_BEGIN(PRINT_FLOW_FUSION_TIMING);
		int iDst = iSrc;
		memcpy(
			&pfDstVolume[0], 
			&ppfTempVolume[iDst][0], 
			sizeof(ppfTempVolume[iDst][0]) * ppfTempVolume[iDst].USize());
		CLOCK_END(PRINT_FLOW_FUSION_TIMING, true);

		CLOCK_PRINT(PRINT_FLOW_FUSION_TIMING);
	}
#endif	// DEL-BY-LEETEN 12/07/2009-END

} // extern "C"

/*

$Log: not supported by cvs2svn $
Revision 1.1.1.1  2009/12/07 20:04:02  leeten

[12/07/2009]
1. [1ST] First time checkin.

Revision 1.3  2009/12/05 21:19:51  leeten

[12/05/2009]
1. [ADD] Add a 3D texture glWeightOffset to store the weights and offsets.
2. [ADD] Specify the flag donochange to the w channel in the 3D textures.
3. [ADD] Output the error to a 3D volume.

Revision 1.2  2009/11/04 19:11:13  leeten

[2009/11/04]
1. [ADD] Add a new preprocessor USE_SHARED_MEMORY to decide whether the shared memory is utilized.
2. [ADD] Add a new preprocessor DIFFUSION_BY_FOR_LOOP_ON_HOSTto decide whether the diffusion is executed via a for loop on the host side to scan throught all XY layers.
If it is not zero, the volume will be divided along the Z direction as well, and each block will be assigned as a single block on CUDA.
3. [ADD] Add 2 new preprocessors BLOCK_DIM_X and BLOCK_DIM_Y to control the block dimension.
4. [ADD] Add 1 array and 1 3D textrue for the weight/offset volume.

Revision 1.1.1.1  2009/11/02 15:30:56  leeten

[11/02/2009]
1 [1ST] Firs time checkin.


*/
