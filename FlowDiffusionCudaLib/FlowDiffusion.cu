#include "hip/hip_runtime.h"
////////////////////////////////////////////
	#include <stdio.h>
	#include <>

	#if	0	// DEL-BY-LEETEN 2009/12/17-BEGIN
	#endif	// DEL-BY-LEETEN 2009/12/17-END


	// ADD-BY-LEETEN 12/07/2009-BEGIN
	#include "FlowDiffusion_cuda.h"
	#include "FlowDiffusion.h"
	// ADD-BY-LEETEN 12/07/2009-END	

////////////////////////////////////////////
	static hipExtent cVolumeExtent;
	static float4 *pf4Volume_host;
	static hipPitchedPtr pcVolumePtrs_global[2];
	static texture<float4, 2, hipReadModeElementType> t2dSrc;
	// ADD-BY-LEETEN 10/02/2009-BEGIN
						// array and texture reference for the weight and offset volume
	static hipExtent cVolumeExtent_array;
	static float4 *pf4WeightOffsetVolume_host;
	static hipArray *cWeightOffsetVolume_array = 0;
	static texture<float4, 3, hipReadModeElementType> t3dWeightOffset;
    static hipChannelFormatDesc cWeightOffsetChannelDesc = hipCreateChannelDesc<float4>();
	// ADD-BY-LEETEN 10/02/2009-END

	// ADD-BY-LEETEN 2009/11/25-BEGIN
	#if	CHECK_ERROR_CONVERGENCE_BY_CUDPP
    static CUDPPConfiguration cConfig;
    static CUDPPHandle cScanplan = 0;
	#endif	// #if	CHECK_ERROR_CONVERGENCE_BY_CUDPP

	// ADD-BY-LEETEN 12/18/2009-BEGIN
	#if	CHECK_ERROR_CONVERGENCE_BY_CUDPP				
	// ADD-BY-LEETEN 12/18/2009-END
	static hipExtent cErrorVolumeExtent;
	// ADD-BY-LEETEN 12/18/2009-BEGIN
	#endif	// #if	CHECK_ERROR_CONVERGENCE_BY_CUDPP	
	// ADD-BY-LEETEN 12/18/2009-END

	static hipPitchedPtr cErrorVolume_device;
	static hipPitchedPtr cErrorSum_device;
	// ADD-BY-LEETEN 2009/11/25-END

	// ADD-BY-LEETEN 12/14/2009-BEGIN
	int iNrOfVoxels;

	static texture<int, 2, hipReadModeElementType> t2dAngleMap;
	static hipArray *pcAngleMap_array;	// cuda array to hold the Gaussian kernels

	static hipExtent cBinVolumeExtent;
	#if	0	// MOD-BY-LEETEN 12/17/2009-FROM:
		static hipPitchedPtr cSrcBinVolumePtr_global;
		static hipPitchedPtr cDstBinVolumePtr_global;
	#else	// MOD-BY-LEETEN 12/17/2009-TO:
	static hipPitchedPtr cSrcBinVolume_pitched;
	static hipPitchedPtr cDstBinVolume_pitched;
	#endif	// MOD-BY-LEETEN 12/17/2009-END

	#if	0	// MOD-BY-LEETEN 2009/12/17-FROM:
		static int *piSrcAngleBinVolume_host;
		static int *piDstAngleBinVolume_host;
	#else	// MOD-BY-LEETEN 2009/12/17-TO:
						// a temp. volume in the host side to stored the 
	static int *piBinVolume_host;
	#endif	// MOD-BY-LEETEN 2009/12/17-END

						// a texture tha represents a vector field to be converted into bins 
	static texture<float4, 2, hipReadModeElementType> t2dVectorVolume;
	// ADD-BY-LEETEN 12/14/2009-END

// ADD-BY-LEETEN 12/07/2009-BEGIN
#if	0	// MOD-BY-LEETEN 2009/12/17-FROM:
#else	// MOD-BY-LEETEN 2009/12/17-TO:
	#include "FlowDiffusion2D_kernel.cu"
#endif	// MOD-BY-LEETEN 2009/12/17-END

// ADD-BY-LEETEN 12/07/2009-END

#if	0		// DEL-BY-LEETEN 2009/12/17-BEGIN
#endif	// DEL-BY-LEETEN 2009/12/17-END

// ADD-BY-LEETEN 12/17/2009-BEGIN
////////////////////////////////////////////

#include "EntropyField_kernel.cu"

__host__
static 
int 
IMirrorCoord_host(int iCoord, int iSize)
{
	int iMirroredCoord = iCoord;
	if( iCoord < 0 )
		iMirroredCoord = -iCoord;
	if( iCoord > iSize - 1 )
		iMirroredCoord = iSize - (iCoord - (iSize - 1));
	return iMirroredCoord;
}

static 
void 
_UpdateSliceToHistogram_host
(
	int iU,
	int iV,

	int3 i3Center, 
	int	iUpdateDir, 
	int iHisotgramOp, 

	int3 i3UDir, 
	int3 i3VDir,
	int iUKernelSize, 
	int iVKernelSize,

	int3 i3VolumeSize,
	int *piBinVolume_host,

	int iNrOfBins,
	int *piHistorgram_host
)
{
	int3 i3Point;
	i3Point.x = i3Center.x + (iU - iUKernelSize) * i3UDir.x + (iV - iVKernelSize) * i3VDir.x;
	i3Point.y = i3Center.y + (iU - iUKernelSize) * i3UDir.y + (iV - iVKernelSize) * i3VDir.y;
	i3Point.z = i3Center.z + (iU - iUKernelSize) * i3UDir.z + (iV - iVKernelSize) * i3VDir.z;

	// read the bin
	int3 i3TexCoord;
	i3TexCoord.x = IMirrorCoord_host(i3Point.x, i3VolumeSize.x);
	i3TexCoord.y = IMirrorCoord_host(i3Point.y, i3VolumeSize.y);
	i3TexCoord.z = IMirrorCoord_host(i3Point.z, i3VolumeSize.z);

	int iSrcBin = piBinVolume_host[i3TexCoord.x + i3TexCoord.y * i3VolumeSize.x + i3TexCoord.z * i3VolumeSize.x * i3VolumeSize.y];

	// update the histogram
	piHistorgram_host[iSrcBin] += iHisotgramOp; 
}

static 
void
_BuildHistogram_host
(
	int3 i3Center,
						// res. of the neighboring region
	int3 i3KernelSize,	
						// the bin volume
	int3 i3VolumeSize,	
	int *piBinVolume_host,								
						// res. of the joint histogram
	int iNrOfBins,		
	int *piHistorgram_host
)
{
	int3 i3Point = i3Center;
	i3Point.x -= i3KernelSize.x;
	for(int			xi = 0; xi < 2 * i3KernelSize.x + 1; xi++, i3Point.x++)
		for(int		iV = 0; iV < 2 * i3KernelSize.z + 1; iV++)
			for(int iU = 0; iU < 2 * i3KernelSize.y + 1; iU++)
				_UpdateSliceToHistogram_host(
					iU,
					iV,

					i3Point , 
					UPDATE_DIR_X, 
					HISTOGRAM_OP_ADD_SLICE, 

					make_int3(0, 1, 0), 
					make_int3(0, 0, 1),
					i3KernelSize.y, 
					i3KernelSize.z,

					i3VolumeSize,
					piBinVolume_host,

					iNrOfBins,
					piHistorgram_host
				);
}

static 
void 
_UpdateSliceToHistogram_host
(
	int3 i3Point,										// the coordinate to be computed

	int iUpdateDir, 
	int iDir,

	int3 i3KernelSize,	// res. of the neighboring region

	int3 i3VolumeSize,	// res. of the volume
	int *piBinVolume_host,								


	int iNrOfBins,		// res. of the joint histogram
	int *piHistogram_host
)
{
	int3 i3Prev = i3Point;
	int3 i3Next = i3Point;
	int3 i3UDir;
	int3 i3VDir;
	int iUKernelSize ;
	int iVKernelSize ;

						// according the direction to divide the new slice into blocks
	switch(iUpdateDir)
	{
	case UPDATE_DIR_X:	
		i3UDir = make_int3(0, 1, 0);
		i3VDir = make_int3(0, 0, 1);
		iUKernelSize = i3KernelSize.y;
		iVKernelSize = i3KernelSize.z;
		i3Prev.x -= iDir * (i3KernelSize.x + 1);
		i3Next.x += iDir * i3KernelSize.x;
		break;
	case UPDATE_DIR_Y:	
		i3UDir = make_int3(1, 0, 0);
		i3VDir = make_int3(0, 0, 1);
		iUKernelSize = i3KernelSize.x;
		iVKernelSize = i3KernelSize.z;
		i3Prev.y -= iDir * (i3KernelSize.y + 1);
		i3Next.y += iDir * i3KernelSize.y;
		break;
	case UPDATE_DIR_Z:	
		i3UDir = make_int3(1, 0, 0);
		i3VDir = make_int3(0, 1, 0);
		iUKernelSize = i3KernelSize.x;
		iVKernelSize = i3KernelSize.y;
		i3Prev.z -= iDir * (i3KernelSize.z + 1);
		i3Next.z += iDir * i3KernelSize.z;
		break;
	} // switch

	for(int		iV = 0; iV < 2 * iVKernelSize + 1; iV++)
		for(int iU = 0; iU < 2 * iUKernelSize + 1; iU++)
		{
			_UpdateSliceToHistogram_host
			(
				iU,
				iV,

				i3Next, 
				iUpdateDir, 
				HISTOGRAM_OP_ADD_SLICE, 

				i3UDir, 
				i3VDir,
				iUKernelSize, 
				iVKernelSize,

				i3VolumeSize,
				piBinVolume_host,

				iNrOfBins,
				piHistogram_host
			);

			_UpdateSliceToHistogram_host
			(
				iU,
				iV,

				i3Prev, 
				iUpdateDir, 
				HISTOGRAM_OP_SUB_SLICE, 

				i3UDir, 
				i3VDir,
				iUKernelSize, 
				iVKernelSize,

				i3VolumeSize,
				piBinVolume_host,

				iNrOfBins,
				piHistogram_host
			);
		}
}

static 
void
_ComputeEntropy_host
(
	int3 i3Point,
	int3 i3KernelSize,

						// the histogram
	int iNrOfBins,
	int *piHistogram_host,

	int3 i3VolumeSize,
	// MOD-BY-LEETEN 12/19/2009-FROM:
		// hipPitchedPtr cEntropyVolume_pitched
	// TO:
	float* pfEntropyVolume_host
	// MOD-BY-LEETEN 12/19/2009-END
)
{
	float fEntropy = 0.0f;
	for(int b = 0; b < iNrOfBins; b++)
	{
		if( 0 == piHistogram_host[b] )
			continue;

		float fProb = 
			float(piHistogram_host[b]) / 
			float(
				(2 * i3KernelSize.x + 1) * 
				(2 * i3KernelSize.y + 1) * 
				(2 * i3KernelSize.z + 1) );
		fEntropy += fProb * log2f(fProb);
	}
	fEntropy = -fEntropy;
	// ADD-BY-LEETEN 12/19/2009-BEGIN
	fEntropy = max(fEntropy, 0.0f);
	// ADD-BY-LEETEN 12/19/2009-END

	// fprintf(stderr, "Entropy(%d, %d, %d) = %f\n", i3Point.x, i3Point.y, i3Point.z, fEntropy); // TEST-DEBUG

	#if	0	// MOD-BY-LEETEN 12/19/2009-FROM:
		CUDA_SAFE_CALL(
			hipMemcpy(
				ADDRESS_2D(
					float, cEntropyVolume_pitched.ptr, 
					sizeof(float), cEntropyVolume_pitched.pitch, 
					i3Point.x, i3Point.y + i3Point.z * i3VolumeSize.y),
				&fEntropy,
				sizeof(fEntropy),
				hipMemcpyHostToDevice) );
	#else	// MOD-BY-LEETEN 12/19/2009-TO:
	pfEntropyVolume_host[i3Point.x + i3Point.y * i3VolumeSize.x + i3Point.z * i3VolumeSize.x * i3VolumeSize.y] = fEntropy;
	#endif	// MOD-BY-LEETEN 12/19/2009-END
}

// ADD-BY-LEETEN 12/18/2009-BEGIN

void 
_ComputeEntropyVolumeWithSorting_cuda
(
						// res. of the neighboring region
	int3 i3KernelSize,

						// the histogram
	int iNrOfBins,
	int *piHistogram_global,
	float *pfLogHistogram_global,

						// res. of the volume																
	int3 i3VolumeSize,	
						// bin volume																	
	hipPitchedPtr		cBinVolume_pitched,								
	hipPitchedPtr		cEntropyVolume_pitched
)
{
CLOCK_INIT(_ComputeEntropyVolume_PRINT_TIMING, __FUNCTION__ ": ");

CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_TIMING);
	const int iNrOfRows = (2 * i3KernelSize.x + 1) * (2 * i3KernelSize.y + 1) * (2 * i3KernelSize.z + 1);

	dim3 v3Blk = dim3(BLOCK_DIM_X, BLOCK_DIM_Y);
	/*
	const unsigned int iMaxMemorySpace = MAX_MEMORY_SIZE;
	int iMaxNrOfBlocks = int(floorf(
		float(iMaxMemorySpace/2) / 
		float(sizeof(int) * iNrOfRows * v3Blk.x * v3Blk.y)));
	*/
	int iNrOfXBlocks = int(ceilf(float(i3VolumeSize.x) / float(v3Blk.x)));
	#if	0	// MOD-BY-LEETEN 12/19/2009-FROM:
		int iNrOfZBlocks = int(ceilf(float(i3VolumeSize.z) / float(v3Blk.y)));
		int iMaxNrOfThreads = int(floor(double(1<<(RADIX_SORT_BITS-1)) / double(iNrOfBins)));
		int iMaxNrOfBlocks	= int(floorf(float(iMaxNrOfThreads) / float(v3Blk.x * v3Blk.y)));
		iMaxNrOfBlocks = min(iMaxNrOfBlocks, iNrOfXBlocks * iNrOfZBlocks);
		iNrOfXBlocks = min(iNrOfXBlocks, iMaxNrOfBlocks);
		iNrOfZBlocks = int(ceilf(float(iMaxNrOfBlocks) / float(iNrOfXBlocks)));
		dim3 v3Grid = dim3(iNrOfXBlocks, iNrOfZBlocks);
	#else	// MOD-BY-LEETEN 12/19/2009-TO:
	int iNrOfYBlocks = int(ceilf(float(i3VolumeSize.y) / float(v3Blk.y)));
	int iMaxNrOfThreads = int(floor(double(1<<(RADIX_SORT_BITS-1)) / double(iNrOfBins)));
	int iMaxNrOfBlocks	= int(floorf(float(iMaxNrOfThreads) / float(v3Blk.x * v3Blk.y)));
	iMaxNrOfBlocks = min(iMaxNrOfBlocks, iNrOfXBlocks * iNrOfYBlocks);
	iNrOfXBlocks = min(iNrOfXBlocks, iMaxNrOfBlocks);
	iNrOfYBlocks = int(ceilf(float(iMaxNrOfBlocks) / float(iNrOfXBlocks)));
	dim3 v3Grid = dim3(iNrOfXBlocks, iNrOfYBlocks);
	#endif	// MOD-BY-LEETEN 12/19/2009-END

	fprintf(stderr, "#BLOCKS = %d x %d\n", v3Grid.x, v3Grid.y);

	hipPitchedPtr cActiveVoxelNeighbors_pitched;
	cActiveVoxelNeighbors_pitched.xsize = v3Grid.x * v3Grid.y * v3Blk.x * v3Blk.y;
	cActiveVoxelNeighbors_pitched.ysize = iNrOfRows;
	CUDA_SAFE_CALL_NO_SYNC(
		hipMallocPitch(
			(void**)&cActiveVoxelNeighbors_pitched.ptr, 
			&cActiveVoxelNeighbors_pitched.pitch,
			cActiveVoxelNeighbors_pitched.xsize * sizeof(int), 
			cActiveVoxelNeighbors_pitched.ysize)	);

	CUDA_SAFE_CALL_NO_SYNC(
		hipMemset2D(
			cActiveVoxelNeighbors_pitched.ptr, 
			cActiveVoxelNeighbors_pitched.pitch,
			0, 
			cActiveVoxelNeighbors_pitched.xsize * sizeof(int), 
			cActiveVoxelNeighbors_pitched.ysize)	);

	hipPitchedPtr cActiveVoxelSortedNeighbors_pitched = cActiveVoxelNeighbors_pitched;
	CUDA_SAFE_CALL_NO_SYNC(
		hipMallocPitch(
			(void**)&cActiveVoxelSortedNeighbors_pitched.ptr, 
			&cActiveVoxelSortedNeighbors_pitched.pitch,
			cActiveVoxelSortedNeighbors_pitched.xsize * sizeof(int), 
			cActiveVoxelSortedNeighbors_pitched.ysize)	);

	// bind the input vin volume to the texture that represents the src. bin volume 
	t2dSrcBinVolume.addressMode[0] = hipAddressModeClamp;
	t2dSrcBinVolume.addressMode[1] = hipAddressModeClamp;
	t2dSrcBinVolume.filterMode =	hipFilterModePoint;
	t2dSrcBinVolume.normalized =	false;
	CUDA_SAFE_CALL_NO_SYNC(
		hipBindTexture2D(
			0, 
			t2dSrcBinVolume, 
			cBinVolume_pitched.ptr, 
			hipCreateChannelDesc<int>(),
			cVolumeExtent_array.width, 
			cVolumeExtent_array.height * cVolumeExtent_array.depth, 
			cBinVolume_pitched.pitch) );

	// bind the histogram as a texture
	#if	0	// MOD-BY-LEETEN 12/18/2009-FROM:
		t1dActiveVoxelSortedNeighbors.addressMode[0] = hipAddressModeClamp;
		t1dActiveVoxelSortedNeighbors.filterMode =	hipFilterModePoint;
		t1dActiveVoxelSortedNeighbors.normalized =	false;

		CUDA_SAFE_CALL_NO_SYNC(
			hipBindTexture(
				0, 
				t1dActiveVoxelSortedNeighbors, 
				cActiveVoxelSortedNeighbors_pitched.ptr, 
				hipCreateChannelDesc<unsigned int>(),
				// hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindUnsigned),
				cActiveVoxelSortedNeighbors_pitched.pitch * cActiveVoxelSortedNeighbors_pitched.ysize) );
	#else	// MOD-BY-LEETEN 12/18/2009-TO:
	t2dActiveVoxelSortedNeighbors.addressMode[0] = hipAddressModeClamp;
	t2dActiveVoxelSortedNeighbors.addressMode[1] = hipAddressModeClamp;
	t2dActiveVoxelSortedNeighbors.filterMode =	hipFilterModePoint;
	t2dActiveVoxelSortedNeighbors.normalized =	false;

	CUDA_SAFE_CALL_NO_SYNC(
		hipBindTexture2D(
			0, 
			t2dActiveVoxelSortedNeighbors, 
			cActiveVoxelSortedNeighbors_pitched.ptr, 
			hipCreateChannelDesc<unsigned int>(),
			cActiveVoxelSortedNeighbors_pitched.xsize,
			cActiveVoxelSortedNeighbors_pitched.ysize, 
			cActiveVoxelSortedNeighbors_pitched.pitch) );
	#endif	// MOD-BY-LEETEN 12/18/2009-END

	CUDPPHandle hScanPlan = 0;

	CUDPPConfiguration	cConfig;
	cConfig.op =		CUDPP_ADD;
	cConfig.datatype =	CUDPP_UINT;
	cConfig.algorithm =	CUDPP_SORT_RADIX;
	cConfig.options =	CUDPP_OPTION_KEYS_ONLY;

	assert( 
		CUDPP_SUCCESS  == cudppPlan(
			&hScanPlan,	
			cConfig, 
			cActiveVoxelSortedNeighbors_pitched.pitch * cActiveVoxelSortedNeighbors_pitched.ysize / sizeof(unsigned int),
			1, 
			0) );

CLOCK_END(_ComputeEntropyVolume_PRINT_TIMING, false);

CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_TIMING);

	#if	0	// MOD-BY-LEETEN 12/19/2009-FROM:
		for(int			z = 0; z < i3VolumeSize.z; z += v3Grid.y * v3Blk.y)
			for(int		x = 0; x < i3VolumeSize.x; x += v3Grid.x * v3Blk.x)
				for(int y = 0; y < i3VolumeSize.y; y++)
	#else	// MOD-BY-LEETEN 12/19/2009-TO:
	for(int			y = 0; y < i3VolumeSize.y; y += v3Grid.y * v3Blk.y)
		for(int		x = 0; x < i3VolumeSize.x; x += v3Grid.x * v3Blk.x)
			for(int	z = 0; z < i3VolumeSize.z; z ++ )
	#endif	// MOD-BY-LEETEN 12/19/2009-END
			{
				CLOCK_INIT(_ComputeEntropyVolume_PRINT_LOOP_TIMING, __FUNCTION__ " (main loop): ");

				CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_LOOP_TIMING);

				int3 i3BlockCorner = make_int3(x, y, z);
				_CollectNeighbors_kernel<<<v3Grid, v3Blk, 0>>>
				(
					i3BlockCorner,
					i3KernelSize,
					i3VolumeSize,
					iNrOfBins,
					cActiveVoxelNeighbors_pitched
				);
				CUT_CHECK_ERROR("_CollectNeighbors_kernel() failed");

				#if	0	// TEST-DEBUG
											unsigned int *puActiveVoxelNeighbors_host;
											CUDA_SAFE_CALL_NO_SYNC(
												hipHostMalloc(
													(void**)&puActiveVoxelNeighbors_host,
													cActiveVoxelNeighbors_pitched.pitch * cActiveVoxelSortedNeighbors_pitched.ysize) );

											CUDA_SAFE_CALL_NO_SYNC(
												hipMemcpy(
													puActiveVoxelNeighbors_host, 
													cActiveVoxelNeighbors_pitched.ptr,
													cActiveVoxelSortedNeighbors_pitched.pitch * cActiveVoxelSortedNeighbors_pitched.ysize,
													hipMemcpyDeviceToHost) );
											for(unsigned int p = 0,	n = 0; n < cActiveVoxelSortedNeighbors_pitched.ysize; n++)
												for(unsigned int	i = 0; i < cActiveVoxelNeighbors_pitched.pitch / sizeof(unsigned int); i++, p++)
												{
													unsigned int uTemp = puActiveVoxelNeighbors_host[p];
													unsigned int uId = uTemp / unsigned int(iNrOfBins);
													unsigned int uBin = uTemp % unsigned int(iNrOfBins);

													fprintf(stderr, "%d, %d, %d\n", n, uId, uBin);
												}
											FREE_MEMORY_ON_HOST(puActiveVoxelNeighbors_host);
				#endif

CLOCK_END(_ComputeEntropyVolume_PRINT_LOOP_TIMING, false);

CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_LOOP_TIMING);

				// copy the memory from the buffer _CollectNeighbors_kernel to another buffer _CollectNeighborsToBeSorted_kernel
				#if	0	// MOD-BY-LEETEN 12/18/2009-FROM:
					CUDA_SAFE_CALL_NO_SYNC(
						hipMemcpy(
							cActiveVoxelSortedNeighbors_pitched.ptr, 
							cActiveVoxelNeighbors_pitched.ptr, 
							cActiveVoxelSortedNeighbors_pitched.pitch * cActiveVoxelSortedNeighbors_pitched.ysize,
							hipMemcpyDeviceToDevice));
				#else	// MOD-BY-LEETEN 12/18/2009-TO:
				CUDA_SAFE_CALL_NO_SYNC(
					hipMemcpy2D(
						cActiveVoxelSortedNeighbors_pitched.ptr, 
						cActiveVoxelSortedNeighbors_pitched.pitch,
						cActiveVoxelNeighbors_pitched.ptr, 
						cActiveVoxelNeighbors_pitched.pitch,
						cActiveVoxelSortedNeighbors_pitched.pitch,
						cActiveVoxelSortedNeighbors_pitched.ysize,
						hipMemcpyDeviceToDevice) );
				#endif	// MOD-BY-LEETEN 12/18/2009-END

CLOCK_END(_ComputeEntropyVolume_PRINT_LOOP_TIMING, false);

CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_LOOP_TIMING);

				// call cudpp to sort the buffer _CollectNeighborsToBeSorted_kernel
				cudppSort(
					hScanPlan,
					cActiveVoxelSortedNeighbors_pitched.ptr,
					NULL,
					RADIX_SORT_BITS,
					cActiveVoxelSortedNeighbors_pitched.pitch * cActiveVoxelSortedNeighbors_pitched.ysize / sizeof(unsigned int)) ;
				CUT_CHECK_ERROR("cudppSort() failed");
CLOCK_END(_ComputeEntropyVolume_PRINT_LOOP_TIMING, false);

CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_LOOP_TIMING);

				#if	0	// TEST-DEBUG
										unsigned int *puActiveVoxelNeighbors_host;
										CUDA_SAFE_CALL_NO_SYNC(
											hipHostMalloc(
												(void**)&puActiveVoxelNeighbors_host,
												cActiveVoxelSortedNeighbors_pitched.pitch * cActiveVoxelSortedNeighbors_pitched.ysize) );
										CUDA_SAFE_CALL_NO_SYNC(
											hipMemcpy(
												puActiveVoxelNeighbors_host, 
												cActiveVoxelSortedNeighbors_pitched.ptr,
												cActiveVoxelSortedNeighbors_pitched.pitch * cActiveVoxelSortedNeighbors_pitched.ysize,
												hipMemcpyDeviceToHost) );
										for(unsigned int n = 0, i = 0; i < cActiveVoxelSortedNeighbors_pitched.pitch / sizeof(unsigned int);	i++)
										{
											unsigned int uPrevBin = 0;
											int iCount = 0;
											float fEntropy = 0.0f;
											for(unsigned		r = 0; r < cActiveVoxelSortedNeighbors_pitched.ysize;					r++,	n++)
											{
												unsigned int uBin = puActiveVoxelNeighbors_host[n];
												unsigned int uId = uBin / unsigned int(iNrOfBins);
												// unsigned int uBin = uTemp % unsigned int(iNrOfBins);

												if( uId != i )
													fprintf(stderr, "%d, %d, %d\n", n, uId, uBin);

												if( r == 0 || uBin == uPrevBin )
												{
													iCount++;
												}
												else
												{
													assert( uPrevBin < uBin );


													{
														float fProb = float(iCount)/float(iNrOfRows);
														fEntropy += fProb * log2f(fProb);
													}
													
													iCount = 1;
												}
												uPrevBin = uBin;
											}
											if( iCount > 0 )
											{
												float fProb = float(iCount)/iNrOfRows;
												fEntropy += fProb * log2f(fProb);
											}

											fEntropy *= -1.0f;
											fEntropy = max(0.0f, fEntropy);

											fprintf(stderr, "%f\n", fEntropy);
										}
										FREE_MEMORY_ON_HOST(puActiveVoxelNeighbors_host);
				#endif

CLOCK_END(_ComputeEntropyVolume_PRINT_LOOP_TIMING, false);

CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_LOOP_TIMING);
				// 
				_ComputeEntropyOnSortedNeighbors_kernel<<<v3Grid, v3Blk, 0>>>
				(
					i3BlockCorner,
					iNrOfBins,
					i3KernelSize,
					(unsigned int *)cActiveVoxelSortedNeighbors_pitched.ptr,
					i3VolumeSize,
					cEntropyVolume_pitched
				);
				CUT_CHECK_ERROR("_ComputeEntropyOnSortedNeighbors_kernel() failed");
CLOCK_END(_ComputeEntropyVolume_PRINT_LOOP_TIMING, false);

CLOCK_PRINT(_ComputeEntropyVolume_PRINT_LOOP_TIMING);
			}


CLOCK_END(_ComputeEntropyVolume_PRINT_TIMING, false);

CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_TIMING);

	cudppDestroyPlan(hScanPlan);
	FREE_MEMORY(cActiveVoxelNeighbors_pitched.ptr);

CLOCK_END(_ComputeEntropyVolume_PRINT_TIMING, false);

CLOCK_PRINT(_ComputeEntropyVolume_PRINT_TIMING);
}

void 
_ComputeEntropyVolume_cuda
(
						// res. of the neighboring region
	int3 i3KernelSize,

						// the histogram
	int iNrOfBins,
	int *piHistogram_global,
	float *pfLogHistogram_global,

						// res. of the volume																
	int3 i3VolumeSize,	
						// bin volume																	
	hipPitchedPtr		cBinVolume_pitched,								
	hipPitchedPtr		cEntropyVolume_pitched
)
{
CLOCK_INIT(_ComputeEntropyVolume_PRINT_TIMING, __FUNCTION__ ": ");

CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_TIMING);
	dim3 v3Blk = dim3(BLOCK_DIM_X, BLOCK_DIM_Y);
	const unsigned int iMaxMemorySpace = MAX_MEMORY_SIZE;
	int iMaxNrOfBlocks = int(floorf(float(iMaxMemorySpace) / float(sizeof(int) * iNrOfBins * v3Blk.x * v3Blk.y)));
	int iNrOfXBlocks = int(ceilf(float(i3VolumeSize.x) / float(v3Blk.x)));
	#if	0	// MOD-BY-LEETEN 12/19/2009-FROM:
		int iNrOfZBlocks = int(ceilf(float(i3VolumeSize.z) / float(v3Blk.y)));
		iMaxNrOfBlocks = min(iMaxNrOfBlocks, iNrOfXBlocks * iNrOfZBlocks);
		iNrOfXBlocks = min(iNrOfXBlocks, iMaxNrOfBlocks);
		iNrOfZBlocks = int(ceilf(float(iMaxNrOfBlocks) / float(iNrOfXBlocks)));
		dim3 v3Grid = dim3(iNrOfXBlocks, iNrOfZBlocks);
	#else	// MOD-BY-LEETEN 12/19/2009-TO:
	int iNrOfYBlocks = int(ceilf(float(i3VolumeSize.y) / float(v3Blk.y)));
	iMaxNrOfBlocks = min(iMaxNrOfBlocks, iNrOfXBlocks * iNrOfYBlocks);
	iNrOfXBlocks = min(iNrOfXBlocks, iMaxNrOfBlocks);
	iNrOfYBlocks = int(ceilf(float(iMaxNrOfBlocks) / float(iNrOfXBlocks)));
	dim3 v3Grid = dim3(iNrOfXBlocks, iNrOfYBlocks);
	#endif	// MOD-BY-LEETEN 12/19/2009-END
	fprintf(stderr, "MEM = %d MB; #BLOCKS = %d x %d\n", iMaxMemorySpace/(1<<20), v3Grid.x, v3Grid.y);

	hipPitchedPtr cActiveVoxelHistorgrams_pitched;
	cActiveVoxelHistorgrams_pitched.xsize = v3Grid.x * v3Grid.y * v3Blk.x * v3Blk.y;
	cActiveVoxelHistorgrams_pitched.ysize = iNrOfBins;
	CUDA_SAFE_CALL_NO_SYNC(
		hipMallocPitch(
			(void**)&cActiveVoxelHistorgrams_pitched.ptr, 
			&cActiveVoxelHistorgrams_pitched.pitch,
			cActiveVoxelHistorgrams_pitched.xsize * sizeof(int), 
			cActiveVoxelHistorgrams_pitched.ysize)	);

	CUDA_SAFE_CALL_NO_SYNC(
		hipMemset2D(
			cActiveVoxelHistorgrams_pitched.ptr, 
			cActiveVoxelHistorgrams_pitched.pitch,
			0, 
			cActiveVoxelHistorgrams_pitched.xsize * sizeof(int), 
			cActiveVoxelHistorgrams_pitched.ysize)	);

	// bind the input vin volume to the texture that represents the src. bin volume 
	t2dSrcBinVolume.addressMode[0] = hipAddressModeClamp;
	t2dSrcBinVolume.addressMode[1] = hipAddressModeClamp;
	t2dSrcBinVolume.filterMode =	hipFilterModePoint;
	t2dSrcBinVolume.normalized =	false;
	CUDA_SAFE_CALL_NO_SYNC(
		hipBindTexture2D(
			0, 
			t2dSrcBinVolume, 
			cBinVolume_pitched.ptr, 
			hipCreateChannelDesc<int>(),
			cVolumeExtent_array.width, 
			cVolumeExtent_array.height * cVolumeExtent_array.depth, 
			cBinVolume_pitched.pitch) );

	// bind the histogram as a texture
	t2dActiveVoxelHistorgrams.addressMode[0] = hipAddressModeClamp;
	t2dActiveVoxelHistorgrams.addressMode[1] = hipAddressModeClamp;
	t2dActiveVoxelHistorgrams.filterMode =	hipFilterModePoint;
	t2dActiveVoxelHistorgrams.normalized =	false;
	CUDA_SAFE_CALL_NO_SYNC(
		hipBindTexture2D(
			0, 
			t2dActiveVoxelHistorgrams, 
			cActiveVoxelHistorgrams_pitched.ptr, 
			hipCreateChannelDesc<int>(),
			cActiveVoxelHistorgrams_pitched.xsize, 
			cActiveVoxelHistorgrams_pitched.ysize,
			cActiveVoxelHistorgrams_pitched.pitch) );

CLOCK_END(_ComputeEntropyVolume_PRINT_TIMING, false);

CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_TIMING);

	#if	0	// MOD-BY-LEETEN 12/19/2009-FROM:
		for(int			z = 0; z < i3VolumeSize.z; z += v3Grid.y * v3Blk.y)
			for(int		x = 0; x < i3VolumeSize.x; x += v3Grid.x * v3Blk.x)
				for(int y = 0; y < i3VolumeSize.y; y++)
	#else	// MOD-BY-LEETEN 12/19/2009-TO:
	for(int			y = 0; y < i3VolumeSize.y; y += v3Grid.y * v3Blk.y)
		for(int		x = 0; x < i3VolumeSize.x; x += v3Grid.x * v3Blk.x)
			for(int z = 0; z < i3VolumeSize.z; z++)
	#endif	// MOD-BY-LEETEN 12/19/2009-END
			{
				int3 i3BlockCorner = make_int3(x, y, z);
				// MOD-BY-LEETEN 12/19/2009-FROM:
					// if( 0 == y )
				// TO:
				if( 0 == z )
				// MOD-BY-LEETEN 12/19/2009-END
				{
					_CreateHistogram_kernel<<<v3Grid, v3Blk, 0>>>
					(
						i3BlockCorner,
						i3KernelSize,
						i3VolumeSize,
						iNrOfBins,
						cActiveVoxelHistorgrams_pitched
					);
					CUT_CHECK_ERROR("_CreateHistogram_kernel() failed");
				}
				else
				{
					_UpdateHistogram_kernel<<<v3Grid, v3Blk, 0>>>
					(
						i3BlockCorner,
						i3KernelSize,
						i3VolumeSize,
						iNrOfBins,
						cActiveVoxelHistorgrams_pitched
					);
					CUT_CHECK_ERROR("_UpdateHistogram_kernel() failed");
				}

				_ComputeEntropy_kernel<<<v3Grid, v3Blk, 0>>>
				(
					i3BlockCorner,
					iNrOfBins,
					i3KernelSize,
					i3VolumeSize,
					cEntropyVolume_pitched
				);
				CUT_CHECK_ERROR("_ComputeEntropy_kernel() failed");
			}

CLOCK_END(_ComputeEntropyVolume_PRINT_TIMING, false);

CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_TIMING);

	FREE_MEMORY(cActiveVoxelHistorgrams_pitched.ptr);

CLOCK_END(_ComputeEntropyVolume_PRINT_TIMING, false);

CLOCK_PRINT(_ComputeEntropyVolume_PRINT_TIMING);
}
// ADD-BY-LEETEN 12/18/2009-END

void 
// MOD-BY-LEETEN 12/18/2009-FROM:
	// _ComputeEntropyVolume
// TO:
_ComputeEntropyVolume_host
// MOD-BY-LEETEN 12/18/2009-END
(
						// res. of the neighboring region
	int3 i3KernelSize,

						// the histogram
	int iNrOfBins,
	int *piHistogram_global,
	float *pfLogHistogram_global,

	// DEL-BY-LEETEN 12/18/2009-BEGIN
		// CUDPPHandle cScanPlanSum,
	// DEL-BY-LEETEN 12/18/2009-END

						// res. of the volume																
	int3 i3VolumeSize,	
						// bin volume																	
	hipPitchedPtr		cBinVolume_pitched,								
	hipPitchedPtr		cEntropyVolume_pitched
)
{
CLOCK_INIT(_ComputeEntropyVolume_PRINT_TIMING, __FUNCTION__ ": ");

CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_TIMING);

	int *piHistogram_host;
	piHistogram_host = (int*)calloc(sizeof(piHistogram_host[0]), iNrOfBins);
	assert(piHistogram_host);

	int *piBinVolume_host;
	piBinVolume_host = (int*)calloc(sizeof(piBinVolume_host[0]), i3VolumeSize.x * i3VolumeSize.y * i3VolumeSize.z);
	assert(piBinVolume_host);

	_GetSrcBinVolume(piBinVolume_host);

	// ADD-BY-LEETEN 12/19/2009-BEGIN
	float* pfEntropyVolume_host;
	pfEntropyVolume_host = (float*)calloc(sizeof(pfEntropyVolume_host[0]), i3VolumeSize.x * i3VolumeSize.y * i3VolumeSize.z);
	assert( pfEntropyVolume_host );
	// ADD-BY-LEETEN 12/19/2009-END

CLOCK_END(_ComputeEntropyVolume_PRINT_TIMING, false);

CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_TIMING);

	int xdir = +1;
	int ydir = +1;
	int zdir = +1;
	int x = 0;
	int y = 0;
	int z = 0;

	_BuildHistogram_host
	(
		make_int3(x, y, z),
							// res. of the neighboring region
		i3KernelSize,	
							// the bin volume
		i3VolumeSize,	
		piBinVolume_host,
							// the joint histogram
		iNrOfBins,		
		piHistogram_host
	);
CLOCK_END(_ComputeEntropyVolume_PRINT_TIMING, false);

CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_TIMING);

	for(int				zi = 0; zi < i3VolumeSize.z;	zi++, z += zdir)
	{
		for(int			yi = 0; yi < i3VolumeSize.y;	yi++, y += ydir)
		{
			for(int		xi = 0; xi < i3VolumeSize.x;	xi++, x += xdir)
			{
				// fprintf(stderr, "%d, %d, %d\n", x, y, z);
				// (x,y,z): center of the volume
				// before enter this loop, the histogram in the region centering at (x, y, z) should have been available
				_ComputeEntropy_host
				(
					make_int3(x, y, z),
					i3KernelSize,

										// the joint histogram
					iNrOfBins,
					piHistogram_host,

					i3VolumeSize,
					// MOD-BY-LEETEN 12/19/2009-FROM:
						// cEntropyVolume_pitched
					// TO:
					pfEntropyVolume_host
					// MOD-BY-LEETEN 12/19/2009-END
				);

				if( xi < i3VolumeSize.x - 1 )
					_UpdateSliceToHistogram_host(
						make_int3(x + xdir, y, z), 
						UPDATE_DIR_X,	
						xdir, 

						i3KernelSize, 

											// the bin volume
						i3VolumeSize,	
						piBinVolume_host,
											// the joint histogram
						iNrOfBins,		
						piHistogram_host
						);
			}
			x -= xdir;
			xdir *= -1;
			if( yi < i3VolumeSize.y - 1 )
				_UpdateSliceToHistogram_host(
					make_int3(x, y + ydir, z), 
					UPDATE_DIR_Y,	
					ydir, 
					
					i3KernelSize, 

										// the bin volume
					i3VolumeSize,	
					piBinVolume_host,
										// the joint histogram
					iNrOfBins,		
					piHistogram_host
					);
		}
		y -= ydir;
		ydir *= -1;
		if( zi < i3VolumeSize.z - 1 )
			_UpdateSliceToHistogram_host(
				make_int3(x, y, z + zdir), 
				UPDATE_DIR_Z,	
				zdir, 

				i3KernelSize, 

											// the bin volume
				i3VolumeSize,	
				piBinVolume_host,
									// the joint histogram
				iNrOfBins,		
				piHistogram_host
			);
	}
CLOCK_END(_ComputeEntropyVolume_PRINT_TIMING, false);

CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_TIMING);
	// ADD-BY-LEETEN 12/19/2009-BEGIN
	CUDA_SAFE_CALL_NO_SYNC(
		hipMemcpy2D(
			cEntropyVolume_pitched.ptr, 
			cEntropyVolume_pitched.pitch,
			pfEntropyVolume_host,
			i3VolumeSize.x * sizeof(pfEntropyVolume_host[0]),
			i3VolumeSize.x * sizeof(pfEntropyVolume_host[0]),
			i3VolumeSize.y * i3VolumeSize.z,
			hipMemcpyHostToDevice) );
	free(pfEntropyVolume_host);
	// ADD-BY-LEETEN 12/19/2009-END

	free(piHistogram_host);
	free(piBinVolume_host);
CLOCK_END(_ComputeEntropyVolume_PRINT_TIMING, false);

CLOCK_PRINT(_ComputeEntropyVolume_PRINT_TIMING);
}
// ADD-BY-LEETEN 12/18/2009-END

#if	0	// DEL-BY-LEETEN 12/18/2009-BEGIN
	CUDPPHandle 
	CInitEntropyField
	(
		int iNrOfBins
	)
	{
		CUDPPHandle cScanPlan = 0;

		CUDPPConfiguration	cConfig;
		cConfig.op =		CUDPP_ADD;
		cConfig.datatype =	CUDPP_FLOAT;
		cConfig.algorithm =	CUDPP_SCAN;
		cConfig.options =	CUDPP_OPTION_BACKWARD | CUDPP_OPTION_INCLUSIVE;

		assert( CUDPP_SUCCESS  == cudppPlan(&cScanPlan,	cConfig, iNrOfBins, 1, 0) );

		return cScanPlan;
	}
#endif	// DEL-BY-LEETEN 12/18/2009-END

void
_GetSrcEntropyVolume
(
	int iNrOfBins,
	int iKernelWidth, int iKernelHeight, int iKernelDepth
)
{
	int *piHistogram_global;
	CUDA_SAFE_CALL_NO_SYNC(
		hipMalloc(	(void**)&piHistogram_global, sizeof(piHistogram_global[0]) * iNrOfBins) );
	CUDA_SAFE_CALL_NO_SYNC(
		hipMemset(	(void*)piHistogram_global, 0, sizeof(piHistogram_global[0]) * iNrOfBins) );

	float *pfLogHistogram_global;
	CUDA_SAFE_CALL_NO_SYNC(
		hipMalloc(	(void**)&pfLogHistogram_global, sizeof(pfLogHistogram_global[0]) * iNrOfBins) );
	CUDA_SAFE_CALL_NO_SYNC(
		hipMemset(	(void*)pfLogHistogram_global, 0, sizeof(pfLogHistogram_global[0]) * iNrOfBins) );

	hipExtent cEntropyVolumeExtent	= make_hipExtent(
		cVolumeExtent_array.width * sizeof(float),
		cVolumeExtent_array.height,
		cVolumeExtent_array.depth);
	hipPitchedPtr cEntropyVolume_pitched;
	CUDA_SAFE_CALL( 
		hipMalloc3D(&cEntropyVolume_pitched, cEntropyVolumeExtent) );

	// DEL-BY-LEETEN 12/18/2009-BEGIN
		// CUDPPHandle hScanPlanEntropy = CInitEntropyField(iNrOfBins);
	// DEL-BY-LEETEN 12/18/2009-END

	#if	0	// MOD-BY-LEETEN 12/18/2009-FROM:
		_ComputeEntropyVolume
		(
							// res. of the neighboring region
			make_int3(iKernelWidth, iKernelHeight, iKernelDepth),

							// the histogram
			iNrOfBins,
			piHistogram_global,
			pfLogHistogram_global,

			hScanPlanEntropy,
							// res. of the volume																
			make_int3(cVolumeExtent_array.width, cVolumeExtent_array.height, cVolumeExtent_array.depth),

							// bin volume																	
			cSrcBinVolume_pitched,
			cEntropyVolume_pitched
		);
	#else	// MOD-BY-LEETEN 12/18/2009-TO:

	#if	COMPUTE_ENTROPY_VOLUME == COMPUTE_ENTROPY_VOLUME_CUDA
		_ComputeEntropyVolume_cuda
	#endif

	#if	COMPUTE_ENTROPY_VOLUME == COMPUTE_ENTROPY_VOLUME_HOST
		_ComputeEntropyVolume_host
	#endif

	#if	COMPUTE_ENTROPY_VOLUME == COMPUTE_ENTROPY_VOLUME_WITH_SORTING_CUDA
		_ComputeEntropyVolumeWithSorting_cuda
	#endif	
	(
						// res. of the neighboring region
		make_int3(iKernelWidth, iKernelHeight, iKernelDepth),

						// the histogram
		iNrOfBins,
		piHistogram_global,
		pfLogHistogram_global,

						// res. of the volume																
		make_int3(int(cVolumeExtent_array.width), int(cVolumeExtent_array.height), int(cVolumeExtent_array.depth)),

						// bin volume																	
		cSrcBinVolume_pitched,
		cEntropyVolume_pitched
	);
	#endif	// MOD-BY-LEETEN 12/18/2009-END

	// ADD-BY-LEETEN 12/19/2009-BEGIN
	float *pfEntropyVolume_host;
	CUDA_SAFE_CALL(
		hipHostMalloc(
			(void**)&pfEntropyVolume_host,
			sizeof(pfEntropyVolume_host[0]) * iNrOfVoxels) );
	CUDA_SAFE_CALL(
		hipMemcpy2D(
			pfEntropyVolume_host, 
			cVolumeExtent_array.width * sizeof(pfEntropyVolume_host[0]),
			cEntropyVolume_pitched.ptr,
			cEntropyVolume_pitched.pitch,
			cVolumeExtent_array.width * sizeof(pfEntropyVolume_host[0]),
			cVolumeExtent_array.height * cVolumeExtent_array.depth,
			hipMemcpyDeviceToHost) );

	#if	1	// TEST-DEBUG
	FILE *fpFile;
	fpFile = fopen( __FUNCTION__ "_" SRC_ENTROPY_VOLUME_POSTFIX ".txt", "wt");
	assert(fpFile);

	for(int	v = 0,		z = 0; z < int(cVolumeExtent_array.depth);	z++)
		for(int			y = 0; y < int(cVolumeExtent_array.height);	y++)
			for(int		x = 0; x < int(cVolumeExtent_array.width);	x++, v++)
				fprintf(fpFile, "E(%d, %d, %d) = %f\n", x, y, z, pfEntropyVolume_host[v]);

	fclose(fpFile);
	#endif
	FREE_MEMORY_ON_HOST(pfEntropyVolume_host);
	// ADD-BY-LEETEN 12/19/2009-END

	FREE_MEMORY(piHistogram_global);
	FREE_MEMORY(pfLogHistogram_global);
	FREE_MEMORY(cEntropyVolume_pitched.ptr);
	// DEL-BY-LEETEN 12/18/2009-BEGIN
		// cudppDestroyPlan(hScanPlanEntropy);
	// DEL-BY-LEETEN 12/18/2009-END
}

void
_GetJointEntropyVolume
(
	int iNrOfSrcBins,
	int iNrOfDstBins,
	int iKernelWidth, int iKernelHeight, int iKernelDepth
)
{
	int iNrOfBins = iNrOfSrcBins * iNrOfDstBins;

	int *piHistogram_global;
	CUDA_SAFE_CALL_NO_SYNC(
		hipMalloc(	(void**)&piHistogram_global, sizeof(piHistogram_global[0]) * iNrOfBins) );
	CUDA_SAFE_CALL_NO_SYNC(
		hipMemset(	(void*)piHistogram_global, 0, sizeof(piHistogram_global[0]) * iNrOfBins) );

	float *pfLogHistogram_global;
	CUDA_SAFE_CALL_NO_SYNC(
		hipMalloc(	(void**)&pfLogHistogram_global, sizeof(pfLogHistogram_global[0]) * iNrOfBins) );
	CUDA_SAFE_CALL_NO_SYNC(
		hipMemset(	(void*)pfLogHistogram_global, 0, sizeof(pfLogHistogram_global[0]) * iNrOfBins) );

	hipExtent cEntropyVolumeExtent	= make_hipExtent(
		cVolumeExtent_array.width * sizeof(float),
		cVolumeExtent_array.height,
		cVolumeExtent_array.depth);
	hipPitchedPtr cEntropyVolume_pitched;
	CUDA_SAFE_CALL( 
		hipMalloc3D(&cEntropyVolume_pitched, cEntropyVolumeExtent) );

	// DEL-BY-LEETEN 12/18/2009-BEGIN
		// CUDPPHandle hScanPlanEntropy = CInitEntropyField(iNrOfBins);
	// DEL-BY-LEETEN 12/18/2009-END

	// bind the bin volume as a 2D texture
	t2dSrcBinVolume.addressMode[0] = hipAddressModeClamp;
	t2dSrcBinVolume.addressMode[1] = hipAddressModeClamp;
	t2dSrcBinVolume.filterMode =	hipFilterModePoint;
	t2dSrcBinVolume.normalized =	false;
	CUDA_SAFE_CALL_NO_SYNC(
		hipBindTexture2D(
			0, 
			t2dSrcBinVolume, 
			cSrcBinVolume_pitched.ptr, 
			hipCreateChannelDesc<int>(),
			cVolumeExtent_array.width, 
			cVolumeExtent_array.height * cVolumeExtent_array.depth, 
			cSrcBinVolume_pitched.pitch) );

	t2dDstBinVolume.addressMode[0] = hipAddressModeClamp;
	t2dDstBinVolume.addressMode[1] = hipAddressModeClamp;
	t2dDstBinVolume.filterMode =	hipFilterModePoint;
	t2dDstBinVolume.normalized =	false;
	CUDA_SAFE_CALL_NO_SYNC(
		hipBindTexture2D(
			0, 
			t2dDstBinVolume, 
			cDstBinVolume_pitched.ptr, 
			hipCreateChannelDesc<int>(),
			cVolumeExtent_array.width, 
			cVolumeExtent_array.height * cVolumeExtent_array.depth, 
			cDstBinVolume_pitched.pitch) );

	// bind the histogram as a 1D texture
	// ...
	hipPitchedPtr cJointBinVolume_pitched;
	CUDA_SAFE_CALL( 
		hipMalloc3D(
			&cJointBinVolume_pitched, 
			make_hipExtent(
				cVolumeExtent_array.width * sizeof(int),
				cVolumeExtent_array.height,
				cVolumeExtent_array.depth) ) );

	dim3 v3Blk, v3Grid;
	v3Blk = dim3(BLOCK_DIM_X, BLOCK_DIM_Y);
	v3Grid = dim3(
		size_t(ceilf(float(cVolumeExtent_array.width) / float(v3Blk.x))),
		size_t(ceilf(float(cVolumeExtent_array.height * cVolumeExtent_array.depth) / float(v3Blk.y)))
	);

	_JointSrcDst_kernel<<<v3Grid, v3Blk, 0>>>
	(
		iNrOfSrcBins, 
		iNrOfDstBins,

		make_int3(
			int(cVolumeExtent_array.width), 
			int(cVolumeExtent_array.height), 
			int(cVolumeExtent_array.depth)),

		cJointBinVolume_pitched
	);
	CUT_CHECK_ERROR("_JointSrcDst_kernel() failed");

// MOD-BY-LEETEN 12/18/2009-FROM:
	// _ComputeEntropyVolume
// TO:

	#if	COMPUTE_ENTROPY_VOLUME == COMPUTE_ENTROPY_VOLUME_CUDA
		_ComputeEntropyVolume_cuda
	#endif

	#if	COMPUTE_ENTROPY_VOLUME == COMPUTE_ENTROPY_VOLUME_HOST
		_ComputeEntropyVolume_host
	#endif

	#if	COMPUTE_ENTROPY_VOLUME == COMPUTE_ENTROPY_VOLUME_WITH_SORTING_CUDA
		_ComputeEntropyVolumeWithSorting_cuda
	#endif

// MOD-BY-LEETEN 12/18/2009-END
	(
						// res. of the neighboring region
		make_int3(iKernelWidth, iKernelHeight, iKernelDepth),

						// the histogram
		iNrOfBins,
		piHistogram_global,
		pfLogHistogram_global,

		// DEL-BY-LEETEN 12/18/2009-BEGIN
			// hScanPlanEntropy,
		// DEL-BY-LEETEN 12/18/2009-END

						// res. of the volume																
		make_int3(int(cVolumeExtent_array.width), int(cVolumeExtent_array.height), int(cVolumeExtent_array.depth)),

						// bin volume																	
		cJointBinVolume_pitched,
		cEntropyVolume_pitched
	);

	// ADD-BY-LEETEN 12/19/2009-BEGIN
	float *pfEntropyVolume_host;
	CUDA_SAFE_CALL(
		hipHostMalloc(
			(void**)&pfEntropyVolume_host,
			sizeof(pfEntropyVolume_host[0]) * iNrOfVoxels) );
	CUDA_SAFE_CALL_NO_SYNC(
		hipMemcpy2D(
			pfEntropyVolume_host, 
			cVolumeExtent_array.width * sizeof(pfEntropyVolume_host[0]),
			cEntropyVolume_pitched.ptr,
			cEntropyVolume_pitched.pitch,
			cVolumeExtent_array.width * sizeof(pfEntropyVolume_host[0]),
			cVolumeExtent_array.height * cVolumeExtent_array.depth,
			hipMemcpyDeviceToHost) );
	 /*
	for(int	v = 0,		z = 0; z < i3VolumeSize.z; z++)
		for(int			y = 0; y < i3VolumeSize.y; y++)
			for(int		x = 0; x < i3VolumeSize.x; x++, v++)
				fprintf(stderr, "E(%d, %d, %d) = %f\n", x, y, z, pfEntropyVolume_host[v]);
	*/
	FREE_MEMORY_ON_HOST(pfEntropyVolume_host);
	// ADD-BY-LEETEN 12/19/2009-END

	FREE_MEMORY(cJointBinVolume_pitched.ptr);
	FREE_MEMORY(piHistogram_global);
	FREE_MEMORY(pfLogHistogram_global);
	FREE_MEMORY(cEntropyVolume_pitched.ptr);
	// DEL-BY-LEETEN 12/18/2009-BEGIN
		// cudppDestroyPlan(hScanPlanEntropy);
	// DEL-BY-LEETEN 12/18/2009-END
}

// ADD-BY-LEETEN 12/17/2009-END

////////////////////////////////////////////
#if	0	// MOD-BY-LEETEN 12/16/2009-FROM:
#else	// MOD-BY-LEETEN 12/16/2009-TO:
	#include "FlowDiffusion3D_kernel.cu"
#endif	// MOD-BY-LEETEN 12/16/2009-END

extern "C" {

// MOD-BY-LEETEN 12/07/2009-FROM:
	// void _FlowFusionFree()
// TO:
void 
_FlowDiffusionFree()
// MOD-BY-LEETEN 12/07/2009-END
{
	FREE_MEMORY_ON_HOST(pf4Volume_host);
	for(int i = 0; i < 2; i++)
		FREE_MEMORY(pcVolumePtrs_global[i].ptr);
	FREE_ARRAY(cWeightOffsetVolume_array);
	FREE_MEMORY_ON_HOST(pf4WeightOffsetVolume_host);
	// ADD-BY-LEETEN 2009/11/25-BEGIN
	FREE_MEMORY(cErrorVolume_device.ptr);
	FREE_MEMORY(cErrorSum_device.ptr);

	#if	CHECK_ERROR_CONVERGENCE_BY_CUDPP
	cudppDestroyPlan(cScanplan);
	#endif	// #if	CHECK_ERROR_CONVERGENCE_BY_CUDPP
	// ADD-BY-LEETEN 2009/11/25-END

	// ADD-BY-LEETEN 12/14/2009-BEGIN
	FREE_ARRAY(pcAngleMap_array);	

	FREE_MEMORY(cSrcBinVolume_pitched.ptr);
	FREE_MEMORY(cDstBinVolume_pitched.ptr);
	// ADD-BY-LEETEN 12/17/2009-BEGIN
	FREE_MEMORY(cSrcDstBinVolumePtr_global.ptr);
	// ADD-BY-LEETEN 12/17/2009-END

	#if	0	// MOD-BY-LEETEN 12/16/2009-FROM:
		FREE_MEMORY_ON_HOST(piSrcAngleBinVolume_host);
		FREE_MEMORY_ON_HOST(piDstAngleBinVolume_host);
	#else	// MOD-BY-LEETEN 12/16/2009-TO:
	FREE_MEMORY_ON_HOST(piBinVolume_host);
	#endif	// MOD-BY-LEETEN 12/16/2009-END
	// ADD-BY-LEETEN 12/14/2009-END
}

void
// MOD-BY-LEETEN 12/07/2009-FROM:
	// _FlowFusionInit(
// TO:
_FlowDiffusionInit(
// MOD-BY-LEETEN 12/07/2009-END
	int iVolumeWidth,
	int iVolumeHeight,
	int iVolumeDepth
)
{
	// MOD-BY-LEETEN 12/07/2009-FROM:
		// CLOCK_INIT(PRINT_FLOW_FUSION_TIMING, "_FlowFusionInit(): ");
	// TO:
	CLOCK_INIT(PRINT_FLOW_FUSION_TIMING, __FUNCTION__ ": ");
	// MOD-BY-LEETEN 12/07/2009-END

	// MOD-BY-LEETEN 12/14/2009-FROM:
		// int iNrOfVoxels = iVolumeWidth * iVolumeHeight * iVolumeDepth;
	// TO:
	iNrOfVoxels = iVolumeWidth * iVolumeHeight * iVolumeDepth;
	// MOD-BY-LEETEN 12/14/2009-END

	cVolumeExtent	= make_hipExtent(
		iVolumeWidth * sizeof(float4),
		iVolumeHeight,
		iVolumeDepth);
	// allocate a local copy in the CPU side
	CLOCK_BEGIN(PRINT_FLOW_FUSION_TIMING);
	CUDA_SAFE_CALL(
		hipHostMalloc(
			(void**)&pf4Volume_host,
			sizeof(pf4Volume_host[0]) * iNrOfVoxels) );
	CLOCK_END(PRINT_FLOW_FUSION_TIMING, true);

	// Allocate 2 volumes on the  GPU's global memory.
	// in each iteration, one of them will be used as the source
	// 3D texture, and the other will be served as the dst.
	CLOCK_BEGIN(PRINT_FLOW_FUSION_TIMING);

	for(int i = 0; i < 2; i++)
	{
		CUDA_SAFE_CALL( 
			hipMalloc3D(&pcVolumePtrs_global[i], cVolumeExtent) );
	}

	// ADD-BY-LEETEN 2009/11/25-BEGIN
	// ADD-BY-LEETEN 2009/12/17-BEGIN
	#if	CHECK_ERROR_CONVERGENCE_BY_CUDPP	
	// ADD-BY-LEETEN 2009/12/17-END
	cErrorVolumeExtent	= make_hipExtent(
		// MOD-BY-LEETEN 12/16/2009-FROM:
			// iVolumeWidth,
		// TO:
		iVolumeWidth * sizeof(float),
		// MOD-BY-LEETEN 12/16/2009-END
		iVolumeHeight,
		iVolumeDepth);
	CUDA_SAFE_CALL( 
		hipMalloc3D(&cErrorVolume_device,	cErrorVolumeExtent) );
	CUDA_SAFE_CALL( 
		hipMalloc3D(&cErrorSum_device,		cErrorVolumeExtent) );

	// DEL-BY-LEETEN 12/16/2009-BEGIN
	// #if	CHECK_ERROR_CONVERGENCE_BY_CUDPP
	// DEL-BY-LEETEN 12/16/2009-END
	cConfig.op = CUDPP_ADD;
	// cConfig.op = CUDPP_MAX;
	cConfig.datatype = CUDPP_FLOAT;
	cConfig.algorithm = CUDPP_SCAN;
	#if	0	// MOD-BY-LEETEN 12/16/2009-FROM:
		cConfig.options = CUDPP_OPTION_FORWARD | CUDPP_OPTION_INCLUSIVE;
		cudppPlan(&cScanplan, cConfig, iVolumeWidth * iVolumeHeight * iVolumeDepth, iVolumeHeight * iVolumeDepth, cErrorVolume_device.pitch);  
	#else	// MOD-BY-LEETEN 12/16/2009-TO:
	cConfig.options = CUDPP_OPTION_BACKWARD | CUDPP_OPTION_INCLUSIVE;
	assert(CUDPP_SUCCESS  == cudppPlan(&cScanplan, cConfig, iVolumeWidth * iVolumeHeight * iVolumeDepth, iVolumeHeight * iVolumeDepth, cErrorVolume_device.pitch) );  
	#endif	// MOD-BY-LEETEN 12/16/2009-END
	// DEL-BY-LEETEN 2009/12/17-BEGIN
	// #endif	// #if	CHECK_ERROR_CONVERGENCE_BY_CUDPP
	// DEL-BY-LEETEN 2009/12/17-END
	// ADD-BY-LEETEN 2009/12/17-BEGIN
	#endif	// #if	CHECK_ERROR_CONVERGENCE_BY_CUDPP
	// ADD-BY-LEETEN 2009/12/17-END
	// ADD-BY-LEETEN 2009/11/25-END

	// ADD-BY-LEETEN 10/02/2009-BEGIN
	cVolumeExtent_array = make_hipExtent(
		iVolumeWidth,
		iVolumeHeight,
		iVolumeDepth);

	CUDA_SAFE_CALL( 
		hipMalloc3DArray(&cWeightOffsetVolume_array, &cWeightOffsetChannelDesc, cVolumeExtent_array) );

	CUDA_SAFE_CALL(
		hipHostMalloc(
			(void**)&pf4WeightOffsetVolume_host,
			sizeof(pf4WeightOffsetVolume_host[0]) * iNrOfVoxels) );
	// ADD-BY-LEETEN 10/02/2009-END

	CLOCK_END(PRINT_FLOW_FUSION_TIMING, true);

	CLOCK_PRINT(PRINT_FLOW_FUSION_TIMING);
}

// ADD-BY-LEETEN 12/14/2009-BEGIN
void 
_FlowDiffusionSetAngleMap(int *piAngleMap, int iNrOfPhis, int iNrOfThetas)
{
	// allocate an array on the GPU side

	// upload the angle map to the araray

	// setup the texture for the angle map
	/*
    t2dAngleMap.addressMode[0] =	hipAddressModeClamp;
    t2dAngleMap.addressMode[1] =	hipAddressModeClamp;
    t2dAngleMap.filterMode =		hipFilterModePoint;
    t2dAngleMap.normalized =		true;
	*/
	SETUP_ARRAY(pcAngleMap_array, iNrOfPhis, iNrOfThetas, 32, 0, 0, 0, hipChannelFormatKindSigned);	
	BIND_ARRAY_AS_TEXTURE(
		t2dAngleMap, pcAngleMap_array, 
		hipAddressModeClamp, hipAddressModeClamp, hipFilterModePoint, true);	

    CUDA_SAFE_CALL( 
		hipMemcpy2DToArray(
			pcAngleMap_array, 
			0, 
			0, 
			(void*)piAngleMap, 
			iNrOfPhis * sizeof(piAngleMap), 
			iNrOfPhis * sizeof(piAngleMap), 
			iNrOfThetas, 
			hipMemcpyHostToDevice) );

	// check whether the volume extent has been setup
	assert( NULL != pcVolumePtrs_global[0].ptr );

	// allocate the volume of bins in the global memorty on GPUs
	cBinVolumeExtent	= make_hipExtent(
		cVolumeExtent_array.width * sizeof(int),
		cVolumeExtent_array.height,
		cVolumeExtent_array.depth);
	CUDA_SAFE_CALL( 
		hipMalloc3D(&cSrcBinVolume_pitched, cBinVolumeExtent) );
	CUDA_SAFE_CALL( 
		hipMalloc3D(&cDstBinVolume_pitched, cBinVolumeExtent) );

	// ADD-BY-LEETEN 12/17/2009-BEGIN
	CUDA_SAFE_CALL( 
		hipMalloc3D(&cSrcDstBinVolumePtr_global, cBinVolumeExtent) );

	t2dSrcBinVolume.addressMode[0] = hipAddressModeClamp;
	t2dSrcBinVolume.addressMode[1] = hipAddressModeClamp;
	t2dSrcBinVolume.filterMode =	hipFilterModePoint;
	t2dSrcBinVolume.normalized =	false;

	t2dDstBinVolume.addressMode[0] = hipAddressModeClamp;
	t2dDstBinVolume.addressMode[1] = hipAddressModeClamp;
	t2dDstBinVolume.filterMode =	hipFilterModePoint;
	t2dDstBinVolume.normalized =	false;
	// ADD-BY-LEETEN 12/17/2009-END


	// allocate the volume of bins in the host side
	#if	0	// MOD-BY-LEETEN 2009/12/17-FROM:
	#else	// MOD-BY-LEETEN 2009/12/17-TO:
	CUDA_SAFE_CALL(
		hipHostMalloc(
			(void**)&piBinVolume_host,
			sizeof(piBinVolume_host[0]) * iNrOfVoxels) );
	#endif	// MOD-BY-LEETEN 2009/12/17-END

	// setup the src. texture for bin lookup
	t2dVectorVolume.addressMode[0] = hipAddressModeClamp;
	t2dVectorVolume.addressMode[1] = hipAddressModeClamp;
	t2dVectorVolume.filterMode =	hipFilterModePoint;
	t2dVectorVolume.normalized =	false;
}

void
_Vector3DToVolume(
	int iVolumeWidth,
	int iVolumeHeight,
	int iVolumeDepth,
	int iBlockZSize,
	hipPitchedPtr cVolumePtr_global,	// input
	hipPitchedPtr cBinVolumePtr_global	// output
)
{
	CUDA_SAFE_CALL_NO_SYNC(
		hipBindTexture2D(
			0, 
			t2dVectorVolume, 
			cVolumePtr_global.ptr, 
			hipCreateChannelDesc<float4>(),
			iVolumeWidth, 
			iVolumeHeight * iVolumeDepth, 
			cVolumePtr_global.pitch) );

	dim3 v3Blk = dim3(BLOCK_DIM_X, BLOCK_DIM_Y);
	dim3 v3Grid = dim3(
		(unsigned int)ceilf((float)iVolumeWidth	 / (float)v3Blk.x),
		unsigned int(ceilf( float(iVolumeHeight) / float(v3Blk.y) ) ) *
		unsigned int(ceilf( float(iVolumeDepth) /  float(iBlockZSize) ) ) );
	// MOD-BY-LEETEN 12/18/2009-FROM:
		// _Vector3DToVolume_kernel<<<v3Grid, v3Blk, 0>>>
	// TO:
	_Vector3DToBinVolume_kernel<<<v3Grid, v3Blk, 0>>>
	// MOD-BY-LEETEN 12/18/2009-END
	(
		iVolumeWidth,
		iVolumeHeight,
		iVolumeDepth,
		int(ceilf(float(iVolumeHeight) / float(BLOCK_DIM_Y))),
		iBlockZSize,
		cBinVolumePtr_global
	);
	CUT_CHECK_ERROR("_Vector3DToVolume_kernel() failed");
}

void
_ComputeSrcBinVolume
(
	int iVolumeWidth,
	int iVolumeHeight,
	int iVolumeDepth,
	int iBlockZSize,
	float *pfSrcVolume
)
{
	CLOCK_INIT(SHOW_COMPUTE_SRC_BIN_VOLUME_TIMING, __FUNCTION__ ": ");

	CLOCK_BEGIN(SHOW_COMPUTE_SRC_BIN_VOLUME_TIMING);
	for(int v = 0,		d = 0; d < iVolumeDepth;	d++)
		for(int			h = 0; h < iVolumeHeight;	h++)
			for(int		w = 0; w < iVolumeWidth;	w++, v++)
			{
				pf4Volume_host[v].x = pfSrcVolume[v * 3 + 0];
				pf4Volume_host[v].y = pfSrcVolume[v * 3 + 1];
				pf4Volume_host[v].z = pfSrcVolume[v * 3 + 2];
				pf4Volume_host[v].w = 0.0f;
			}
	CLOCK_END(SHOW_COMPUTE_SRC_BIN_VOLUME_TIMING, false);

	CLOCK_BEGIN(SHOW_COMPUTE_SRC_BIN_VOLUME_TIMING);
    hipMemcpy3DParms cCopyParamsHostToDevice = {0};

    cCopyParamsHostToDevice.srcPtr   = make_hipPitchedPtr(
		(void*)pf4Volume_host, 
		cVolumeExtent.width, 
		iVolumeWidth * sizeof(float4), 
		iVolumeHeight);
    cCopyParamsHostToDevice.dstPtr		= pcVolumePtrs_global[0];
    cCopyParamsHostToDevice.extent		= cVolumeExtent;
    cCopyParamsHostToDevice.kind		= hipMemcpyHostToDevice;
    CUDA_SAFE_CALL( hipMemcpy3D(&cCopyParamsHostToDevice) );  

	CLOCK_END(SHOW_COMPUTE_SRC_BIN_VOLUME_TIMING, false);

	CLOCK_BEGIN(SHOW_COMPUTE_SRC_BIN_VOLUME_TIMING);
	_Vector3DToVolume(
		iVolumeWidth,
		iVolumeHeight,
		iVolumeDepth,
		iBlockZSize,
		pcVolumePtrs_global[0],
		cSrcBinVolume_pitched);
	CLOCK_END(SHOW_COMPUTE_SRC_BIN_VOLUME_TIMING, false);

	CLOCK_PRINT(SHOW_COMPUTE_SRC_BIN_VOLUME_TIMING);
}

void 
_GetSrcBinVolume(int *piBinVolume)
{
	hipMemcpy3DParms cCopyParamsDeviceToHost = {0};
	cCopyParamsDeviceToHost.srcPtr		= cSrcBinVolume_pitched;
	cCopyParamsDeviceToHost.dstPtr		= make_hipPitchedPtr(
		// MOD-BY-LEETEN 2009/12/17-FROM:
			// (void*)piSrcAngleBinVolume_host, 
		// TO:
		(void*)piBinVolume_host, 
		// MOD-BY-LEETEN 2009/12/17-END
		cVolumeExtent_array.width * sizeof(int),	// cSrcBinVolume_pitched.width, 
		cVolumeExtent_array.width * sizeof(int), 
		cVolumeExtent_array.height);
	cCopyParamsDeviceToHost.extent		= cBinVolumeExtent;
	cCopyParamsDeviceToHost.kind		= hipMemcpyDeviceToHost;

	CUDA_SAFE_CALL( hipMemcpy3D(&cCopyParamsDeviceToHost) );  

	// MOD-BY-LEETEN 2009/12/17-FROM:
		// memcpy(piBinVolume, piSrcAngleBinVolume_host, sizeof(piBinVolume) * iNrOfVoxels);
	// TO:
	memcpy(piBinVolume, piBinVolume_host, sizeof(piBinVolume) * iNrOfVoxels);
	// MOD-BY-LEETEN 2009/12/17-END
}

void 
_GetDstBinVolume(int *piBinVolume)
{
	hipMemcpy3DParms cCopyParamsDeviceToHost = {0};
	cCopyParamsDeviceToHost.srcPtr		= cDstBinVolume_pitched;
	cCopyParamsDeviceToHost.dstPtr		= make_hipPitchedPtr(
		// MOD-BY-LEETEN 2009/12/17-FROM:
			// (void*)piDstAngleBinVolume_host,		
		// TO:
		(void*)piBinVolume_host, 
		// MOD-BY-LEETEN 2009/12/17-END

		cVolumeExtent_array.width * sizeof(int),	// cDstBinVolume_pitched.width, 
		cVolumeExtent_array.width * sizeof(int), 
		cVolumeExtent_array.height);
	cCopyParamsDeviceToHost.extent		= cBinVolumeExtent;
	cCopyParamsDeviceToHost.kind		= hipMemcpyDeviceToHost;

	CUDA_SAFE_CALL( hipMemcpy3D(&cCopyParamsDeviceToHost) );  

	// MOD-BY-LEETEN 2009/12/17-FROM:
		// memcpy(piBinVolume, piDstAngleBinVolume_host, sizeof(piBinVolume) * iNrOfVoxels);
	// TO:
	memcpy(piBinVolume, piBinVolume_host, sizeof(piBinVolume) * iNrOfVoxels);
	// MOD-BY-LEETEN 2009/12/17-END
}

// ADD-BY-LEETEN 12/14/2009-END

void
// MOD-BY-LEETEN 12/07/2009-FROM:
	// _FlowFusion(
// TO:
_FlowDiffusion(
// MOD-BY-LEETEN 12/07/2009-END
	float fAttenuation,
	int iNrOfIterations,
	int iVolumeWidth,
	int iVolumeHeight,
	int iVolumeDepth,
	// ADD-BY-LEETEN 10/02/2009-BEGIN
	int iBlockZSize,
	float *pfWeightVolume,
	// MOD-BY-LEETEN 2009/11/10-FROM:
		// float *pfOffsetVolume,
	// TO:
	float *pfXOffsetVolume,
	float *pfYOffsetVolume,
	float *pfZOffsetVolume,
	// MOD-BY-LEETEN 2009/11/10-END
	// ADD-BY-LEETEN 10/02/2009-END
	float *pfSrcVolume,
	float *pfDstVolume,
	// ADD-BY-LEETEN 2009/11/10-BEGIN
	int *piFlagVolume
	// ADD-BY-LEETEN 2009/11/10-END
)
{
	// ADD-BY-LEETEN 12/07/2009-BEGIN
	#if	CHECK_ERROR_CONVERGENCE_BY_CUDPP
	iNrOfIterations = 4 * max(max(iVolumeWidth, iVolumeHeight), iVolumeDepth);
	// LOG(printf("Warning! iNrOfIterations is changed to %d", iNrOfIterations));
	#endif
	// ADD-BY-LEETEN 12/07/2009-END
	

	// MOD-BY-LEETEN 12/07/2009-FROM:
		// CLOCK_INIT(PRINT_FLOW_FUSION_TIMING, "_FlowFusion(): ");
	// TO:
	// MOD-BY-LEETEN 12/14/2009-FROM:
		// CLOCK_INIT(PRINT_FLOW_FUSION_TIMING, __FUNCTION__);
	// TO:
	CLOCK_INIT(PRINT_FLOW_FUSION_TIMING, __FUNCTION__ ": ");
	// MOD-BY-LEETEN 12/07/2009-END

	// MOD-BY-LEETEN 10/02/2009-FROM:
		// dim3 v3Blk = dim3(16, 8);
	// TO:
	dim3 v3Blk = dim3(BLOCK_DIM_X, BLOCK_DIM_Y);
	// MOD-BY-LEETEN 10/02/2009-END
	dim3 v3Grid = dim3(
		(unsigned int)ceilf((float)iVolumeWidth	 / (float)v3Blk.x),
		// ADD-BY-LEETEN 11/04/2009-BEGIN
		#if					DIFFUSION_BY_FOR_LOOP_ON_HOST	
		// ADD-BY-LEETEN 11/04/2009-END

		(unsigned int)ceilf((float)iVolumeHeight / (float)v3Blk.y));

		// ADD-BY-LEETEN 11/04/2009-BEGIN
		#else	// #if		DIFFUSION_BY_FOR_LOOP_ON_HOST
		unsigned int(ceilf( float(iVolumeHeight) / float(v3Blk.y) ) ) *
		unsigned int(ceilf( float(iVolumeDepth) /  float(iBlockZSize) ) ) );
		#endif	// #if		DIFFUSION_BY_FOR_LOOP_ON_HOST
		// ADD-BY-LEETEN 11/04/2009-END

	// convert the #channels in the src volume from 3 to 4
	CLOCK_BEGIN(PRINT_FLOW_FUSION_TIMING);
	for(int v = 0,		d = 0; d < iVolumeDepth;	d++)
		for(int			h = 0; h < iVolumeHeight;	h++)
			for(int		w = 0; w < iVolumeWidth;	w++, v++)
			{
				pf4Volume_host[v].x = pfSrcVolume[v * 3 + 0];
				pf4Volume_host[v].y = pfSrcVolume[v * 3 + 1];
				pf4Volume_host[v].z = pfSrcVolume[v * 3 + 2];

				// ADD-BY-LEETEN 2009/11/10-BEGIN
				if( piFlagVolume )
					pf4Volume_host[v].w = float(piFlagVolume[v]);
				else
				// ADD-BY-LEETEN 2009/11/10-END
				pf4Volume_host[v].w = 0.0f;
			}

	// ADD-BY-LEETEN 10/02/2009-BEGIN
	for(int v = 0,		d = 0; d < iVolumeDepth;	d++)
		for(int			h = 0; h < iVolumeHeight;	h++)
			for(int		w = 0; w < iVolumeWidth;	w++, v++)
			{
				#if	0	// MOD-BY-LEETEN 2009/11/10-FROM:
				#else	// MOD-BY-LEETEN 2009/11/10-TO:
				// ADD-BY-LEETEN 12/07/2009-BEGIN
				if( pfXOffsetVolume )
				// ADD-BY-LEETEN 12/07/2009-END
				pf4WeightOffsetVolume_host[v].x = pfXOffsetVolume[v];
				// ADD-BY-LEETEN 12/07/2009-BEGIN
				if( pfYOffsetVolume )
				// ADD-BY-LEETEN 12/07/2009-END
				pf4WeightOffsetVolume_host[v].y = pfYOffsetVolume[v];
				// ADD-BY-LEETEN 12/07/2009-BEGIN
				if( pfZOffsetVolume )
				// ADD-BY-LEETEN 12/07/2009-END
				pf4WeightOffsetVolume_host[v].z = pfZOffsetVolume[v];
				pf4WeightOffsetVolume_host[v].w = 1.0f - pfWeightVolume[v];
				#endif	// MOD-BY-LEETEN 2009/11/10-END
			}
	// ADD-BY-LEETEN 10/02/2009-END

	CLOCK_END(PRINT_FLOW_FUSION_TIMING, true);

	// Upload the input volume to the first volume on the GPU
	CLOCK_BEGIN(PRINT_FLOW_FUSION_TIMING);

    hipMemcpy3DParms cCopyParamsHostToDevice = {0};

    cCopyParamsHostToDevice.srcPtr   = make_hipPitchedPtr(
		(void*)pf4Volume_host, 
		cVolumeExtent.width, 
		iVolumeWidth * sizeof(float4), 
		iVolumeHeight);
    cCopyParamsHostToDevice.dstPtr		= pcVolumePtrs_global[0];
    cCopyParamsHostToDevice.extent		= cVolumeExtent;
    cCopyParamsHostToDevice.kind		= hipMemcpyHostToDevice;
    CUDA_SAFE_CALL( hipMemcpy3D(&cCopyParamsHostToDevice) );  

	// ADD-BY-LEETEN 10/02/2009-BEGIN
    hipMemcpy3DParms cCopyParamsHostToArray = {0};
    cCopyParamsHostToArray.srcPtr   = make_hipPitchedPtr(
		(void*)pf4WeightOffsetVolume_host, 
		iVolumeWidth * sizeof(float4), 
		iVolumeWidth, 
		iVolumeHeight);
    cCopyParamsHostToArray.dstArray = cWeightOffsetVolume_array;
    cCopyParamsHostToArray.extent   = cVolumeExtent_array;
    cCopyParamsHostToArray.kind     = hipMemcpyHostToDevice;
    CUDA_SAFE_CALL( 
		hipMemcpy3D(&cCopyParamsHostToArray) );  

    t3dWeightOffset.addressMode[0] = hipAddressModeClamp;
    t3dWeightOffset.addressMode[1] = hipAddressModeClamp;
    t3dWeightOffset.addressMode[2] = hipAddressModeClamp;
    t3dWeightOffset.filterMode		= hipFilterModePoint;
    t3dWeightOffset.normalized		= false; 

    // bind array to 3D texture
    CUDA_SAFE_CALL(
		hipBindTextureToArray(t3dWeightOffset, cWeightOffsetVolume_array, cWeightOffsetChannelDesc));
	// ADD-BY-LEETEN 10/02/2009-END

	CLOCK_END(PRINT_FLOW_FUSION_TIMING, true);

	// calcuate the fusion operator on GPUs
	CLOCK_BEGIN(PRINT_FLOW_FUSION_TIMING);

    t2dSrc.addressMode[0] = hipAddressModeClamp;
    t2dSrc.addressMode[1] = hipAddressModeClamp;
    t2dSrc.filterMode =		hipFilterModePoint;
    t2dSrc.normalized =		false;

	int iSrc = 0;

	for(int i = 0; 
			i < iNrOfIterations; 
			i++,				iSrc = 1 - iSrc)
	{
		// bind the src. volume as the 3D texture

		hipChannelFormatDesc cChannelDesc =
			hipCreateChannelDesc<float4>();

		CUDA_SAFE_CALL_NO_SYNC(
			hipBindTexture2D(
				0, 
				t2dSrc, 
				pcVolumePtrs_global[iSrc].ptr, 
				cChannelDesc,
				iVolumeWidth, 
				iVolumeHeight * iVolumeDepth, 
				pcVolumePtrs_global[iSrc].pitch) );


		// ADD-BY-LEETEN 12/07/2009-BEGIN
		if( iVolumeDepth <= 2 )
			_FlowDiffusion2D_kernel<<<v3Grid, v3Blk, 0>>>
			(
				fAttenuation,
				iVolumeWidth,
				iVolumeHeight,
				pcVolumePtrs_global[1 - iSrc],
				cErrorVolume_device
			);	
		else
		{
		// ADD-BY-LEETEN 12/07/2009-END

		// launch the kernel to compute the diffusion operator for 1 iteration
		// MOD-BY-LEETEN 11/04/2009-FROM:
			// #if	0	// MOD-BY-LEETEN 10/02/2009-FROM:
		// TO:
		#if	DIFFUSION_BY_FOR_LOOP_ON_HOST
		// MOD-BY-LEETEN 11/04/2009-END

			for(int z = 0; z < iVolumeDepth; z++)
				// MOD-BY-LEETEN 12/07/2009-FROM:
					// _FlowFusion_kernel<<<v3Grid, v3Blk, 0>>>
				// TO:
				_FlowDiffusion3D_kernel<<<v3Grid, v3Blk, 0>>>
				// MOD-BY-LEETEN 12/07/2009-END
				(
					fAttenuation,
					iVolumeWidth,
					iVolumeHeight,
					iVolumeDepth,
					z,

					// output
					pcVolumePtrs_global[1 - iSrc],
					// ADD-BY-LEETEN 2009/11/25-BEGIN
					cErrorVolume_device
					// ADD-BY-LEETEN 2009/11/25-END
				);	
		// MOD-BY-LEETEN 11/04/2009-FROM:
			// #else	// MOD-BY-LEETEN 10/02/2009-TO:
		// TO:
		#else	// #if	DIFFUSION_BY_FOR_LOOP_ON_HOST
		// MOD-BY-LEETEN 11/04/2009-END

		// _FlowFusion_kernel<<<v3Grid, v3Blk, v3Blk.x * v3Blk.y * sizeof(float4)>>>
		// MOD-BY-LEETEN 12/07/2009-FROM:
			// _FlowFusion_kernel<<<v3Grid, v3Blk, 0>>>
		// TO:
		_FlowDiffusion3D_kernel<<<v3Grid, v3Blk, 0>>>
		// MOD-BY-LEETEN 12/07/2009-END
		(
			fAttenuation,
			iVolumeWidth,
			iVolumeHeight,
			iVolumeDepth,
			0,
			// ADD-BY-LEETEN 11/04/2009-BEGIN
			int(ceilf(float(iVolumeHeight) / float(BLOCK_DIM_Y))),
			iBlockZSize,
			// ADD-BY-LEETEN 11/04/2009-END
			// output
			pcVolumePtrs_global[1 - iSrc],
			// ADD-BY-LEETEN 2009/11/25-BEGIN
			cErrorVolume_device
			// ADD-BY-LEETEN 2009/11/25-END
		);
		#endif	// MOD-BY-LEETEN 10/02/2009-END
		// ADD-BY-LEETEN 12/07/2009-BEGIN
		}
		// ADD-BY-LEETEN 12/07/2009-END
		CUT_CHECK_ERROR("_FlowFusion_kernel() failed");

		// ADD-BY-LEETEN 2009/11/25-BEGIN
		#if	CHECK_ERROR_CONVERGENCE_BY_CUDPP
		cudppScan(cScanplan, cErrorSum_device.ptr, cErrorVolume_device.ptr, iVolumeWidth * iVolumeHeight * iVolumeDepth);

		float fError;
		#if	0	// MOD-BY-LEETEN 2009/12/17-FROM:
			CUDA_SAFE_CALL( 
				hipMemcpy(
					&fError, 
					ADDRESS_2D(
						float, cErrorSum_device.ptr, 
						sizeof(float), cErrorSum_device.pitch, 
						iVolumeWidth - 1, iVolumeHeight - 1 + (iVolumeDepth - 1 ) * iVolumeHeight),
					sizeof(float), 
					hipMemcpyDeviceToHost));
		#else	// MOD-BY-LEETEN 2009/12/17-TO:
	    CUDA_SAFE_CALL_NO_SYNC( 
			hipMemcpy(
				&fError, 
				cErrorSum_device.ptr, 
				sizeof(float), 
				hipMemcpyDeviceToHost));
		#endif	// MOD-BY-LEETEN 2009/12/17-END

		/*
		fError /= float(iVolumeWidth * iVolumeHeight * iVolumeDepth * 3);
		fError = sqrtf(fError);
		*/
		#if	0	// MOD-BY-LEETEN 2009/12/17-FROM:
			fprintf(stderr, "Error: %f\n", fError);
			if( fError < 1e-10 )
				break;
		#else	// MOD-BY-LEETEN 2009/12/17-TO:
		fError /= float(iVolumeWidth * iVolumeHeight * iVolumeDepth * 3);
		fError = sqrtf(fError);
		// fprintf(stderr, "Error: %f\n", fError);

		static float fPrevError = -1.0f;
		float fErrorDif = fabsf(fPrevError - fError);
		if( i > max(max(iVolumeWidth, iVolumeHeight), iVolumeDepth) && (fErrorDif/fPrevError < 0.01f || fErrorDif < 0.000001f) )
		{
			printf("\t#iters = %d;", i);
			break;
		}
		fPrevError = fError;
		#endif	// MOD-BY-LEETEN 2009/12/17-END

		#endif	// #if	CHECK_ERROR_CONVERGENCE_BY_CUDPP
		// ADD-BY-LEETEN 2009/11/25-END

	}

	// ADD-BY-LEETEN 12/14/2009-BEGIN
	if( iVolumeDepth > 2 )
	{
		_Vector3DToVolume(
			iVolumeWidth,
			iVolumeHeight,
			iVolumeDepth,
			iBlockZSize,
			pcVolumePtrs_global[iSrc],
			cDstBinVolume_pitched);
			// piDstAngleBinVolume_host);
	}
	// ADD-BY-LEETEN 12/14/2009-END

	CLOCK_END(PRINT_FLOW_FUSION_TIMING, true);

	int iDst = iSrc;

	// read the result from latest iteration back to the CPU side
	CLOCK_BEGIN(PRINT_FLOW_FUSION_TIMING);

	hipMemcpy3DParms cCopyParamsDeviceToHost = {0};
    cCopyParamsDeviceToHost.srcPtr		= pcVolumePtrs_global[iDst];
    cCopyParamsDeviceToHost.dstPtr		= make_hipPitchedPtr(
		(void*)pf4Volume_host, 
		cVolumeExtent.width, 
		iVolumeWidth * sizeof(float4), 
		iVolumeHeight);
    cCopyParamsDeviceToHost.extent		= cVolumeExtent;
    cCopyParamsDeviceToHost.kind		= hipMemcpyDeviceToHost;

	CUDA_SAFE_CALL( hipMemcpy3D(&cCopyParamsDeviceToHost) );  

	CLOCK_END(PRINT_FLOW_FUSION_TIMING, true);

	// convert the #channels in the dst volume from 4 to 3
	CLOCK_BEGIN(PRINT_FLOW_FUSION_TIMING);
	for(int v = 0,		d = 0; d < iVolumeDepth;	d++)
		for(int			h = 0; h < iVolumeHeight;	h++)
			for(int		w = 0; w < iVolumeWidth;	w++, v++)
			{
				pfDstVolume[v * 3 + 0] = pf4Volume_host[v].x;
				pfDstVolume[v * 3 + 1] = pf4Volume_host[v].y;
				pfDstVolume[v * 3 + 2] = pf4Volume_host[v].z;
			}
	CLOCK_END(PRINT_FLOW_FUSION_TIMING, true);

	CLOCK_PRINT(PRINT_FLOW_FUSION_TIMING);
}

} // extern "C"

/*

$Log: not supported by cvs2svn $
Revision 1.3  2009/12/17 20:20:32  leeten

[12/17/2009]
1. [MOD] Change the preprocessor USE_CUDPP to CHECK_ERROR_CONVERGENCE_BY_CUDPP.
2. [MOD] Move several preprocessors to a new header FlowDiffusion_cuda.h
3. [MOD] Change the variable t2dBinVolumeSrc to t2dVectorVolume.
4. [MOD] Move CUDA kernels for 2D diffusion to FlowDiffusion2D_kernel.cu.
5. [MOD] Move CUDA kernels for 3D diffusion to FlowDiffusion3D_kernel.cu.
6. [MOD] Move CUDA kernels for 3D entropy field computation to EntropyField_kernel.cu.
7. [ADD] Define host functions to compute the entropy field by CPUs.
8. [DEL] Remove useless codesegments.

Revision 1.2  2009/12/15 20:05:57  leeten

[12/15/2009]
1. [ADD] Define new function _FlowDiffusionSetAngleMap() to setup the lookup table that converts a vector to a discrete bin.
2. [ADD] Define new function _GetSrcBinVolume() to download the bin of the orginal input vector field.
3. [ADD] Define new function _GetDstBinVolume() to download the bin for the diffused vector field.
4. [ADD] Define new function _ComputeSrcBinVolume() to download the bin of the orginal input vector field.
5. [ADD] Define a host function _Vector3DToVolume to convert each vector in a vector field into a bin.
6. [ADD] Define a kernel function _Vector3DToVolume_kernel() to convert each vector in a vector field into a bin on GPUs.

2. [ADD] Define new functions

Revision 1.1.1.1  2009/12/07 20:04:02  leeten

[12/07/2009]
1. [1ST] First time checkin.

Revision 1.3  2009/12/05 21:19:51  leeten

[12/05/2009]
1. [ADD] Add a 3D texture glWeightOffset to store the weights and offsets.
2. [ADD] Specify the flag donochange to the w channel in the 3D textures.
3. [ADD] Output the error to a 3D volume.

Revision 1.2  2009/11/04 19:11:13  leeten

[2009/11/04]
1. [ADD] Add a new preprocessor USE_SHARED_MEMORY to decide whether the shared memory is utilized.
2. [ADD] Add a new preprocessor DIFFUSION_BY_FOR_LOOP_ON_HOSTto decide whether the diffusion is executed via a for loop on the host side to scan throught all XY layers.
If it is not zero, the volume will be divided along the Z direction as well, and each block will be assigned as a single block on CUDA.
3. [ADD] Add 2 new preprocessors BLOCK_DIM_X and BLOCK_DIM_Y to control the block dimension.
4. [ADD] Add 1 array and 1 3D textrue for the weight/offset volume.

Revision 1.1.1.1  2009/11/02 15:30:56  leeten

[11/02/2009]
1 [1ST] Firs time checkin.


*/
