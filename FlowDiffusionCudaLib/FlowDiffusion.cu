#include "hip/hip_runtime.h"
////////////////////////////////////////////
	#include <stdio.h>
	#include <>

	#include "FlowDiffusion_cuda.h"
	#include "FlowDiffusion.h"

////////////////////////////////////////////
	static hipExtent cVolumeExtent;
	static float4 *pf4Volume_host;
	static hipPitchedPtr pcVolumePtrs_global[2];
	static texture<float4, 2, hipReadModeElementType> t2dSrc;
						// array and texture reference for the weight and offset volume
	static hipExtent cVolumeExtent_array;
	static float4 *pf4WeightOffsetVolume_host;
	static hipArray *cWeightOffsetVolume_array = 0;
	static texture<float4, 3, hipReadModeElementType> t3dWeightOffset;
    static hipChannelFormatDesc cWeightOffsetChannelDesc = hipCreateChannelDesc<float4>();

	#if	CHECK_ERROR_CONVERGENCE_BY_CUDPP
    static CUDPPConfiguration cConfig;
    static CUDPPHandle cScanplan = 0;
	#endif	// #if	CHECK_ERROR_CONVERGENCE_BY_CUDPP

	#if	CHECK_ERROR_CONVERGENCE_BY_CUDPP				
	static hipExtent cErrorVolumeExtent;
	#endif	// #if	CHECK_ERROR_CONVERGENCE_BY_CUDPP	

	static hipPitchedPtr cErrorVolume_device;
	static hipPitchedPtr cErrorSum_device;

	int iNrOfVoxels;

	static texture<int, 2, hipReadModeElementType> t2dAngleMap;
	static hipArray *pcAngleMap_array;	// cuda array to hold the Gaussian kernels

	static hipExtent cBinVolumeExtent;
	static hipPitchedPtr cSrcBinVolume_pitched;
	static hipPitchedPtr cDstBinVolume_pitched;

						// a temp. volume in the host side to stored the 
	static int *piBinVolume_host;

						// a texture tha represents a vector field to be converted into bins 
	static texture<float4, 2, hipReadModeElementType> t2dVectorVolume;

////////////////////////////////////////////
	#include "FlowDiffusion2D_kernel.cu"

////////////////////////////////////////////

#include "EntropyField_kernel.cu"

	#if	COMPUTE_ENTROPY_VOLUME == COMPUTE_ENTROPY_VOLUME_HOST
		#include	"EntropyField_host.cu"
	#endif

	#if	COMPUTE_ENTROPY_VOLUME == COMPUTE_ENTROPY_VOLUME_CUDA
		#include	"EntropyField_cuda.cu"
	#endif

	#if	COMPUTE_ENTROPY_VOLUME == COMPUTE_ENTROPY_VOLUME_WITH_SORTING_CUDA	
		#include	"EntropyFieldWithSorting_cuda.cu"
	#endif

	#if	COMPUTE_ENTROPY_VOLUME == COMPUTE_ENTROPY_VOLUME_PER_SCANLINE_CUDA
		#include	"EntropyFieldPerScanline_cuda.cu"
	#endif

	#if	COMPUTE_ENTROPY_VOLUME == COMPUTE_ENTROPY_VOLUME_SORT_PER_VOXEL_CUDA	
		#include	"EntropyFieldSortPerVoxel_cuda.cu"
	#endif

	#if	COMPUTE_ENTROPY_VOLUME == COMPUTE_ENTROPY_VOLUME_ON_SPARSE_HISTOGRAM_CUDA	
		#include	"EntropyFieldOnSparseHistogram_cuda.cu"
	#endif

	#if	COMPUTE_ENTROPY_VOLUME == COMPUTE_ENTROPY_VOLUME_WITH_MARGINAL_HISTOGRAM	
		#include	"EntropyFieldWithMarginalHistogram_cuda.cu"
	#endif

	#if	COMPUTE_ENTROPY_VOLUME == COMPUTE_ENTROPY_VOLUME_WITH_SORTING_IN_VECTORS_CUDA	
		#include	"EntropyFieldWithSortingInVectors_cuda.cu"
	#endif

////////////////////////////////////////////

void
_ComputeSrcEntropyVolume
(
	int iNrOfBins,
	int iKernelWidth, int iKernelHeight, int iKernelDepth,
	float *pfEntropyVolume_host
)
{
	int *piHistogram_global;
	CUDA_SAFE_CALL_NO_SYNC(
		hipMalloc(	(void**)&piHistogram_global, sizeof(piHistogram_global[0]) * iNrOfBins) );
	CUDA_SAFE_CALL_NO_SYNC(
		hipMemset(	(void*)piHistogram_global, 0, sizeof(piHistogram_global[0]) * iNrOfBins) );

	float *pfLogHistogram_global;
	CUDA_SAFE_CALL_NO_SYNC(
		hipMalloc(	(void**)&pfLogHistogram_global, sizeof(pfLogHistogram_global[0]) * iNrOfBins) );
	CUDA_SAFE_CALL_NO_SYNC(
		hipMemset(	(void*)pfLogHistogram_global, 0, sizeof(pfLogHistogram_global[0]) * iNrOfBins) );

	hipExtent cEntropyVolumeExtent	= make_hipExtent(
		cVolumeExtent_array.width * sizeof(float),
		cVolumeExtent_array.height,
		cVolumeExtent_array.depth);
	hipPitchedPtr cEntropyVolume_pitched;
	CUDA_SAFE_CALL( 
		hipMalloc3D(&cEntropyVolume_pitched, cEntropyVolumeExtent) );

	#if	COMPUTE_ENTROPY_VOLUME == COMPUTE_ENTROPY_VOLUME_CUDA
		_ComputeEntropyVolume_cuda
	#endif

	#if	COMPUTE_ENTROPY_VOLUME == COMPUTE_ENTROPY_VOLUME_HOST
		_ComputeEntropyVolume_host
	#endif

	#if	COMPUTE_ENTROPY_VOLUME == COMPUTE_ENTROPY_VOLUME_WITH_SORTING_CUDA
		_ComputeEntropyVolumeWithSorting_cuda
	#endif	

	#if	COMPUTE_ENTROPY_VOLUME == COMPUTE_ENTROPY_VOLUME_PER_SCANLINE_CUDA	
		_ComputeEntropyVolumePerScanline_cuda
	#endif	

	#if	COMPUTE_ENTROPY_VOLUME == COMPUTE_ENTROPY_VOLUME_SORT_PER_VOXEL_CUDA	
		_ComputeEntropyVolumePerVoxel_cuda
	#endif	

	#if	COMPUTE_ENTROPY_VOLUME == COMPUTE_ENTROPY_VOLUME_ON_SPARSE_HISTOGRAM_CUDA
		_ComputeEntropyVolumeOnSparseHistogram_cuda
	#endif	

	#if	COMPUTE_ENTROPY_VOLUME == COMPUTE_ENTROPY_VOLUME_WITH_SORTING_IN_VECTORS_CUDA
		_ComputeEntropyVolumeWithSortingInVectors_cuda
	#endif	

	(
						// res. of the neighboring region
		make_int3(iKernelWidth, iKernelHeight, iKernelDepth),

						// the histogram
		iNrOfBins,
		piHistogram_global,
		pfLogHistogram_global,

						// res. of the volume																
		make_int3(int(cVolumeExtent_array.width), int(cVolumeExtent_array.height), int(cVolumeExtent_array.depth)),

						// bin volume																	
		cSrcBinVolume_pitched,
		cEntropyVolume_pitched
	);

	if( NULL != pfEntropyVolume_host )
		CUDA_SAFE_CALL(
			hipMemcpy2D(
				pfEntropyVolume_host, 
				cVolumeExtent_array.width * sizeof(pfEntropyVolume_host[0]),
				cEntropyVolume_pitched.ptr,
				cEntropyVolume_pitched.pitch,
				cVolumeExtent_array.width * sizeof(pfEntropyVolume_host[0]),
				cVolumeExtent_array.height * cVolumeExtent_array.depth,
				hipMemcpyDeviceToHost) );

	FREE_MEMORY(piHistogram_global);
	FREE_MEMORY(pfLogHistogram_global);
	FREE_MEMORY(cEntropyVolume_pitched.ptr);
}	

void
_ComputeJointEntropyVolume
(
	int iNrOfSrcBins,
	int iNrOfDstBins,
	int iKernelWidth, int iKernelHeight, int iKernelDepth,
	float *pfEntropyVolume_host
)
{
	int iNrOfBins = iNrOfSrcBins * iNrOfDstBins;

	int *piHistogram_global;
	CUDA_SAFE_CALL_NO_SYNC(
		hipMalloc(	(void**)&piHistogram_global, sizeof(piHistogram_global[0]) * iNrOfBins) );
	CUDA_SAFE_CALL_NO_SYNC(
		hipMemset(	(void*)piHistogram_global, 0, sizeof(piHistogram_global[0]) * iNrOfBins) );

	float *pfLogHistogram_global;
	CUDA_SAFE_CALL_NO_SYNC(
		hipMalloc(	(void**)&pfLogHistogram_global, sizeof(pfLogHistogram_global[0]) * iNrOfBins) );
	CUDA_SAFE_CALL_NO_SYNC(
		hipMemset(	(void*)pfLogHistogram_global, 0, sizeof(pfLogHistogram_global[0]) * iNrOfBins) );

	hipExtent cEntropyVolumeExtent	= make_hipExtent(
		cVolumeExtent_array.width * sizeof(float),
		cVolumeExtent_array.height,
		cVolumeExtent_array.depth);
	hipPitchedPtr cEntropyVolume_pitched;
	CUDA_SAFE_CALL( 
		hipMalloc3D(&cEntropyVolume_pitched, cEntropyVolumeExtent) );

	// bind the bin volume as a 2D texture
	t2dSrcBinVolume.addressMode[0] = hipAddressModeClamp;
	t2dSrcBinVolume.addressMode[1] = hipAddressModeClamp;
	t2dSrcBinVolume.filterMode =	hipFilterModePoint;
	t2dSrcBinVolume.normalized =	false;
	CUDA_SAFE_CALL_NO_SYNC(
		hipBindTexture2D(
			0, 
			t2dSrcBinVolume, 
			cSrcBinVolume_pitched.ptr, 
			hipCreateChannelDesc<int>(),
			cVolumeExtent_array.width, 
			cVolumeExtent_array.height * cVolumeExtent_array.depth, 
			cSrcBinVolume_pitched.pitch) );

	t2dDstBinVolume.addressMode[0] = hipAddressModeClamp;
	t2dDstBinVolume.addressMode[1] = hipAddressModeClamp;
	t2dDstBinVolume.filterMode =	hipFilterModePoint;
	t2dDstBinVolume.normalized =	false;
	CUDA_SAFE_CALL_NO_SYNC(
		hipBindTexture2D(
			0, 
			t2dDstBinVolume, 
			cDstBinVolume_pitched.ptr, 
			hipCreateChannelDesc<int>(),
			cVolumeExtent_array.width, 
			cVolumeExtent_array.height * cVolumeExtent_array.depth, 
			cDstBinVolume_pitched.pitch) );

	// bind the histogram as a 1D texture
	// ...
	hipPitchedPtr cJointBinVolume_pitched;
	CUDA_SAFE_CALL( 
		hipMalloc3D(
			&cJointBinVolume_pitched, 
			make_hipExtent(
				cVolumeExtent_array.width * sizeof(int),
				cVolumeExtent_array.height,
				cVolumeExtent_array.depth) ) );

	dim3 v3Blk, v3Grid;
	v3Blk = dim3(BLOCK_DIM_X, BLOCK_DIM_Y);
	v3Grid = dim3(
		size_t(ceilf(float(cVolumeExtent_array.width) / float(v3Blk.x))),
		size_t(ceilf(float(cVolumeExtent_array.height * cVolumeExtent_array.depth) / float(v3Blk.y)))
	);

	_JointSrcDst_kernel<<<v3Grid, v3Blk, 0>>>
	(
		iNrOfSrcBins, 
		iNrOfDstBins,

		make_int3(
			int(cVolumeExtent_array.width), 
			int(cVolumeExtent_array.height), 
			int(cVolumeExtent_array.depth)),

		cJointBinVolume_pitched
	);
	CUT_CHECK_ERROR("_JointSrcDst_kernel() failed");

	#if	COMPUTE_ENTROPY_VOLUME == COMPUTE_ENTROPY_VOLUME_CUDA
		_ComputeEntropyVolume_cuda
	#endif

	#if	COMPUTE_ENTROPY_VOLUME == COMPUTE_ENTROPY_VOLUME_HOST
		_ComputeEntropyVolume_host
	#endif

	#if	COMPUTE_ENTROPY_VOLUME == COMPUTE_ENTROPY_VOLUME_WITH_SORTING_CUDA
		_ComputeEntropyVolumeWithSorting_cuda
	#endif

	#if	COMPUTE_ENTROPY_VOLUME == COMPUTE_ENTROPY_VOLUME_PER_SCANLINE_CUDA	
		_ComputeEntropyVolumePerScanline_cuda
	#endif	

	#if	COMPUTE_ENTROPY_VOLUME == COMPUTE_ENTROPY_VOLUME_SORT_PER_VOXEL_CUDA	
		_ComputeEntropyVolumePerVoxel_cuda
	#endif	

	#if	COMPUTE_ENTROPY_VOLUME == COMPUTE_ENTROPY_VOLUME_ON_SPARSE_HISTOGRAM_CUDA
		_ComputeEntropyVolumeOnSparseHistogram_cuda
	#endif	 

	#if	COMPUTE_ENTROPY_VOLUME == COMPUTE_ENTROPY_VOLUME_WITH_MARGINAL_HISTOGRAM
		_ComputeEntropyVolumeWithMarginalHistogram_cuda
	#endif	

	#if	COMPUTE_ENTROPY_VOLUME == COMPUTE_ENTROPY_VOLUME_WITH_SORTING_IN_VECTORS_CUDA
		_ComputeEntropyVolumeWithSortingInVectors_cuda
	#endif	
	(
						// res. of the neighboring region
		make_int3(iKernelWidth, iKernelHeight, iKernelDepth),

						// the histogram
		iNrOfBins,
		piHistogram_global,
		pfLogHistogram_global,

						// res. of the volume																
		make_int3(int(cVolumeExtent_array.width), int(cVolumeExtent_array.height), int(cVolumeExtent_array.depth)),

						// bin volume																	
		cJointBinVolume_pitched,
		cEntropyVolume_pitched
	);

	if( NULL != pfEntropyVolume_host )
		CUDA_SAFE_CALL_NO_SYNC(
			hipMemcpy2D(
				pfEntropyVolume_host, 
				cVolumeExtent_array.width * sizeof(pfEntropyVolume_host[0]),
				cEntropyVolume_pitched.ptr,
				cEntropyVolume_pitched.pitch,
				cVolumeExtent_array.width * sizeof(pfEntropyVolume_host[0]),
				cVolumeExtent_array.height * cVolumeExtent_array.depth,
				hipMemcpyDeviceToHost) );

	FREE_MEMORY(cJointBinVolume_pitched.ptr);
	FREE_MEMORY(piHistogram_global);
	FREE_MEMORY(pfLogHistogram_global);
	FREE_MEMORY(cEntropyVolume_pitched.ptr);
}

////////////////////////////////////////////
	#include "FlowDiffusion3D_kernel.cu"

extern "C" {

void 
_FlowDiffusionFree()
{
	FREE_MEMORY_ON_HOST(pf4Volume_host);
	for(int i = 0; i < 2; i++)
		FREE_MEMORY(pcVolumePtrs_global[i].ptr);
	FREE_ARRAY(cWeightOffsetVolume_array);
	FREE_MEMORY_ON_HOST(pf4WeightOffsetVolume_host);
	FREE_MEMORY(cErrorVolume_device.ptr);
	FREE_MEMORY(cErrorSum_device.ptr);

	#if	CHECK_ERROR_CONVERGENCE_BY_CUDPP
	cudppDestroyPlan(cScanplan);
	#endif	// #if	CHECK_ERROR_CONVERGENCE_BY_CUDPP

	FREE_ARRAY(pcAngleMap_array);	

	FREE_MEMORY(cSrcBinVolume_pitched.ptr);
	FREE_MEMORY(cDstBinVolume_pitched.ptr);
	FREE_MEMORY(cSrcDstBinVolumePtr_global.ptr);
}

void
_FlowDiffusionInit(
	int iVolumeWidth,
	int iVolumeHeight,
	int iVolumeDepth
)
{
	CLOCK_INIT(PRINT_FLOW_DIFFUSION_TIMING, __FUNCTION__ ": ");

	iNrOfVoxels = iVolumeWidth * iVolumeHeight * iVolumeDepth;

	cVolumeExtent	= make_hipExtent(
		iVolumeWidth * sizeof(float4),
		iVolumeHeight,
		iVolumeDepth);

	// allocate a local copy in the CPU side
	CLOCK_BEGIN(PRINT_FLOW_DIFFUSION_TIMING);
	CUDA_SAFE_CALL(
		hipHostMalloc(
			(void**)&pf4Volume_host,
			sizeof(pf4Volume_host[0]) * iNrOfVoxels) );
	CLOCK_END(PRINT_FLOW_DIFFUSION_TIMING, true);

	// Allocate 2 volumes on the  GPU's global memory.
	// in each iteration, one of them will be used as the source
	// 3D texture, and the other will be served as the dst.
	CLOCK_BEGIN(PRINT_FLOW_DIFFUSION_TIMING);

	for(int i = 0; i < 2; i++)
	{
		CUDA_SAFE_CALL( 
			hipMalloc3D(&pcVolumePtrs_global[i], cVolumeExtent) );
	}

	#if	CHECK_ERROR_CONVERGENCE_BY_CUDPP	
	cErrorVolumeExtent	= make_hipExtent(
		iVolumeWidth * sizeof(float),
		iVolumeHeight,
		iVolumeDepth);
	CUDA_SAFE_CALL( 
		hipMalloc3D(&cErrorVolume_device,	cErrorVolumeExtent) );
	CUDA_SAFE_CALL( 
		hipMalloc3D(&cErrorSum_device,		cErrorVolumeExtent) );

	cConfig.op = CUDPP_ADD;
	// cConfig.op = CUDPP_MAX;
	cConfig.datatype = CUDPP_FLOAT;
	cConfig.algorithm = CUDPP_SCAN;
	cConfig.options = CUDPP_OPTION_BACKWARD | CUDPP_OPTION_INCLUSIVE;
	unsigned int uPlanStatus = cudppPlan(&cScanplan, cConfig, iVolumeWidth * iVolumeHeight * iVolumeDepth, iVolumeHeight * iVolumeDepth, cErrorVolume_device.pitch);
	assert(CUDPP_SUCCESS  == uPlanStatus);  
	#endif	// #if	CHECK_ERROR_CONVERGENCE_BY_CUDPP

	cVolumeExtent_array = make_hipExtent(
		iVolumeWidth,
		iVolumeHeight,
		iVolumeDepth);

	CUDA_SAFE_CALL( 
		hipMalloc3DArray(&cWeightOffsetVolume_array, &cWeightOffsetChannelDesc, cVolumeExtent_array) );

	CUDA_SAFE_CALL(
		hipHostMalloc(
			(void**)&pf4WeightOffsetVolume_host,
			sizeof(pf4WeightOffsetVolume_host[0]) * iNrOfVoxels) );

	CLOCK_END(PRINT_FLOW_DIFFUSION_TIMING, true);

	CLOCK_PRINT(PRINT_FLOW_DIFFUSION_TIMING);
}

void 
_FlowDiffusionSetAngleMap(int *piAngleMap, int iNrOfPhis, int iNrOfThetas)
{
	// allocate an array on the GPU side

	// upload the angle map to the araray

	// setup the texture for the angle map
	/*
    t2dAngleMap.addressMode[0] =	hipAddressModeClamp;
    t2dAngleMap.addressMode[1] =	hipAddressModeClamp;
    t2dAngleMap.filterMode =		hipFilterModePoint;
    t2dAngleMap.normalized =		true;
	*/
	SETUP_ARRAY(pcAngleMap_array, iNrOfPhis, iNrOfThetas, 32, 0, 0, 0, hipChannelFormatKindSigned);	
	BIND_ARRAY_AS_TEXTURE(
		t2dAngleMap, pcAngleMap_array, 
		hipAddressModeClamp, hipAddressModeClamp, hipFilterModePoint, true);	

    CUDA_SAFE_CALL( 
		hipMemcpy2DToArray(
			pcAngleMap_array, 
			0, 
			0, 
			(void*)piAngleMap, 
			iNrOfPhis * sizeof(piAngleMap[0]), 
			iNrOfPhis * sizeof(piAngleMap[0]), 
			iNrOfThetas, 
			hipMemcpyHostToDevice) );

	// check whether the volume extent has been setup
	assert( NULL != pcVolumePtrs_global[0].ptr );

	// allocate the volume of bins in the global memorty on GPUs
	cBinVolumeExtent	= make_hipExtent(
		cVolumeExtent_array.width * sizeof(int),
		cVolumeExtent_array.height,
		cVolumeExtent_array.depth);
	CUDA_SAFE_CALL( 
		hipMalloc3D(&cSrcBinVolume_pitched, cBinVolumeExtent) );
	CUDA_SAFE_CALL( 
		hipMalloc3D(&cDstBinVolume_pitched, cBinVolumeExtent) );

	CUDA_SAFE_CALL( 
		hipMalloc3D(&cSrcDstBinVolumePtr_global, cBinVolumeExtent) );

	t2dSrcBinVolume.addressMode[0] = hipAddressModeClamp;
	t2dSrcBinVolume.addressMode[1] = hipAddressModeClamp;
	t2dSrcBinVolume.filterMode =	hipFilterModePoint;
	t2dSrcBinVolume.normalized =	false;

	t2dDstBinVolume.addressMode[0] = hipAddressModeClamp;
	t2dDstBinVolume.addressMode[1] = hipAddressModeClamp;
	t2dDstBinVolume.filterMode =	hipFilterModePoint;
	t2dDstBinVolume.normalized =	false;

	// allocate the volume of bins in the host side

	// setup the src. texture for bin lookup
	t2dVectorVolume.addressMode[0] = hipAddressModeClamp;
	t2dVectorVolume.addressMode[1] = hipAddressModeClamp;
	t2dVectorVolume.filterMode =	hipFilterModePoint;
	t2dVectorVolume.normalized =	false;
}

void
_Vector3DToVolume(
	int iVolumeWidth,
	int iVolumeHeight,
	int iVolumeDepth,
	int iBlockZSize,
	hipPitchedPtr cVolumePtr_global,	// input
	hipPitchedPtr cBinVolumePtr_global	// output
)
{
	CUDA_SAFE_CALL_NO_SYNC(
		hipBindTexture2D(
			0, 
			t2dVectorVolume, 
			cVolumePtr_global.ptr, 
			hipCreateChannelDesc<float4>(),
			iVolumeWidth, 
			iVolumeHeight * iVolumeDepth, 
			cVolumePtr_global.pitch) );

	dim3 v3Blk = dim3(BLOCK_DIM_X, BLOCK_DIM_Y);
	dim3 v3Grid = dim3(
		(unsigned int)ceilf((float)iVolumeWidth	 / (float)v3Blk.x),
		unsigned int(ceilf( float(iVolumeHeight) / float(v3Blk.y) ) ) *
		unsigned int(ceilf( float(iVolumeDepth) /  float(iBlockZSize) ) ) );
	_Vector3DToBinVolume_kernel<<<v3Grid, v3Blk, 0>>>
	(
		iVolumeWidth,
		iVolumeHeight,
		iVolumeDepth,
		int(ceilf(float(iVolumeHeight) / float(BLOCK_DIM_Y))),
		iBlockZSize,
		cBinVolumePtr_global
	);
	CUT_CHECK_ERROR("_Vector3DToVolume_kernel() failed");
}

void
_ComputeSrcBinVolume
(
	int iVolumeWidth,
	int iVolumeHeight,
	int iVolumeDepth,
	int iBlockZSize,
	float *pfSrcVolume
)
{
	CLOCK_INIT(SHOW_COMPUTE_SRC_BIN_VOLUME_TIMING, __FUNCTION__ ": ");

	CLOCK_BEGIN(SHOW_COMPUTE_SRC_BIN_VOLUME_TIMING);
	for(int v = 0,		d = 0; d < iVolumeDepth;	d++)
		for(int			h = 0; h < iVolumeHeight;	h++)
			for(int		w = 0; w < iVolumeWidth;	w++, v++)
			{
				pf4Volume_host[v].x = pfSrcVolume[v * 3 + 0];
				pf4Volume_host[v].y = pfSrcVolume[v * 3 + 1];
				pf4Volume_host[v].z = pfSrcVolume[v * 3 + 2];
				pf4Volume_host[v].w = 0.0f;
			}
	CLOCK_END(SHOW_COMPUTE_SRC_BIN_VOLUME_TIMING, false);

	CLOCK_BEGIN(SHOW_COMPUTE_SRC_BIN_VOLUME_TIMING);
    hipMemcpy3DParms cCopyParamsHostToDevice = {0};

    cCopyParamsHostToDevice.srcPtr   = make_hipPitchedPtr(
		(void*)pf4Volume_host, 
		cVolumeExtent.width, 
		iVolumeWidth * sizeof(float4), 
		iVolumeHeight);
    cCopyParamsHostToDevice.dstPtr		= pcVolumePtrs_global[0];
    cCopyParamsHostToDevice.extent		= cVolumeExtent;
    cCopyParamsHostToDevice.kind		= hipMemcpyHostToDevice;
    CUDA_SAFE_CALL( hipMemcpy3D(&cCopyParamsHostToDevice) );  

	CLOCK_END(SHOW_COMPUTE_SRC_BIN_VOLUME_TIMING, false);

	CLOCK_BEGIN(SHOW_COMPUTE_SRC_BIN_VOLUME_TIMING);
	_Vector3DToVolume(
		iVolumeWidth,
		iVolumeHeight,
		iVolumeDepth,
		iBlockZSize,
		pcVolumePtrs_global[0],
		cSrcBinVolume_pitched);
	CLOCK_END(SHOW_COMPUTE_SRC_BIN_VOLUME_TIMING, false);

	CLOCK_PRINT(SHOW_COMPUTE_SRC_BIN_VOLUME_TIMING);
}

void 
_GetSrcBinVolume(int *piBinVolume)
{
	CUDA_SAFE_CALL(
		hipHostMalloc(
			(void**)&piBinVolume_host,
			sizeof(piBinVolume_host[0]) * iNrOfVoxels) );

	hipMemcpy3DParms cCopyParamsDeviceToHost = {0};
	cCopyParamsDeviceToHost.srcPtr		= cSrcBinVolume_pitched;
	cCopyParamsDeviceToHost.dstPtr		= make_hipPitchedPtr(
		(void*)piBinVolume_host, 
		cVolumeExtent_array.width * sizeof(int),	// cSrcBinVolume_pitched.width, 
		cVolumeExtent_array.width * sizeof(int), 
		cVolumeExtent_array.height);
	cCopyParamsDeviceToHost.extent		= cBinVolumeExtent;
	cCopyParamsDeviceToHost.kind		= hipMemcpyDeviceToHost;

	CUDA_SAFE_CALL( hipMemcpy3D(&cCopyParamsDeviceToHost) );  

	memcpy(piBinVolume, piBinVolume_host, sizeof(piBinVolume) * iNrOfVoxels);

	FREE_MEMORY_ON_HOST(piBinVolume_host);
}

void 
_GetDstBinVolume(int *piBinVolume)
{
	CUDA_SAFE_CALL(
		hipHostMalloc(
			(void**)&piBinVolume_host,
			sizeof(piBinVolume_host[0]) * iNrOfVoxels) );


	hipMemcpy3DParms cCopyParamsDeviceToHost = {0};
	cCopyParamsDeviceToHost.srcPtr		= cDstBinVolume_pitched;
	cCopyParamsDeviceToHost.dstPtr		= make_hipPitchedPtr(
		(void*)piBinVolume_host, 

		cVolumeExtent_array.width * sizeof(int),	// cDstBinVolume_pitched.width, 
		cVolumeExtent_array.width * sizeof(int), 
		cVolumeExtent_array.height);
	cCopyParamsDeviceToHost.extent		= cBinVolumeExtent;
	cCopyParamsDeviceToHost.kind		= hipMemcpyDeviceToHost;

	CUDA_SAFE_CALL( hipMemcpy3D(&cCopyParamsDeviceToHost) );  

	memcpy(piBinVolume, piBinVolume_host, sizeof(piBinVolume) * iNrOfVoxels);

	FREE_MEMORY_ON_HOST(piBinVolume_host);
}

void
_FlowDiffusion(
	float fAttenuation,
	int iNrOfIterations,
	int iVolumeWidth,
	int iVolumeHeight,
	int iVolumeDepth,
	int iBlockZSize,
	float *pfWeightVolume,
	float *pfXOffsetVolume,
	float *pfYOffsetVolume,
	float *pfZOffsetVolume,
	float *pfSrcVolume,
	float *pfDstVolume,
	int *piFlagVolume
)
{
	#if	CHECK_ERROR_CONVERGENCE_BY_CUDPP
	iNrOfIterations = 9 * max(max(iVolumeWidth, iVolumeHeight), iVolumeDepth);
	// LOG(printf("Warning! iNrOfIterations is changed to %d", iNrOfIterations));
	#endif
	
	CLOCK_INIT(PRINT_FLOW_DIFFUSION_TIMING, __FUNCTION__ ": ");

	dim3 v3Blk = dim3(BLOCK_DIM_X, BLOCK_DIM_Y);
	dim3 v3Grid = dim3(
		(unsigned int)ceilf((float)iVolumeWidth	 / (float)v3Blk.x),
		#if					DIFFUSION_BY_FOR_LOOP_ON_HOST	

		(unsigned int)ceilf((float)iVolumeHeight / (float)v3Blk.y));

		#else	// #if		DIFFUSION_BY_FOR_LOOP_ON_HOST
		unsigned int(ceilf( float(iVolumeHeight) / float(v3Blk.y) ) ) *
		unsigned int(ceilf( float(iVolumeDepth) /  float(iBlockZSize) ) ) );
		#endif	// #if		DIFFUSION_BY_FOR_LOOP_ON_HOST

	// convert the #channels in the src volume from 3 to 4
	CLOCK_BEGIN(PRINT_FLOW_DIFFUSION_TIMING);
	for(int v = 0,		d = 0; d < iVolumeDepth;	d++)
		for(int			h = 0; h < iVolumeHeight;	h++)
			for(int		w = 0; w < iVolumeWidth;	w++, v++)
			{
				pf4Volume_host[v].x = pfSrcVolume[v * 3 + 0];
				pf4Volume_host[v].y = pfSrcVolume[v * 3 + 1];
				pf4Volume_host[v].z = pfSrcVolume[v * 3 + 2];

				if( piFlagVolume )
					pf4Volume_host[v].w = float(piFlagVolume[v]);
				else
				pf4Volume_host[v].w = 0.0f;
			}

	for(int v = 0,		d = 0; d < iVolumeDepth;	d++)
		for(int			h = 0; h < iVolumeHeight;	h++)
			for(int		w = 0; w < iVolumeWidth;	w++, v++)
			{
				if( pfXOffsetVolume )
				pf4WeightOffsetVolume_host[v].x = pfXOffsetVolume[v];
				if( pfYOffsetVolume )
				pf4WeightOffsetVolume_host[v].y = pfYOffsetVolume[v];
				if( pfZOffsetVolume )
				pf4WeightOffsetVolume_host[v].z = pfZOffsetVolume[v];
				pf4WeightOffsetVolume_host[v].w = 1.0f - pfWeightVolume[v];
			}

	CLOCK_END(PRINT_FLOW_DIFFUSION_TIMING, true);

	// Upload the input volume to the first volume on the GPU
	CLOCK_BEGIN(PRINT_FLOW_DIFFUSION_TIMING);

    hipMemcpy3DParms cCopyParamsHostToDevice = {0};

    cCopyParamsHostToDevice.srcPtr   = make_hipPitchedPtr(
		(void*)pf4Volume_host, 
		cVolumeExtent.width, 
		iVolumeWidth * sizeof(float4), 
		iVolumeHeight);
    cCopyParamsHostToDevice.dstPtr		= pcVolumePtrs_global[0];
    cCopyParamsHostToDevice.extent		= cVolumeExtent;
    cCopyParamsHostToDevice.kind		= hipMemcpyHostToDevice;
    CUDA_SAFE_CALL( hipMemcpy3D(&cCopyParamsHostToDevice) );  

    hipMemcpy3DParms cCopyParamsHostToArray = {0};
    cCopyParamsHostToArray.srcPtr   = make_hipPitchedPtr(
		(void*)pf4WeightOffsetVolume_host, 
		iVolumeWidth * sizeof(float4), 
		iVolumeWidth, 
		iVolumeHeight);
    cCopyParamsHostToArray.dstArray = cWeightOffsetVolume_array;
    cCopyParamsHostToArray.extent   = cVolumeExtent_array;
    cCopyParamsHostToArray.kind     = hipMemcpyHostToDevice;
    CUDA_SAFE_CALL( 
		hipMemcpy3D(&cCopyParamsHostToArray) );  

    t3dWeightOffset.addressMode[0] = hipAddressModeClamp;
    t3dWeightOffset.addressMode[1] = hipAddressModeClamp;
    t3dWeightOffset.addressMode[2] = hipAddressModeClamp;
    t3dWeightOffset.filterMode		= hipFilterModePoint;
    t3dWeightOffset.normalized		= false; 

    // bind array to 3D texture
    CUDA_SAFE_CALL(
		hipBindTextureToArray(t3dWeightOffset, cWeightOffsetVolume_array, cWeightOffsetChannelDesc));

	CLOCK_END(PRINT_FLOW_DIFFUSION_TIMING, true);

	// calcuate the fusion operator on GPUs
	CLOCK_BEGIN(PRINT_FLOW_DIFFUSION_TIMING);

    t2dSrc.addressMode[0] = hipAddressModeClamp;
    t2dSrc.addressMode[1] = hipAddressModeClamp;
    t2dSrc.filterMode =		hipFilterModePoint;
    t2dSrc.normalized =		false;

	int iSrc = 0;

	for(int i = 0; 
			i < iNrOfIterations; 
			i++,				iSrc = 1 - iSrc)
	{
		// bind the src. volume as the 3D texture

		hipChannelFormatDesc cChannelDesc =
			hipCreateChannelDesc<float4>();

		CUDA_SAFE_CALL_NO_SYNC(
			hipBindTexture2D(
				0, 
				t2dSrc, 
				pcVolumePtrs_global[iSrc].ptr, 
				cChannelDesc,
				iVolumeWidth, 
				iVolumeHeight * iVolumeDepth, 
				pcVolumePtrs_global[iSrc].pitch) );


		if( iVolumeDepth <= 2 )
			_FlowDiffusion2D_kernel<<<v3Grid, v3Blk, 0>>>
			(
				fAttenuation,
				iVolumeWidth,
				iVolumeHeight,
				pcVolumePtrs_global[1 - iSrc],
				cErrorVolume_device
			);	
		else
		{

		// launch the kernel to compute the diffusion operator for 1 iteration
		#if	DIFFUSION_BY_FOR_LOOP_ON_HOST

			for(int z = 0; z < iVolumeDepth; z++)
				_FlowDiffusion3D_kernel<<<v3Grid, v3Blk, 0>>>
				(
					fAttenuation,
					iVolumeWidth,
					iVolumeHeight,
					iVolumeDepth,
					z,

					// output
					pcVolumePtrs_global[1 - iSrc],
					cErrorVolume_device
				);	
		#else	// #if	DIFFUSION_BY_FOR_LOOP_ON_HOST

		// _FlowFusion_kernel<<<v3Grid, v3Blk, v3Blk.x * v3Blk.y * sizeof(float4)>>>
		_FlowDiffusion3D_kernel<<<v3Grid, v3Blk, 0>>>
		(
			fAttenuation,
			iVolumeWidth,
			iVolumeHeight,
			iVolumeDepth,
			0,
			int(ceilf(float(iVolumeHeight) / float(BLOCK_DIM_Y))),
			iBlockZSize,
			// output
			pcVolumePtrs_global[1 - iSrc],
			cErrorVolume_device
		);
		#endif
		}
		CUT_CHECK_ERROR("_FlowFusion_kernel() failed");

		#if	CHECK_ERROR_CONVERGENCE_BY_CUDPP
		cudppScan(cScanplan, cErrorSum_device.ptr, cErrorVolume_device.ptr, iVolumeWidth * iVolumeHeight * iVolumeDepth);

		float fError;
	    CUDA_SAFE_CALL_NO_SYNC( 
			hipMemcpy(
				&fError, 
				cErrorSum_device.ptr, 
				sizeof(float), 
				hipMemcpyDeviceToHost));
		/*
		fError /= float(iVolumeWidth * iVolumeHeight * iVolumeDepth * 3);
		fError = sqrtf(fError);
		*/
		fError /= float(iVolumeWidth * iVolumeHeight * iVolumeDepth * 3);
		fError = sqrtf(fError);
		// fprintf(stderr, "Error: %f\n", fError);

		static float fPrevError = -1.0f;
		float fErrorRate = fError/fPrevError;
		if( i > max(max(iVolumeWidth, iVolumeHeight), iVolumeDepth) && 
			(DIFFUSION_CONVERGE_THRESHOLD <= fErrorRate && fErrorRate <= 1.00f ) )
		{
			printf("\t#iters = %d;", i);
			break;
		}
		if( i > 1 && fErrorRate > 1.0f )
		{
			printf("Error increase. Stop.");
			break;
		}
		fPrevError = fError;

		#endif	// #if	CHECK_ERROR_CONVERGENCE_BY_CUDPP
	}

	if( iVolumeDepth > 2 )
	{
		_Vector3DToVolume(
			iVolumeWidth,
			iVolumeHeight,
			iVolumeDepth,
			iBlockZSize,
			pcVolumePtrs_global[iSrc],
			cDstBinVolume_pitched);
			// piDstAngleBinVolume_host);
	}

	CLOCK_END(PRINT_FLOW_DIFFUSION_TIMING, true);

	int iDst = iSrc;

	// read the result from latest iteration back to the CPU side
	CLOCK_BEGIN(PRINT_FLOW_DIFFUSION_TIMING);

	hipMemcpy3DParms cCopyParamsDeviceToHost = {0};
    cCopyParamsDeviceToHost.srcPtr		= pcVolumePtrs_global[iDst];
    cCopyParamsDeviceToHost.dstPtr		= make_hipPitchedPtr(
		(void*)pf4Volume_host, 
		cVolumeExtent.width, 
		iVolumeWidth * sizeof(float4), 
		iVolumeHeight);
    cCopyParamsDeviceToHost.extent		= cVolumeExtent;
    cCopyParamsDeviceToHost.kind		= hipMemcpyDeviceToHost;

	CUDA_SAFE_CALL( hipMemcpy3D(&cCopyParamsDeviceToHost) );  

	CLOCK_END(PRINT_FLOW_DIFFUSION_TIMING, true);

	// convert the #channels in the dst volume from 4 to 3
	CLOCK_BEGIN(PRINT_FLOW_DIFFUSION_TIMING);
	for(int v = 0,		d = 0; d < iVolumeDepth;	d++)
		for(int			h = 0; h < iVolumeHeight;	h++)
			for(int		w = 0; w < iVolumeWidth;	w++, v++)
			{
				pfDstVolume[v * 3 + 0] = pf4Volume_host[v].x;
				pfDstVolume[v * 3 + 1] = pf4Volume_host[v].y;
				pfDstVolume[v * 3 + 2] = pf4Volume_host[v].z;
			}
	CLOCK_END(PRINT_FLOW_DIFFUSION_TIMING, true);

	CLOCK_PRINT(PRINT_FLOW_DIFFUSION_TIMING);
}

} // extern "C"

