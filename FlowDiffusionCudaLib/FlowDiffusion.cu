#include "hip/hip_runtime.h"
////////////////////////////////////////////
	#include <stdio.h>
	#include <>

	#if	0	// DEL-BY-LEETEN 2009/12/17-BEGIN
		// ADD-BY-LEETEN 2009/11/25-BEGIN
		#define CHECK_ERROR_CONVERGENCE_BY_CUDPP	0

		#if	CHECK_ERROR_CONVERGENCE_BY_CUDPP	
			#include "cudpp/cudpp.h"
			#pragma comment (lib, "cudpp32.lib")
		#endif
		// ADD-BY-LEETEN 2009/11/25-END

		#include "cuda_macro.h"

		// ADD-BY-LEETEN 12/14/2009-BEGIN
		#include "liblog.h"
		using namespace std;

		#define	M_PI	3.1415926535897932384626433832795f
		// ADD-BY-LEETEN 12/14/2009-END

		#include "libbuf.h"

	////////////////////////////////////////////
		#define PRINT_FLOW_FUSION_TIMING	1
		#define USE_SHARED_MEMORY			0

		// ADD-BY-LEETEN 12/14/2009-BEGIN
		#define SHOW_COMPUTE_SRC_BIN_VOLUME_TIMING	1	
		// ADD-BY-LEETEN 12/14/2009-END

		// ADD-BY-LEETEN 11/04/2009-BEGIN
							// if this preprocessor is non zero, the volume is scanned via a for loop on the host
		#define DIFFUSION_BY_FOR_LOOP_ON_HOST		0
		// ADD-BY-LEETEN 11/04/2009-END

		// ADD-BY-LEETEN 10/02/2009-BEGIN
		#define BLOCK_DIM_X	16
		#define BLOCK_DIM_Y	12
		// ADD-BY-LEETEN 10/02/2009-END
	#endif	// DEL-BY-LEETEN 2009/12/17-END


	// ADD-BY-LEETEN 12/07/2009-BEGIN
	#include "FlowDiffusion_cuda.h"
	#include "FlowDiffusion.h"
	// ADD-BY-LEETEN 12/07/2009-END	

////////////////////////////////////////////
	static hipExtent cVolumeExtent;
	static float4 *pf4Volume_host;
	static hipPitchedPtr pcVolumePtrs_global[2];
	static texture<float4, 2, hipReadModeElementType> t2dSrc;
	// ADD-BY-LEETEN 10/02/2009-BEGIN
						// array and texture reference for the weight and offset volume
	static hipExtent cVolumeExtent_array;
	static float4 *pf4WeightOffsetVolume_host;
	static hipArray *cWeightOffsetVolume_array = 0;
	static texture<float4, 3, hipReadModeElementType> t3dWeightOffset;
    static hipChannelFormatDesc cWeightOffsetChannelDesc = hipCreateChannelDesc<float4>();
	// ADD-BY-LEETEN 10/02/2009-END

	// ADD-BY-LEETEN 2009/11/25-BEGIN
	#if	CHECK_ERROR_CONVERGENCE_BY_CUDPP
    static CUDPPConfiguration cConfig;
    static CUDPPHandle cScanplan = 0;
	#endif	// #if	CHECK_ERROR_CONVERGENCE_BY_CUDPP

	static hipExtent cErrorVolumeExtent;
	static hipPitchedPtr cErrorVolume_device;
	static hipPitchedPtr cErrorSum_device;
	// ADD-BY-LEETEN 2009/11/25-END

	// ADD-BY-LEETEN 12/14/2009-BEGIN
	int iNrOfVoxels;

	static texture<int, 2, hipReadModeElementType> t2dAngleMap;
	static hipArray *pcAngleMap_array;	// cuda array to hold the Gaussian kernels

	static hipExtent cBinVolumeExtent;
	#if	0	// MOD-BY-LEETEN 12/17/2009-FROM:
		static hipPitchedPtr cSrcBinVolumePtr_global;
		static hipPitchedPtr cDstBinVolumePtr_global;
	#else	// MOD-BY-LEETEN 12/17/2009-TO:
	static hipPitchedPtr cSrcBinVolume_pitched;
	static hipPitchedPtr cDstBinVolume_pitched;
	#endif	// MOD-BY-LEETEN 12/17/2009-END

	#if	0	// MOD-BY-LEETEN 2009/12/17-FROM:
		static int *piSrcAngleBinVolume_host;
		static int *piDstAngleBinVolume_host;
	#else	// MOD-BY-LEETEN 2009/12/17-TO:
						// a temp. volume in the host side to stored the 
	static int *piBinVolume_host;
	#endif	// MOD-BY-LEETEN 2009/12/17-END

						// a texture tha represents a vector field to be converted into bins 
	static texture<float4, 2, hipReadModeElementType> t2dVectorVolume;
	// ADD-BY-LEETEN 12/14/2009-END

// ADD-BY-LEETEN 12/07/2009-BEGIN
#if	0	// MOD-BY-LEETEN 2009/12/17-FROM:
	__global__ 
	static 
	void 
	_FlowDiffusion2D_kernel
	(
		// INPUT
		float fAttenuation,

		int iVolumeWidth,
		int iVolumeHeight,

		hipPitchedPtr cDstPitchedPtr,

		// ADD-BY-LEETEN 2009/11/25-BEGIN
		hipPitchedPtr cErrorPitchedPtr
		// ADD-BY-LEETEN 2009/11/25-END
	)
	{
 		int iVoxelX = blockIdx.x * blockDim.x + threadIdx.x;
		int iVoxelY = blockIdx.y * blockDim.y + threadIdx.y;

		// compute the central differnece
		float4 f4Value	= tex2D(t2dSrc, iVoxelX,		iVoxelY);
		float4 f4PX		= tex2D(t2dSrc, min(iVoxelX + 1, iVolumeWidth - 1),	iVoxelY);
		float4 f4NX		= tex2D(t2dSrc, max(iVoxelX - 1, 0),				iVoxelY);
		float4 f4PY		= tex2D(t2dSrc, iVoxelX,		min(iVoxelY + 1, iVolumeHeight - 1));
		float4 f4NY		= tex2D(t2dSrc, iVoxelX,		max(iVoxelY - 1, 0)				);

		float4 f4WeightOffset = tex3D(t3dWeightOffset, iVoxelX,	iVoxelY, 0);

		float4 f4Result;

		f4Result = make_float4(
			f4WeightOffset.w * f4Value.x + (f4PX.x + f4NX.x + f4PY.x + f4NY.x - 4.0f * f4Value.x) * fAttenuation + f4WeightOffset.x,
			f4WeightOffset.w * f4Value.y + (f4PX.y + f4NX.y + f4PY.y + f4NY.y - 4.0f * f4Value.y) * fAttenuation + f4WeightOffset.y,
			f4WeightOffset.w * f4Value.z + (f4PX.z + f4NX.z + f4PY.z + f4NY.z - 4.0f * f4Value.z) * fAttenuation + f4WeightOffset.z,
			0);

		if( iVoxelX < iVolumeWidth && iVoxelY < iVolumeHeight )
		{
			*ADDRESS_2D(
				float4, cDstPitchedPtr.ptr, 
				sizeof(float4), cDstPitchedPtr.pitch, 
				iVoxelX, iVoxelY) = f4Result;

			float4 f4Diff;
			f4Diff.x = f4Value.x - f4Result.x;
			f4Diff.y = f4Value.y - f4Result.y;
			f4Diff.z = f4Value.z - f4Result.z;
			float fDiff = f4Diff.x * f4Diff.x + f4Diff.y * f4Diff.y + f4Diff.z * f4Diff.z;
			*ADDRESS_2D(
				float, cErrorPitchedPtr.ptr, 
				sizeof(float), cErrorPitchedPtr.pitch, 
				iVoxelX, iVoxelY) = fDiff;
		}
	}
#else	// MOD-BY-LEETEN 2009/12/17-TO:
	#include "FlowDiffusion2D_kernel.cu"
#endif	// MOD-BY-LEETEN 2009/12/17-END

// ADD-BY-LEETEN 12/07/2009-END

#if	0		// DEL-BY-LEETEN 2009/12/17-BEGIN
	// ADD-BY-LEETEN 12/14/2009-BEGIN
	__global__ 
	static 
	void 
	// MOD-BY-LEETEN 12/07/2009-FROM:
		// _FlowFusion_kernel
	// TO:
	_Vector3DToVolume_kernel
	// MOD-BY-LEETEN 12/07/2009-END
	(
		int iVolumeWidth,
		int iVolumeHeight,
		int iVolumeDepth,

		int iNrOfYBlocks,
		int iBlockZSize,

		// texture<float4, 2, hipReadModeElementType> t2dSrc,
		hipPitchedPtr cBinVolumePtr_global
	)
	{
 		int iVoxelX = blockIdx.x * blockDim.x + threadIdx.x;
		int iVoxelY = (blockIdx.y % iNrOfYBlocks) * blockDim.y + threadIdx.y;
		int iBeginZ = (blockIdx.y / iNrOfYBlocks) * iBlockZSize;
		int iEndZ = min(iBeginZ + iBlockZSize, iVolumeDepth);

		for(int z = iBeginZ; z < iEndZ; z++)
		{
			float4 f4Vector = tex2D(t2dVectorVolume, iVoxelX,		iVoxelY + z				* iVolumeHeight);;
			
			float fLength	= sqrt(f4Vector.x * f4Vector.x + f4Vector.y * f4Vector.y + f4Vector.z * f4Vector.z);
			float fTheta	= 0.0f;
			float fPhi		= 0.0f;
			int iBin = 0;

			if( 0 < fLength )
			{
				f4Vector.x /= fLength;
				f4Vector.y /= fLength;
				f4Vector.z /= fLength;
				fTheta = ( 0.0f == f4Vector.x && 0.0f == f4Vector.y )?0.0f:(M_PI+(atan2(f4Vector.y, f4Vector.x)));
				fTheta /= 2.0f * M_PI;

				float fLength2D = sqrt(f4Vector.x * f4Vector.x + f4Vector.y * f4Vector.y);
				fPhi = ((0.0f == fLength2D)&&(0.0f == f4Vector.z))?0.0f:fabs(M_PI/2.0f-(atan2(f4Vector.z, fLength2D)));
				fPhi /= M_PI;
				iBin = tex2D(t2dAngleMap, fPhi, fTheta);
			}

			if( iVoxelX < iVolumeWidth && iVoxelY < iVolumeHeight )
			{
				*ADDRESS_2D(
					int,			cBinVolumePtr_global.ptr, 
					sizeof(int),	cBinVolumePtr_global.pitch, 
					iVoxelX, iVoxelY + z * iVolumeHeight) = iBin;
			}
		}
	}
	// ADD-BY-LEETEN 12/14/2009-END
#endif	// DEL-BY-LEETEN 2009/12/17-END

// ADD-BY-LEETEN 12/17/2009-BEGIN
////////////////////////////////////////////

#include "EntropyField_kernel.cu"

__host__
static 
int 
IMirrorCoord_host(int iCoord, int iSize)
{
	int iMirroredCoord = iCoord;
	if( iCoord < 0 )
		iMirroredCoord = -iCoord;
	if( iCoord > iSize - 1 )
		iMirroredCoord = iSize - (iCoord - (iSize - 1));
	return iMirroredCoord;
}

static 
void 
_UpdateSliceToHistogram_host
(
	int iU,
	int iV,

	int3 i3Center, 
	int	iUpdateDir, 
	int iHisotgramOp, 

	int3 i3UDir, 
	int3 i3VDir,
	int iUKernelSize, 
	int iVKernelSize,

	int3 i3VolumeSize,
	int *piBinVolume_host,

	int iNrOfBins,
	int *piHistorgram_host
)
{
	int3 i3Point;
	i3Point.x = i3Center.x + (iU - iUKernelSize) * i3UDir.x + (iV - iVKernelSize) * i3VDir.x;
	i3Point.y = i3Center.y + (iU - iUKernelSize) * i3UDir.y + (iV - iVKernelSize) * i3VDir.y;
	i3Point.z = i3Center.z + (iU - iUKernelSize) * i3UDir.z + (iV - iVKernelSize) * i3VDir.z;

	// read the bin
	int3 i3TexCoord;
	i3TexCoord.x = IMirrorCoord_host(i3Point.x, i3VolumeSize.x);
	i3TexCoord.y = IMirrorCoord_host(i3Point.y, i3VolumeSize.y);
	i3TexCoord.z = IMirrorCoord_host(i3Point.z, i3VolumeSize.z);

	int iSrcBin = piBinVolume_host[i3TexCoord.x + i3TexCoord.y * i3VolumeSize.x + i3TexCoord.z * i3VolumeSize.x * i3VolumeSize.y];

	// update the histogram
	piHistorgram_host[iSrcBin] += iHisotgramOp; 
}

static 
void
_BuildHistogram_host
(
	int3 i3Center,
						// res. of the neighboring region
	int3 i3KernelSize,	
						// the bin volume
	int3 i3VolumeSize,	
	int *piBinVolume_host,								
						// res. of the joint histogram
	int iNrOfBins,		
	int *piHistorgram_host
)
{
	int3 i3Point = i3Center;
	i3Point.x -= i3KernelSize.x;
	for(int xi = 0; xi < 2 * i3KernelSize.x + 1; xi++, i3Point.x++)
		for(int		iV = 0; iV < 2 * i3KernelSize.z + 1; iV++)
			for(int iU = 0; iU < 2 * i3KernelSize.y + 1; iU++)
				_UpdateSliceToHistogram_host(
					iU,
					iV,

					i3Point , 
					UPDATE_DIR_X, 
					HISTOGRAM_OP_ADD_SLICE, 

					make_int3(0, 1, 0), 
					make_int3(0, 0, 1),
					i3KernelSize.y, 
					i3KernelSize.z,

					i3VolumeSize,
					piBinVolume_host,

					iNrOfBins,
					piHistorgram_host
				);
}

static 
void 
_UpdateSliceToHistogram_host
(
	int3 i3Point,										// the coordinate to be computed

	int iUpdateDir, 
	int iDir,

	int3 i3KernelSize,	// res. of the neighboring region

	int3 i3VolumeSize,	// res. of the volume
	int *piBinVolume_host,								


	int iNrOfBins,		// res. of the joint histogram
	int *piHistogram_host
)
{
	int3 i3Prev = i3Point;
	int3 i3Next = i3Point;
	int3 i3UDir;
	int3 i3VDir;
	int iUKernelSize ;
	int iVKernelSize ;

						// according the direction to divide the new slice into blocks
	switch(iUpdateDir)
	{
	case UPDATE_DIR_X:	
		i3UDir = make_int3(0, 1, 0);
		i3VDir = make_int3(0, 0, 1);
		iUKernelSize = i3KernelSize.y;
		iVKernelSize = i3KernelSize.z;
		i3Prev.x -= iDir * i3KernelSize.x;
		i3Next.x += iDir * i3KernelSize.x;
		break;
	case UPDATE_DIR_Y:	
		i3UDir = make_int3(1, 0, 0);
		i3VDir = make_int3(0, 0, 1);
		iUKernelSize = i3KernelSize.x;
		iVKernelSize = i3KernelSize.z;
		i3Prev.y -= iDir * i3KernelSize.y;
		i3Next.y += iDir * i3KernelSize.y;
		break;
	case UPDATE_DIR_Z:	
		i3UDir = make_int3(1, 0, 0);
		i3VDir = make_int3(0, 1, 0);
		iUKernelSize = i3KernelSize.y;
		iVKernelSize = i3KernelSize.z;
		i3Prev.z -= iDir * i3KernelSize.z;
		i3Next.z += iDir * i3KernelSize.z;
		break;
	} // switch

	for(int		iV = 0; iV < 2 * iVKernelSize + 1; iV++)
		for(int iU = 0; iU < 2 * iUKernelSize + 1; iU++)
		{
			_UpdateSliceToHistogram_host
			(
				iU,
				iV,

				i3Next, 
				iUpdateDir, 
				HISTOGRAM_OP_ADD_SLICE, 

				i3UDir, 
				i3VDir,
				iUKernelSize, 
				iVKernelSize,

				i3VolumeSize,
				piBinVolume_host,

				iNrOfBins,
				piHistogram_host
			);

			_UpdateSliceToHistogram_host
			(
				iU,
				iV,

				i3Prev, 
				iUpdateDir, 
				HISTOGRAM_OP_SUB_SLICE, 

				i3UDir, 
				i3VDir,
				iUKernelSize, 
				iVKernelSize,

				i3VolumeSize,
				piBinVolume_host,

				iNrOfBins,
				piHistogram_host
			);
		}
}

static 
void
_ComputeEntropy_host
(
	int3 i3Point,
	int3 i3KernelSize,

						// the histogram
	int iNrOfBins,
	int *piHistogram_host,

	int3 i3VolumeSize,
	hipPitchedPtr cEntropyVolume_pitched
)
{
	float fEntropy = 0.0f;
	for(int b = 0; b < iNrOfBins; b++)
	{
		if( 0 == piHistogram_host[b] )
			continue;

		float fProb = float(piHistogram_host[b]) / float(i3KernelSize.x * i3KernelSize.y * i3KernelSize.z);
		fEntropy += fProb * log2f(fProb);
	}

	CUDA_SAFE_CALL(
		hipMemcpy(
			ADDRESS_2D(
				float, cEntropyVolume_pitched.ptr, 
				sizeof(float), cEntropyVolume_pitched.pitch, 
				i3Point.x, i3Point.y + i3Point.z * i3VolumeSize.y),
			&fEntropy,
			sizeof(fEntropy),
			hipMemcpyHostToDevice) );
}

void 
_ComputeEntropyVolume
(
						// res. of the neighboring region
	int3 i3KernelSize,

						// the histogram
	int iNrOfBins,
	int *piHistogram_global,
	float *pfLogHistogram_global,

    CUDPPHandle cScanPlanSum,

						// res. of the volume																
	int3 i3VolumeSize,	
						// bin volume																	
	hipPitchedPtr		cBinVolume_pitched,								
	hipPitchedPtr		cEntropyVolume_pitched
)
{
CLOCK_INIT(_ComputeEntropyVolume_PRINT_TIMING, __FUNCTION__ ": ");

CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_TIMING);

	int *piHistogram_host;
	piHistogram_host = (int*)calloc(sizeof(piHistogram_host[0]), iNrOfBins);
	assert(piHistogram_host);

	int *piBinVolume_host;
	piBinVolume_host = (int*)calloc(sizeof(piBinVolume_host[0]), i3VolumeSize.x * i3VolumeSize.y * i3VolumeSize.z);
	assert(piBinVolume_host);

	_GetSrcBinVolume(piBinVolume_host);

CLOCK_END(_ComputeEntropyVolume_PRINT_TIMING, false);

CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_TIMING);

	int xdir = +1;
	int ydir = +1;
	int zdir = +1;
	int x = 0;
	int y = 0;
	int z = 0;

	_BuildHistogram_host
	(
		make_int3(x, y, z),
							// res. of the neighboring region
		i3KernelSize,	
							// the bin volume
		i3VolumeSize,	
		piBinVolume_host,
							// the joint histogram
		iNrOfBins,		
		piHistogram_host
	);
CLOCK_END(_ComputeEntropyVolume_PRINT_TIMING, false);

CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_TIMING);

	for(int				zi = 0; zi < i3VolumeSize.z;	zi++, z += zdir)
	{
		for(int			yi = 0; yi < i3VolumeSize.y;	yi++, y += ydir)
		{
			for(int		xi = 0; xi < i3VolumeSize.x;	xi++, x += xdir)
			{
				// fprintf(stderr, "%d, %d, %d\n", x, y, z);
				// (x,y,z): center of the volume
				// before enter this loop, the histogram in the region centering at (x, y, z) should have been available
				_ComputeEntropy_host
				(
					make_int3(x, y, z),
					i3KernelSize,

										// the joint histogram
					iNrOfBins,
					piHistogram_host,

					i3VolumeSize,
					cEntropyVolume_pitched
				);

				if( xi < i3VolumeSize.x - 1 )
					_UpdateSliceToHistogram_host(
						make_int3(x + xdir, y, z), 
						UPDATE_DIR_X,	
						xdir, 

						i3KernelSize, 

											// the bin volume
						i3VolumeSize,	
						piBinVolume_host,
											// the joint histogram
						iNrOfBins,		
						piHistogram_host
						);
			}
			x -= xdir;
			xdir *= -1;
			if( yi < i3VolumeSize.y - 1 )
				_UpdateSliceToHistogram_host(
					make_int3(x, y + ydir, z), 
					UPDATE_DIR_Y,	
					ydir, 
					
					i3KernelSize, 

										// the bin volume
					i3VolumeSize,	
					piBinVolume_host,
										// the joint histogram
					iNrOfBins,		
					piHistogram_host
					);
		}
		y -= ydir;
		ydir *= -1;
		if( zi < i3VolumeSize.z - 1 )
			_UpdateSliceToHistogram_host(
				make_int3(x, y, z + zdir), 
				UPDATE_DIR_Z,	
				zdir, 

				i3KernelSize, 

											// the bin volume
				i3VolumeSize,	
				piBinVolume_host,
									// the joint histogram
				iNrOfBins,		
				piHistogram_host
			);
	}
CLOCK_END(_ComputeEntropyVolume_PRINT_TIMING, false);

CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_TIMING);
	free(piHistogram_host);
	free(piBinVolume_host);
CLOCK_END(_ComputeEntropyVolume_PRINT_TIMING, false);

CLOCK_PRINT(_ComputeEntropyVolume_PRINT_TIMING);
}

CUDPPHandle 
CInitEntropyField
(
	int iNrOfBins
)
{
	CUDPPHandle cScanPlan = 0;

	CUDPPConfiguration	cConfig;
	cConfig.op =		CUDPP_ADD;
	cConfig.datatype =	CUDPP_FLOAT;
	cConfig.algorithm =	CUDPP_SCAN;
	cConfig.options =	CUDPP_OPTION_BACKWARD | CUDPP_OPTION_INCLUSIVE;

	assert( CUDPP_SUCCESS  == cudppPlan(&cScanPlan,	cConfig, iNrOfBins, 1, 0) );

	return cScanPlan;
}

void
_GetSrcEntropyVolume
(
	int iNrOfBins,
	int iKernelWidth, int iKernelHeight, int iKernelDepth
)
{
	int *piHistogram_global;
	CUDA_SAFE_CALL_NO_SYNC(
		hipMalloc(	(void**)&piHistogram_global, sizeof(piHistogram_global[0]) * iNrOfBins) );
	CUDA_SAFE_CALL_NO_SYNC(
		hipMemset(	(void*)piHistogram_global, 0, sizeof(piHistogram_global[0]) * iNrOfBins) );

	float *pfLogHistogram_global;
	CUDA_SAFE_CALL_NO_SYNC(
		hipMalloc(	(void**)&pfLogHistogram_global, sizeof(pfLogHistogram_global[0]) * iNrOfBins) );
	CUDA_SAFE_CALL_NO_SYNC(
		hipMemset(	(void*)pfLogHistogram_global, 0, sizeof(pfLogHistogram_global[0]) * iNrOfBins) );

	hipExtent cEntropyVolumeExtent	= make_hipExtent(
		cVolumeExtent_array.width * sizeof(float),
		cVolumeExtent_array.height,
		cVolumeExtent_array.depth);
	hipPitchedPtr cEntropyVolume_pitched;
	CUDA_SAFE_CALL( 
		hipMalloc3D(&cEntropyVolume_pitched, cEntropyVolumeExtent) );

	CUDPPHandle hScanPlanEntropy = CInitEntropyField(iNrOfBins);

	_ComputeEntropyVolume
	(
						// res. of the neighboring region
		make_int3(iKernelWidth, iKernelHeight, iKernelDepth),

						// the histogram
		iNrOfBins,
		piHistogram_global,
		pfLogHistogram_global,

		hScanPlanEntropy,
						// res. of the volume																
		make_int3(cVolumeExtent_array.width, cVolumeExtent_array.height, cVolumeExtent_array.depth),

						// bin volume																	
		cSrcBinVolume_pitched,
		cEntropyVolume_pitched
	);

	FREE_MEMORY(piHistogram_global);
	FREE_MEMORY(pfLogHistogram_global);
	FREE_MEMORY(cEntropyVolume_pitched.ptr);
	cudppDestroyPlan(hScanPlanEntropy);
}

void
_GetJointEntropyVolume
(
	int iNrOfSrcBins,
	int iNrOfDstBins,
	int iKernelWidth, int iKernelHeight, int iKernelDepth
)
{
	int iNrOfBins = iNrOfSrcBins * iNrOfDstBins;

	int *piHistogram_global;
	CUDA_SAFE_CALL_NO_SYNC(
		hipMalloc(	(void**)&piHistogram_global, sizeof(piHistogram_global[0]) * iNrOfBins) );
	CUDA_SAFE_CALL_NO_SYNC(
		hipMemset(	(void*)piHistogram_global, 0, sizeof(piHistogram_global[0]) * iNrOfBins) );

	float *pfLogHistogram_global;
	CUDA_SAFE_CALL_NO_SYNC(
		hipMalloc(	(void**)&pfLogHistogram_global, sizeof(pfLogHistogram_global[0]) * iNrOfBins) );
	CUDA_SAFE_CALL_NO_SYNC(
		hipMemset(	(void*)pfLogHistogram_global, 0, sizeof(pfLogHistogram_global[0]) * iNrOfBins) );

	hipExtent cEntropyVolumeExtent	= make_hipExtent(
		cVolumeExtent_array.width * sizeof(float),
		cVolumeExtent_array.height,
		cVolumeExtent_array.depth);
	hipPitchedPtr cEntropyVolume_pitched;
	CUDA_SAFE_CALL( 
		hipMalloc3D(&cEntropyVolume_pitched, cEntropyVolumeExtent) );

	CUDPPHandle hScanPlanEntropy = CInitEntropyField(iNrOfBins);

	// bind the bin volume as a 2D texture
	t2dSrcBinVolume.addressMode[0] = hipAddressModeClamp;
	t2dSrcBinVolume.addressMode[1] = hipAddressModeClamp;
	t2dSrcBinVolume.filterMode =	hipFilterModePoint;
	t2dSrcBinVolume.normalized =	false;
	CUDA_SAFE_CALL_NO_SYNC(
		hipBindTexture2D(
			0, 
			t2dSrcBinVolume, 
			cSrcBinVolume_pitched.ptr, 
			hipCreateChannelDesc<int>(),
			cVolumeExtent_array.width, 
			cVolumeExtent_array.height * cVolumeExtent_array.depth, 
			cSrcBinVolume_pitched.pitch) );

	t2dDstBinVolume.addressMode[0] = hipAddressModeClamp;
	t2dDstBinVolume.addressMode[1] = hipAddressModeClamp;
	t2dDstBinVolume.filterMode =	hipFilterModePoint;
	t2dDstBinVolume.normalized =	false;
	CUDA_SAFE_CALL_NO_SYNC(
		hipBindTexture2D(
			0, 
			t2dDstBinVolume, 
			cDstBinVolume_pitched.ptr, 
			hipCreateChannelDesc<int>(),
			cVolumeExtent_array.width, 
			cVolumeExtent_array.height * cVolumeExtent_array.depth, 
			cDstBinVolume_pitched.pitch) );

	// bind the histogram as a 1D texture
	// ...
	hipPitchedPtr cJointBinVolume_pitched;
	CUDA_SAFE_CALL( 
		hipMalloc3D(
			&cJointBinVolume_pitched, 
			make_hipExtent(
				cVolumeExtent_array.width * sizeof(int),
				cVolumeExtent_array.height,
				cVolumeExtent_array.depth) ) );

	dim3 v3Blk, v3Grid;
	v3Blk = dim3(BLOCK_DIM_X, BLOCK_DIM_Y);
	v3Grid = dim3(
		size_t(ceilf(float(cVolumeExtent_array.width) / float(v3Blk.x))),
		size_t(ceilf(float(cVolumeExtent_array.height * cVolumeExtent_array.depth) / float(v3Blk.y)))
	);

	_JointSrcDst_kernel<<<v3Grid, v3Blk, 0>>>
	(
		iNrOfSrcBins, 
		iNrOfDstBins,

		make_int3(
			cVolumeExtent_array.width, 
			cVolumeExtent_array.height, 
			cVolumeExtent_array.depth),

		cJointBinVolume_pitched
	);
	CUT_CHECK_ERROR("_JointSrcDst_kernel() failed");

	_ComputeEntropyVolume
	(
						// res. of the neighboring region
		make_int3(iKernelWidth, iKernelHeight, iKernelDepth),

						// the histogram
		iNrOfBins,
		piHistogram_global,
		pfLogHistogram_global,

		hScanPlanEntropy,
						// res. of the volume																
		make_int3(cVolumeExtent_array.width, cVolumeExtent_array.height, cVolumeExtent_array.depth),

						// bin volume																	
		cJointBinVolume_pitched,
		cEntropyVolume_pitched
	);

	FREE_MEMORY(cJointBinVolume_pitched.ptr);
	FREE_MEMORY(piHistogram_global);
	FREE_MEMORY(pfLogHistogram_global);
	FREE_MEMORY(cEntropyVolume_pitched.ptr);
	cudppDestroyPlan(hScanPlanEntropy);
}

// ADD-BY-LEETEN 12/17/2009-END

////////////////////////////////////////////
#if	0	// MOD-BY-LEETEN 12/16/2009-FROM:
	__global__ 
	static 
	void 
	// MOD-BY-LEETEN 12/07/2009-FROM:
		// _FlowFusion_kernel
	// TO:
	_FlowDiffusion3D_kernel
	// MOD-BY-LEETEN 12/07/2009-END
	(
		// INPUT
		float fAttenuation,

		int iVolumeWidth,
		int iVolumeHeight,
		int iVolumeDepth,
		int iZ,

		// ADD-BY-LEETEN 11/04/2009-BEGIN
		#if				!DIFFUSION_BY_FOR_LOOP_ON_HOST
		// ADD-BY-LEETEN 11/04/2009-END

		int iNrOfYBlocks,
		int iBlockZSize,

		// ADD-BY-LEETEN 11/04/2009-BEGIN
		#endif // #if	!DIFFUSION_BY_FOR_LOOP_ON_HOST
		// ADD-BY-LEETEN 11/04/2009-END

		// output
		hipPitchedPtr cDstPitchedPtr,

		// ADD-BY-LEETEN 2009/11/25-BEGIN
		hipPitchedPtr cErrorPitchedPtr
		// ADD-BY-LEETEN 2009/11/25-END
	)
	{
 		int iVoxelX = blockIdx.x * blockDim.x + threadIdx.x;
		// ADD-BY-LEETEN 11/04/2009-BEGIN
		#if				DIFFUSION_BY_FOR_LOOP_ON_HOST
		// ADD-BY-LEETEN 11/04/2009-BEGIN

		int iVoxelY = blockIdx.y * blockDim.y + threadIdx.y;

		// ADD-BY-LEETEN 11/04/2009-BEGIN
		#else	// #if	DIFFUSION_BY_FOR_LOOP_ON_HOST
		int iVoxelY = (blockIdx.y % iNrOfYBlocks) * blockDim.y + threadIdx.y;
		int iBeginZ = (blockIdx.y / iNrOfYBlocks) * iBlockZSize;
		int iEndZ = min(iBeginZ + iBlockZSize, iVolumeDepth);
		#endif	// #if	DIFFUSION_BY_FOR_LOOP_ON_HOST
		// ADD-BY-LEETEN 11/04/2009-END

		#if	DIFFUSION_BY_FOR_LOOP_ON_HOST

			// compute the central differnece
			float4 f4Value	= tex2D(t2dSrc, iVoxelX,		iVoxelY + iZ * iVolumeHeight);
			float4 f4PX		= tex2D(t2dSrc, iVoxelX + 1,	iVoxelY + iZ * iVolumeHeight);
			float4 f4NX		= tex2D(t2dSrc, iVoxelX - 1,	iVoxelY + iZ * iVolumeHeight);
			float4 f4PY		= tex2D(t2dSrc, iVoxelX,		min(iVoxelY + 1, iVolumeHeight - 1) + iZ * iVolumeHeight);
			float4 f4NY		= tex2D(t2dSrc, iVoxelX,		max(iVoxelY - 1, 0)					+ iZ * iVolumeHeight);
			float4 f4PZ		= tex2D(t2dSrc, iVoxelX,		iVoxelY + min(iZ + 1, iVolumeDepth - 1) * iVolumeHeight);
			float4 f4NZ		= tex2D(t2dSrc, iVoxelX,		iVoxelY + max(iZ - 1, 0)				* iVolumeHeight);

		#if	0	// MOD-BY-LEETEN 11/04/2009-FROM:
		#else	// MOD-BY-LEETEN 11/04/2009-TO:
			float4 f4WeightOffset = tex3D(t3dWeightOffset, iVoxelX,	iVoxelY, iZ);

			#if	0	// MOD-BY-LEETEN 2009/11/25-FROM:
			#else	// MOD-BY-LEETEN 2009/11/25-TO:
			float4 f4Result;

			#if	0	// DEL-BY-LEETEN 12/07/2009-BEGIN
			#endif	// DEL-BY-LEETEN 12/07/2009-END

			#if	0	// MOD-BY-LEETEN 12/07/2009-FROM:
				f4Result = make_float4(
					f4WeightOffset.w * f4Value.x + (f4PX.x + f4NX.x + f4PY.x + f4NY.x + f4PZ.x + f4NZ.x - 6.0f * f4Value.x) * fAttenuationDividedBy6 + f4WeightOffset.x,
					f4WeightOffset.w * f4Value.y + (f4PX.y + f4NX.y + f4PY.y + f4NY.y + f4PZ.y + f4NZ.y - 6.0f * f4Value.y) * fAttenuationDividedBy6 + f4WeightOffset.y,
					f4WeightOffset.w * f4Value.z + (f4PX.z + f4NX.z + f4PY.z + f4NY.z + f4PZ.z + f4NZ.z - 6.0f * f4Value.z) * fAttenuationDividedBy6 + f4WeightOffset.z,
					0);
			#else	// MOD-BY-LEETEN 12/07/2009-TO:
			f4Result = make_float4(
				f4WeightOffset.w * f4Value.x + (f4PX.x + f4NX.x + f4PY.x + f4NY.x + f4PZ.x + f4NZ.x - 6.0f * f4Value.x) * fAttenuation + f4WeightOffset.x,
				f4WeightOffset.w * f4Value.y + (f4PX.y + f4NX.y + f4PY.y + f4NY.y + f4PZ.y + f4NZ.y - 6.0f * f4Value.y) * fAttenuation + f4WeightOffset.y,
				f4WeightOffset.w * f4Value.z + (f4PX.z + f4NX.z + f4PY.z + f4NY.z + f4PZ.z + f4NZ.z - 6.0f * f4Value.z) * fAttenuation + f4WeightOffset.z,
				0);
			#endif	// MOD-BY-LEETEN 12/07/2009-END

			if( iVoxelX < iVolumeWidth && iVoxelY < iVolumeHeight )
			{
				*ADDRESS_2D(
					float4, cDstPitchedPtr.ptr, 
					sizeof(float4), cDstPitchedPtr.pitch, 
					iVoxelX, iVoxelY + iZ * iVolumeHeight) = f4Result;

				float4 f4Diff;
				f4Diff.x = f4Value.x - f4Result.x;
				f4Diff.y = f4Value.y - f4Result.y;
				f4Diff.z = f4Value.z - f4Result.z;
				float fDiff = f4Diff.x * f4Diff.x + f4Diff.y * f4Diff.y + f4Diff.z * f4Diff.z;
				*ADDRESS_2D(
					float, cErrorPitchedPtr.ptr, 
					sizeof(float), cErrorPitchedPtr.pitch, 
					iVoxelX, iVoxelY + iZ * iVolumeHeight) = fDiff;
			}
		#endif	// MOD-BY-LEETEN 2009/11/25-END

		#endif	// MOD-BY-LEETEN 11/04/2009-END

		// MOD-BY-LEETEN 11/04/2009-FROM:
			// #else	// MOD-BY-LEETEN 10/02/2009-TO:
		// TO:
		#else	// #if	DIFFUSION_BY_FOR_LOOP_ON_HOST
		// MOD-BY-LEETEN 11/04/2009-END

		float4 f4Value;
		float4 f4PX;
		float4 f4NX;
		float4 f4PY;
		float4 f4NY;
		float4 f4PZ;
		float4 f4NZ;

		#if	0	// MOD-BY-LEETEN 11/04/2009-FROM:
		#else	// MOD-BY-LEETEN 11/04/2009-TO:

		float4 *pf4DstPtr = ADDRESS_2D(
								float4, cDstPitchedPtr.ptr, 
								sizeof(float4), cDstPitchedPtr.pitch, 
								iVoxelX, iVoxelY + iBeginZ * iVolumeHeight);

		// ADD-BY-LEETEN 2009/11/25-BEGIN
		float *pfErrorPtr = ADDRESS_2D(
								float, cErrorPitchedPtr.ptr, 
								sizeof(float), cErrorPitchedPtr.pitch, 
								iVoxelX, iVoxelY + iBeginZ * iVolumeHeight);
		// ADD-BY-LEETEN 2009/11/25-END

		#endif	// MOD-BY-LEETEN 11/04/2009-END

			#if	0	// MOD-BY-LEETEN 11/04/2009-FROM:
			#else	// MOD-BY-LEETEN 11/04/2009-TO:
			f4Value		= tex2D(t2dSrc, iVoxelX,		iVoxelY + iBeginZ				* iVolumeHeight);
			f4NZ		= tex2D(t2dSrc, iVoxelX,		iVoxelY + max(iBeginZ - 1, 0)	* iVolumeHeight);
			#endif	// MOD-BY-LEETEN 11/04/2009-END

			for(int z = iBeginZ; z < iEndZ; z++, f4NZ = f4Value, f4Value = f4PZ)
			{
				// ADD-BY-LEETEN 10/02/2009-BEGIN
				float4 f4WeightOffset = tex3D(t3dWeightOffset, iVoxelX,	iVoxelY, z);
				// ADD-BY-LEETEN 10/02/2009-END
				// f4Value		= tex2D(t2dSrc, iVoxelX,		iVoxelY + z * iVolumeHeight);
				#if	!USE_SHARED_MEMORY
				f4PX		= tex2D(t2dSrc, iVoxelX + 1,	iVoxelY + z * iVolumeHeight);
				f4NX		= tex2D(t2dSrc, iVoxelX - 1,	iVoxelY + z * iVolumeHeight);
				f4PY		= tex2D(t2dSrc, iVoxelX,		min(iVoxelY + 1, iVolumeHeight - 1) + z * iVolumeHeight);
				f4NY		= tex2D(t2dSrc, iVoxelX,		max(iVoxelY - 1, 0)					+ z * iVolumeHeight);

				#else	// #if	!USE_SHARED_MEMORY
				#endif	// #if	!USE_SHARED_MEMORY
				f4PZ		= tex2D(t2dSrc, iVoxelX,		iVoxelY + min(z + 1, iVolumeDepth - 1) * iVolumeHeight);
				// f4NZ		= tex2D(t2dSrc, iVoxelX,		iVoxelY + max(z - 1, 0)				* iVolumeHeight);

				// store the result back to the dst.
				#if	0	// MOD-BY-LEETEN 2009/11/10-FROM:
				#else	// MOD-BY-LEETEN 2009/11/10-TO:
				float4 f4Result;

				#if	0	// DEL-BY-LEETEN 12/07/2009-BEGIN
				#endif	// DEL-BY-LEETEN 12/07/2009-END	

					#if	0	// MOD-BY-LEETEN 12/07/2009-FROM:
						f4Result = make_float4(
							f4WeightOffset.w * f4Value.x + (f4PX.x + f4NX.x + f4PY.x + f4NY.x + f4PZ.x + f4NZ.x - 6.0f * f4Value.x) * fAttenuationDividedBy6 + f4WeightOffset.x,
							f4WeightOffset.w * f4Value.y + (f4PX.y + f4NX.y + f4PY.y + f4NY.y + f4PZ.y + f4NZ.y - 6.0f * f4Value.y) * fAttenuationDividedBy6 + f4WeightOffset.y,
							f4WeightOffset.w * f4Value.z + (f4PX.z + f4NX.z + f4PY.z + f4NY.z + f4PZ.z + f4NZ.z - 6.0f * f4Value.z) * fAttenuationDividedBy6 + f4WeightOffset.z,
							0);
					#else	// MOD-BY-LEETEN 12/07/2009-TO:
					f4Result = make_float4(	
						f4WeightOffset.w * f4Value.x + (f4PX.x + f4NX.x + f4PY.x + f4NY.x + f4PZ.x + f4NZ.x - 6.0f * f4Value.x) * fAttenuation + f4WeightOffset.x,
						f4WeightOffset.w * f4Value.y + (f4PX.y + f4NX.y + f4PY.y + f4NY.y + f4PZ.y + f4NZ.y - 6.0f * f4Value.y) * fAttenuation + f4WeightOffset.y,
						f4WeightOffset.w * f4Value.z + (f4PX.z + f4NX.z + f4PY.z + f4NY.z + f4PZ.z + f4NZ.z - 6.0f * f4Value.z) * fAttenuation + f4WeightOffset.z,
						0.0);
					#endif	// MOD-BY-LEETEN 12/07/2009-END
				#endif	// MOD-BY-LEETEN 2009/11/10-TO:

				// ADD-BY-LEETEN 11/04/2009-BEGIN
				if( iVoxelX < iVolumeWidth && iVoxelY < iVolumeHeight )
				// ADD-BY-LEETEN 11/04/2009-END
				{
					pf4DstPtr[0] = f4Result;

					// ADD-BY-LEETEN 2009/11/25-BEGIN
					float4 f4Diff;
					f4Diff.x = f4Value.x - f4Result.x;
					f4Diff.y = f4Value.y - f4Result.y;
					f4Diff.z = f4Value.z - f4Result.z;
					float fDiff;
					// float fDiff = f4Diff.x * f4Diff.x + f4Diff.y * f4Diff.y + f4Diff.z * f4Diff.z;
					// MOD-BY-LEETEN 12/16/2009-FROM:
						// fDiff = max(abs(f4Diff.x), max(abs(f4Diff.y), abs(f4Diff.z)));
					// TO:
					fDiff = f4Diff.x * f4Diff.x + f4Diff.y * f4Diff.y + f4Diff.z * f4Diff.z;
					// MOD-BY-LEETEN 12/16/2009-END
					pfErrorPtr[0] = fDiff;
					// ADD-BY-LEETEN 2009/11/25-END
				}

				pf4DstPtr += iVolumeHeight * cDstPitchedPtr.pitch / sizeof(float4);

				// ADD-BY-LEETEN 2009/11/25-BEGIN
				pfErrorPtr += iVolumeHeight * cErrorPitchedPtr.pitch / sizeof(float);
				// ADD-BY-LEETEN 2009/11/25-END
			}

		#endif	// MOD-BY-LEETEN 10/02/2009-END
	}
#else	// MOD-BY-LEETEN 12/16/2009-TO:
	#include "FlowDiffusion3D_kernel.cu"
#endif	// MOD-BY-LEETEN 12/16/2009-END

extern "C" {

// MOD-BY-LEETEN 12/07/2009-FROM:
	// void _FlowFusionFree()
// TO:
void 
_FlowDiffusionFree()
// MOD-BY-LEETEN 12/07/2009-END
{
	FREE_MEMORY_ON_HOST(pf4Volume_host);
	for(int i = 0; i < 2; i++)
		FREE_MEMORY(pcVolumePtrs_global[i].ptr);
	FREE_ARRAY(cWeightOffsetVolume_array);
	FREE_MEMORY_ON_HOST(pf4WeightOffsetVolume_host);
	// ADD-BY-LEETEN 2009/11/25-BEGIN
	FREE_MEMORY(cErrorVolume_device.ptr);
	FREE_MEMORY(cErrorSum_device.ptr);

	#if	CHECK_ERROR_CONVERGENCE_BY_CUDPP
	cudppDestroyPlan(cScanplan);
	#endif	// #if	CHECK_ERROR_CONVERGENCE_BY_CUDPP
	// ADD-BY-LEETEN 2009/11/25-END

	// ADD-BY-LEETEN 12/14/2009-BEGIN
	FREE_ARRAY(pcAngleMap_array);	

	FREE_MEMORY(cSrcBinVolume_pitched.ptr);
	FREE_MEMORY(cDstBinVolume_pitched.ptr);
	// ADD-BY-LEETEN 12/17/2009-BEGIN
	FREE_MEMORY(cSrcDstBinVolumePtr_global.ptr);
	// ADD-BY-LEETEN 12/17/2009-END

	#if	0	// MOD-BY-LEETEN 12/16/2009-FROM:
		FREE_MEMORY_ON_HOST(piSrcAngleBinVolume_host);
		FREE_MEMORY_ON_HOST(piDstAngleBinVolume_host);
	#else	// MOD-BY-LEETEN 12/16/2009-TO:
	FREE_MEMORY_ON_HOST(piBinVolume_host);
	#endif	// MOD-BY-LEETEN 12/16/2009-END
	// ADD-BY-LEETEN 12/14/2009-END
}

void
// MOD-BY-LEETEN 12/07/2009-FROM:
	// _FlowFusionInit(
// TO:
_FlowDiffusionInit(
// MOD-BY-LEETEN 12/07/2009-END
	int iVolumeWidth,
	int iVolumeHeight,
	int iVolumeDepth
)
{
	// MOD-BY-LEETEN 12/07/2009-FROM:
		// CLOCK_INIT(PRINT_FLOW_FUSION_TIMING, "_FlowFusionInit(): ");
	// TO:
	CLOCK_INIT(PRINT_FLOW_FUSION_TIMING, __FUNCTION__ ": ");
	// MOD-BY-LEETEN 12/07/2009-END

	// MOD-BY-LEETEN 12/14/2009-FROM:
		// int iNrOfVoxels = iVolumeWidth * iVolumeHeight * iVolumeDepth;
	// TO:
	iNrOfVoxels = iVolumeWidth * iVolumeHeight * iVolumeDepth;
	// MOD-BY-LEETEN 12/14/2009-END

	cVolumeExtent	= make_hipExtent(
		iVolumeWidth * sizeof(float4),
		iVolumeHeight,
		iVolumeDepth);
	// allocate a local copy in the CPU side
	CLOCK_BEGIN(PRINT_FLOW_FUSION_TIMING);
	CUDA_SAFE_CALL(
		hipHostMalloc(
			(void**)&pf4Volume_host,
			sizeof(pf4Volume_host[0]) * iNrOfVoxels) );
	CLOCK_END(PRINT_FLOW_FUSION_TIMING, true);

	// Allocate 2 volumes on the  GPU's global memory.
	// in each iteration, one of them will be used as the source
	// 3D texture, and the other will be served as the dst.
	CLOCK_BEGIN(PRINT_FLOW_FUSION_TIMING);

	for(int i = 0; i < 2; i++)
	{
		CUDA_SAFE_CALL( 
			hipMalloc3D(&pcVolumePtrs_global[i], cVolumeExtent) );
	}

	// ADD-BY-LEETEN 2009/11/25-BEGIN
	// ADD-BY-LEETEN 2009/12/17-BEGIN
	#if	CHECK_ERROR_CONVERGENCE_BY_CUDPP	
	// ADD-BY-LEETEN 2009/12/17-END
	cErrorVolumeExtent	= make_hipExtent(
		// MOD-BY-LEETEN 12/16/2009-FROM:
			// iVolumeWidth,
		// TO:
		iVolumeWidth * sizeof(float),
		// MOD-BY-LEETEN 12/16/2009-END
		iVolumeHeight,
		iVolumeDepth);
	CUDA_SAFE_CALL( 
		hipMalloc3D(&cErrorVolume_device,	cErrorVolumeExtent) );
	CUDA_SAFE_CALL( 
		hipMalloc3D(&cErrorSum_device,		cErrorVolumeExtent) );

	// DEL-BY-LEETEN 12/16/2009-BEGIN
	// #if	CHECK_ERROR_CONVERGENCE_BY_CUDPP
	// DEL-BY-LEETEN 12/16/2009-END
	cConfig.op = CUDPP_ADD;
	// cConfig.op = CUDPP_MAX;
	cConfig.datatype = CUDPP_FLOAT;
	cConfig.algorithm = CUDPP_SCAN;
	#if	0	// MOD-BY-LEETEN 12/16/2009-FROM:
		cConfig.options = CUDPP_OPTION_FORWARD | CUDPP_OPTION_INCLUSIVE;
		cudppPlan(&cScanplan, cConfig, iVolumeWidth * iVolumeHeight * iVolumeDepth, iVolumeHeight * iVolumeDepth, cErrorVolume_device.pitch);  
	#else	// MOD-BY-LEETEN 12/16/2009-TO:
	cConfig.options = CUDPP_OPTION_BACKWARD | CUDPP_OPTION_INCLUSIVE;
	assert(CUDPP_SUCCESS  == cudppPlan(&cScanplan, cConfig, iVolumeWidth * iVolumeHeight * iVolumeDepth, iVolumeHeight * iVolumeDepth, cErrorVolume_device.pitch) );  
	#endif	// MOD-BY-LEETEN 12/16/2009-END
	// DEL-BY-LEETEN 2009/12/17-BEGIN
	// #endif	// #if	CHECK_ERROR_CONVERGENCE_BY_CUDPP
	// DEL-BY-LEETEN 2009/12/17-END
	// ADD-BY-LEETEN 2009/12/17-BEGIN
	#endif	// #if	CHECK_ERROR_CONVERGENCE_BY_CUDPP
	// ADD-BY-LEETEN 2009/12/17-END
	// ADD-BY-LEETEN 2009/11/25-END

	// ADD-BY-LEETEN 10/02/2009-BEGIN
	cVolumeExtent_array = make_hipExtent(
		iVolumeWidth,
		iVolumeHeight,
		iVolumeDepth);

	CUDA_SAFE_CALL( 
		hipMalloc3DArray(&cWeightOffsetVolume_array, &cWeightOffsetChannelDesc, cVolumeExtent_array) );

	CUDA_SAFE_CALL(
		hipHostMalloc(
			(void**)&pf4WeightOffsetVolume_host,
			sizeof(pf4WeightOffsetVolume_host[0]) * iNrOfVoxels) );
	// ADD-BY-LEETEN 10/02/2009-END

	CLOCK_END(PRINT_FLOW_FUSION_TIMING, true);

	CLOCK_PRINT(PRINT_FLOW_FUSION_TIMING);
}

// ADD-BY-LEETEN 12/14/2009-BEGIN
void 
_FlowDiffusionSetAngleMap(int *piAngleMap, int iNrOfPhis, int iNrOfThetas)
{
	// allocate an array on the GPU side

	// upload the angle map to the araray

	// setup the texture for the angle map
	/*
    t2dAngleMap.addressMode[0] =	hipAddressModeClamp;
    t2dAngleMap.addressMode[1] =	hipAddressModeClamp;
    t2dAngleMap.filterMode =		hipFilterModePoint;
    t2dAngleMap.normalized =		true;
	*/
	SETUP_ARRAY(pcAngleMap_array, iNrOfPhis, iNrOfThetas, 32, 0, 0, 0, hipChannelFormatKindSigned);	
	BIND_ARRAY_AS_TEXTURE(
		t2dAngleMap, pcAngleMap_array, 
		hipAddressModeClamp, hipAddressModeClamp, hipFilterModePoint, true);	

    CUDA_SAFE_CALL( 
		hipMemcpy2DToArray(
			pcAngleMap_array, 
			0, 
			0, 
			(void*)piAngleMap, 
			iNrOfPhis * sizeof(piAngleMap), 
			iNrOfPhis * sizeof(piAngleMap), 
			iNrOfThetas, 
			hipMemcpyHostToDevice) );

	// check whether the volume extent has been setup
	assert( NULL != pcVolumePtrs_global[0].ptr );

	// allocate the volume of bins in the global memorty on GPUs
	cBinVolumeExtent	= make_hipExtent(
		cVolumeExtent_array.width * sizeof(int),
		cVolumeExtent_array.height,
		cVolumeExtent_array.depth);
	CUDA_SAFE_CALL( 
		hipMalloc3D(&cSrcBinVolume_pitched, cBinVolumeExtent) );
	CUDA_SAFE_CALL( 
		hipMalloc3D(&cDstBinVolume_pitched, cBinVolumeExtent) );

	// ADD-BY-LEETEN 12/17/2009-BEGIN
	CUDA_SAFE_CALL( 
		hipMalloc3D(&cSrcDstBinVolumePtr_global, cBinVolumeExtent) );

	t2dSrcBinVolume.addressMode[0] = hipAddressModeClamp;
	t2dSrcBinVolume.addressMode[1] = hipAddressModeClamp;
	t2dSrcBinVolume.filterMode =	hipFilterModePoint;
	t2dSrcBinVolume.normalized =	false;

	t2dDstBinVolume.addressMode[0] = hipAddressModeClamp;
	t2dDstBinVolume.addressMode[1] = hipAddressModeClamp;
	t2dDstBinVolume.filterMode =	hipFilterModePoint;
	t2dDstBinVolume.normalized =	false;
	// ADD-BY-LEETEN 12/17/2009-END


	// allocate the volume of bins in the host side
	#if	0	// MOD-BY-LEETEN 2009/12/17-FROM:
		CUDA_SAFE_CALL(
			hipHostMalloc(
				(void**)&piSrcAngleBinVolume_host,
				sizeof(piSrcAngleBinVolume_host[0]) * iNrOfVoxels) );

		CUDA_SAFE_CALL(
			hipHostMalloc(
				(void**)&piDstAngleBinVolume_host,
				sizeof(piDstAngleBinVolume_host[0]) * iNrOfVoxels) );
	#else	// MOD-BY-LEETEN 2009/12/17-TO:
	CUDA_SAFE_CALL(
		hipHostMalloc(
			(void**)&piBinVolume_host,
			sizeof(piBinVolume_host[0]) * iNrOfVoxels) );
	#endif	// MOD-BY-LEETEN 2009/12/17-END

	// setup the src. texture for bin lookup
	t2dVectorVolume.addressMode[0] = hipAddressModeClamp;
	t2dVectorVolume.addressMode[1] = hipAddressModeClamp;
	t2dVectorVolume.filterMode =	hipFilterModePoint;
	t2dVectorVolume.normalized =	false;
}

void
_Vector3DToVolume(
	int iVolumeWidth,
	int iVolumeHeight,
	int iVolumeDepth,
	int iBlockZSize,
	hipPitchedPtr cVolumePtr_global,	// input
	hipPitchedPtr cBinVolumePtr_global	// output
)
{
	CUDA_SAFE_CALL_NO_SYNC(
		hipBindTexture2D(
			0, 
			t2dVectorVolume, 
			cVolumePtr_global.ptr, 
			hipCreateChannelDesc<float4>(),
			iVolumeWidth, 
			iVolumeHeight * iVolumeDepth, 
			cVolumePtr_global.pitch) );

	dim3 v3Blk = dim3(BLOCK_DIM_X, BLOCK_DIM_Y);
	dim3 v3Grid = dim3(
		(unsigned int)ceilf((float)iVolumeWidth	 / (float)v3Blk.x),
		unsigned int(ceilf( float(iVolumeHeight) / float(v3Blk.y) ) ) *
		unsigned int(ceilf( float(iVolumeDepth) /  float(iBlockZSize) ) ) );
	_Vector3DToVolume_kernel<<<v3Grid, v3Blk, 0>>>
	(
		iVolumeWidth,
		iVolumeHeight,
		iVolumeDepth,
		int(ceilf(float(iVolumeHeight) / float(BLOCK_DIM_Y))),
		iBlockZSize,
		cBinVolumePtr_global
	);
	CUT_CHECK_ERROR("_Vector3DToVolume_kernel() failed");
}

void
_ComputeSrcBinVolume
(
	int iVolumeWidth,
	int iVolumeHeight,
	int iVolumeDepth,
	int iBlockZSize,
	float *pfSrcVolume
)
{
	CLOCK_INIT(SHOW_COMPUTE_SRC_BIN_VOLUME_TIMING, __FUNCTION__ ": ");

	CLOCK_BEGIN(SHOW_COMPUTE_SRC_BIN_VOLUME_TIMING);
	for(int v = 0,		d = 0; d < iVolumeDepth;	d++)
		for(int			h = 0; h < iVolumeHeight;	h++)
			for(int		w = 0; w < iVolumeWidth;	w++, v++)
			{
				pf4Volume_host[v].x = pfSrcVolume[v * 3 + 0];
				pf4Volume_host[v].y = pfSrcVolume[v * 3 + 1];
				pf4Volume_host[v].z = pfSrcVolume[v * 3 + 2];
				pf4Volume_host[v].w = 0.0f;
			}
	CLOCK_END(SHOW_COMPUTE_SRC_BIN_VOLUME_TIMING, false);

	CLOCK_BEGIN(SHOW_COMPUTE_SRC_BIN_VOLUME_TIMING);
    hipMemcpy3DParms cCopyParamsHostToDevice = {0};

    cCopyParamsHostToDevice.srcPtr   = make_hipPitchedPtr(
		(void*)pf4Volume_host, 
		cVolumeExtent.width, 
		iVolumeWidth * sizeof(float4), 
		iVolumeHeight);
    cCopyParamsHostToDevice.dstPtr		= pcVolumePtrs_global[0];
    cCopyParamsHostToDevice.extent		= cVolumeExtent;
    cCopyParamsHostToDevice.kind		= hipMemcpyHostToDevice;
    CUDA_SAFE_CALL( hipMemcpy3D(&cCopyParamsHostToDevice) );  

	CLOCK_END(SHOW_COMPUTE_SRC_BIN_VOLUME_TIMING, false);

	CLOCK_BEGIN(SHOW_COMPUTE_SRC_BIN_VOLUME_TIMING);
	_Vector3DToVolume(
		iVolumeWidth,
		iVolumeHeight,
		iVolumeDepth,
		iBlockZSize,
		pcVolumePtrs_global[0],
		cSrcBinVolume_pitched);
	CLOCK_END(SHOW_COMPUTE_SRC_BIN_VOLUME_TIMING, false);

	CLOCK_PRINT(SHOW_COMPUTE_SRC_BIN_VOLUME_TIMING);
}

void 
_GetSrcBinVolume(int *piBinVolume)
{
	hipMemcpy3DParms cCopyParamsDeviceToHost = {0};
	cCopyParamsDeviceToHost.srcPtr		= cSrcBinVolume_pitched;
	cCopyParamsDeviceToHost.dstPtr		= make_hipPitchedPtr(
		// MOD-BY-LEETEN 2009/12/17-FROM:
			// (void*)piSrcAngleBinVolume_host, 
		// TO:
		(void*)piBinVolume_host, 
		// MOD-BY-LEETEN 2009/12/17-END
		cVolumeExtent_array.width * sizeof(int),	// cSrcBinVolume_pitched.width, 
		cVolumeExtent_array.width * sizeof(int), 
		cVolumeExtent_array.height);
	cCopyParamsDeviceToHost.extent		= cBinVolumeExtent;
	cCopyParamsDeviceToHost.kind		= hipMemcpyDeviceToHost;

	CUDA_SAFE_CALL( hipMemcpy3D(&cCopyParamsDeviceToHost) );  

	// MOD-BY-LEETEN 2009/12/17-FROM:
		// memcpy(piBinVolume, piSrcAngleBinVolume_host, sizeof(piBinVolume) * iNrOfVoxels);
	// TO:
	memcpy(piBinVolume, piBinVolume_host, sizeof(piBinVolume) * iNrOfVoxels);
	// MOD-BY-LEETEN 2009/12/17-END
}

void 
_GetDstBinVolume(int *piBinVolume)
{
	hipMemcpy3DParms cCopyParamsDeviceToHost = {0};
	cCopyParamsDeviceToHost.srcPtr		= cDstBinVolume_pitched;
	cCopyParamsDeviceToHost.dstPtr		= make_hipPitchedPtr(
		// MOD-BY-LEETEN 2009/12/17-FROM:
			// (void*)piDstAngleBinVolume_host,		
		// TO:
		(void*)piBinVolume_host, 
		// MOD-BY-LEETEN 2009/12/17-END

		cVolumeExtent_array.width * sizeof(int),	// cDstBinVolume_pitched.width, 
		cVolumeExtent_array.width * sizeof(int), 
		cVolumeExtent_array.height);
	cCopyParamsDeviceToHost.extent		= cBinVolumeExtent;
	cCopyParamsDeviceToHost.kind		= hipMemcpyDeviceToHost;

	CUDA_SAFE_CALL( hipMemcpy3D(&cCopyParamsDeviceToHost) );  

	// MOD-BY-LEETEN 2009/12/17-FROM:
		// memcpy(piBinVolume, piDstAngleBinVolume_host, sizeof(piBinVolume) * iNrOfVoxels);
	// TO:
	memcpy(piBinVolume, piBinVolume_host, sizeof(piBinVolume) * iNrOfVoxels);
	// MOD-BY-LEETEN 2009/12/17-END
}

// ADD-BY-LEETEN 12/14/2009-END

void
// MOD-BY-LEETEN 12/07/2009-FROM:
	// _FlowFusion(
// TO:
_FlowDiffusion(
// MOD-BY-LEETEN 12/07/2009-END
	float fAttenuation,
	int iNrOfIterations,
	int iVolumeWidth,
	int iVolumeHeight,
	int iVolumeDepth,
	// ADD-BY-LEETEN 10/02/2009-BEGIN
	int iBlockZSize,
	float *pfWeightVolume,
	// MOD-BY-LEETEN 2009/11/10-FROM:
		// float *pfOffsetVolume,
	// TO:
	float *pfXOffsetVolume,
	float *pfYOffsetVolume,
	float *pfZOffsetVolume,
	// MOD-BY-LEETEN 2009/11/10-END
	// ADD-BY-LEETEN 10/02/2009-END
	float *pfSrcVolume,
	float *pfDstVolume,
	// ADD-BY-LEETEN 2009/11/10-BEGIN
	int *piFlagVolume
	// ADD-BY-LEETEN 2009/11/10-END
)
{
	// ADD-BY-LEETEN 12/07/2009-BEGIN
	#if	CHECK_ERROR_CONVERGENCE_BY_CUDPP
	iNrOfIterations = 4 * max(max(iVolumeWidth, iVolumeHeight), iVolumeDepth);
	// LOG(printf("Warning! iNrOfIterations is changed to %d", iNrOfIterations));
	#endif
	// ADD-BY-LEETEN 12/07/2009-END
	

	// MOD-BY-LEETEN 12/07/2009-FROM:
		// CLOCK_INIT(PRINT_FLOW_FUSION_TIMING, "_FlowFusion(): ");
	// TO:
	// MOD-BY-LEETEN 12/14/2009-FROM:
		// CLOCK_INIT(PRINT_FLOW_FUSION_TIMING, __FUNCTION__);
	// TO:
	CLOCK_INIT(PRINT_FLOW_FUSION_TIMING, __FUNCTION__ ": ");
	// MOD-BY-LEETEN 12/07/2009-END

	// MOD-BY-LEETEN 10/02/2009-FROM:
		// dim3 v3Blk = dim3(16, 8);
	// TO:
	dim3 v3Blk = dim3(BLOCK_DIM_X, BLOCK_DIM_Y);
	// MOD-BY-LEETEN 10/02/2009-END
	dim3 v3Grid = dim3(
		(unsigned int)ceilf((float)iVolumeWidth	 / (float)v3Blk.x),
		// ADD-BY-LEETEN 11/04/2009-BEGIN
		#if					DIFFUSION_BY_FOR_LOOP_ON_HOST	
		// ADD-BY-LEETEN 11/04/2009-END

		(unsigned int)ceilf((float)iVolumeHeight / (float)v3Blk.y));

		// ADD-BY-LEETEN 11/04/2009-BEGIN
		#else	// #if		DIFFUSION_BY_FOR_LOOP_ON_HOST
		unsigned int(ceilf( float(iVolumeHeight) / float(v3Blk.y) ) ) *
		unsigned int(ceilf( float(iVolumeDepth) /  float(iBlockZSize) ) ) );
		#endif	// #if		DIFFUSION_BY_FOR_LOOP_ON_HOST
		// ADD-BY-LEETEN 11/04/2009-END

	// convert the #channels in the src volume from 3 to 4
	CLOCK_BEGIN(PRINT_FLOW_FUSION_TIMING);
	for(int v = 0,		d = 0; d < iVolumeDepth;	d++)
		for(int			h = 0; h < iVolumeHeight;	h++)
			for(int		w = 0; w < iVolumeWidth;	w++, v++)
			{
				pf4Volume_host[v].x = pfSrcVolume[v * 3 + 0];
				pf4Volume_host[v].y = pfSrcVolume[v * 3 + 1];
				pf4Volume_host[v].z = pfSrcVolume[v * 3 + 2];

				// ADD-BY-LEETEN 2009/11/10-BEGIN
				if( piFlagVolume )
					pf4Volume_host[v].w = float(piFlagVolume[v]);
				else
				// ADD-BY-LEETEN 2009/11/10-END
				pf4Volume_host[v].w = 0.0f;
			}

	// ADD-BY-LEETEN 10/02/2009-BEGIN
	for(int v = 0,		d = 0; d < iVolumeDepth;	d++)
		for(int			h = 0; h < iVolumeHeight;	h++)
			for(int		w = 0; w < iVolumeWidth;	w++, v++)
			{
				#if	0	// MOD-BY-LEETEN 2009/11/10-FROM:
				#else	// MOD-BY-LEETEN 2009/11/10-TO:
				// ADD-BY-LEETEN 12/07/2009-BEGIN
				if( pfXOffsetVolume )
				// ADD-BY-LEETEN 12/07/2009-END
				pf4WeightOffsetVolume_host[v].x = pfXOffsetVolume[v];
				// ADD-BY-LEETEN 12/07/2009-BEGIN
				if( pfYOffsetVolume )
				// ADD-BY-LEETEN 12/07/2009-END
				pf4WeightOffsetVolume_host[v].y = pfYOffsetVolume[v];
				// ADD-BY-LEETEN 12/07/2009-BEGIN
				if( pfZOffsetVolume )
				// ADD-BY-LEETEN 12/07/2009-END
				pf4WeightOffsetVolume_host[v].z = pfZOffsetVolume[v];
				pf4WeightOffsetVolume_host[v].w = 1.0f - pfWeightVolume[v];
				#endif	// MOD-BY-LEETEN 2009/11/10-END
			}
	// ADD-BY-LEETEN 10/02/2009-END

	CLOCK_END(PRINT_FLOW_FUSION_TIMING, true);

	// Upload the input volume to the first volume on the GPU
	CLOCK_BEGIN(PRINT_FLOW_FUSION_TIMING);

    hipMemcpy3DParms cCopyParamsHostToDevice = {0};

    cCopyParamsHostToDevice.srcPtr   = make_hipPitchedPtr(
		(void*)pf4Volume_host, 
		cVolumeExtent.width, 
		iVolumeWidth * sizeof(float4), 
		iVolumeHeight);
    cCopyParamsHostToDevice.dstPtr		= pcVolumePtrs_global[0];
    cCopyParamsHostToDevice.extent		= cVolumeExtent;
    cCopyParamsHostToDevice.kind		= hipMemcpyHostToDevice;
    CUDA_SAFE_CALL( hipMemcpy3D(&cCopyParamsHostToDevice) );  

	// ADD-BY-LEETEN 10/02/2009-BEGIN
    hipMemcpy3DParms cCopyParamsHostToArray = {0};
    cCopyParamsHostToArray.srcPtr   = make_hipPitchedPtr(
		(void*)pf4WeightOffsetVolume_host, 
		iVolumeWidth * sizeof(float4), 
		iVolumeWidth, 
		iVolumeHeight);
    cCopyParamsHostToArray.dstArray = cWeightOffsetVolume_array;
    cCopyParamsHostToArray.extent   = cVolumeExtent_array;
    cCopyParamsHostToArray.kind     = hipMemcpyHostToDevice;
    CUDA_SAFE_CALL( 
		hipMemcpy3D(&cCopyParamsHostToArray) );  

    t3dWeightOffset.addressMode[0] = hipAddressModeClamp;
    t3dWeightOffset.addressMode[1] = hipAddressModeClamp;
    t3dWeightOffset.addressMode[2] = hipAddressModeClamp;
    t3dWeightOffset.filterMode		= hipFilterModePoint;
    t3dWeightOffset.normalized		= false; 

    // bind array to 3D texture
    CUDA_SAFE_CALL(
		hipBindTextureToArray(t3dWeightOffset, cWeightOffsetVolume_array, cWeightOffsetChannelDesc));
	// ADD-BY-LEETEN 10/02/2009-END

	CLOCK_END(PRINT_FLOW_FUSION_TIMING, true);

	// calcuate the fusion operator on GPUs
	CLOCK_BEGIN(PRINT_FLOW_FUSION_TIMING);

    t2dSrc.addressMode[0] = hipAddressModeClamp;
    t2dSrc.addressMode[1] = hipAddressModeClamp;
    t2dSrc.filterMode =		hipFilterModePoint;
    t2dSrc.normalized =		false;

	int iSrc = 0;

	for(int i = 0; 
			i < iNrOfIterations; 
			i++,				iSrc = 1 - iSrc)
	{
		// bind the src. volume as the 3D texture

		hipChannelFormatDesc cChannelDesc =
			hipCreateChannelDesc<float4>();

		CUDA_SAFE_CALL_NO_SYNC(
			hipBindTexture2D(
				0, 
				t2dSrc, 
				pcVolumePtrs_global[iSrc].ptr, 
				cChannelDesc,
				iVolumeWidth, 
				iVolumeHeight * iVolumeDepth, 
				pcVolumePtrs_global[iSrc].pitch) );


		// ADD-BY-LEETEN 12/07/2009-BEGIN
		if( iVolumeDepth <= 2 )
			_FlowDiffusion2D_kernel<<<v3Grid, v3Blk, 0>>>
			(
				fAttenuation,
				iVolumeWidth,
				iVolumeHeight,
				pcVolumePtrs_global[1 - iSrc],
				cErrorVolume_device
			);	
		else
		{
		// ADD-BY-LEETEN 12/07/2009-END

		// launch the kernel to compute the diffusion operator for 1 iteration
		// MOD-BY-LEETEN 11/04/2009-FROM:
			// #if	0	// MOD-BY-LEETEN 10/02/2009-FROM:
		// TO:
		#if	DIFFUSION_BY_FOR_LOOP_ON_HOST
		// MOD-BY-LEETEN 11/04/2009-END

			for(int z = 0; z < iVolumeDepth; z++)
				// MOD-BY-LEETEN 12/07/2009-FROM:
					// _FlowFusion_kernel<<<v3Grid, v3Blk, 0>>>
				// TO:
				_FlowDiffusion3D_kernel<<<v3Grid, v3Blk, 0>>>
				// MOD-BY-LEETEN 12/07/2009-END
				(
					fAttenuation,
					iVolumeWidth,
					iVolumeHeight,
					iVolumeDepth,
					z,

					// output
					pcVolumePtrs_global[1 - iSrc],
					// ADD-BY-LEETEN 2009/11/25-BEGIN
					cErrorVolume_device
					// ADD-BY-LEETEN 2009/11/25-END
				);	
		// MOD-BY-LEETEN 11/04/2009-FROM:
			// #else	// MOD-BY-LEETEN 10/02/2009-TO:
		// TO:
		#else	// #if	DIFFUSION_BY_FOR_LOOP_ON_HOST
		// MOD-BY-LEETEN 11/04/2009-END

		// _FlowFusion_kernel<<<v3Grid, v3Blk, v3Blk.x * v3Blk.y * sizeof(float4)>>>
		// MOD-BY-LEETEN 12/07/2009-FROM:
			// _FlowFusion_kernel<<<v3Grid, v3Blk, 0>>>
		// TO:
		_FlowDiffusion3D_kernel<<<v3Grid, v3Blk, 0>>>
		// MOD-BY-LEETEN 12/07/2009-END
		(
			fAttenuation,
			iVolumeWidth,
			iVolumeHeight,
			iVolumeDepth,
			0,
			// ADD-BY-LEETEN 11/04/2009-BEGIN
			int(ceilf(float(iVolumeHeight) / float(BLOCK_DIM_Y))),
			iBlockZSize,
			// ADD-BY-LEETEN 11/04/2009-END
			// output
			pcVolumePtrs_global[1 - iSrc],
			// ADD-BY-LEETEN 2009/11/25-BEGIN
			cErrorVolume_device
			// ADD-BY-LEETEN 2009/11/25-END
		);
		#endif	// MOD-BY-LEETEN 10/02/2009-END
		// ADD-BY-LEETEN 12/07/2009-BEGIN
		}
		// ADD-BY-LEETEN 12/07/2009-END
		CUT_CHECK_ERROR("_FlowFusion_kernel() failed");

		// ADD-BY-LEETEN 2009/11/25-BEGIN
		#if	CHECK_ERROR_CONVERGENCE_BY_CUDPP
		cudppScan(cScanplan, cErrorSum_device.ptr, cErrorVolume_device.ptr, iVolumeWidth * iVolumeHeight * iVolumeDepth);

		float fError;
		#if	0	// MOD-BY-LEETEN 2009/12/17-FROM:
			CUDA_SAFE_CALL( 
				hipMemcpy(
					&fError, 
					ADDRESS_2D(
						float, cErrorSum_device.ptr, 
						sizeof(float), cErrorSum_device.pitch, 
						iVolumeWidth - 1, iVolumeHeight - 1 + (iVolumeDepth - 1 ) * iVolumeHeight),
					sizeof(float), 
					hipMemcpyDeviceToHost));
		#else	// MOD-BY-LEETEN 2009/12/17-TO:
	    CUDA_SAFE_CALL_NO_SYNC( 
			hipMemcpy(
				&fError, 
				cErrorSum_device.ptr, 
				sizeof(float), 
				hipMemcpyDeviceToHost));
		#endif	// MOD-BY-LEETEN 2009/12/17-END

		/*
		fError /= float(iVolumeWidth * iVolumeHeight * iVolumeDepth * 3);
		fError = sqrtf(fError);
		*/
		#if	0	// MOD-BY-LEETEN 2009/12/17-FROM:
			fprintf(stderr, "Error: %f\n", fError);
			if( fError < 1e-10 )
				break;
		#else	// MOD-BY-LEETEN 2009/12/17-TO:
		fError /= float(iVolumeWidth * iVolumeHeight * iVolumeDepth * 3);
		fError = sqrtf(fError);
		// fprintf(stderr, "Error: %f\n", fError);

		static float fPrevError = -1.0f;
		float fErrorDif = fabsf(fPrevError - fError);
		if( i > max(max(iVolumeWidth, iVolumeHeight), iVolumeDepth) && (fErrorDif/fPrevError < 0.01f || fErrorDif < 0.000001f) )
		{
			printf("\t#iters = %d;", i);
			break;
		}
		fPrevError = fError;
		#endif	// MOD-BY-LEETEN 2009/12/17-END

		#endif	// #if	CHECK_ERROR_CONVERGENCE_BY_CUDPP
		// ADD-BY-LEETEN 2009/11/25-END

	}

	// ADD-BY-LEETEN 12/14/2009-BEGIN
	if( iVolumeDepth > 2 )
	{
		_Vector3DToVolume(
			iVolumeWidth,
			iVolumeHeight,
			iVolumeDepth,
			iBlockZSize,
			pcVolumePtrs_global[iSrc],
			cDstBinVolume_pitched);
			// piDstAngleBinVolume_host);
	}
	// ADD-BY-LEETEN 12/14/2009-END

	CLOCK_END(PRINT_FLOW_FUSION_TIMING, true);

	int iDst = iSrc;

	// read the result from latest iteration back to the CPU side
	CLOCK_BEGIN(PRINT_FLOW_FUSION_TIMING);

	hipMemcpy3DParms cCopyParamsDeviceToHost = {0};
    cCopyParamsDeviceToHost.srcPtr		= pcVolumePtrs_global[iDst];
    cCopyParamsDeviceToHost.dstPtr		= make_hipPitchedPtr(
		(void*)pf4Volume_host, 
		cVolumeExtent.width, 
		iVolumeWidth * sizeof(float4), 
		iVolumeHeight);
    cCopyParamsDeviceToHost.extent		= cVolumeExtent;
    cCopyParamsDeviceToHost.kind		= hipMemcpyDeviceToHost;

	CUDA_SAFE_CALL( hipMemcpy3D(&cCopyParamsDeviceToHost) );  

	CLOCK_END(PRINT_FLOW_FUSION_TIMING, true);

	// convert the #channels in the dst volume from 4 to 3
	CLOCK_BEGIN(PRINT_FLOW_FUSION_TIMING);
	for(int v = 0,		d = 0; d < iVolumeDepth;	d++)
		for(int			h = 0; h < iVolumeHeight;	h++)
			for(int		w = 0; w < iVolumeWidth;	w++, v++)
			{
				pfDstVolume[v * 3 + 0] = pf4Volume_host[v].x;
				pfDstVolume[v * 3 + 1] = pf4Volume_host[v].y;
				pfDstVolume[v * 3 + 2] = pf4Volume_host[v].z;
			}
	CLOCK_END(PRINT_FLOW_FUSION_TIMING, true);

	CLOCK_PRINT(PRINT_FLOW_FUSION_TIMING);
}

} // extern "C"

/*

$Log: not supported by cvs2svn $
Revision 1.2  2009/12/15 20:05:57  leeten

[12/15/2009]
1. [ADD] Define new function _FlowDiffusionSetAngleMap() to setup the lookup table that converts a vector to a discrete bin.
2. [ADD] Define new function _GetSrcBinVolume() to download the bin of the orginal input vector field.
3. [ADD] Define new function _GetDstBinVolume() to download the bin for the diffused vector field.
4. [ADD] Define new function _ComputeSrcBinVolume() to download the bin of the orginal input vector field.
5. [ADD] Define a host function _Vector3DToVolume to convert each vector in a vector field into a bin.
6. [ADD] Define a kernel function _Vector3DToVolume_kernel() to convert each vector in a vector field into a bin on GPUs.

2. [ADD] Define new functions

Revision 1.1.1.1  2009/12/07 20:04:02  leeten

[12/07/2009]
1. [1ST] First time checkin.

Revision 1.3  2009/12/05 21:19:51  leeten

[12/05/2009]
1. [ADD] Add a 3D texture glWeightOffset to store the weights and offsets.
2. [ADD] Specify the flag donochange to the w channel in the 3D textures.
3. [ADD] Output the error to a 3D volume.

Revision 1.2  2009/11/04 19:11:13  leeten

[2009/11/04]
1. [ADD] Add a new preprocessor USE_SHARED_MEMORY to decide whether the shared memory is utilized.
2. [ADD] Add a new preprocessor DIFFUSION_BY_FOR_LOOP_ON_HOSTto decide whether the diffusion is executed via a for loop on the host side to scan throught all XY layers.
If it is not zero, the volume will be divided along the Z direction as well, and each block will be assigned as a single block on CUDA.
3. [ADD] Add 2 new preprocessors BLOCK_DIM_X and BLOCK_DIM_Y to control the block dimension.
4. [ADD] Add 1 array and 1 3D textrue for the weight/offset volume.

Revision 1.1.1.1  2009/11/02 15:30:56  leeten

[11/02/2009]
1 [1ST] Firs time checkin.


*/
