#include "hip/hip_runtime.h"
#include "FlowDiffusion_cuda.h"

__global__ 
void 
// MOD-BY-LEETEN 12/07/2009-FROM:
	// _FlowFusion_kernel
// TO:
_FlowDiffusion3D_kernel
// MOD-BY-LEETEN 12/07/2009-END
(
	// INPUT
	float fAttenuation,

	int iVolumeWidth,
	int iVolumeHeight,
	int iVolumeDepth,
	int iZ,

	// ADD-BY-LEETEN 11/04/2009-BEGIN
	#if				!DIFFUSION_BY_FOR_LOOP_ON_HOST
	// ADD-BY-LEETEN 11/04/2009-END

	int iNrOfYBlocks,
	int iBlockZSize,

	// ADD-BY-LEETEN 11/04/2009-BEGIN
	#endif // #if	!DIFFUSION_BY_FOR_LOOP_ON_HOST
	// ADD-BY-LEETEN 11/04/2009-END

	// output
	hipPitchedPtr cDstPitchedPtr,

	// ADD-BY-LEETEN 2009/11/25-BEGIN
	hipPitchedPtr cErrorPitchedPtr
	// ADD-BY-LEETEN 2009/11/25-END
)
{
	int iVoxelX = blockIdx.x * blockDim.x + threadIdx.x;
	// ADD-BY-LEETEN 11/04/2009-BEGIN
	#if				DIFFUSION_BY_FOR_LOOP_ON_HOST
	// ADD-BY-LEETEN 11/04/2009-BEGIN

	int iVoxelY = blockIdx.y * blockDim.y + threadIdx.y;

	// ADD-BY-LEETEN 11/04/2009-BEGIN
	#else	// #if	DIFFUSION_BY_FOR_LOOP_ON_HOST
	int iVoxelY = (blockIdx.y % iNrOfYBlocks) * blockDim.y + threadIdx.y;
	int iBeginZ = (blockIdx.y / iNrOfYBlocks) * iBlockZSize;
	int iEndZ = min(iBeginZ + iBlockZSize, iVolumeDepth);
	#endif	// #if	DIFFUSION_BY_FOR_LOOP_ON_HOST
	// ADD-BY-LEETEN 11/04/2009-END

	#if	DIFFUSION_BY_FOR_LOOP_ON_HOST

		// compute the central differnece
		float4 f4Value	= tex2D(t2dSrc, iVoxelX,		iVoxelY + iZ * iVolumeHeight);
		float4 f4PX		= tex2D(t2dSrc, iVoxelX + 1,	iVoxelY + iZ * iVolumeHeight);
		float4 f4NX		= tex2D(t2dSrc, iVoxelX - 1,	iVoxelY + iZ * iVolumeHeight);
		float4 f4PY		= tex2D(t2dSrc, iVoxelX,		min(iVoxelY + 1, iVolumeHeight - 1) + iZ * iVolumeHeight);
		float4 f4NY		= tex2D(t2dSrc, iVoxelX,		max(iVoxelY - 1, 0)					+ iZ * iVolumeHeight);
		float4 f4PZ		= tex2D(t2dSrc, iVoxelX,		iVoxelY + min(iZ + 1, iVolumeDepth - 1) * iVolumeHeight);
		float4 f4NZ		= tex2D(t2dSrc, iVoxelX,		iVoxelY + max(iZ - 1, 0)				* iVolumeHeight);

	#if	0	// MOD-BY-LEETEN 11/04/2009-FROM:
	#else	// MOD-BY-LEETEN 11/04/2009-TO:
		float4 f4WeightOffset = tex3D(t3dWeightOffset, iVoxelX,	iVoxelY, iZ);

		#if	0	// MOD-BY-LEETEN 2009/11/25-FROM:
		#else	// MOD-BY-LEETEN 2009/11/25-TO:
		float4 f4Result;

		#if	0	// DEL-BY-LEETEN 12/07/2009-BEGIN
		#endif	// DEL-BY-LEETEN 12/07/2009-END

		#if	0	// MOD-BY-LEETEN 12/07/2009-FROM:
			f4Result = make_float4(
				f4WeightOffset.w * f4Value.x + (f4PX.x + f4NX.x + f4PY.x + f4NY.x + f4PZ.x + f4NZ.x - 6.0f * f4Value.x) * fAttenuationDividedBy6 + f4WeightOffset.x,
				f4WeightOffset.w * f4Value.y + (f4PX.y + f4NX.y + f4PY.y + f4NY.y + f4PZ.y + f4NZ.y - 6.0f * f4Value.y) * fAttenuationDividedBy6 + f4WeightOffset.y,
				f4WeightOffset.w * f4Value.z + (f4PX.z + f4NX.z + f4PY.z + f4NY.z + f4PZ.z + f4NZ.z - 6.0f * f4Value.z) * fAttenuationDividedBy6 + f4WeightOffset.z,
				0);
		#else	// MOD-BY-LEETEN 12/07/2009-TO:
		f4Result = make_float4(
			f4WeightOffset.w * f4Value.x + (f4PX.x + f4NX.x + f4PY.x + f4NY.x + f4PZ.x + f4NZ.x - 6.0f * f4Value.x) * fAttenuation + f4WeightOffset.x,
			f4WeightOffset.w * f4Value.y + (f4PX.y + f4NX.y + f4PY.y + f4NY.y + f4PZ.y + f4NZ.y - 6.0f * f4Value.y) * fAttenuation + f4WeightOffset.y,
			f4WeightOffset.w * f4Value.z + (f4PX.z + f4NX.z + f4PY.z + f4NY.z + f4PZ.z + f4NZ.z - 6.0f * f4Value.z) * fAttenuation + f4WeightOffset.z,
			0);
		#endif	// MOD-BY-LEETEN 12/07/2009-END

		if( iVoxelX < iVolumeWidth && iVoxelY < iVolumeHeight )
		{
			*ADDRESS_2D(
				float4, cDstPitchedPtr.ptr, 
				sizeof(float4), cDstPitchedPtr.pitch, 
				iVoxelX, iVoxelY + iZ * iVolumeHeight) = f4Result;

			// ADD-BY-LEETEN 12/16/2009-BEGIN
			#if	CHECK_ERROR_CONVERGENCE_BY_CUDPP
			// ADD-BY-LEETEN 12/16/2009-END
			float4 f4Diff;
			f4Diff.x = f4Value.x - f4Result.x;
			f4Diff.y = f4Value.y - f4Result.y;
			f4Diff.z = f4Value.z - f4Result.z;
			float fDiff = f4Diff.x * f4Diff.x + f4Diff.y * f4Diff.y + f4Diff.z * f4Diff.z;
			*ADDRESS_2D(
				float, cErrorPitchedPtr.ptr, 
				sizeof(float), cErrorPitchedPtr.pitch, 
				iVoxelX, iVoxelY + iZ * iVolumeHeight) = fDiff;
			// ADD-BY-LEETEN 12/16/2009-BEGIN
			#endif	// CHECK_ERROR_CONVERGENCE_BY_CUDPP	
			// ADD-BY-LEETEN 12/16/2009-END
		}
	#endif	// MOD-BY-LEETEN 2009/11/25-END

	#endif	// MOD-BY-LEETEN 11/04/2009-END

	// MOD-BY-LEETEN 11/04/2009-FROM:
		// #else	// MOD-BY-LEETEN 10/02/2009-TO:
	// TO:
	#else	// #if	DIFFUSION_BY_FOR_LOOP_ON_HOST
	// MOD-BY-LEETEN 11/04/2009-END

	float4 f4Value;
	float4 f4PX;
	float4 f4NX;
	float4 f4PY;
	float4 f4NY;
	float4 f4PZ;
	float4 f4NZ;

	#if	0	// MOD-BY-LEETEN 11/04/2009-FROM:
	#else	// MOD-BY-LEETEN 11/04/2009-TO:

	float4 *pf4DstPtr = ADDRESS_2D(
							float4, cDstPitchedPtr.ptr, 
							sizeof(float4), cDstPitchedPtr.pitch, 
							iVoxelX, iVoxelY + iBeginZ * iVolumeHeight);

	// ADD-BY-LEETEN 2009/11/25-BEGIN
	// ADD-BY-LEETEN 12/16/2009-BEGIN
	#if	CHECK_ERROR_CONVERGENCE_BY_CUDPP	
	// ADD-BY-LEETEN 12/16/2009-END
	float *pfErrorPtr = ADDRESS_2D(
							float, cErrorPitchedPtr.ptr, 
							sizeof(float), cErrorPitchedPtr.pitch, 
							iVoxelX, iVoxelY + iBeginZ * iVolumeHeight);
	// ADD-BY-LEETEN 12/16/2009-BEGIN
	#endif	// CHECK_ERROR_CONVERGENCE_BY_CUDPP	
	// ADD-BY-LEETEN 12/16/2009-END

	// ADD-BY-LEETEN 2009/11/25-END

	#endif	// MOD-BY-LEETEN 11/04/2009-END

		#if	0	// MOD-BY-LEETEN 11/04/2009-FROM:
		#else	// MOD-BY-LEETEN 11/04/2009-TO:
		f4Value		= tex2D(t2dSrc, iVoxelX,		iVoxelY + iBeginZ				* iVolumeHeight);
		f4NZ		= tex2D(t2dSrc, iVoxelX,		iVoxelY + max(iBeginZ - 1, 0)	* iVolumeHeight);
		#endif	// MOD-BY-LEETEN 11/04/2009-END

		for(int z = iBeginZ; z < iEndZ; z++, f4NZ = f4Value, f4Value = f4PZ)
		{
			// ADD-BY-LEETEN 10/02/2009-BEGIN
			float4 f4WeightOffset = tex3D(t3dWeightOffset, iVoxelX,	iVoxelY, z);
			// ADD-BY-LEETEN 10/02/2009-END
			// f4Value		= tex2D(t2dSrc, iVoxelX,		iVoxelY + z * iVolumeHeight);
			#if	!USE_SHARED_MEMORY
			f4PX		= tex2D(t2dSrc, iVoxelX + 1,	iVoxelY + z * iVolumeHeight);
			f4NX		= tex2D(t2dSrc, iVoxelX - 1,	iVoxelY + z * iVolumeHeight);
			f4PY		= tex2D(t2dSrc, iVoxelX,		min(iVoxelY + 1, iVolumeHeight - 1) + z * iVolumeHeight);
			f4NY		= tex2D(t2dSrc, iVoxelX,		max(iVoxelY - 1, 0)					+ z * iVolumeHeight);

			#else	// #if	!USE_SHARED_MEMORY
			#endif	// #if	!USE_SHARED_MEMORY
			f4PZ		= tex2D(t2dSrc, iVoxelX,		iVoxelY + min(z + 1, iVolumeDepth - 1) * iVolumeHeight);
			// f4NZ		= tex2D(t2dSrc, iVoxelX,		iVoxelY + max(z - 1, 0)				* iVolumeHeight);

			// store the result back to the dst.
			#if	0	// MOD-BY-LEETEN 2009/11/10-FROM:
			#else	// MOD-BY-LEETEN 2009/11/10-TO:
			float4 f4Result;

			#if	0	// DEL-BY-LEETEN 12/07/2009-BEGIN
			#endif	// DEL-BY-LEETEN 12/07/2009-END	

				#if	0	// MOD-BY-LEETEN 12/07/2009-FROM:
					f4Result = make_float4(
						f4WeightOffset.w * f4Value.x + (f4PX.x + f4NX.x + f4PY.x + f4NY.x + f4PZ.x + f4NZ.x - 6.0f * f4Value.x) * fAttenuationDividedBy6 + f4WeightOffset.x,
						f4WeightOffset.w * f4Value.y + (f4PX.y + f4NX.y + f4PY.y + f4NY.y + f4PZ.y + f4NZ.y - 6.0f * f4Value.y) * fAttenuationDividedBy6 + f4WeightOffset.y,
						f4WeightOffset.w * f4Value.z + (f4PX.z + f4NX.z + f4PY.z + f4NY.z + f4PZ.z + f4NZ.z - 6.0f * f4Value.z) * fAttenuationDividedBy6 + f4WeightOffset.z,
						0);
				#else	// MOD-BY-LEETEN 12/07/2009-TO:
				f4Result = make_float4(	
					f4WeightOffset.w * f4Value.x + (f4PX.x + f4NX.x + f4PY.x + f4NY.x + f4PZ.x + f4NZ.x - 6.0f * f4Value.x) * fAttenuation + f4WeightOffset.x,
					f4WeightOffset.w * f4Value.y + (f4PX.y + f4NX.y + f4PY.y + f4NY.y + f4PZ.y + f4NZ.y - 6.0f * f4Value.y) * fAttenuation + f4WeightOffset.y,
					f4WeightOffset.w * f4Value.z + (f4PX.z + f4NX.z + f4PY.z + f4NY.z + f4PZ.z + f4NZ.z - 6.0f * f4Value.z) * fAttenuation + f4WeightOffset.z,
					0.0);
				#endif	// MOD-BY-LEETEN 12/07/2009-END
			#endif	// MOD-BY-LEETEN 2009/11/10-TO:

			// ADD-BY-LEETEN 11/04/2009-BEGIN
			if( iVoxelX < iVolumeWidth && iVoxelY < iVolumeHeight )
			// ADD-BY-LEETEN 11/04/2009-END
			{
				pf4DstPtr[0] = f4Result;

				// ADD-BY-LEETEN 12/16/2009-BEGIN
				#if	CHECK_ERROR_CONVERGENCE_BY_CUDPP
				// ADD-BY-LEETEN 12/16/2009-END

				// ADD-BY-LEETEN 2009/11/25-BEGIN
				float4 f4Diff;
				f4Diff.x = f4Value.x - f4Result.x;
				f4Diff.y = f4Value.y - f4Result.y;
				f4Diff.z = f4Value.z - f4Result.z;
				float fDiff;
				// float fDiff = f4Diff.x * f4Diff.x + f4Diff.y * f4Diff.y + f4Diff.z * f4Diff.z;
				// MOD-BY-LEETEN 2009/12/17-FROM:
					// fDiff = max(abs(f4Diff.x), max(abs(f4Diff.y), abs(f4Diff.z)));
				// TO:
				fDiff = f4Diff.x * f4Diff.x + f4Diff.y * f4Diff.y + f4Diff.z * f4Diff.z;
				// MOD-BY-LEETEN 2009/12/17-END
				pfErrorPtr[0] = fDiff;
				// ADD-BY-LEETEN 2009/11/25-END
				// ADD-BY-LEETEN 12/16/2009-BEGIN
				#endif	// #if	CHECK_ERROR_CONVERGENCE_BY_CUDPP	
				// ADD-BY-LEETEN 12/16/2009-END
			}

			pf4DstPtr += iVolumeHeight * cDstPitchedPtr.pitch / sizeof(float4);

			// ADD-BY-LEETEN 12/16/2009-BEGIN
			#if	CHECK_ERROR_CONVERGENCE_BY_CUDPP	
			// ADD-BY-LEETEN 12/16/2009-END
			// ADD-BY-LEETEN 2009/11/25-BEGIN
			pfErrorPtr += iVolumeHeight * cErrorPitchedPtr.pitch / sizeof(float);
			// ADD-BY-LEETEN 2009/11/25-END
			// ADD-BY-LEETEN 12/16/2009-BEGIN
			#endif	// #if	CHECK_ERROR_CONVERGENCE_BY_CUDPP
			// ADD-BY-LEETEN 12/16/2009-END
		}

	#endif	// MOD-BY-LEETEN 10/02/2009-END
}

/*

$Log: not supported by cvs2svn $

*/

