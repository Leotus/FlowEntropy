
#include <hip/hip_runtime.h>
#if	COMPUTE_ENTROPY_VOLUME_PER_SCANLINE_CUDA

// ADD-BY-LEETEN 12/20/2009-BEGIN
__global__
void
_ComputeEntropyAlongScanline_kernel
(
	int3 i3BlockCorner,
	int iNrOfBins,
	int3 i3KernelSize,
	int iRoundKernelSizeXToHalfWarp,
	int3 i3Step,
	cudaPitchedPtr cActiveVoxelNeighbors_pitched,
	int3 i3VolumeSize,
	cudaPitchedPtr cEntropyVolume_pitched
)
{
	int iActiveVoxelX = threadIdx.x + blockIdx.x * blockDim.x;
	int iActiveVoxelID = blockIdx.x + blockIdx.y * gridDim.x;
	int iActiveVoxelYZ = blockIdx.y;
	int iActiveVoxelY = iActiveVoxelYZ % i3Step.y;
	int iActiveVoxelZ = iActiveVoxelYZ / i3Step.y;
 	int iVoxelX = i3BlockCorner.x + iActiveVoxelX;
	int iVoxelY = i3BlockCorner.y + iActiveVoxelY;
	int iVoxelZ = i3BlockCorner.z + iActiveVoxelZ;

	int iNrOfNeighbors = (2 * i3KernelSize.x + 1) * (2 * i3KernelSize.y + 1) * (2 * i3KernelSize.z + 1);
	float fNrOfNeighbors = float(iNrOfNeighbors);

	int iNrOfYZNeighbors = (2 * i3KernelSize.y + 1) * (2 * i3KernelSize.z + 1);

	float fEntropy = 0.0f;
	int iPrevBin = -1;
	float fCount = 0.0f; 

	for(int		n = 0,	r = 0; r < iNrOfYZNeighbors;					r++)
	{
		for(int			x = 0; x < cActiveVoxelNeighbors_pitched.xsize; x++, n++)
		{

			unsigned int uBin = 
				tex2D(t2dActiveVoxelSortedNeighbors, 
					x, 
					iNrOfYZNeighbors * iActiveVoxelID + r);
			/*
			unsigned int uX = uBin / unsigned int (iNrOfBins);
			uBin = uBin % unsigned int (iNrOfBins);
			*/
			unsigned int uX = uBin % (i3VolumeSize.x + 2 * iRoundKernelSizeXToHalfWarp);
			uBin = uBin / (i3VolumeSize.x + 2 * iRoundKernelSizeXToHalfWarp);

			int iX = int(uX) - iRoundKernelSizeXToHalfWarp;
			if( iX > iVoxelX + i3KernelSize.x || iX < iVoxelX - i3KernelSize.x )
				continue;

			if( iPrevBin < 0 || int(uBin) == iPrevBin  )
				fCount += 1.0f;
			else
			{
				if( fCount > 0.0f )
					fEntropy += fCount * log2(fCount);
				
				fCount = 1.0f;
			}
			iPrevBin = int(uBin);
		}
	}

	if( fCount > 0.0f )
		fEntropy += fCount * log2(fCount);

	fEntropy = -fEntropy / fNrOfNeighbors + log2(fNrOfNeighbors);
	fEntropy = max(fEntropy, 0.0f);

	if( iVoxelX < i3VolumeSize.x && iVoxelY < i3VolumeSize.y && iVoxelZ < i3VolumeSize.z )
		*ADDRESS_2D(
			float, cEntropyVolume_pitched.ptr, 
			sizeof(float), cEntropyVolume_pitched.pitch, 
			iVoxelX, iVoxelY + iVoxelZ * i3VolumeSize.y) = fEntropy;
}

__global__ 
void 
_CollectNeighborsAlongScanline_kernel
(
	int3 i3BlockCorner,
	int3 i3KernelSize,
	int iRoundKernelSizeXToHalfWarp,
	int3 i3VolumeSize,
	int3 i3Step,
	int iNrOfBins,
	cudaPitchedPtr cActiveVoxelNeighbors_pitched
)
{
	// NOTE!! here blockDim.x is the actuall #threads; 
	// but in order to get the idea #threads per block, 
	// the tails in the 2-sided should be substratected
	int iActiveVoxelX = threadIdx.x + blockIdx.x * (blockDim.x - 2 * iRoundKernelSizeXToHalfWarp);	
	int iActiveVoxelID = blockIdx.x + blockIdx.y * gridDim.x;
	int iActiveVoxelYZ = blockIdx.y;
	int iActiveVoxelY = iActiveVoxelYZ % i3Step.y;
	int iActiveVoxelZ = iActiveVoxelYZ / i3Step.y;
 	int iVoxelX = i3BlockCorner.x + iActiveVoxelX - iRoundKernelSizeXToHalfWarp;
	int iVoxelY = i3BlockCorner.y + iActiveVoxelY;
	int iVoxelZ = i3BlockCorner.z + iActiveVoxelZ;

	int iNrOfRows = (i3KernelSize.y * 2 + 1) * (i3KernelSize.z * 2 + 1);

	int iXOffset = 0;
	for(int	iOffset = 0,iZOffset = -i3KernelSize.z; iZOffset <= i3KernelSize.z; iZOffset++)
		for(int			iYOffset = -i3KernelSize.y; iYOffset <= i3KernelSize.y; iYOffset++, iOffset++)
		{
			int iX, iY, iZ;
			iX = iVoxelX + iXOffset;
			iY = iVoxelY + iYOffset;
			iZ = iVoxelZ + iZOffset;

			int3 i3TexCoord;
			i3TexCoord.x = IMirrorCoord(iX, i3VolumeSize.x);
			i3TexCoord.y = IMirrorCoord(iY, i3VolumeSize.y);
			i3TexCoord.z = IMirrorCoord(iZ, i3VolumeSize.z);

			int iSrcBin = tex2D(t2dSrcBinVolume, i3TexCoord.x, i3TexCoord.y + i3TexCoord.z * i3VolumeSize.y);

			*ADDRESS_2D(
				unsigned int,			cActiveVoxelNeighbors_pitched.ptr, 
				sizeof(unsigned int),	cActiveVoxelNeighbors_pitched.pitch, 
				threadIdx.x, 
				iActiveVoxelID * iNrOfRows + iOffset) 
					= unsigned int(iSrcBin * (i3VolumeSize.x + 2 * iRoundKernelSizeXToHalfWarp) + (iVoxelX + iRoundKernelSizeXToHalfWarp));
		}
}
// ADD-BY-LEETEN 12/20/2009-END

void 
_ComputeEntropyVolumePerScanline_cuda
(
						// res. of the neighboring region
	int3 i3KernelSize,

						// the histogram
	int iNrOfBins,
	int *piHistogram_global,
	float *pfLogHistogram_global,

						// res. of the volume																
	int3 i3VolumeSize,	
						// bin volume																	
	cudaPitchedPtr		cBinVolume_pitched,								
	cudaPitchedPtr		cEntropyVolume_pitched
)
{
CLOCK_INIT(_ComputeEntropyVolume_PRINT_TIMING, __FUNCTION__ ": ");

CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_TIMING);
	dim3 v3Blk, v3Grid;
	int3 i3Step;
	int3 i3NrOfSteps;

	const int iNrOfRows = (2 * i3KernelSize.y + 1) * (2 * i3KernelSize.z + 1);

	int iRoundKernelSizeXToHalfWarp = NR_OF_THREADS_PER_HALF_WARP * int(ceilf( float(i3KernelSize.x) / float(NR_OF_THREADS_PER_HALF_WARP) ));

	int iNrOfNeededThreadsPerBlock = min(MAX_THREADS - 2 * iRoundKernelSizeXToHalfWarp, i3VolumeSize.x);
	int iNrOfActualThreadsPerBlock = iNrOfNeededThreadsPerBlock + 2 * iRoundKernelSizeXToHalfWarp;		// #threads (including the real ones and the tailing ones)
	v3Blk = dim3(iNrOfActualThreadsPerBlock, 1, 1);
	v3Grid = dim3(1, i3VolumeSize.y, 1);
	i3Step.x = iNrOfNeededThreadsPerBlock;
	i3Step.y = i3VolumeSize.y;
	i3Step.z = 1;

	fprintf(stderr, "MEM = %d MB; #BLOCKS = %d x %d\n", MAX_MEMORY_SIZE / (1<<20), v3Grid.x, v3Grid.y);

	cudaPitchedPtr cActiveVoxelNeighbors_pitched;
	cActiveVoxelNeighbors_pitched.xsize = v3Blk.x;
	cActiveVoxelNeighbors_pitched.ysize = iNrOfRows * v3Grid.x * v3Grid.y;
	CUDA_SAFE_CALL_NO_SYNC(
		cudaMallocPitch(
			(void**)&cActiveVoxelNeighbors_pitched.ptr, 
			&cActiveVoxelNeighbors_pitched.pitch,
			cActiveVoxelNeighbors_pitched.xsize * sizeof(int), 
			cActiveVoxelNeighbors_pitched.ysize)	);

	// bind the input vin volume to the texture that represents the src. bin volume 
	t2dSrcBinVolume.addressMode[0] = cudaAddressModeClamp;
	t2dSrcBinVolume.addressMode[1] = cudaAddressModeClamp;
	t2dSrcBinVolume.filterMode =	cudaFilterModePoint;
	t2dSrcBinVolume.normalized =	false;
	CUDA_SAFE_CALL_NO_SYNC(
		cudaBindTexture2D(
			0, 
			t2dSrcBinVolume, 
			cBinVolume_pitched.ptr, 
			cudaCreateChannelDesc<int>(),
			cVolumeExtent_array.width, 
			cVolumeExtent_array.height * cVolumeExtent_array.depth, 
			cBinVolume_pitched.pitch) );

	// bind the histogram as a texture
	t2dActiveVoxelSortedNeighbors.addressMode[0] = cudaAddressModeClamp;
	t2dActiveVoxelSortedNeighbors.addressMode[1] = cudaAddressModeClamp;
	t2dActiveVoxelSortedNeighbors.filterMode =	cudaFilterModePoint;
	t2dActiveVoxelSortedNeighbors.normalized =	false;

	CUDA_SAFE_CALL_NO_SYNC(
		cudaBindTexture2D(
			0, 
			t2dActiveVoxelSortedNeighbors, 
			cActiveVoxelNeighbors_pitched.ptr, 
			cudaCreateChannelDesc<unsigned int>(),
			cActiveVoxelNeighbors_pitched.xsize,
			cActiveVoxelNeighbors_pitched.ysize, 
			cActiveVoxelNeighbors_pitched.pitch) );

	CUDPPHandle hScanPlan = 0;

	CUDPPConfiguration	cConfig;
	cConfig.op =		CUDPP_ADD;
	cConfig.datatype =	CUDPP_UINT;
	cConfig.algorithm =	CUDPP_SORT_RADIX;
	cConfig.options =	CUDPP_OPTION_KEYS_ONLY;

	assert( 
		CUDPP_SUCCESS  == cudppPlan(
			&hScanPlan,	
			cConfig, 
			iNrOfRows * cActiveVoxelNeighbors_pitched.xsize,
			1, 
			0) );

CLOCK_END(_ComputeEntropyVolume_PRINT_TIMING, false);

CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_TIMING);

	for(int			z = 0; z < i3VolumeSize.z; z += i3Step.z)
		for(int		y = 0; y < i3VolumeSize.y; y += i3Step.y)
			for(int	x = 0; x < i3VolumeSize.x; x += i3Step.x)
			{
				CLOCK_INIT(_ComputeEntropyVolume_PRINT_LOOP_TIMING, __FUNCTION__ " (main loop): ");

				CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_LOOP_TIMING);

				int3 i3BlockCorner = make_int3(x, y, z);
				_CollectNeighborsAlongScanline_kernel<<<
					v3Grid, 
					v3Blk, 
					0>>>
				(
					i3BlockCorner,
					i3KernelSize,
					iRoundKernelSizeXToHalfWarp,
					i3VolumeSize,
					#if	1	// TEST-ADD
					i3Step,
					#endif
					iNrOfBins,
					cActiveVoxelNeighbors_pitched
				);
				CUT_CHECK_ERROR("_CollectNeighborsAlongScanline_kernel() failed");

CLOCK_END(_ComputeEntropyVolume_PRINT_LOOP_TIMING, false);

CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_LOOP_TIMING);

				// call cudpp to sort the buffer _CollectNeighborsToBeSorted_kernel
				for(int y2 = 0; y2 < int(v3Grid.y); y2++)
				{
					cudppSort(
						hScanPlan,
						((unsigned int*)cActiveVoxelNeighbors_pitched.ptr) + y2 * iNrOfRows * v3Blk.x,
						NULL,
						32,	// RADIX_SORT_BITS,
						iNrOfRows * v3Blk.x) ;
					CUT_CHECK_ERROR("cudppSort() failed");
				}

CLOCK_END(_ComputeEntropyVolume_PRINT_LOOP_TIMING, false);

CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_LOOP_TIMING);
				// 
				_ComputeEntropyAlongScanline_kernel<<<
					v3Grid, 
					dim3(iNrOfNeededThreadsPerBlock, 1, 1),
					0>>>
				(
					i3BlockCorner,
					iNrOfBins,
					i3KernelSize,
					iRoundKernelSizeXToHalfWarp,
					i3Step,
					cActiveVoxelNeighbors_pitched,
					i3VolumeSize,
					cEntropyVolume_pitched
				);
				CUT_CHECK_ERROR("_ComputeEntropyOnSortedNeighbors_kernel() failed");
CLOCK_END(_ComputeEntropyVolume_PRINT_LOOP_TIMING, false);

CLOCK_PRINT(_ComputeEntropyVolume_PRINT_LOOP_TIMING);
			}


CLOCK_END(_ComputeEntropyVolume_PRINT_TIMING, false);

CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_TIMING);

	cudppDestroyPlan(hScanPlan);
	FREE_MEMORY(cActiveVoxelNeighbors_pitched.ptr);

CLOCK_END(_ComputeEntropyVolume_PRINT_TIMING, false);

CLOCK_PRINT(_ComputeEntropyVolume_PRINT_TIMING);
}
	
#endif	// #if	COMPUTE_ENTROPY_VOLUME_PER_SCANLINE_CUDA

/*

$Log: not supported by cvs2svn $

*/
