
#include <hip/hip_runtime.h>
#if	COMPUTE_ENTROPY_VOLUME_PER_SCANLINE_CUDA

// ADD-BY-LEETEN 12/27/2009-BEGIN
__global__
void
_ComputeEntropyAlongScanlineInParallel_kernel
(
	int3 i3BlockCorner,
	int iNrOfBins,
	int3 i3KernelSize,
	int iRoundKernelSizeXToHalfWarp,
	int3 i3Step,
	dim3 v3Grid,
	cudaPitchedPtr cActiveVoxelNeighbors_pitched,
	int3 i3VolumeSize,
	cudaPitchedPtr cEntropyVolume_pitched
)
{
	__shared__ int	piBin_shared[MAX_THREADS];
	__shared__ int	piCounter_shared[MAX_THREADS];
	__shared__ int iMinBin_shared;
	__shared__ int iMaxBin_shared;
	__shared__ int iNextBin_shared;
	__shared__ int iMinInd_shared;

	// initialize the tables
	piBin_shared[threadIdx.x] = -1;
	piCounter_shared[threadIdx.x] = 0;
	__syncthreads();

	int iBlockIdxX = blockIdx.x;
	int iBlockIdxY = blockIdx.y % v3Grid.y;
	int iBlockIdxZ = blockIdx.y / v3Grid.y;
	int iActiveVoxelX =	threadIdx.x + iBlockIdxX * (blockDim.x - 2 * iRoundKernelSizeXToHalfWarp);
	int iActiveVoxelY = threadIdx.y + iBlockIdxY * blockDim.y;
	int iActiveVoxelZ = threadIdx.z + iBlockIdxZ * blockDim.z;
 	int iVoxelX = i3BlockCorner.x + iActiveVoxelX - iRoundKernelSizeXToHalfWarp;
	int iVoxelY = i3BlockCorner.y + iActiveVoxelY;
	int iVoxelZ = i3BlockCorner.z + iActiveVoxelZ;
	int iActiveVoxelID = iBlockIdxX + iActiveVoxelY * v3Grid.x + iActiveVoxelZ * v3Grid.x * v3Grid.y;

	int iNrOfYZNeighbors = (2 * i3KernelSize.y + 1) * (2 * i3KernelSize.z + 1);

	float fEntropy = 0.0f;
	for(int r = 0; r < iNrOfYZNeighbors; r++)
	{
		unsigned int uBin = 
			tex2D(t2dActiveVoxelSortedNeighbors, 
				threadIdx.x, 
				iNrOfYZNeighbors * iActiveVoxelID + r);
		#if	0	// MOD-BY-LEETEN 12/28/2009-FROM:
			unsigned int uX = uBin % (i3VolumeSize.x + 2 * iRoundKernelSizeXToHalfWarp);
			int iX = int(uX) - iRoundKernelSizeXToHalfWarp;
			int iBin = uBin / (i3VolumeSize.x + 2 * iRoundKernelSizeXToHalfWarp);
		#else	// MOD-BY-LEETEN 12/28/2009-TO:
		unsigned int uX = uBin % blockDim.x;
		int iX = int(uX);
		int iBin = uBin / blockDim.x;
		#endif	// MOD-BY-LEETEN 12/28/2009-END

		if( 0 == threadIdx.x )
			iMinBin_shared = iBin;

		if( blockDim.x - 1 == threadIdx.x )
			iMaxBin_shared = iBin;

		__syncthreads();

		// iNextBin_shared = iMinBin_shared;

		for(int b = iMinBin_shared; b <= iMaxBin_shared; b = iNextBin_shared)
		{
			// pass-1: each thread updates the corresponding entropy
			int iPrevBin = piBin_shared[threadIdx.x];
			int iCounter = piCounter_shared[threadIdx.x];
			if( iCounter > 0 && b != iPrevBin  )
			{
				fEntropy += float(iCounter) * log2f(float(iCounter));
				piCounter_shared[threadIdx.x] = 0; // reset the counter
			}
			__syncthreads();

			// pass-2: each thread updates its neighbors
			if( b == iBin )
			{
				#if	0	// MOD-BY-LEETEN 12/28/2009-FROM:
					int iLeftT	= int(threadIdx.x) + ((iX - i3KernelSize.x) - iVoxelX);
					iLeftT = max(iRoundKernelSizeXToHalfWarp, iLeftT);
					int iRightT = int(threadIdx.x) + ((iX + i3KernelSize.x) - iVoxelX);
					iRightT = min(blockDim.x - iRoundKernelSizeXToHalfWarp, iRightT);
				#else	// MOD-BY-LEETEN 12/28/2009-TO:
				int iLeftT	= iX - i3KernelSize.x;
				iLeftT = max(iRoundKernelSizeXToHalfWarp, iLeftT);
				int iRightT = iX + i3KernelSize.x + 1;
				iRightT = min(blockDim.x - iRoundKernelSizeXToHalfWarp, iRightT);
				#endif	// MOD-BY-LEETEN 12/28/2009-END

				int iNrOfThreadsToBeUpdated = iRightT - iLeftT;
				for(int ti = 0; ti < iNrOfThreadsToBeUpdated; ti++)
				{
					// shuffer the index to reduce band conflict
					int t = iLeftT + (threadIdx.x + ti) % iNrOfThreadsToBeUpdated;

					atomicAdd(&piCounter_shared[t], 1);
					piBin_shared[t] = b;
				}
			}
			__syncthreads();

			// pass-3: update the next bin to search
			if( 0 == threadIdx.x )
				iMinInd_shared = blockDim.x;
			__syncthreads();

			if( iBin > b )
				atomicMin(&iMinInd_shared, threadIdx.x);
			__syncthreads();

			// if no new bin is found, exit the loop
			if( iMinInd_shared == blockDim.x )
				break;

			if( threadIdx.x == iMinInd_shared ) 
				iNextBin_shared = iBin;
			__syncthreads();
		}
	}

	int iCounter = piCounter_shared[threadIdx.x];
	if( iCounter > 0 )
		fEntropy += float(iCounter) * log2f(float(iCounter));

	int iNrOfNeighbors = (2 * i3KernelSize.x + 1) * (2 * i3KernelSize.y + 1) * (2 * i3KernelSize.z + 1);
	float fNrOfNeighbors = float(iNrOfNeighbors);

	fEntropy = -fEntropy / fNrOfNeighbors + log2(fNrOfNeighbors);
	fEntropy = max(fEntropy, 0.0f);

	if( iRoundKernelSizeXToHalfWarp <=	threadIdx.x && 
										threadIdx.x < blockDim.x - iRoundKernelSizeXToHalfWarp &&
		0 <= iVoxelX && iVoxelX < i3VolumeSize.x && 
		0 <= iVoxelY && iVoxelY < i3VolumeSize.y && 
		0 <= iVoxelZ && iVoxelZ < i3VolumeSize.z )
		*ADDRESS_2D(
			float, cEntropyVolume_pitched.ptr, 
			sizeof(float), cEntropyVolume_pitched.pitch, 
			iVoxelX, iVoxelY + iVoxelZ * i3VolumeSize.y) = fEntropy;
}
// ADD-BY-LEETEN 12/27/2009-END

// ADD-BY-LEETEN 12/20/2009-BEGIN
__global__
void
_ComputeEntropyAlongScanline_kernel
(
	int3 i3BlockCorner,
	int iNrOfBins,
	int3 i3KernelSize,
	int iRoundKernelSizeXToHalfWarp,
	int3 i3Step,
	// ADD-BY-LEETEN 12/20/2009-BEGIN
	dim3 v3Grid,
	// ADD-BY-LEETEN 12/20/2009-END
	cudaPitchedPtr cActiveVoxelNeighbors_pitched,
	int3 i3VolumeSize,
	cudaPitchedPtr cEntropyVolume_pitched
)
{
	#if	0	// MOD-BY-LEETEN 12/20/2009-FROM:
		int iActiveVoxelX = threadIdx.x + blockIdx.x * blockDim.x;
		int iActiveVoxelID = blockIdx.x + blockIdx.y * gridDim.x;
		int iActiveVoxelYZ = blockIdx.y;
		int iActiveVoxelY = iActiveVoxelYZ % i3Step.y;
		int iActiveVoxelZ = iActiveVoxelYZ / i3Step.y;
 		int iVoxelX = i3BlockCorner.x + iActiveVoxelX;
		int iVoxelY = i3BlockCorner.y + iActiveVoxelY;
		int iVoxelZ = i3BlockCorner.z + iActiveVoxelZ;
	#else	// MOD-BY-LEETEN 12/20/2009-TO:
	int iBlockIdxX = blockIdx.x;
	int iBlockIdxY = blockIdx.y % v3Grid.y;
	int iBlockIdxZ = blockIdx.y / v3Grid.y;
	int iActiveVoxelX =	threadIdx.x + iBlockIdxX * blockDim.x;
	int iActiveVoxelY = threadIdx.y + iBlockIdxY * blockDim.y;
	int iActiveVoxelZ = threadIdx.z + iBlockIdxZ * blockDim.z;
 	int iVoxelX = i3BlockCorner.x + iActiveVoxelX;
	int iVoxelY = i3BlockCorner.y + iActiveVoxelY;
	int iVoxelZ = i3BlockCorner.z + iActiveVoxelZ;
	int iActiveVoxelID = iBlockIdxX + iActiveVoxelY * v3Grid.x + iActiveVoxelZ * v3Grid.x * v3Grid.y;
	#endif	// MOD-BY-LEETEN 12/20/2009-END

	int iNrOfNeighbors = (2 * i3KernelSize.x + 1) * (2 * i3KernelSize.y + 1) * (2 * i3KernelSize.z + 1);
	float fNrOfNeighbors = float(iNrOfNeighbors);

	int iNrOfYZNeighbors = (2 * i3KernelSize.y + 1) * (2 * i3KernelSize.z + 1);

	float fEntropy = 0.0f;
	int iPrevBin = -1;
	float fCount = 0.0f; 

	for(int		n = 0,	r = 0; r < iNrOfYZNeighbors;					r++)
	{
		for(int			x = 0; x < cActiveVoxelNeighbors_pitched.xsize; x++, n++)
		{

			unsigned int uBin = 
				tex2D(t2dActiveVoxelSortedNeighbors, 
					x, 
					iNrOfYZNeighbors * iActiveVoxelID + r);
			/*
			unsigned int uX = uBin / unsigned int (iNrOfBins);
			uBin = uBin % unsigned int (iNrOfBins);
			*/
			#if	0	// MOD-BY-LEETEN 12/28/2009-FROM:
				unsigned int uX = uBin % (i3VolumeSize.x + 2 * iRoundKernelSizeXToHalfWarp);
				uBin = uBin / (i3VolumeSize.x + 2 * iRoundKernelSizeXToHalfWarp);
				int iX = int(uX) - iRoundKernelSizeXToHalfWarp;
				if( iX > iVoxelX + i3KernelSize.x || iX < iVoxelX - i3KernelSize.x )
					continue;
			#else	// MOD-BY-LEETEN 12/28/2009-TO:
			unsigned int uX = uBin % (blockDim.x + 2 * iRoundKernelSizeXToHalfWarp);
			uBin = uBin / (blockDim.x + 2 * iRoundKernelSizeXToHalfWarp);
			int iX = int(uX) - iRoundKernelSizeXToHalfWarp;
			if( iX > int(threadIdx.x) + i3KernelSize.x || iX < int(threadIdx.x) - i3KernelSize.x )
				continue;
			#endif	// MOD-BY-LEETEN 12/28/2009-END


			if( iPrevBin < 0 || int(uBin) == iPrevBin  )
				fCount += 1.0f;
			else
			{
				if( fCount > 0.0f )
					fEntropy += fCount * log2(fCount);
				
				fCount = 1.0f;
			}
			iPrevBin = int(uBin);
		}
	}

	if( fCount > 0.0f )
		fEntropy += fCount * log2(fCount);

	fEntropy = -fEntropy / fNrOfNeighbors + log2(fNrOfNeighbors);
	fEntropy = max(fEntropy, 0.0f);

	// MOD-BY-LEETEN 12/20/2009-FROM:
		// if( iVoxelX < i3VolumeSize.x && iVoxelY < i3VolumeSize.y && iVoxelZ < i3VolumeSize.z )
	// TO:
	if( 0 <= iVoxelX && iVoxelX < i3VolumeSize.x && 
		0 <= iVoxelY && iVoxelY < i3VolumeSize.y && 
		0 <= iVoxelZ && iVoxelZ < i3VolumeSize.z )
	// MOD-BY-LEETEN 12/20/2009-END
		*ADDRESS_2D(
			float, cEntropyVolume_pitched.ptr, 
			sizeof(float), cEntropyVolume_pitched.pitch, 
			iVoxelX, iVoxelY + iVoxelZ * i3VolumeSize.y) = fEntropy;
}

__global__ 
void 
_CollectNeighborsAlongScanline_kernel
(
	int3 i3BlockCorner,
	int3 i3KernelSize,
	int iRoundKernelSizeXToHalfWarp,
	int3 i3VolumeSize,
	int3 i3Step,
	// ADD-BY-LEETEN 12/20/2009-BEGIN
	dim3 v3Grid,
	// ADD-BY-LEETEN 12/20/2009-END
	int iNrOfBins,
	cudaPitchedPtr cActiveVoxelNeighbors_pitched
)
{
	// NOTE!! here blockDim.x is the actuall #threads; 
	// but in order to get the idea #threads per block, 
	// the tails in the 2-sided should be substratected
	#if	0	// MOD-BY-LEETEN 12/20/2009-FROM:
		int iActiveVoxelX = threadIdx.x + blockIdx.x * (blockDim.x - 2 * iRoundKernelSizeXToHalfWarp);	
		int iActiveVoxelID = blockIdx.x + blockIdx.y * gridDim.x;
		int iActiveVoxelYZ = blockIdx.y;
		int iActiveVoxelY = iActiveVoxelYZ % i3Step.y;
		int iActiveVoxelZ = iActiveVoxelYZ / i3Step.y;
 		int iVoxelX = i3BlockCorner.x + iActiveVoxelX - iRoundKernelSizeXToHalfWarp;
		int iVoxelY = i3BlockCorner.y + iActiveVoxelY;
		int iVoxelZ = i3BlockCorner.z + iActiveVoxelZ;
	#else	// MOD-BY-LEETEN 12/20/2009-TO:
	int iBlockIdxX = blockIdx.x;
	int iBlockIdxY = blockIdx.y % v3Grid.y;
	int iBlockIdxZ = blockIdx.y / v3Grid.y;
	int iActiveVoxelX =	threadIdx.x + iBlockIdxX * (blockDim.x - 2 * iRoundKernelSizeXToHalfWarp);
	int iActiveVoxelY = threadIdx.y + iBlockIdxY * blockDim.y;
	int iActiveVoxelZ = threadIdx.z + iBlockIdxZ * blockDim.z;
 	int iVoxelX = i3BlockCorner.x + iActiveVoxelX - iRoundKernelSizeXToHalfWarp;
	int iVoxelY = i3BlockCorner.y + iActiveVoxelY;
	int iVoxelZ = i3BlockCorner.z + iActiveVoxelZ;
	int iActiveVoxelID = iBlockIdxX + iActiveVoxelY * v3Grid.x + iActiveVoxelZ * v3Grid.x * v3Grid.y;
	#endif	// MOD-BY-LEETEN 12/20/2009-END

	int iNrOfRows = (i3KernelSize.y * 2 + 1) * (i3KernelSize.z * 2 + 1);

	int iXOffset = 0;
	for(int	iOffset = 0,iZOffset = -i3KernelSize.z; iZOffset <= i3KernelSize.z; iZOffset++)
		for(int			iYOffset = -i3KernelSize.y; iYOffset <= i3KernelSize.y; iYOffset++, iOffset++)
		{
			int iX, iY, iZ;
			iX = iVoxelX + iXOffset;
			iY = iVoxelY + iYOffset;
			iZ = iVoxelZ + iZOffset;

			int3 i3TexCoord;
			i3TexCoord.x = IMirrorCoord(iX, i3VolumeSize.x);
			i3TexCoord.y = IMirrorCoord(iY, i3VolumeSize.y);
			i3TexCoord.z = IMirrorCoord(iZ, i3VolumeSize.z);

			int iSrcBin = tex2D(t2dSrcBinVolume, i3TexCoord.x, i3TexCoord.y + i3TexCoord.z * i3VolumeSize.y);

			#if	0	// MOD-BY-LEETEN 12/20/2009-FROM:
				*ADDRESS_2D(
					unsigned int,			cActiveVoxelNeighbors_pitched.ptr, 
					sizeof(unsigned int),	cActiveVoxelNeighbors_pitched.pitch, 
					threadIdx.x, 
					iActiveVoxelID * iNrOfRows + iOffset) 
			#else	// MOD-BY-LEETEN 12/20/2009-TO:
			*ADDRESS_2D(
				unsigned int,			cActiveVoxelNeighbors_pitched.ptr, 
				sizeof(unsigned int),	cActiveVoxelNeighbors_pitched.pitch, 
				threadIdx.x, 
				iActiveVoxelID * iNrOfRows + iOffset) 
			#endif	// MOD-BY-LEETEN 12/20/2009-END
					// MOD-BY-LEETEN 12/28/2009-FROM:
						// = unsigned int(iSrcBin * (i3VolumeSize.x + 2 * iRoundKernelSizeXToHalfWarp) + (iVoxelX + iRoundKernelSizeXToHalfWarp));
					// TO:
					= unsigned int(iSrcBin * blockDim.x  + threadIdx.x);
					// MOD-BY-LEETEN 12/28/2009-END
		}
}
// ADD-BY-LEETEN 12/20/2009-END

void 
_ComputeEntropyVolumePerScanline_cuda
(
						// res. of the neighboring region
	int3 i3KernelSize,

						// the histogram
	int iNrOfBins,
	int *piHistogram_global,
	float *pfLogHistogram_global,

						// res. of the volume																
	int3 i3VolumeSize,	
						// bin volume																	
	cudaPitchedPtr		cBinVolume_pitched,								
	cudaPitchedPtr		cEntropyVolume_pitched
)
{
CLOCK_INIT(_ComputeEntropyVolume_PRINT_TIMING, __FUNCTION__ ": ");

CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_TIMING);
	dim3 v3Blk, v3Grid;
	int3 i3Step;
	// DEL-BY-LEETEN 12/28/2009-BEGIN
		// int3 i3NrOfSteps;
	// DEL-BY-LEETEN 12/28/2009-END

	const int iNrOfRows = (2 * i3KernelSize.y + 1) * (2 * i3KernelSize.z + 1);

	int iRoundKernelSizeXToHalfWarp = NR_OF_THREADS_PER_HALF_WARP * int(ceilf( float(i3KernelSize.x) / float(NR_OF_THREADS_PER_HALF_WARP) ));

	#if	0	// MOD-BY-LEETEN 12/20/2009-FROM:
		int iNrOfNeededThreadsPerBlock = min(MAX_THREADS - 2 * iRoundKernelSizeXToHalfWarp, i3VolumeSize.x);
		int iNrOfActualThreadsPerBlock = iNrOfNeededThreadsPerBlock + 2 * iRoundKernelSizeXToHalfWarp;		// #threads (including the real ones and the tailing ones)
		v3Blk = dim3(iNrOfActualThreadsPerBlock, 1, 1);
		v3Grid = dim3(1, i3VolumeSize.y, 1);
		i3Step.x = iNrOfNeededThreadsPerBlock;
		i3Step.y = i3VolumeSize.y;
		i3Step.z = 1;

		fprintf(stderr, "MEM = %d MB; #BLOCKS = %d x %d\n", MAX_MEMORY_SIZE / (1<<20), v3Grid.x, v3Grid.y);
	#else	// MOD-BY-LEETEN 12/20/2009-TO:
	int iNrOfNeededThreadsPerBlock = min(MAX_THREADS - 2 * iRoundKernelSizeXToHalfWarp, i3VolumeSize.x);
	int iNrOfActualThreadsPerBlock = iNrOfNeededThreadsPerBlock + 2 * iRoundKernelSizeXToHalfWarp;		// #threads (including the real ones and the tailing ones)

	v3Blk = dim3(iNrOfActualThreadsPerBlock, 1, 1);

	int iTotalNrOfXBlocks = int(ceilf(float(i3VolumeSize.x) / float(v3Blk.x)));
	int iTotalNrOfYBlocks = int(ceilf(float(i3VolumeSize.y) / float(v3Blk.y)));
	int iTotalNrOfZBlocks = int(ceilf(float(i3VolumeSize.z) / float(v3Blk.z)));
	int iNrOfBlocksInMemory = MAX_MEMORY_SIZE / (sizeof(int) * iNrOfRows * iNrOfActualThreadsPerBlock);
	// ADD-BY-LEETEN 12/20/2009-BEGIN
	int iNrOfAllowableBlocks = int(pow(2.0, double(RADIX_SORT_BITS)) / (iNrOfBins * (i3VolumeSize.x + 2 * iRoundKernelSizeXToHalfWarp)));
	iNrOfBlocksInMemory = min(iNrOfBlocksInMemory, iNrOfAllowableBlocks);
	// ADD-BY-LEETEN 12/20/2009-END
	int iNrOfXBlocksInMemory = min(iTotalNrOfXBlocks, iNrOfBlocksInMemory);
	int iNrOfYBlocksInMemory = min(iTotalNrOfYBlocks, max(1, iNrOfBlocksInMemory / iTotalNrOfXBlocks));
	#if	0	// MOD-BY-LEETEN 12/28/2009-FROM:
		int iNrOfZBlocksInMemory = min(
			MAX_Z_BLOCKS_IN_MEMORY, min(
				iTotalNrOfZBlocks, max(
					1, iNrOfBlocksInMemory / (iTotalNrOfXBlocks * iTotalNrOfYBlocks))));
	#else	// MOD-BY-LEETEN 12/28/2009-TO:
	int iMaxNrOfAllowableZBlocks = max(1, (1<<16) / (iNrOfRows * iNrOfXBlocksInMemory * iNrOfYBlocksInMemory));
	int iNrOfZBlocksInMemory = min(
		iMaxNrOfAllowableZBlocks, min(
			iTotalNrOfZBlocks, max(
				1, iNrOfBlocksInMemory / (iTotalNrOfXBlocks * iTotalNrOfYBlocks))));
	#endif	// MOD-BY-LEETEN 12/28/2009-END

	v3Grid = dim3(iNrOfXBlocksInMemory, iNrOfYBlocksInMemory, iNrOfZBlocksInMemory);
	fprintf(stderr, "#BLOCKS = %d x %d x %d\n", v3Grid.x, v3Grid.y, v3Grid.z);

	i3Step.x = iNrOfXBlocksInMemory * v3Blk.x;
	i3Step.y = iNrOfYBlocksInMemory * v3Blk.y;
	i3Step.z = iNrOfZBlocksInMemory * v3Blk.z;

	fprintf(stderr, "MEM = %d MB; #BLOCKS = %d x %d x %d\n", MAX_MEMORY_SIZE / (1<<20), v3Grid.x, v3Grid.y, v3Grid.z);
	#endif	// MOD-BY-LEETEN 12/20/2009-END
	cudaPitchedPtr cActiveVoxelNeighbors_pitched;
	cActiveVoxelNeighbors_pitched.xsize = v3Blk.x;
	// MOD-BY-LEETEN 12/20/2009-FROM:
		// cActiveVoxelNeighbors_pitched.ysize = iNrOfRows * v3Grid.x * v3Grid.y;
	// TO:
	cActiveVoxelNeighbors_pitched.ysize = iNrOfRows * v3Grid.x * v3Grid.y * v3Grid.z;
	// MOD-BY-LEETEN 12/20/2009-END
	CUDA_SAFE_CALL_NO_SYNC(
		cudaMallocPitch(
			(void**)&cActiveVoxelNeighbors_pitched.ptr, 
			&cActiveVoxelNeighbors_pitched.pitch,
			cActiveVoxelNeighbors_pitched.xsize * sizeof(int), 
			cActiveVoxelNeighbors_pitched.ysize)	);

	// TEST-DEBUG
	fprintf(stderr, "%d = 2 ^ %f", cActiveVoxelNeighbors_pitched.ysize, log2f(float(cActiveVoxelNeighbors_pitched.ysize)));

	// bind the input vin volume to the texture that represents the src. bin volume 
	t2dSrcBinVolume.addressMode[0] = cudaAddressModeClamp;
	t2dSrcBinVolume.addressMode[1] = cudaAddressModeClamp;
	t2dSrcBinVolume.filterMode =	cudaFilterModePoint;
	t2dSrcBinVolume.normalized =	false;
	CUDA_SAFE_CALL_NO_SYNC(
		cudaBindTexture2D(
			0, 
			t2dSrcBinVolume, 
			cBinVolume_pitched.ptr, 
			cudaCreateChannelDesc<int>(),
			cVolumeExtent_array.width, 
			cVolumeExtent_array.height * cVolumeExtent_array.depth, 
			cBinVolume_pitched.pitch) );

	// bind the histogram as a texture
	t2dActiveVoxelSortedNeighbors.addressMode[0] = cudaAddressModeClamp;
	t2dActiveVoxelSortedNeighbors.addressMode[1] = cudaAddressModeClamp;
	t2dActiveVoxelSortedNeighbors.filterMode =	cudaFilterModePoint;
	t2dActiveVoxelSortedNeighbors.normalized =	false;

	CUDA_SAFE_CALL_NO_SYNC(
		cudaBindTexture2D(
			0, 
			t2dActiveVoxelSortedNeighbors, 
			cActiveVoxelNeighbors_pitched.ptr, 
			cudaCreateChannelDesc<unsigned int>(),
			cActiveVoxelNeighbors_pitched.xsize,
			cActiveVoxelNeighbors_pitched.ysize, 
			cActiveVoxelNeighbors_pitched.pitch) );

	CUDPPHandle hScanPlan = 0;

	CUDPPConfiguration	cConfig;
	cConfig.op =		CUDPP_ADD;
	cConfig.datatype =	CUDPP_UINT;
	cConfig.algorithm =	CUDPP_SORT_RADIX;
	cConfig.options =	CUDPP_OPTION_KEYS_ONLY;

	assert( 
		CUDPP_SUCCESS  == cudppPlan(
			&hScanPlan,	
			cConfig, 
			iNrOfRows * cActiveVoxelNeighbors_pitched.xsize,
			1, 
			0) );

CLOCK_END(_ComputeEntropyVolume_PRINT_TIMING, false);

CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_TIMING);

	for(int			z = 0; z < i3VolumeSize.z; z += i3Step.z)
		for(int		y = 0; y < i3VolumeSize.y; y += i3Step.y)
			for(int	x = 0; x < i3VolumeSize.x; x += i3Step.x)
			{
				CLOCK_INIT(_ComputeEntropyVolume_PRINT_LOOP_TIMING, __FUNCTION__ " (main loop): ");

				CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_LOOP_TIMING);

				int3 i3BlockCorner = make_int3(x, y, z);
				_CollectNeighborsAlongScanline_kernel<<<
					// MOD-BY-LEETEN 12/20/2009-FROM:
						// v3Grid, 
					// TO:
					dim3(v3Grid.x, v3Grid.y * v3Grid.z),
					// MOD-BY-LEETEN 12/20/2009-END
					v3Blk, 
					0>>>
				(
					i3BlockCorner,
					i3KernelSize,
					iRoundKernelSizeXToHalfWarp,
					i3VolumeSize,
					// ADD-BY-LEETEN 12/20/2009-BEGIN
					i3Step,
					v3Grid,
					// ADD-BY-LEETEN 12/20/2009-END
					iNrOfBins,
					cActiveVoxelNeighbors_pitched
				);
				CUT_CHECK_ERROR("_CollectNeighborsAlongScanline_kernel() failed");

CLOCK_END(_ComputeEntropyVolume_PRINT_LOOP_TIMING, false);

CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_LOOP_TIMING);

				// call cudpp to sort the buffer _CollectNeighborsToBeSorted_kernel
				// MOD-BY-LEETEN 12/20/2009-FROM:
					// for(int y2 = 0; y2 < int(v3Grid.y); y2++)
				// TO:
				for(int y2 = 0; y2 < int(v3Grid.x * v3Grid.y * v3Grid.z); y2++)
				// MOD-BY-LEETEN 12/20/2009-END
				{
					cudppSort(
						hScanPlan,
						((unsigned int*)cActiveVoxelNeighbors_pitched.ptr) + y2 * iNrOfRows * v3Blk.x,
						NULL,
						// MOD-BY-LEETEN 12/27/2009-FROM:
							// RADIX_SORT_BITS,
						// TO:
						// MOD-BY-LEETEN 12/28/2009-FROM:
							// int(ceilf(log2f(float(iNrOfBins * (i3VolumeSize.x + 2 * iRoundKernelSizeXToHalfWarp))))),
						// TO:
						int(ceilf(log2f(float(iNrOfBins * v3Blk.x)))),
						// MOD-BY-LEETEN 12/28/2009-END
						// MOD-BY-LEETEN 12/27/2009-END
						iNrOfRows * v3Blk.x) ;
					CUT_CHECK_ERROR("cudppSort() failed");
				}

CLOCK_END(_ComputeEntropyVolume_PRINT_LOOP_TIMING, false);

CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_LOOP_TIMING);
				// 
				// ADD-BY-LEETEN 12/27/2009-BEGIN
				#if	SCANNING_METHOD == SCANNING_METHOD_SCAN_WHOLE_HISTOGRAM
				// ADD-BY-LEETEN 12/27/2009-END

				_ComputeEntropyAlongScanline_kernel<<<
					// MOD-BY-LEETEN 12/20/2009-FROM:
						// v3Grid, 
					// TO:
					dim3(v3Grid.x, v3Grid.y * v3Grid.z),
					// MOD-BY-LEETEN 12/20/2009-END
					dim3(iNrOfNeededThreadsPerBlock, 1, 1),
					0>>>
				(
					i3BlockCorner,
					iNrOfBins,
					i3KernelSize,
					iRoundKernelSizeXToHalfWarp,
					i3Step,
					// ADD-BY-LEETEN 12/20/2009-BEGIN
					v3Grid,
					// ADD-BY-LEETEN 12/20/2009-END
					cActiveVoxelNeighbors_pitched,
					i3VolumeSize,
					cEntropyVolume_pitched
				);
				CUT_CHECK_ERROR("_ComputeEntropyOnSortedNeighbors_kernel() failed");

				// ADD-BY-LEETEN 12/27/2009-BEGIN
				#endif	// #if	SCANNING_METHOD == SCANNING_METHOD_SCAN_WHOLE_HISTOGRAM

				#if	SCANNING_METHOD == SCANNING_METHOD_SCAN_ROWS_IN_PARALLEL

				_ComputeEntropyAlongScanlineInParallel_kernel<<<
					dim3(v3Grid.x, v3Grid.y * v3Grid.z),
					dim3(iNrOfActualThreadsPerBlock, 1, 1)
					>>>
 				(
					i3BlockCorner,
					iNrOfBins,
					i3KernelSize,
					iRoundKernelSizeXToHalfWarp,
					i3Step,
					v3Grid,
					cActiveVoxelNeighbors_pitched,
					i3VolumeSize,
					cEntropyVolume_pitched
				);
				CUT_CHECK_ERROR("_ComputeEntropyAlongScanlineInParallel_kernel() failed");
				#endif	// #if	SCANNING_METHOD == SCANNING_METHOD_SCAN_ROWS_IN_PARALLEL

CLOCK_END(_ComputeEntropyVolume_PRINT_LOOP_TIMING, false);

CLOCK_PRINT(_ComputeEntropyVolume_PRINT_LOOP_TIMING);
			}


CLOCK_END(_ComputeEntropyVolume_PRINT_TIMING, false);

CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_TIMING);

	cudppDestroyPlan(hScanPlan);
	FREE_MEMORY(cActiveVoxelNeighbors_pitched.ptr);

CLOCK_END(_ComputeEntropyVolume_PRINT_TIMING, false);

CLOCK_PRINT(_ComputeEntropyVolume_PRINT_TIMING);
}
	
#endif	// #if	COMPUTE_ENTROPY_VOLUME_PER_SCANLINE_CUDA

/*

$Log: not supported by cvs2svn $
Revision 1.1  2009/12/27 19:35:53  leeten

[12/27/2009]
1. [1ST] First time checkin. Ths file define the host and kernel functions to compute the netropy field by sorting all the YZ neighbors of all threads along the X scanline via CUDPP.


*/
