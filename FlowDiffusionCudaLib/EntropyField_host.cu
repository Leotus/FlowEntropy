
#include <hip/hip_runtime.h>
#if	COMPUTE_ENTROPY_VOLUME_HOST

int iNrOfMargins;
int iNrOfMarginalBins;
static 	TBuffer<int> piMarginalHistogram_host;

__host__
static 
int 
IMirrorCoord_host(int iCoord, int iSize)
{
	int iMirroredCoord = iCoord;
	if( iCoord < 0 )
		iMirroredCoord = -iCoord;
	if( iCoord > iSize - 1 )
		iMirroredCoord = iSize - (iCoord - (iSize - 1));
	return iMirroredCoord;
}

static 
void 
_UpdateSliceToHistogram_host
(
	int iU,
	int iV,

	int3 i3Center, 
	int	iUpdateDir, 
	int iHisotgramOp, 

	int3 i3UDir, 
	int3 i3VDir,
	int iUKernelSize, 
	int iVKernelSize,

	int3 i3VolumeSize,
	int *piBinVolume_host,

	int iNrOfBins,
	int *piHistorgram_host
)
{
	int3 i3Point;
	i3Point.x = i3Center.x + (iU - iUKernelSize) * i3UDir.x + (iV - iVKernelSize) * i3VDir.x;
	i3Point.y = i3Center.y + (iU - iUKernelSize) * i3UDir.y + (iV - iVKernelSize) * i3VDir.y;
	i3Point.z = i3Center.z + (iU - iUKernelSize) * i3UDir.z + (iV - iVKernelSize) * i3VDir.z;

	// read the bin
	int3 i3TexCoord;
	i3TexCoord.x = IMirrorCoord_host(i3Point.x, i3VolumeSize.x);
	i3TexCoord.y = IMirrorCoord_host(i3Point.y, i3VolumeSize.y);
	i3TexCoord.z = IMirrorCoord_host(i3Point.z, i3VolumeSize.z);

	int iSrcBin = piBinVolume_host[i3TexCoord.x + i3TexCoord.y * i3VolumeSize.x + i3TexCoord.z * i3VolumeSize.x * i3VolumeSize.y];

	// update the histogram
	piHistorgram_host[iSrcBin] += iHisotgramOp; 

	piMarginalHistogram_host[iSrcBin / iNrOfMargins] += iHisotgramOp; 
}

static 
void
_BuildHistogram_host
(
	int3 i3Center,
						// res. of the neighboring region
	int3 i3KernelSize,	
						// the bin volume
	int3 i3VolumeSize,	
	int *piBinVolume_host,								
						// res. of the joint histogram
	int iNrOfBins,		
	int *piHistorgram_host
)
{
	int3 i3Point = i3Center;
	i3Point.x -= i3KernelSize.x;
	for(int			xi = 0; xi < 2 * i3KernelSize.x + 1; xi++, i3Point.x++)
		for(int		iV = 0; iV < 2 * i3KernelSize.z + 1; iV++)
			for(int iU = 0; iU < 2 * i3KernelSize.y + 1; iU++)
				_UpdateSliceToHistogram_host(
					iU,
					iV,

					i3Point , 
					UPDATE_DIR_X, 
					HISTOGRAM_OP_ADD_SLICE, 

					make_int3(0, 1, 0), 
					make_int3(0, 0, 1),
					i3KernelSize.y, 
					i3KernelSize.z,

					i3VolumeSize,
					piBinVolume_host,

					iNrOfBins,
					piHistorgram_host
				);
}

static 
void 
_UpdateSliceToHistogram_host
(
	int3 i3Point,										// the coordinate to be computed

	int iUpdateDir, 
	int iDir,

	int3 i3KernelSize,	// res. of the neighboring region

	int3 i3VolumeSize,	// res. of the volume
	int *piBinVolume_host,								


	int iNrOfBins,		// res. of the joint histogram
	int *piHistogram_host
)
{
	int3 i3Prev = i3Point;
	int3 i3Next = i3Point;
	int3 i3UDir;
	int3 i3VDir;
	int iUKernelSize ;
	int iVKernelSize ;

						// according the direction to divide the new slice into blocks
	switch(iUpdateDir)
	{
	case UPDATE_DIR_X:	
		i3UDir = make_int3(0, 1, 0);
		i3VDir = make_int3(0, 0, 1);
		iUKernelSize = i3KernelSize.y;
		iVKernelSize = i3KernelSize.z;
		i3Prev.x -= iDir * (i3KernelSize.x + 1);
		i3Next.x += iDir * i3KernelSize.x;
		break;
	case UPDATE_DIR_Y:	
		i3UDir = make_int3(1, 0, 0);
		i3VDir = make_int3(0, 0, 1);
		iUKernelSize = i3KernelSize.x;
		iVKernelSize = i3KernelSize.z;
		i3Prev.y -= iDir * (i3KernelSize.y + 1);
		i3Next.y += iDir * i3KernelSize.y;
		break;
	case UPDATE_DIR_Z:	
		i3UDir = make_int3(1, 0, 0);
		i3VDir = make_int3(0, 1, 0);
		iUKernelSize = i3KernelSize.x;
		iVKernelSize = i3KernelSize.y;
		i3Prev.z -= iDir * (i3KernelSize.z + 1);
		i3Next.z += iDir * i3KernelSize.z;
		break;
	} // switch

	for(int		iV = 0; iV < 2 * iVKernelSize + 1; iV++)
		for(int iU = 0; iU < 2 * iUKernelSize + 1; iU++)
		{
			_UpdateSliceToHistogram_host
			(
				iU,
				iV,

				i3Next, 
				iUpdateDir, 
				HISTOGRAM_OP_ADD_SLICE, 

				i3UDir, 
				i3VDir,
				iUKernelSize, 
				iVKernelSize,

				i3VolumeSize,
				piBinVolume_host,

				iNrOfBins,
				piHistogram_host
			);

			_UpdateSliceToHistogram_host
			(
				iU,
				iV,

				i3Prev, 
				iUpdateDir, 
				HISTOGRAM_OP_SUB_SLICE, 

				i3UDir, 
				i3VDir,
				iUKernelSize, 
				iVKernelSize,

				i3VolumeSize,
				piBinVolume_host,

				iNrOfBins,
				piHistogram_host
			);
		}
}

static 
void
_ComputeEntropy_host
(
	int3 i3Point,
	int3 i3KernelSize,

						// the histogram
	int iNrOfBins,
	int *piHistogram_host,

	int3 i3VolumeSize,
	float* pfEntropyVolume_host
)
{
	float fEntropy = 0.0f;

	#if	SCANNING_METHOD == SCANNING_METHOD_SCAN_WHOLE_HISTOGRAM
	for(int b = 0; b < iNrOfBins; b++)
	{
		if( 0 == piHistogram_host[b] )
			continue;

		float fProb = 
			float(piHistogram_host[b]) / 
			float(
				(2 * i3KernelSize.x + 1) * 
				(2 * i3KernelSize.y + 1) * 
				(2 * i3KernelSize.z + 1) );
		fEntropy += fProb * log2f(fProb);
	}
	fEntropy = -fEntropy;

	#endif
	
	#if	SCANNING_METHOD == SCANNING_METHOD_SKIP_WITH_MARGINAL_HISTOGRAM
	for(int mb = 0; mb < iNrOfMarginalBins; mb++)
	{
		if( 0 == piMarginalHistogram_host[mb] )
			continue;

		for(int b = mb * iNrOfMargins, b0 = 0; b < iNrOfBins && b0 < iNrOfMargins; b0++, b++)
		{
			if( 0 == piHistogram_host[b] )
				continue;

			float fCount = float(piHistogram_host[b]);
			fEntropy += fCount * log2f(fCount);
		}
	}
	float fNrOfNeighbors = 	float(
		(2 * i3KernelSize.x + 1) * 
		(2 * i3KernelSize.y + 1) * 
		(2 * i3KernelSize.z + 1) );

	fEntropy = -fEntropy / fNrOfNeighbors + log2(fNrOfNeighbors);
	#endif

	fEntropy = max(fEntropy, 0.0f);

	pfEntropyVolume_host[i3Point.x + i3Point.y * i3VolumeSize.x + i3Point.z * i3VolumeSize.x * i3VolumeSize.y] = fEntropy;
}

void 
_ComputeEntropyVolume_host
(
						// res. of the neighboring region
	int3 i3KernelSize,

						// the histogram
	int iNrOfBins,
	int *piHistogram_global,
	float *pfLogHistogram_global,

						// res. of the volume																
	int3 i3VolumeSize,	
						// bin volume																	
	cudaPitchedPtr		cBinVolume_pitched,								
	cudaPitchedPtr		cEntropyVolume_pitched
)
{
CLOCK_INIT(_ComputeEntropyVolume_PRINT_TIMING, __FUNCTION__ ": ");

CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_TIMING);

	int *piHistogram_host;
	piHistogram_host = (int*)calloc(sizeof(piHistogram_host[0]), iNrOfBins);
	assert(piHistogram_host);

	iNrOfMargins = int(sqrtf(float(iNrOfBins)));
	iNrOfMarginalBins = int(ceilf(float(iNrOfBins) / float(iNrOfMargins)));

	piMarginalHistogram_host.alloc(iNrOfMarginalBins);

	int *piBinVolume_host;
	piBinVolume_host = (int*)calloc(sizeof(piBinVolume_host[0]), i3VolumeSize.x * i3VolumeSize.y * i3VolumeSize.z);
	assert(piBinVolume_host);

	_GetSrcBinVolume(piBinVolume_host);

	float* pfEntropyVolume_host;
	pfEntropyVolume_host = (float*)calloc(sizeof(pfEntropyVolume_host[0]), i3VolumeSize.x * i3VolumeSize.y * i3VolumeSize.z);
	assert( pfEntropyVolume_host );

CLOCK_END(_ComputeEntropyVolume_PRINT_TIMING, false);

CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_TIMING);

	int xdir = +1;
	int ydir = +1;
	int zdir = +1;
	int x = 0;
	int y = 0;
	int z = 0;

	_BuildHistogram_host
	(
		make_int3(x, y, z),
							// res. of the neighboring region
		i3KernelSize,	
							// the bin volume
		i3VolumeSize,	
		piBinVolume_host,
							// the joint histogram
		iNrOfBins,		
		piHistogram_host
	);
CLOCK_END(_ComputeEntropyVolume_PRINT_TIMING, false);

CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_TIMING);

	for(int				zi = 0; zi < i3VolumeSize.z;	zi++, z += zdir)
	{
		for(int			yi = 0; yi < i3VolumeSize.y;	yi++, y += ydir)
		{
			for(int		xi = 0; xi < i3VolumeSize.x;	xi++, x += xdir)
			{
				// fprintf(stderr, "%d, %d, %d\n", x, y, z);
				// (x,y,z): center of the volume
				// before enter this loop, the histogram in the region centering at (x, y, z) should have been available
				_ComputeEntropy_host
				(
					make_int3(x, y, z),
					i3KernelSize,

										// the joint histogram
					iNrOfBins,
					piHistogram_host,

					i3VolumeSize,
					pfEntropyVolume_host
				);

				if( xi < i3VolumeSize.x - 1 )
					_UpdateSliceToHistogram_host(
						make_int3(x + xdir, y, z), 
						UPDATE_DIR_X,	
						xdir, 

						i3KernelSize, 

											// the bin volume
						i3VolumeSize,	
						piBinVolume_host,
											// the joint histogram
						iNrOfBins,		
						piHistogram_host
						);
			}
			x -= xdir;
			xdir *= -1;
			if( yi < i3VolumeSize.y - 1 )
				_UpdateSliceToHistogram_host(
					make_int3(x, y + ydir, z), 
					UPDATE_DIR_Y,	
					ydir, 
					
					i3KernelSize, 

										// the bin volume
					i3VolumeSize,	
					piBinVolume_host,
										// the joint histogram
					iNrOfBins,		
					piHistogram_host
					);
		}
		y -= ydir;
		ydir *= -1;
		if( zi < i3VolumeSize.z - 1 )
			_UpdateSliceToHistogram_host(
				make_int3(x, y, z + zdir), 
				UPDATE_DIR_Z,	
				zdir, 

				i3KernelSize, 

											// the bin volume
				i3VolumeSize,	
				piBinVolume_host,
									// the joint histogram
				iNrOfBins,		
				piHistogram_host
			);
	}
CLOCK_END(_ComputeEntropyVolume_PRINT_TIMING, false);

CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_TIMING);
	CUDA_SAFE_CALL_NO_SYNC(
		cudaMemcpy2D(
			cEntropyVolume_pitched.ptr, 
			cEntropyVolume_pitched.pitch,
			pfEntropyVolume_host,
			i3VolumeSize.x * sizeof(pfEntropyVolume_host[0]),
			i3VolumeSize.x * sizeof(pfEntropyVolume_host[0]),
			i3VolumeSize.y * i3VolumeSize.z,
			cudaMemcpyHostToDevice) );
	free(pfEntropyVolume_host);

	free(piHistogram_host);
	free(piBinVolume_host);
CLOCK_END(_ComputeEntropyVolume_PRINT_TIMING, false);

CLOCK_PRINT(_ComputeEntropyVolume_PRINT_TIMING);
}

#endif	// #if #if	COMPUTE_ENTROPY_VOLUME_HOST

/*

$Log: not supported by cvs2svn $
Revision 1.2  2009/12/31 02:43:30  leeten

[12/30/2009]
1. [DEL] Remvoe the debug statement.

Revision 1.1  2009/12/27 19:05:04  leeten

[12/27/2009]
1. [1ST] First time checkin. Ths file define the functions to compute the entropy field on CPU.


*/
