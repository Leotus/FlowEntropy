
#include <hip/hip_runtime.h>
#if	COMPUTE_ENTROPY_VOLUME_HOST

__host__
static 
int 
IMirrorCoord_host(int iCoord, int iSize)
{
	int iMirroredCoord = iCoord;
	if( iCoord < 0 )
		iMirroredCoord = -iCoord;
	if( iCoord > iSize - 1 )
		iMirroredCoord = iSize - (iCoord - (iSize - 1));
	return iMirroredCoord;
}

static 
void 
_UpdateSliceToHistogram_host
(
	int iU,
	int iV,

	int3 i3Center, 
	int	iUpdateDir, 
	int iHisotgramOp, 

	int3 i3UDir, 
	int3 i3VDir,
	int iUKernelSize, 
	int iVKernelSize,

	int3 i3VolumeSize,
	int *piBinVolume_host,

	int iNrOfBins,
	int *piHistorgram_host
)
{
	int3 i3Point;
	i3Point.x = i3Center.x + (iU - iUKernelSize) * i3UDir.x + (iV - iVKernelSize) * i3VDir.x;
	i3Point.y = i3Center.y + (iU - iUKernelSize) * i3UDir.y + (iV - iVKernelSize) * i3VDir.y;
	i3Point.z = i3Center.z + (iU - iUKernelSize) * i3UDir.z + (iV - iVKernelSize) * i3VDir.z;

	// read the bin
	int3 i3TexCoord;
	i3TexCoord.x = IMirrorCoord_host(i3Point.x, i3VolumeSize.x);
	i3TexCoord.y = IMirrorCoord_host(i3Point.y, i3VolumeSize.y);
	i3TexCoord.z = IMirrorCoord_host(i3Point.z, i3VolumeSize.z);

	int iSrcBin = piBinVolume_host[i3TexCoord.x + i3TexCoord.y * i3VolumeSize.x + i3TexCoord.z * i3VolumeSize.x * i3VolumeSize.y];

	// update the histogram
	piHistorgram_host[iSrcBin] += iHisotgramOp; 
}

static 
void
_BuildHistogram_host
(
	int3 i3Center,
						// res. of the neighboring region
	int3 i3KernelSize,	
						// the bin volume
	int3 i3VolumeSize,	
	int *piBinVolume_host,								
						// res. of the joint histogram
	int iNrOfBins,		
	int *piHistorgram_host
)
{
	int3 i3Point = i3Center;
	i3Point.x -= i3KernelSize.x;
	for(int			xi = 0; xi < 2 * i3KernelSize.x + 1; xi++, i3Point.x++)
		for(int		iV = 0; iV < 2 * i3KernelSize.z + 1; iV++)
			for(int iU = 0; iU < 2 * i3KernelSize.y + 1; iU++)
				_UpdateSliceToHistogram_host(
					iU,
					iV,

					i3Point , 
					UPDATE_DIR_X, 
					HISTOGRAM_OP_ADD_SLICE, 

					make_int3(0, 1, 0), 
					make_int3(0, 0, 1),
					i3KernelSize.y, 
					i3KernelSize.z,

					i3VolumeSize,
					piBinVolume_host,

					iNrOfBins,
					piHistorgram_host
				);
}

static 
void 
_UpdateSliceToHistogram_host
(
	int3 i3Point,										// the coordinate to be computed

	int iUpdateDir, 
	int iDir,

	int3 i3KernelSize,	// res. of the neighboring region

	int3 i3VolumeSize,	// res. of the volume
	int *piBinVolume_host,								


	int iNrOfBins,		// res. of the joint histogram
	int *piHistogram_host
)
{
	int3 i3Prev = i3Point;
	int3 i3Next = i3Point;
	int3 i3UDir;
	int3 i3VDir;
	int iUKernelSize ;
	int iVKernelSize ;

						// according the direction to divide the new slice into blocks
	switch(iUpdateDir)
	{
	case UPDATE_DIR_X:	
		i3UDir = make_int3(0, 1, 0);
		i3VDir = make_int3(0, 0, 1);
		iUKernelSize = i3KernelSize.y;
		iVKernelSize = i3KernelSize.z;
		i3Prev.x -= iDir * (i3KernelSize.x + 1);
		i3Next.x += iDir * i3KernelSize.x;
		break;
	case UPDATE_DIR_Y:	
		i3UDir = make_int3(1, 0, 0);
		i3VDir = make_int3(0, 0, 1);
		iUKernelSize = i3KernelSize.x;
		iVKernelSize = i3KernelSize.z;
		i3Prev.y -= iDir * (i3KernelSize.y + 1);
		i3Next.y += iDir * i3KernelSize.y;
		break;
	case UPDATE_DIR_Z:	
		i3UDir = make_int3(1, 0, 0);
		i3VDir = make_int3(0, 1, 0);
		iUKernelSize = i3KernelSize.x;
		iVKernelSize = i3KernelSize.y;
		i3Prev.z -= iDir * (i3KernelSize.z + 1);
		i3Next.z += iDir * i3KernelSize.z;
		break;
	} // switch

	for(int		iV = 0; iV < 2 * iVKernelSize + 1; iV++)
		for(int iU = 0; iU < 2 * iUKernelSize + 1; iU++)
		{
			_UpdateSliceToHistogram_host
			(
				iU,
				iV,

				i3Next, 
				iUpdateDir, 
				HISTOGRAM_OP_ADD_SLICE, 

				i3UDir, 
				i3VDir,
				iUKernelSize, 
				iVKernelSize,

				i3VolumeSize,
				piBinVolume_host,

				iNrOfBins,
				piHistogram_host
			);

			_UpdateSliceToHistogram_host
			(
				iU,
				iV,

				i3Prev, 
				iUpdateDir, 
				HISTOGRAM_OP_SUB_SLICE, 

				i3UDir, 
				i3VDir,
				iUKernelSize, 
				iVKernelSize,

				i3VolumeSize,
				piBinVolume_host,

				iNrOfBins,
				piHistogram_host
			);
		}
}

static 
void
_ComputeEntropy_host
(
	int3 i3Point,
	int3 i3KernelSize,

						// the histogram
	int iNrOfBins,
	int *piHistogram_host,

	int3 i3VolumeSize,
	// MOD-BY-LEETEN 12/19/2009-FROM:
		// cudaPitchedPtr cEntropyVolume_pitched
	// TO:
	float* pfEntropyVolume_host
	// MOD-BY-LEETEN 12/19/2009-END
)
{
	float fEntropy = 0.0f;
	for(int b = 0; b < iNrOfBins; b++)
	{
		if( 0 == piHistogram_host[b] )
			continue;

		float fProb = 
			float(piHistogram_host[b]) / 
			float(
				(2 * i3KernelSize.x + 1) * 
				(2 * i3KernelSize.y + 1) * 
				(2 * i3KernelSize.z + 1) );
		fEntropy += fProb * log2f(fProb);
	}
	fEntropy = -fEntropy;
	// ADD-BY-LEETEN 12/19/2009-BEGIN
	fEntropy = max(fEntropy, 0.0f);
	// ADD-BY-LEETEN 12/19/2009-END

	// fprintf(stderr, "Entropy(%d, %d, %d) = %f\n", i3Point.x, i3Point.y, i3Point.z, fEntropy); // TEST-DEBUG

	#if	0	// MOD-BY-LEETEN 12/19/2009-FROM:
		CUDA_SAFE_CALL(
			cudaMemcpy(
				ADDRESS_2D(
					float, cEntropyVolume_pitched.ptr, 
					sizeof(float), cEntropyVolume_pitched.pitch, 
					i3Point.x, i3Point.y + i3Point.z * i3VolumeSize.y),
				&fEntropy,
				sizeof(fEntropy),
				cudaMemcpyHostToDevice) );
	#else	// MOD-BY-LEETEN 12/19/2009-TO:
	pfEntropyVolume_host[i3Point.x + i3Point.y * i3VolumeSize.x + i3Point.z * i3VolumeSize.x * i3VolumeSize.y] = fEntropy;
	#endif	// MOD-BY-LEETEN 12/19/2009-END
}

// ADD-BY-LEETEN 12/18/2009-BEGIN

void 
// MOD-BY-LEETEN 12/18/2009-FROM:
	// _ComputeEntropyVolume
// TO:
_ComputeEntropyVolume_host
// MOD-BY-LEETEN 12/18/2009-END
(
						// res. of the neighboring region
	int3 i3KernelSize,

						// the histogram
	int iNrOfBins,
	int *piHistogram_global,
	float *pfLogHistogram_global,

	// DEL-BY-LEETEN 12/18/2009-BEGIN
		// CUDPPHandle cScanPlanSum,
	// DEL-BY-LEETEN 12/18/2009-END

						// res. of the volume																
	int3 i3VolumeSize,	
						// bin volume																	
	cudaPitchedPtr		cBinVolume_pitched,								
	cudaPitchedPtr		cEntropyVolume_pitched
)
{
CLOCK_INIT(_ComputeEntropyVolume_PRINT_TIMING, __FUNCTION__ ": ");

CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_TIMING);

	int *piHistogram_host;
	piHistogram_host = (int*)calloc(sizeof(piHistogram_host[0]), iNrOfBins);
	assert(piHistogram_host);

	int *piBinVolume_host;
	piBinVolume_host = (int*)calloc(sizeof(piBinVolume_host[0]), i3VolumeSize.x * i3VolumeSize.y * i3VolumeSize.z);
	assert(piBinVolume_host);

	_GetSrcBinVolume(piBinVolume_host);

	// ADD-BY-LEETEN 12/19/2009-BEGIN
	float* pfEntropyVolume_host;
	pfEntropyVolume_host = (float*)calloc(sizeof(pfEntropyVolume_host[0]), i3VolumeSize.x * i3VolumeSize.y * i3VolumeSize.z);
	assert( pfEntropyVolume_host );
	// ADD-BY-LEETEN 12/19/2009-END

CLOCK_END(_ComputeEntropyVolume_PRINT_TIMING, false);

CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_TIMING);

	int xdir = +1;
	int ydir = +1;
	int zdir = +1;
	int x = 0;
	int y = 0;
	int z = 0;

	_BuildHistogram_host
	(
		make_int3(x, y, z),
							// res. of the neighboring region
		i3KernelSize,	
							// the bin volume
		i3VolumeSize,	
		piBinVolume_host,
							// the joint histogram
		iNrOfBins,		
		piHistogram_host
	);
CLOCK_END(_ComputeEntropyVolume_PRINT_TIMING, false);

CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_TIMING);

	for(int				zi = 0; zi < i3VolumeSize.z;	zi++, z += zdir)
	{
		for(int			yi = 0; yi < i3VolumeSize.y;	yi++, y += ydir)
		{
			for(int		xi = 0; xi < i3VolumeSize.x;	xi++, x += xdir)
			{
				// fprintf(stderr, "%d, %d, %d\n", x, y, z);
				// (x,y,z): center of the volume
				// before enter this loop, the histogram in the region centering at (x, y, z) should have been available
				_ComputeEntropy_host
				(
					make_int3(x, y, z),
					i3KernelSize,

										// the joint histogram
					iNrOfBins,
					piHistogram_host,

					i3VolumeSize,
					// MOD-BY-LEETEN 12/19/2009-FROM:
						// cEntropyVolume_pitched
					// TO:
					pfEntropyVolume_host
					// MOD-BY-LEETEN 12/19/2009-END
				);

				if( xi < i3VolumeSize.x - 1 )
					_UpdateSliceToHistogram_host(
						make_int3(x + xdir, y, z), 
						UPDATE_DIR_X,	
						xdir, 

						i3KernelSize, 

											// the bin volume
						i3VolumeSize,	
						piBinVolume_host,
											// the joint histogram
						iNrOfBins,		
						piHistogram_host
						);
			}
			x -= xdir;
			xdir *= -1;
			if( yi < i3VolumeSize.y - 1 )
				_UpdateSliceToHistogram_host(
					make_int3(x, y + ydir, z), 
					UPDATE_DIR_Y,	
					ydir, 
					
					i3KernelSize, 

										// the bin volume
					i3VolumeSize,	
					piBinVolume_host,
										// the joint histogram
					iNrOfBins,		
					piHistogram_host
					);
		}
		y -= ydir;
		ydir *= -1;
		if( zi < i3VolumeSize.z - 1 )
			_UpdateSliceToHistogram_host(
				make_int3(x, y, z + zdir), 
				UPDATE_DIR_Z,	
				zdir, 

				i3KernelSize, 

											// the bin volume
				i3VolumeSize,	
				piBinVolume_host,
									// the joint histogram
				iNrOfBins,		
				piHistogram_host
			);
	}
CLOCK_END(_ComputeEntropyVolume_PRINT_TIMING, false);

CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_TIMING);
	// ADD-BY-LEETEN 12/19/2009-BEGIN
	CUDA_SAFE_CALL_NO_SYNC(
		cudaMemcpy2D(
			cEntropyVolume_pitched.ptr, 
			cEntropyVolume_pitched.pitch,
			pfEntropyVolume_host,
			i3VolumeSize.x * sizeof(pfEntropyVolume_host[0]),
			i3VolumeSize.x * sizeof(pfEntropyVolume_host[0]),
			i3VolumeSize.y * i3VolumeSize.z,
			cudaMemcpyHostToDevice) );
	free(pfEntropyVolume_host);
	// ADD-BY-LEETEN 12/19/2009-END

	free(piHistogram_host);
	free(piBinVolume_host);
CLOCK_END(_ComputeEntropyVolume_PRINT_TIMING, false);

CLOCK_PRINT(_ComputeEntropyVolume_PRINT_TIMING);
}
// ADD-BY-LEETEN 12/18/2009-END

#endif	// #if #if	COMPUTE_ENTROPY_VOLUME_HOST

/*

$Log: not supported by cvs2svn $

*/
