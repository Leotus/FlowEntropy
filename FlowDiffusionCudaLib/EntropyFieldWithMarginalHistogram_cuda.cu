
#include <hip/hip_runtime.h>
#if	COMPUTE_ENTROPY_VOLUME_WITH_MARGINAL_HISTOGRAM	

__global__ 
static 
void 
_UpdateHistogramWithMarginalHistogram_kernel
(
	int3 i3BlockCorner,

	int3 i3KernelSize,

	int3 i3VolumeSize,

	int iNrOfBins,
	int iNrOfMarginalBins,

	cudaPitchedPtr cActiveVoxelsHistorgram_pitched
)
{
	int iActiveVoxelX = blockIdx.x * blockDim.x + threadIdx.x;
	int iActiveVoxelY = blockIdx.y * blockDim.y + threadIdx.y;
	int iActiveVoxelId = iActiveVoxelX + iActiveVoxelY * gridDim.x * blockDim.x;
 	int iVoxelX = i3BlockCorner.x + iActiveVoxelX;
	int iVoxelY = i3BlockCorner.y + iActiveVoxelY;
	int iVoxelZ = i3BlockCorner.z;

	if( iVoxelX < i3VolumeSize.x && iVoxelY < i3VolumeSize.y && iVoxelZ < i3VolumeSize.z  )

		for(int iZDir = -1; iZDir <= +1; iZDir+=2)
		{
			int iZOffset = i3KernelSize.z * iZDir;
			switch (iZDir)
			{
			case -1:	iZOffset--;		break;
			}
			for(int		iYOffset = -i3KernelSize.y; iYOffset <= i3KernelSize.y; iYOffset++)
				for(int iXOffset = -i3KernelSize.x; iXOffset <= i3KernelSize.x; iXOffset++)
				{
					int iX, iY, iZ;
					iX = iVoxelX + iXOffset;
					iY = iVoxelY + iYOffset;
					iZ = iVoxelZ + iZOffset;

					int3 i3TexCoord;
					i3TexCoord.x = IMirrorCoord(iX, i3VolumeSize.x);
					i3TexCoord.y = IMirrorCoord(iY, i3VolumeSize.y);
					i3TexCoord.z = IMirrorCoord(iZ, i3VolumeSize.z);

					int iSrcBin = tex2D(t2dSrcBinVolume, i3TexCoord.x, i3TexCoord.y + i3TexCoord.z * i3VolumeSize.y);
					int iMarginalBin = iSrcBin / iNrOfMarginalBins;

										// update the joint histogram
					atomicAdd(
						ADDRESS_2D(
							int,			cActiveVoxelsHistorgram_pitched.ptr, 
							sizeof(int),	cActiveVoxelsHistorgram_pitched.pitch, 
							iActiveVoxelId, iSrcBin + iNrOfMarginalBins),
						iZDir);

					atomicAdd(
						ADDRESS_2D(
							int,			cActiveVoxelsHistorgram_pitched.ptr, 
							sizeof(int),	cActiveVoxelsHistorgram_pitched.pitch, 
							iActiveVoxelId, iMarginalBin),
						iZDir);
				}
		}
}

__global__
void
_CreateHistogramWithMarginalHistogram_kernel
(
	int3 i3BlockCorner,

	int3 i3KernelSize,

	int3 i3VolumeSize,

	int iNrOfBins,
	int iNrOfMarginalBins,
	cudaPitchedPtr cActiveVoxelsHistorgram_pitched
)
{
	int iActiveVoxelX = blockIdx.x * blockDim.x + threadIdx.x;
	int iActiveVoxelY = blockIdx.y * blockDim.y + threadIdx.y;
	int iActiveVoxelId = iActiveVoxelX + iActiveVoxelY * gridDim.x * blockDim.x;
 	int iVoxelX = i3BlockCorner.x + iActiveVoxelX;
	int iVoxelY = i3BlockCorner.y + iActiveVoxelY;
	int iVoxelZ = i3BlockCorner.z;

	if( iVoxelX < i3VolumeSize.x && iVoxelY < i3VolumeSize.y && iVoxelZ < i3VolumeSize.z  )
		for(int				iZOffset = -i3KernelSize.z; iZOffset <= i3KernelSize.z; iZOffset++)
			for(int			iYOffset = -i3KernelSize.y; iYOffset <= i3KernelSize.y; iYOffset++)
				for(int		iXOffset = -i3KernelSize.x; iXOffset <= i3KernelSize.x; iXOffset++)
				{
					int iX, iY, iZ;
					iX = iVoxelX + iXOffset;
					iY = iVoxelY + iYOffset;
					iZ = iVoxelZ + iZOffset;

					int3 i3TexCoord;
					i3TexCoord.x = IMirrorCoord(iX, i3VolumeSize.x);
					i3TexCoord.y = IMirrorCoord(iY, i3VolumeSize.y);
					i3TexCoord.z = IMirrorCoord(iZ, i3VolumeSize.z);

					int iSrcBin = tex2D(t2dSrcBinVolume, i3TexCoord.x, i3TexCoord.y + i3TexCoord.z * i3VolumeSize.y);
					int iMarginalBin = iSrcBin / iNrOfMarginalBins;

										// update the joint histogram
					atomicAdd(
						ADDRESS_2D(
							int,			cActiveVoxelsHistorgram_pitched.ptr, 
							sizeof(int),	cActiveVoxelsHistorgram_pitched.pitch, 
							iActiveVoxelId, iSrcBin + iNrOfMarginalBins),
						+1.0);

					atomicAdd(
						ADDRESS_2D(
							int,			cActiveVoxelsHistorgram_pitched.ptr, 
							sizeof(int),	cActiveVoxelsHistorgram_pitched.pitch, 
							iActiveVoxelId, iMarginalBin),
						+1.0);
				}
}

__global__ 
void 
_ComputeEntropyWithMarginalHistogram_kernel
(
	int3 i3BlockCorner,
	int iNrOfBins,
	int iNrOfMarginalBins,
	int3 i3KernelSize,
	int3 i3VolumeSize,
	cudaPitchedPtr cEntropyVolume_pitched
)
{
	int iActiveVoxelX = blockIdx.x * blockDim.x + threadIdx.x;
	int iActiveVoxelY = blockIdx.y * blockDim.y + threadIdx.y;
	int iActiveVoxelId = iActiveVoxelX + iActiveVoxelY * gridDim.x * blockDim.x;
 	int iVoxelX = i3BlockCorner.x + iActiveVoxelX;
	int iVoxelY = i3BlockCorner.y + iActiveVoxelY;
	int iVoxelZ = i3BlockCorner.z;

	int iNrOfNeighbors = (2 * i3KernelSize.x + 1) * (2 * i3KernelSize.y + 1) * (2 * i3KernelSize.z + 1);
	float fNrOfNeighbors = float(iNrOfNeighbors);

	int iNrOfMargins = iNrOfBins / iNrOfMarginalBins;
	float fEntropy = 0.0f;
	for(int b0 = 0; b0 < iNrOfMarginalBins; b0++)
	{
		int iCount = tex2D(t2dActiveVoxelHistorgrams, iActiveVoxelId, b0);
		if( 0 == iCount )
			continue;

		for(int		b1 = 0; b1 < iNrOfMargins;		b1++)
		{
			int iCount = tex2D(t2dActiveVoxelHistorgrams, iActiveVoxelId, iNrOfMarginalBins + b0 * iNrOfMargins + b1);
			if( iCount )
			{
				float fCount = float(iCount);
				fEntropy += fCount * log2(fCount);
			}
		}
	}
	// fEntropy *= -1.0;
	fEntropy = -fEntropy / fNrOfNeighbors + log2(fNrOfNeighbors);
	fEntropy = max(0.0, fEntropy);

	*ADDRESS_2D(
		float, cEntropyVolume_pitched.ptr, 
		sizeof(float), cEntropyVolume_pitched.pitch, 
		iVoxelX, iVoxelY + iVoxelZ * i3VolumeSize.y) = fEntropy;
}

void 
_ComputeEntropyVolumeWithMarginalHistogram_cuda
(
						// res. of the neighboring region
	int3 i3KernelSize,

						// the histogram
	int iNrOfBins,
	int *piHistogram_global,
	float *pfLogHistogram_global,

						// res. of the volume																
	int3 i3VolumeSize,	
						// bin volume																	
	cudaPitchedPtr		cBinVolume_pitched,								
	cudaPitchedPtr		cEntropyVolume_pitched
)
{
CLOCK_INIT(_ComputeEntropyVolume_PRINT_TIMING, __FUNCTION__ ": ");

CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_TIMING);
	int iNrOfMarginalBins = int(sqrt(double(iNrOfBins)));

	dim3 v3Blk = dim3(BLOCK_DIM_X, BLOCK_DIM_Y);
	const unsigned int iMaxMemorySpace = MAX_MEMORY_SIZE;
	int iMaxNrOfBlocks = int(floorf(float(iMaxMemorySpace) / float(sizeof(int) * (iNrOfMarginalBins + iNrOfBins) * v3Blk.x * v3Blk.y)));
	int iNrOfXBlocks = int(ceilf(float(i3VolumeSize.x) / float(v3Blk.x)));
	int iNrOfYBlocks = int(ceilf(float(i3VolumeSize.y) / float(v3Blk.y)));
	iMaxNrOfBlocks = min(iMaxNrOfBlocks, iNrOfXBlocks * iNrOfYBlocks);
	iNrOfXBlocks = min(iNrOfXBlocks, iMaxNrOfBlocks);
	iNrOfYBlocks = int(ceilf(float(iMaxNrOfBlocks) / float(iNrOfXBlocks)));
	dim3 v3Grid = dim3(iNrOfXBlocks, iNrOfYBlocks);
	fprintf(stderr, "MEM = %d MB; #BLOCKS = %d x %d\n", iMaxMemorySpace/(1<<20), v3Grid.x, v3Grid.y);

	cudaPitchedPtr cActiveVoxelHistorgrams_pitched;
	cActiveVoxelHistorgrams_pitched.xsize = v3Grid.x * v3Grid.y * v3Blk.x * v3Blk.y;
	cActiveVoxelHistorgrams_pitched.ysize = iNrOfMarginalBins + iNrOfBins;
	CUDA_SAFE_CALL_NO_SYNC(
		cudaMallocPitch(
			(void**)&cActiveVoxelHistorgrams_pitched.ptr, 
			&cActiveVoxelHistorgrams_pitched.pitch,
			cActiveVoxelHistorgrams_pitched.xsize * sizeof(int), 
			cActiveVoxelHistorgrams_pitched.ysize)	);

	// bind the input vin volume to the texture that represents the src. bin volume 
	t2dSrcBinVolume.addressMode[0] = cudaAddressModeClamp;
	t2dSrcBinVolume.addressMode[1] = cudaAddressModeClamp;
	t2dSrcBinVolume.filterMode =	cudaFilterModePoint;
	t2dSrcBinVolume.normalized =	false;
	CUDA_SAFE_CALL_NO_SYNC(
		cudaBindTexture2D(
			0, 
			t2dSrcBinVolume, 
			cBinVolume_pitched.ptr, 
			cudaCreateChannelDesc<int>(),
			cVolumeExtent_array.width, 
			cVolumeExtent_array.height * cVolumeExtent_array.depth, 
			cBinVolume_pitched.pitch) );

	// bind the histogram as a texture
	t2dActiveVoxelHistorgrams.addressMode[0] = cudaAddressModeClamp;
	t2dActiveVoxelHistorgrams.addressMode[1] = cudaAddressModeClamp;
	t2dActiveVoxelHistorgrams.filterMode =	cudaFilterModePoint;
	t2dActiveVoxelHistorgrams.normalized =	false;
	CUDA_SAFE_CALL_NO_SYNC(
		cudaBindTexture2D(
			0, 
			t2dActiveVoxelHistorgrams, 
			cActiveVoxelHistorgrams_pitched.ptr, 
			cudaCreateChannelDesc<int>(),
			cActiveVoxelHistorgrams_pitched.xsize, 
			cActiveVoxelHistorgrams_pitched.ysize,
			cActiveVoxelHistorgrams_pitched.pitch) );

CLOCK_END(_ComputeEntropyVolume_PRINT_TIMING, false);

CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_TIMING);

	for(int			y = 0; y < i3VolumeSize.y; y += v3Grid.y * v3Blk.y)
		for(int		x = 0; x < i3VolumeSize.x; x += v3Grid.x * v3Blk.x)
			for(int z = 0; z < i3VolumeSize.z; z++)
			{
				CLOCK_INIT(_ComputeEntropyVolume_PRINT_LOOP_TIMING, __FUNCTION__ " (main loop): ");

				int3 i3BlockCorner = make_int3(x, y, z);

				/*
				CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_LOOP_TIMING);

				CUDA_SAFE_CALL_NO_SYNC(
					cudaMemset2D(
						cActiveVoxelHistorgrams_pitched.ptr, 
						cActiveVoxelHistorgrams_pitched.pitch,
						0, 
						cActiveVoxelHistorgrams_pitched.xsize * sizeof(int), 
						cActiveVoxelHistorgrams_pitched.ysize)	);

				CLOCK_END(_ComputeEntropyVolume_PRINT_LOOP_TIMING, false);
				*/

				CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_LOOP_TIMING);

				if( 0 == z )
				{
					_CreateHistogramWithMarginalHistogram_kernel<<<v3Grid, v3Blk, 0>>>
					(
						i3BlockCorner,
						i3KernelSize,
						i3VolumeSize,
						iNrOfBins,
						iNrOfMarginalBins,
						cActiveVoxelHistorgrams_pitched
					);
					CUT_CHECK_ERROR("_CreateHistogramWithMarginalHistogram_kernel() failed");
				}
				else
				{
					_UpdateHistogramWithMarginalHistogram_kernel<<<v3Grid, v3Blk, 0>>>
					(
						i3BlockCorner,
						i3KernelSize,
						i3VolumeSize,
						iNrOfBins,
						iNrOfMarginalBins,
						cActiveVoxelHistorgrams_pitched
					);
					CUT_CHECK_ERROR("_UpdateHistogramWithMarginalHistogram_kernel() failed");
				}

				CLOCK_END(_ComputeEntropyVolume_PRINT_LOOP_TIMING, false);

				CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_LOOP_TIMING);

				_ComputeEntropyWithMarginalHistogram_kernel<<<v3Grid, v3Blk, 0>>>
				(
					i3BlockCorner,
					iNrOfBins,
					iNrOfMarginalBins,
					i3KernelSize,
					i3VolumeSize,
					cEntropyVolume_pitched
				);
				CUT_CHECK_ERROR("_ComputeEntropy_kernel() failed");

				CLOCK_END(_ComputeEntropyVolume_PRINT_LOOP_TIMING, false);

				CLOCK_PRINT(_ComputeEntropyVolume_PRINT_LOOP_TIMING);

			}

CLOCK_END(_ComputeEntropyVolume_PRINT_TIMING, false);

CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_TIMING);

	FREE_MEMORY(cActiveVoxelHistorgrams_pitched.ptr);

CLOCK_END(_ComputeEntropyVolume_PRINT_TIMING, false);

CLOCK_PRINT(_ComputeEntropyVolume_PRINT_TIMING);
}

#endif	// COMPUTE_ENTROPY_VOLUME_WITH_MARGINAL_HISTOGRAM

/*

$Log: not supported by cvs2svn $

*/
