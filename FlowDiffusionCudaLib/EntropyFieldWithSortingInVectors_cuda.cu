
#include <hip/hip_runtime.h>
#if COMPUTE_ENTROPY_VOLUME_WITH_SORTING_IN_VECTORS_CUDA

static texture<int4, 2, cudaReadModeElementType> t2dActiveVoxelSortedNeighborsInVectors;

__device__
void
_SwapData(int tid, int ixj, int t, int k, int iNrOfBins, int column, cudaPitchedPtr cActiveVoxelNeighbors_pitched)
{
	int4 i4Tid, i4Ixj;
	i4Tid = tex2D(t2dActiveVoxelSortedNeighborsInVectors, column, tid);
	i4Ixj = tex2D(t2dActiveVoxelSortedNeighborsInVectors, column, ixj);

	int4 i4Max = make_int4(
		max(i4Tid.x, i4Ixj.x),
		max(i4Tid.y, i4Ixj.y),
		max(i4Tid.z, i4Ixj.z),
		max(i4Tid.w, i4Ixj.w)
		);
	int4 i4Min = make_int4(
		min(i4Tid.x, i4Ixj.x),
		min(i4Tid.y, i4Ixj.y),
		min(i4Tid.z, i4Ixj.z),
		min(i4Tid.w, i4Ixj.w)
		);
	if ((t & k) == 0)
    {
		/*
        if (shared[tid] < shared[ixj])
        {
            swap(shared[tid], shared[ixj]);
        }
		*/
		i4Tid = i4Max;
		i4Ixj = i4Min;
    }
    else
    {
		/*
        if (shared[tid] > shared[ixj])
        {
            swap(shared[tid], shared[ixj]);
        }
		*/
		i4Tid = i4Min;
		i4Ixj = i4Max;
    }

	if( column < cActiveVoxelNeighbors_pitched.pitch/sizeof(int4) )
	{
		if(tid < cActiveVoxelNeighbors_pitched.ysize)
			*ADDRESS_2D(
				int4, cActiveVoxelNeighbors_pitched.ptr, 
				sizeof(int4), cActiveVoxelNeighbors_pitched.pitch, 
				column, tid) = i4Tid;
		if(ixj < cActiveVoxelNeighbors_pitched.ysize)
			*ADDRESS_2D(
				int4, cActiveVoxelNeighbors_pitched.ptr, 
				sizeof(int4), cActiveVoxelNeighbors_pitched.pitch, 
				column, ixj) = i4Ixj;
	}
}

__device__ 
void
_BitonicSwap(int j, int k, int tid, int iNrOfBins, int column, cudaPitchedPtr cActiveVoxelNeighbors_pitched)
{
    int ixj = tid ^ j;

    if (ixj > tid)
		_SwapData(tid, ixj, tid, k, iNrOfBins, column, cActiveVoxelNeighbors_pitched);
}

__global__
void
_BitonicMergeInVectors_kernel
(
	int k, 
	int j,
	int iNrOfBins,
	cudaPitchedPtr cActiveVoxelNeighbors_pitched
)
{
	int iThreadId = threadIdx.x + blockIdx.x * blockDim.x;

	// size_t tid = blockIdx.y;
	size_t tid = blockIdx.y;
    size_t ixj = tid ^ j;

	if (ixj <= tid)
    {
		tid = ixj + cActiveVoxelNeighbors_pitched.ysize / 2;
		ixj = tid ^ j;
	}

	_BitonicSwap(j, k, tid, iNrOfBins, iThreadId, cActiveVoxelNeighbors_pitched);
}

__global__ 
void
_ComputeEntropyOnSortedNeighborsInVectors_kernel
(
	int3 i3BlockCorner,
	int iNrOfBins,
	int3 i3Step,
	dim3 v3Grid,
	int3 i3KernelSize,
	unsigned int *puActiveVoxelSortedNeighbors_global,
	int3 i3VolumeSize,
	cudaPitchedPtr cEntropyVolume_pitched
)
{
	int iBlockIdX = blockIdx.x;
	int iBlockIdY = blockIdx.y % v3Grid.y;
	int iBlockIdZ = blockIdx.y / v3Grid.y;
	int iThreadX = iBlockIdX * blockDim.x + threadIdx.x;
	int iThreadY = iBlockIdY * blockDim.y + threadIdx.y;
	int iThreadZ = iBlockIdZ * blockDim.z + threadIdx.z;
	int iThreadId = iThreadX + iThreadY * i3Step.x + iThreadZ * i3Step.x * i3Step.y;

 	int iVoxelX = i3BlockCorner.x + iThreadX;
	int iVoxelY = i3BlockCorner.y + iThreadY;
	int iVoxelZ = i3BlockCorner.z + iThreadZ;

	int iNrOfNeighbors = (2 * i3KernelSize.x + 1) * (2 * i3KernelSize.y + 1) * (2 * i3KernelSize.z + 1);
	float fNrOfNeighbors = float(iNrOfNeighbors);

	float fEntropy = 0.0f;
	unsigned int uPrevBin = 0;
	float fCount = 0.0f; 

	for(int n = 0; n < iNrOfNeighbors; n++)
	{
		unsigned int uBin = tex2D(t2dActiveVoxelSortedNeighbors, iThreadId, n);

		if( n == 0 || uBin == uPrevBin )
		{
			fCount += 1.0f;
		}
		else
		{
			if( fCount > 0.0f )
				fEntropy += fCount * log2(fCount);
			
			fCount = 1.0f;
		}
		uPrevBin = uBin;
	}
	if( fCount > 0.0f )
		fEntropy += fCount * log2(fCount);

	fEntropy = -fEntropy / fNrOfNeighbors + log2(fNrOfNeighbors);
	fEntropy = max(fEntropy, 0.0f);

	if( iVoxelX < i3VolumeSize.x && iVoxelY < i3VolumeSize.y && iVoxelZ < i3VolumeSize.z )
		*ADDRESS_2D(
			float, cEntropyVolume_pitched.ptr, 
			sizeof(float), cEntropyVolume_pitched.pitch, 
			iVoxelX, iVoxelY + iVoxelZ * i3VolumeSize.y) = fEntropy;
}

__global__ 
void 
_CollectNeighborsInVectors_kernel
(
	int3 i3BlockCorner,
	int3 i3Step,
	dim3 v3Grid,
	int3 i3KernelSize,
	int3 i3VolumeSize,
	int iNrOfBins,
	cudaPitchedPtr cActiveVoxelNeighbors_pitched
)
{
	int iBlockIdX = blockIdx.x;
	int iBlockIdY = blockIdx.y % v3Grid.y;
	int iBlockIdZ = blockIdx.y / v3Grid.y;
	int iThreadX = iBlockIdX * blockDim.x + threadIdx.x;
	int iThreadY = iBlockIdY * blockDim.y + threadIdx.y;
	int iThreadZ = iBlockIdZ * blockDim.z + threadIdx.z;
	int iThreadId = iThreadX + iThreadY * i3Step.x + iThreadZ * i3Step.x * i3Step.y;

 	int iVoxelX = i3BlockCorner.x + iThreadX;
	int iVoxelY = i3BlockCorner.y + iThreadY;
	int iVoxelZ = i3BlockCorner.z + iThreadZ;

	for(int	iOffset = 0,iZOffset = -i3KernelSize.z; iZOffset <= i3KernelSize.z; iZOffset++)
		for(int			iYOffset = -i3KernelSize.y; iYOffset <= i3KernelSize.y; iYOffset++)
			for(int		iXOffset = -i3KernelSize.x; iXOffset <= i3KernelSize.x; iXOffset++, iOffset++)
			{
				int iX, iY, iZ;
				iX = iVoxelX + iXOffset;
				iY = iVoxelY + iYOffset;
				iZ = iVoxelZ + iZOffset;

				int3 i3TexCoord;
				i3TexCoord.x = IMirrorCoord(iX, i3VolumeSize.x);
				i3TexCoord.y = IMirrorCoord(iY, i3VolumeSize.y);
				i3TexCoord.z = IMirrorCoord(iZ, i3VolumeSize.z);

				int iSrcBin = tex2D(t2dSrcBinVolume, i3TexCoord.x, i3TexCoord.y + i3TexCoord.z * i3VolumeSize.y);

				*ADDRESS_2D(
					int,			cActiveVoxelNeighbors_pitched.ptr, 
					sizeof(int),	cActiveVoxelNeighbors_pitched.pitch, 
					iThreadId, iOffset) = iSrcBin;
			}
}
// ADD-BY-LEETEN 12/18/2009-END

void 
_ComputeEntropyVolumeWithSortingInVectors_cuda
(
						// res. of the neighboring region
	int3 i3KernelSize,

						// the histogram
	int iNrOfBins,
	int *piHistogram_global,
	float *pfLogHistogram_global,

						// res. of the volume																
	int3 i3VolumeSize,	
						// bin volume																	
	cudaPitchedPtr		cBinVolume_pitched,								
	cudaPitchedPtr		cEntropyVolume_pitched
)
{
CLOCK_INIT(_ComputeEntropyVolume_PRINT_TIMING, __FUNCTION__ ": ");

CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_TIMING);
	const int iNrOfRows = (2 * i3KernelSize.x + 1) * (2 * i3KernelSize.y + 1) * (2 * i3KernelSize.z + 1);
	const int iNrOfRowsRoundTo2 = int(pow(2.0, ceil(log2(double(iNrOfRows)))));

	dim3 v3Blk = dim3(BLOCK_DIM_X, BLOCK_DIM_Y);
	int iTotalNrOfXBlocks = int(ceilf(float(i3VolumeSize.x) / float(v3Blk.x)));
	int iTotalNrOfYBlocks = int(ceilf(float(i3VolumeSize.y) / float(v3Blk.y)));
	int iTotalNrOfZBlocks = int(ceilf(float(i3VolumeSize.z) / float(v3Blk.z)));
	int iNrOfBlocksInMemory = MAX_MEMORY_SIZE / (iNrOfRowsRoundTo2 * sizeof(int) * v3Blk.x * v3Blk.y * v3Blk.z);
	int iNrOfXBlocksInMemory = min(iTotalNrOfXBlocks, iNrOfBlocksInMemory);
	int iNrOfYBlocksInMemory = min(iTotalNrOfYBlocks, max(1, iNrOfBlocksInMemory / iTotalNrOfXBlocks));
	int iNrOfZBlocksInMemory = min(iTotalNrOfZBlocks, max(1, iNrOfBlocksInMemory / (iTotalNrOfXBlocks * iTotalNrOfYBlocks)));

	dim3 v3Grid = dim3(iNrOfXBlocksInMemory, iNrOfYBlocksInMemory, iNrOfZBlocksInMemory);
	fprintf(stderr, "#BLOCKS = %d x %d x %d\n", v3Grid.x, v3Grid.y, v3Grid.z);

	int3 i3Step;
	i3Step.x = iNrOfXBlocksInMemory * v3Blk.x;
	i3Step.y = iNrOfYBlocksInMemory * v3Blk.y;
	i3Step.z = iNrOfZBlocksInMemory * v3Blk.z;

	cudaPitchedPtr cActiveVoxelNeighbors_pitched;
	cActiveVoxelNeighbors_pitched.xsize = v3Grid.x * v3Grid.y * v3Grid.z * v3Blk.x * v3Blk.y* v3Blk.z;
	cActiveVoxelNeighbors_pitched.ysize = iNrOfRowsRoundTo2;
	CUDA_SAFE_CALL_NO_SYNC(
		cudaMallocPitch(
			(void**)&cActiveVoxelNeighbors_pitched.ptr, 
			&cActiveVoxelNeighbors_pitched.pitch,
			cActiveVoxelNeighbors_pitched.xsize * sizeof(int), 
			cActiveVoxelNeighbors_pitched.ysize)	);

	CUDA_SAFE_CALL_NO_SYNC(
		cudaMemset2D(
			cActiveVoxelNeighbors_pitched.ptr, 
			cActiveVoxelNeighbors_pitched.pitch,
			0, 
			cActiveVoxelNeighbors_pitched.xsize * sizeof(int), 
			cActiveVoxelNeighbors_pitched.ysize)	);

	// bind the input vin volume to the texture that represents the src. bin volume 
	t2dSrcBinVolume.addressMode[0] = cudaAddressModeClamp;
	t2dSrcBinVolume.addressMode[1] = cudaAddressModeClamp;
	t2dSrcBinVolume.filterMode =	cudaFilterModePoint;
	t2dSrcBinVolume.normalized =	false;
	CUDA_SAFE_CALL_NO_SYNC(
		cudaBindTexture2D(
			0, 
			t2dSrcBinVolume, 
			cBinVolume_pitched.ptr, 
			cudaCreateChannelDesc<int>(),
			cVolumeExtent_array.width, 
			cVolumeExtent_array.height * cVolumeExtent_array.depth, 
			cBinVolume_pitched.pitch) );

	// bind the histogram as a texture
	t2dActiveVoxelSortedNeighbors.addressMode[0] = cudaAddressModeClamp;
	t2dActiveVoxelSortedNeighbors.addressMode[1] = cudaAddressModeClamp;
	t2dActiveVoxelSortedNeighbors.filterMode =	cudaFilterModePoint;
	t2dActiveVoxelSortedNeighbors.normalized =	false;

	CUDA_SAFE_CALL_NO_SYNC(
		cudaBindTexture2D(
			0, 
			t2dActiveVoxelSortedNeighbors, 
			cActiveVoxelNeighbors_pitched.ptr, 
			cudaCreateChannelDesc<unsigned int>(),
			cActiveVoxelNeighbors_pitched.xsize,
			cActiveVoxelNeighbors_pitched.ysize, 
			cActiveVoxelNeighbors_pitched.pitch) );

	// bind the histogram as another texture where the type of each element is int4
	t2dActiveVoxelSortedNeighborsInVectors.addressMode[0] = cudaAddressModeClamp;
	t2dActiveVoxelSortedNeighborsInVectors.addressMode[1] = cudaAddressModeClamp;
	t2dActiveVoxelSortedNeighborsInVectors.filterMode =	cudaFilterModePoint;
	t2dActiveVoxelSortedNeighborsInVectors.normalized =	false;

	CUDA_SAFE_CALL_NO_SYNC(
		cudaBindTexture2D(
			0, 
			t2dActiveVoxelSortedNeighborsInVectors, 
			cActiveVoxelNeighbors_pitched.ptr, 
			cudaCreateChannelDesc<int4>(),
			cActiveVoxelNeighbors_pitched.xsize * sizeof(int) / sizeof(int4),
			cActiveVoxelNeighbors_pitched.ysize, 
			cActiveVoxelNeighbors_pitched.pitch) );
CLOCK_END(_ComputeEntropyVolume_PRINT_TIMING, false);

CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_TIMING);

	for(int			z = 0; z < i3VolumeSize.z; z += i3Step.z)
		for(int		y = 0; y < i3VolumeSize.y; y += i3Step.y)
			for(int	x = 0; x < i3VolumeSize.x; x += i3Step.x)
			{
				CLOCK_INIT(_ComputeEntropyVolume_PRINT_LOOP_TIMING, __FUNCTION__ " (main loop): ");

				CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_LOOP_TIMING);

				int3 i3BlockCorner = make_int3(x, y, z);
				_CollectNeighborsInVectors_kernel<<<
					dim3(v3Grid.x, v3Grid.y * v3Grid.z), 
					v3Blk, 
					0>>>
				(
					i3BlockCorner,
					i3Step,
					v3Grid,
					i3KernelSize,
					i3VolumeSize,
					iNrOfBins,
					cActiveVoxelNeighbors_pitched
				);
				CUT_CHECK_ERROR("_CollectNeighbors_kernel() failed");

CLOCK_END(_ComputeEntropyVolume_PRINT_LOOP_TIMING, false);

CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_LOOP_TIMING);

				// apply bitnoic sort to sort the data in descend order
				for (int k = 2; k <= iNrOfRowsRoundTo2; k *= 2)
					// Bitonic merge:
					for (int j = k / 2; j > 0; j /= 2)
					{
						_BitonicMergeInVectors_kernel<<<
							dim3(
								v3Grid.x * v3Grid.y * v3Grid.z / (sizeof(int4) / sizeof(int)) , 
								// iNrOfRowsRoundTo2),
								iNrOfRowsRoundTo2/2),
							dim3(
								v3Blk.x * v3Blk.y * v3Blk.z), 
							0>>>(
								k, 
								j,
								iNrOfBins,
								cActiveVoxelNeighbors_pitched
								);
						CUT_CHECK_ERROR("BitonicSort failed");
					}

CLOCK_END(_ComputeEntropyVolume_PRINT_LOOP_TIMING, false);

CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_LOOP_TIMING);
				// 
				_ComputeEntropyOnSortedNeighborsInVectors_kernel<<<
					dim3(v3Grid.x, v3Grid.y * v3Grid.z), 
					v3Blk, 
					0>>>
				(
					i3BlockCorner,
					iNrOfBins,
					i3Step,
					v3Grid,
					i3KernelSize,
					(unsigned int *)cActiveVoxelNeighbors_pitched.ptr,
					i3VolumeSize,
					cEntropyVolume_pitched
				);
				CUT_CHECK_ERROR("_ComputeEntropyOnSortedNeighbors_kernel() failed");
CLOCK_END(_ComputeEntropyVolume_PRINT_LOOP_TIMING, false);

CLOCK_PRINT(_ComputeEntropyVolume_PRINT_LOOP_TIMING);
			}


CLOCK_END(_ComputeEntropyVolume_PRINT_TIMING, false);

CLOCK_BEGIN(_ComputeEntropyVolume_PRINT_TIMING);

	FREE_MEMORY(cActiveVoxelNeighbors_pitched.ptr);

CLOCK_END(_ComputeEntropyVolume_PRINT_TIMING, false);

CLOCK_PRINT(_ComputeEntropyVolume_PRINT_TIMING);
}

#endif	// #if COMPUTE_ENTROPY_VOLUME_WITH_SORTING_IN_VECTORS_CUDA

/*

$Log: not supported by cvs2svn $

*/
